
#include <hip/hip_runtime.h>
/*! \file plmc_ctu_cuda.cu
 *  \brief Definitions of the piecewise linear reconstruction functions with 
           liminting applied in the characteristic variables, as decribed
           in Stone et al., 2008. */
#ifdef CUDA
#ifdef PLMC

#include<cuda.h>
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"plmc_ctu_cuda.h"


/*! \fn __global__ void PLMC_CTU(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx, int ny, int nz, int n_ghost, Real dx, Real dt, Real gamma, int dir)
 *  \brief When passed a stencil of conserved variables, returns the left and right 
           boundary values for the interface calculated using plm. */
__global__ void PLMC_CTU(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx, int ny, int nz, int n_ghost, Real dx, Real dt, Real gamma, int dir)
{
  int n_cells = nx*ny*nz;
  int o1, o2, o3;
  if (dir == 0) {
    o1 = 1; o2 = 2; o3 = 3;
  }
  if (dir == 1) {
    o1 = 2; o2 = 3; o3 = 1;
  }
  if (dir == 2) {
    o1 = 3; o2 = 1; o3 = 2;
  }

  // declare primative variables for each stencil
  // these will be placed into registers for each thread
  Real d_i, vx_i, vy_i, vz_i, p_i;
  Real d_imo, vx_imo, vy_imo, vz_imo, p_imo; 
  Real d_ipo, vx_ipo, vy_ipo, vz_ipo, p_ipo;

  Real dtodx = dt/dx;

  // declare other variables to be used
  Real a_i;
  Real lambda_m, lambda_0, lambda_p;
  Real del_d_L, del_vx_L, del_vy_L, del_vz_L, del_p_L;
  Real del_d_R, del_vx_R, del_vy_R, del_vz_R, del_p_R;
  Real del_d_C, del_vx_C, del_vy_C, del_vz_C, del_p_C;
  Real del_d_G, del_vx_G, del_vy_G, del_vz_G, del_p_G;
  Real del_a_0_L, del_a_1_L, del_a_2_L, del_a_3_L, del_a_4_L;
  Real del_a_0_R, del_a_1_R, del_a_2_R, del_a_3_R, del_a_4_R;
  Real del_a_0_C, del_a_1_C, del_a_2_C, del_a_3_C, del_a_4_C;
  Real del_a_0_G, del_a_1_G, del_a_2_G, del_a_3_G, del_a_4_G;
  Real del_a_0_m, del_a_1_m, del_a_2_m, del_a_3_m, del_a_4_m;
  Real lim_slope_a, lim_slope_b;
  Real del_d_m_i, del_vx_m_i, del_vy_m_i, del_vz_m_i, del_p_m_i;
  Real d_L_iph, vx_L_iph, vy_L_iph, vz_L_iph, p_L_iph;
  Real d_R_imh, vx_R_imh, vy_R_imh, vz_R_imh, p_R_imh;
  Real C;
  Real qx;
  Real lamdiff;
  Real sum_0, sum_1, sum_2, sum_3, sum_4;  
  #ifdef DE
  Real ge_i, ge_imo, ge_ipo;
  Real del_ge_L, del_ge_R, del_ge_C, del_ge_G;
  Real del_ge_m_i;
  Real ge_L_iph, ge_R_imh;
  Real sum_5 = 0;
  #endif

  // get a thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId*blockDim.x;
  int id;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;


  //if (xid > n_ghost-3 && xid < nx-n_ghost+2 && yid < ny && zid < nz)
  if (xid < nx && yid < ny && zid < nz)
  {
    // load the 3-cell stencil into registers
    // cell i
    id = xid + yid*nx + zid*nx*ny;
    d_i  =  dev_conserved[            id];
    vx_i =  dev_conserved[o1*n_cells + id] / d_i;
    vy_i =  dev_conserved[o2*n_cells + id] / d_i;
    vz_i =  dev_conserved[o3*n_cells + id] / d_i;
    p_i  = (dev_conserved[4*n_cells + id] - 0.5*d_i*(vx_i*vx_i + vy_i*vy_i + vz_i*vz_i)) * (gamma - 1.0);
    p_i  = fmax(p_i, (Real) TINY_NUMBER);
    #ifdef DE
    ge_i =  dev_conserved[5*n_cells + id] / d_i;
    #endif
    // cell i-1
    if (dir == 0) id = xid-1 + yid*nx + zid*nx*ny;
    if (dir == 1) id = xid + (yid-1)*nx + zid*nx*ny;
    if (dir == 2) id = xid + yid*nx + (zid-1)*nx*ny;
    d_imo  =  dev_conserved[            id];
    vx_imo =  dev_conserved[o1*n_cells + id] / d_imo;
    vy_imo =  dev_conserved[o2*n_cells + id] / d_imo;
    vz_imo =  dev_conserved[o3*n_cells + id] / d_imo;
    p_imo  = (dev_conserved[4*n_cells + id] - 0.5*d_imo*(vx_imo*vx_imo + vy_imo*vy_imo + vz_imo*vz_imo)) * (gamma - 1.0);
    p_imo  = fmax(p_imo, (Real) TINY_NUMBER);
    #ifdef DE
    ge_imo =  dev_conserved[5*n_cells + id] / d_imo;
    #endif
    // cell i+1
    if (dir == 0) id = xid+1 + yid*nx + zid*nx*ny;
    if (dir == 1) id = xid + (yid+1)*nx + zid*nx*ny;
    if (dir == 2) id = xid + yid*nx + (zid+1)*nx*ny;
    d_ipo  =  dev_conserved[            id];
    vx_ipo =  dev_conserved[o1*n_cells + id] / d_ipo;
    vy_ipo =  dev_conserved[o2*n_cells + id] / d_ipo;
    vz_ipo =  dev_conserved[o3*n_cells + id] / d_ipo;
    p_ipo  = (dev_conserved[4*n_cells + id] - 0.5*d_ipo*(vx_ipo*vx_ipo + vy_ipo*vy_ipo + vz_ipo*vz_ipo)) * (gamma - 1.0);
    p_ipo  = fmax(p_ipo, (Real) TINY_NUMBER);
    #ifdef DE
    ge_ipo =  dev_conserved[5*n_cells + id] / d_ipo;
    #endif


    // calculate the adiabatic sound speed in cell i
    a_i   = sqrt(gamma*p_i/d_i);


    // Step 1 - Compute the eigenvalues of the linearized equations in the
    //          primative variables using the cell-centered primative variables

    lambda_m = vx_i-a_i;
    lambda_0 = vx_i;
    lambda_p = vx_i+a_i; 


    // Step 2 - Compute the left, right, centered, and van Leer differences of the primative variables
    //          Note that here L and R refer to locations relative to the cell center
    //          Stone Eqn 36

    // left
    del_d_L  = d_i - d_imo;
    del_vx_L = vx_i - vx_imo;
    del_vy_L = vy_i - vy_imo;
    del_vz_L = vz_i - vz_imo;
    del_p_L  = p_i  - p_imo;

    // right
    del_d_R  = d_ipo  - d_i;
    del_vx_R = vx_ipo - vx_i;
    del_vy_R = vy_ipo - vy_i;
    del_vz_R = vz_ipo - vz_i;
    del_p_R  = p_ipo  - p_i;

    // centered
    del_d_C  = (d_ipo - d_imo) / 2.0;
    del_vx_C = (vx_ipo - vx_imo) / 2.0;
    del_vy_C = (vy_ipo - vy_imo) / 2.0;
    del_vz_C = (vz_ipo - vz_imo) / 2.0;
    del_p_C  = (p_ipo - p_imo) / 2.0;

    // Van Leer
    if (del_d_L*del_d_R > 0.0) { del_d_G = 2.0*del_d_L*del_d_R / (del_d_L+del_d_R); }
    else { del_d_G = 0.0; }
    if (del_vx_L*del_vx_R > 0.0) { del_vx_G = 2.0*del_vx_L*del_vx_R / (del_vx_L+del_vx_R); }
    else { del_vx_G = 0.0; } 
    if (del_vy_L*del_vy_R > 0.0) { del_vy_G = 2.0*del_vy_L*del_vy_R / (del_vy_L+del_vy_R); }
    else { del_vy_G = 0.0; } 
    if (del_vz_L*del_vz_R > 0.0) { del_vz_G = 2.0*del_vz_L*del_vz_R / (del_vz_L+del_vz_R); }
    else { del_vz_G = 0.0; } 
    if (del_p_L*del_p_R > 0.0) { del_p_G = 2.0*del_p_L*del_p_R / (del_p_L+del_p_R); }
    else { del_p_G = 0.0; } 

    #ifdef DE
    del_ge_L = ge_i - ge_imo;
    del_ge_R = ge_ipo - ge_i;
    del_ge_C = 0.5*(ge_ipo - ge_imo);
    if (del_ge_L*del_ge_R > 0.0) { del_ge_G = 2.0*del_ge_L*del_ge_R / (del_ge_L+del_ge_R); }
    else { del_ge_G = 0.0; } 
    #endif


    // Step 3 - Project the left, right, centered and van Leer differences onto the characteristic variables
    //          Stone Eqn 37 (del_a are differences in characteristic variables, see Stone for notation)
    //          Use the eigenvectors given in Stone 2008, Appendix A

    del_a_0_L = -d_i * del_vx_L / (2*a_i) + del_p_L / (2*a_i*a_i);
    del_a_1_L = del_d_L - del_p_L / (a_i*a_i);
    del_a_2_L = del_vy_L;
    del_a_3_L = del_vz_L;
    del_a_4_L = d_i * del_vx_L / (2*a_i) + del_p_L / (2*a_i*a_i);

    del_a_0_R = -d_i * del_vx_R / (2*a_i) + del_p_R / (2*a_i*a_i);
    del_a_1_R = del_d_R - del_p_R / (a_i*a_i);
    del_a_2_R = del_vy_R;
    del_a_3_R = del_vz_R;
    del_a_4_R = d_i * del_vx_R / (2*a_i) + del_p_R / (2*a_i*a_i);

    del_a_0_C = -d_i * del_vx_C / (2*a_i) + del_p_C / (2*a_i*a_i);
    del_a_1_C = del_d_C - del_p_C / (a_i*a_i);
    del_a_2_C = del_vy_C;
    del_a_3_C = del_vz_C;
    del_a_4_C = d_i * del_vx_C / (2*a_i) + del_p_C / (2*a_i*a_i);

    del_a_0_G = -d_i * del_vx_G / (2*a_i) + del_p_G / (2*a_i*a_i);
    del_a_1_G = del_d_G - del_p_G / (a_i*a_i);
    del_a_2_G = del_vy_G;
    del_a_3_G = del_vz_G;
    del_a_4_G = d_i * del_vx_G / (2*a_i) + del_p_G / (2*a_i*a_i); 


    // Step 4 - Apply monotonicity constraints to the differences in the characteristic variables
    //          Stone Eqn 38

    /*
      del_a_0_m = SIGN(del_a_0_C) * minof3(2*fabs(del_a_0_L), 2*fabs(del_a_0_R), fabs(del_a_0_C));
      del_a_1_m = SIGN(del_a_1_C) * minof3(2*fabs(del_a_1_L), 2*fabs(del_a_1_R), fabs(del_a_1_C));
      del_a_2_m = SIGN(del_a_2_C) * minof3(2*fabs(del_a_2_L), 2*fabs(del_a_2_R), fabs(del_a_2_C));
      del_a_3_m = SIGN(del_a_3_C) * minof3(2*fabs(del_a_3_L), 2*fabs(del_a_3_R), fabs(del_a_3_C));
      del_a_4_m = SIGN(del_a_4_C) * minof3(2*fabs(del_a_4_L), 2*fabs(del_a_4_R), fabs(del_a_4_C));
    */

    del_a_0_m = del_a_1_m = del_a_2_m = del_a_3_m = del_a_4_m = 0.0;
  
    if (del_a_0_L*del_a_0_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_0_L), fabs(del_a_0_R));
      lim_slope_b = fmin(fabs(del_a_0_C), fabs(del_a_0_G));
      del_a_0_m = sgn(del_a_0_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_1_L*del_a_1_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_1_L), fabs(del_a_1_R));
      lim_slope_b = fmin(fabs(del_a_1_C), fabs(del_a_1_G));
      del_a_1_m = sgn(del_a_1_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_2_L*del_a_2_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_2_L), fabs(del_a_2_R));
      lim_slope_b = fmin(fabs(del_a_2_C), fabs(del_a_2_G));
      del_a_2_m = sgn(del_a_2_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_3_L*del_a_3_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_3_L), fabs(del_a_3_R));
      lim_slope_b = fmin(fabs(del_a_3_C), fabs(del_a_3_G));
      del_a_3_m = sgn(del_a_3_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_4_L*del_a_4_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_4_L), fabs(del_a_4_R));
      lim_slope_b = fmin(fabs(del_a_4_C), fabs(del_a_4_G));
      del_a_4_m = sgn(del_a_4_C) * fmin(2.0*lim_slope_a, lim_slope_b); 
    }
    #ifdef DE
    if (del_ge_L*del_ge_R > 0.0) {
      lim_slope_a = fmin(fabs(del_ge_L), fabs(del_ge_R));
      lim_slope_b = fmin(fabs(del_ge_C), fabs(del_ge_G));
      del_ge_m_i = sgn(del_ge_C) * fmin(2.0*lim_slope_a, lim_slope_b); 
    }
    else del_ge_m_i = 0.0;
    #endif


    // Step 5 - Project the monotonized difference in the characteristic variables back onto the 
    //          primative variables
    //          Stone Eqn 39

    del_d_m_i  = del_a_0_m + del_a_1_m + del_a_4_m;
    del_vx_m_i = -a_i*del_a_0_m / d_i + a_i* del_a_4_m / d_i;
    del_vy_m_i = del_a_2_m;
    del_vz_m_i = del_a_3_m;
    del_p_m_i  = a_i*a_i*del_a_0_m + a_i*a_i*del_a_4_m;  


    // Step 6 - Compute the left and right interface values using the monotonized difference in the
    //          primative variables
    //          Stone Eqns 40 & 41
      
    /*
    d_R_imh  = d_i  - (0.5 - fmin(lambda_m, 0) * 0.5*dtodx) * del_d_m_i;
    vx_R_imh = vx_i - (0.5 - fmin(lambda_m, 0) * 0.5*dtodx) * del_vx_m_i;
    vy_R_imh = vy_i - (0.5 - fmin(lambda_m, 0) * 0.5*dtodx) * del_vy_m_i;
    vz_R_imh = vz_i - (0.5 - fmin(lambda_m, 0) * 0.5*dtodx) * del_vz_m_i;
    p_R_imh  = p_i  - (0.5 - fmin(lambda_m, 0) * 0.5*dtodx) * del_p_m_i;

    d_L_iph  = d_i  + (0.5 - fmax(lambda_p, 0) * 0.5*dtodx) * del_d_m_i;
    vx_L_iph = vx_i + (0.5 - fmax(lambda_p, 0) * 0.5*dtodx) * del_vx_m_i;
    vy_L_iph = vy_i + (0.5 - fmax(lambda_p, 0) * 0.5*dtodx) * del_vy_m_i;
    vz_L_iph = vz_i + (0.5 - fmax(lambda_p, 0) * 0.5*dtodx) * del_vz_m_i;
    p_L_iph  = p_i  + (0.5 - fmax(lambda_p, 0) * 0.5*dtodx) * del_p_m_i;
    */

    // Step 7 Compute L/R values, ensure they lie between neighboring cell-centered values
    d_R_imh  = d_i  - 0.5*del_d_m_i; 
    vx_R_imh = vx_i - 0.5*del_vx_m_i;
    vy_R_imh = vy_i - 0.5*del_vy_m_i;
    vz_R_imh = vz_i - 0.5*del_vz_m_i;
    p_R_imh  = p_i  - 0.5*del_p_m_i;
 
    d_L_iph  = d_i  + 0.5*del_d_m_i; 
    vx_L_iph = vx_i + 0.5*del_vx_m_i;
    vy_L_iph = vy_i + 0.5*del_vy_m_i;
    vz_L_iph = vz_i + 0.5*del_vz_m_i;
    p_L_iph  = p_i  + 0.5*del_p_m_i; 

    #ifdef DE
    ge_R_imh = ge_i - 0.5*del_ge_m_i;
    ge_L_iph = ge_i + 0.5*del_ge_m_i;
    #endif


    C = d_R_imh + d_L_iph;
    d_R_imh = fmax( fmin(d_i, d_imo), d_R_imh );
    d_R_imh = fmin( fmax(d_i, d_imo), d_R_imh );
    d_L_iph = C - d_R_imh;
    d_L_iph = fmax( fmin(d_i, d_ipo), d_L_iph );
    d_L_iph = fmin( fmax(d_i, d_ipo), d_L_iph );
    d_R_imh = C - d_L_iph;

    C = vx_R_imh + vx_L_iph;
    vx_R_imh = fmax( fmin(vx_i, vx_imo), vx_R_imh );
    vx_R_imh = fmin( fmax(vx_i, vx_imo), vx_R_imh );
    vx_L_iph = C - vx_R_imh;
    vx_L_iph = fmax( fmin(vx_i, vx_ipo), vx_L_iph );
    vx_L_iph = fmin( fmax(vx_i, vx_ipo), vx_L_iph );
    vx_R_imh = C - vx_L_iph;  

    C = vy_R_imh + vy_L_iph;
    vy_R_imh = fmax( fmin(vy_i, vy_imo), vy_R_imh );
    vy_R_imh = fmin( fmax(vy_i, vy_imo), vy_R_imh );
    vy_L_iph = C - vy_R_imh;
    vy_L_iph = fmax( fmin(vy_i, vy_ipo), vy_L_iph );
    vy_L_iph = fmin( fmax(vy_i, vy_ipo), vy_L_iph );
    vy_R_imh = C - vy_L_iph;
 
    C = vz_R_imh + vz_L_iph;
    vz_R_imh = fmax( fmin(vz_i, vz_imo), vz_R_imh );
    vz_R_imh = fmin( fmax(vz_i, vz_imo), vz_R_imh );
    vz_L_iph = C - vz_R_imh; 
    vz_L_iph = fmax( fmin(vz_i, vz_ipo), vz_L_iph );
    vz_L_iph = fmin( fmax(vz_i, vz_ipo), vz_L_iph );
    vz_R_imh = C - vz_L_iph;

    C = p_R_imh + p_L_iph;
    p_R_imh = fmax( fmin(p_i, p_imo), p_R_imh );
    p_R_imh = fmin( fmax(p_i, p_imo), p_R_imh );
    p_L_iph = C - p_R_imh;
    p_L_iph = fmax( fmin(p_i, p_ipo), p_L_iph );
    p_L_iph = fmin( fmax(p_i, p_ipo), p_L_iph );
    p_R_imh = C - p_L_iph;

    del_d_m_i  = d_L_iph  - d_R_imh;
    del_vx_m_i = vx_L_iph - vx_R_imh;
    del_vy_m_i = vy_L_iph - vy_R_imh;
    del_vz_m_i = vz_L_iph - vz_R_imh;
    del_p_m_i  = p_L_iph  - p_R_imh;

    #ifdef DE
    C = ge_R_imh + ge_L_iph;
    ge_R_imh = fmax( fmin(ge_i, ge_imo), ge_R_imh );
    ge_R_imh = fmin( fmax(ge_i, ge_imo), ge_R_imh );
    ge_L_iph = C - ge_R_imh; 
    ge_L_iph = fmax( fmin(ge_i, ge_ipo), ge_L_iph );
    ge_L_iph = fmin( fmax(ge_i, ge_ipo), ge_L_iph );
    ge_R_imh = C - ge_L_iph;    
    del_ge_m_i = ge_L_iph - ge_R_imh;
    #endif


    // Step 8 - Integrate linear interpolation function over domain of dependence
    //          defined by max(min) eigenvalue
    qx = -0.5*fmin(lambda_m, 0)*dtodx;
    d_R_imh  = d_R_imh  + qx * del_d_m_i;
    vx_R_imh = vx_R_imh + qx * del_vx_m_i;
    vy_R_imh = vy_R_imh + qx * del_vy_m_i;
    vz_R_imh = vz_R_imh + qx * del_vz_m_i;
    p_R_imh  = p_R_imh  + qx * del_p_m_i;

    qx = 0.5*fmax(lambda_p, 0)*dtodx;
    d_L_iph  = d_L_iph  - qx * del_d_m_i;
    vx_L_iph = vx_L_iph - qx * del_vx_m_i;
    vy_L_iph = vy_L_iph - qx * del_vy_m_i;
    vz_L_iph = vz_L_iph - qx * del_vz_m_i;
    p_L_iph  = p_L_iph  - qx * del_p_m_i;

    #ifdef DE
    ge_R_imh = ge_R_imh + qx * del_ge_m_i;
    ge_L_iph = ge_L_iph - qx * del_ge_m_i;
    #endif


    // Step 7 - Perform the characteristic tracing
    //          Stone Eqns 42 & 43

    // left-hand interface value, i+1/2
    sum_0 = sum_1 = sum_2 = sum_3 = sum_4 = 0;
    if (lambda_m >= 0)
    {
      lamdiff = lambda_p - lambda_m;

      sum_0 += lamdiff * (-d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 - del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (-d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }
    if (lambda_0 >= 0)
    {
      lamdiff = lambda_p - lambda_0;
  
      sum_0 += lamdiff * (del_d_m_i - del_p_m_i/(a_i*a_i));
      sum_2 += lamdiff * del_vy_m_i;
      sum_3 += lamdiff * del_vz_m_i;
      #ifdef DE
      sum_5 += lamdiff * del_ge_m_i;
      #endif
    }
    if (lambda_p >= 0)
    {
      lamdiff = lambda_p - lambda_p;

      sum_0 += lamdiff * (d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 + del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }

    // add the corrections to the initial guesses for the interface values
    d_L_iph  += 0.5*dtodx*sum_0;
    vx_L_iph += 0.5*dtodx*sum_1;
    vy_L_iph += 0.5*dtodx*sum_2;
    vz_L_iph += 0.5*dtodx*sum_3;
    p_L_iph  += 0.5*dtodx*sum_4;
    #ifdef DE
    ge_L_iph += 0.5*dtodx*sum_5;
    #endif


    // right-hand interface value, i-1/2
    sum_0 = sum_1 = sum_2 = sum_3 = sum_4 = 0;
    if (lambda_m <= 0)
    {
      lamdiff = lambda_m - lambda_m; 

      sum_0 += lamdiff * (-d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 - del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (-d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }
    if (lambda_0 <= 0)
    {
      lamdiff = lambda_m - lambda_0;
  
      sum_0 += lamdiff * (del_d_m_i - del_p_m_i/(a_i*a_i));
      sum_2 += lamdiff * del_vy_m_i;
      sum_3 += lamdiff * del_vz_m_i;
      #ifdef DE
      sum_5 += lamdiff * del_ge_m_i;
      #endif
    }
    if (lambda_p <= 0)
    {
      lamdiff = lambda_m - lambda_p;

      sum_0 += lamdiff * (d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 + del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }

    // add the corrections
    d_R_imh  += 0.5*dtodx*sum_0;
    vx_R_imh += 0.5*dtodx*sum_1;
    vy_R_imh += 0.5*dtodx*sum_2;
    vz_R_imh += 0.5*dtodx*sum_3;
    p_R_imh  += 0.5*dtodx*sum_4;
    #ifdef DE
    ge_R_imh += 0.5*dtodx*sum_5;
    #endif

    // apply minimum constraints
    d_R_imh = fmax(d_R_imh, (Real) TINY_NUMBER);
    d_L_iph = fmax(d_L_iph, (Real) TINY_NUMBER);
    p_R_imh = fmax(p_R_imh, (Real) TINY_NUMBER);
    p_L_iph = fmax(p_L_iph, (Real) TINY_NUMBER);

    // Step 8 - Convert the left and right states in the primitive to the conserved variables
    // send final values back from kernel
    // bounds_R refers to the right side of the i-1/2 interface
    if (dir == 0) id = xid-1 + yid*nx + zid*nx*ny;
    if (dir == 1) id = xid + (yid-1)*nx + zid*nx*ny;
    if (dir == 2) id = xid + yid*nx + (zid-1)*nx*ny;
    dev_bounds_R[            id] = d_R_imh;
    dev_bounds_R[o1*n_cells + id] = d_R_imh*vx_R_imh;
    dev_bounds_R[o2*n_cells + id] = d_R_imh*vy_R_imh;
    dev_bounds_R[o3*n_cells + id] = d_R_imh*vz_R_imh;
    dev_bounds_R[4*n_cells + id] = (p_R_imh/(gamma-1.0)) + 0.5*d_R_imh*(vx_R_imh*vx_R_imh + vy_R_imh*vy_R_imh + vz_R_imh*vz_R_imh);    
    #ifdef DE
    dev_bounds_R[5*n_cells + id] = d_R_imh*ge_R_imh;
    #endif
    // bounds_L refers to the left side of the i+1/2 interface
    id = xid + yid*nx + zid*nx*ny;
    dev_bounds_L[            id] = d_L_iph;
    dev_bounds_L[o1*n_cells + id] = d_L_iph*vx_L_iph;
    dev_bounds_L[o2*n_cells + id] = d_L_iph*vy_L_iph;
    dev_bounds_L[o3*n_cells + id] = d_L_iph*vz_L_iph;
    dev_bounds_L[4*n_cells + id] = (p_L_iph/(gamma-1.0)) + 0.5*d_L_iph*(vx_L_iph*vx_L_iph + vy_L_iph*vy_L_iph + vz_L_iph*vz_L_iph);
    #ifdef DE
    dev_bounds_L[5*n_cells + id] = d_L_iph*ge_L_iph;
    #endif

  }
}
    


#endif //PLMC
#endif //CUDA
