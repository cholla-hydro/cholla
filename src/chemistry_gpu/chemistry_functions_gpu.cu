
#include <hip/hip_runtime.h>
#ifdef CHEMISTRY_GPU

  #include "../global/global_cuda.h"
  #include "../grid/grid_enum.h"
  #include "../hydro/hydro_cuda.h"
  #include "../io/io.h"
  #include "chemistry_gpu.h"
  #include "rates.cuh"
  #include "rates_Katz95.cuh"

  #define eV_to_K 1.160451812e4
  #define K_to_eV 8.617333263e-5
  #define n_min   1e-20
  #define tiny    1e-20

  #define TPB_CHEM 256

void Chem_GPU::Allocate_Array_GPU_float(float **array_dev, int size)
{
  GPU_Error_Check(cudaMalloc((void **)array_dev, size * sizeof(float)));
}

void Chem_GPU::Copy_Float_Array_to_Device(int size, float *array_h, float *array_d)
{
  GPU_Error_Check(cudaMemcpy(array_d, array_h, size * sizeof(float), cudaMemcpyHostToDevice));
  cudaDeviceSynchronize();
}

void Chem_GPU::Free_Array_GPU_float(float *array_dev) { GPU_Error_Check(cudaFree(array_dev)); }

void Chem_GPU::Allocate_Array_GPU_Real(Real **array_dev, int size)
{
  GPU_Error_Check(cudaMalloc((void **)array_dev, size * sizeof(Real)));
  GPU_Error_Check();
}

void Chem_GPU::Copy_Real_Array_to_Device(int size, Real *array_h, Real *array_d)
{
  GPU_Error_Check(cudaMemcpy(array_d, array_h, size * sizeof(Real), cudaMemcpyHostToDevice));
  cudaDeviceSynchronize();
}

void Chem_GPU::Free_Array_GPU_Real(Real *array_dev)
{
  GPU_Error_Check(cudaFree(array_dev));
  GPU_Error_Check();
}

class Thermal_State
{
 public:
  Real U;
  Real d;
  Real d_HI;
  Real d_HII;
  Real d_HeI;
  Real d_HeII;
  Real d_HeIII;
  Real d_e;

  // Constructor
  __host__ __device__ Thermal_State(Real U_0 = 1, Real d_0 = 1, Real d_HI_0 = 1, Real d_HII_0 = 0, Real d_HeI_0 = 1,
                                    Real d_HeII_0 = 0, Real d_HeIII_0 = 1, Real d_e_0 = 0)
      : U(U_0), d(d_0), d_HI(d_HI_0), d_HII(d_HII_0), d_HeI(d_HeI_0), d_HeII(d_HeII_0), d_HeIII(d_HeIII_0), d_e(d_e_0)
  {
  }

  __host__ __device__ Real get_MMW()
  {
    // Real m_tot = d_HI + d_HII + d_HeI + d_HeII + d_HeIII;
    Real n_tot = d_HI + d_HII + 0.25 * (d_HeI + d_HeII + d_HeIII) + d_e;
    return d / n_tot;
    // return m_tot / n_tot;
  }

  __host__ __device__ Real get_temperature(Real gamma)
  {
    Real mu, temp;
    mu   = get_MMW();
    temp = (gamma - 1) * mu * U * MP / KB * 1e10;
    return temp;
  }

  __host__ __device__ Real compute_U(Real temp, Real gamma)
  {
    Real mu, U_local;
    mu      = get_MMW();
    U_local = temp / (gamma - 1) / mu / MP * KB / 1e10;
    return U_local;
  }
};

__device__ void get_temperature_indx(Real T, Chemistry_Header &Chem_H, int &temp_indx, Real &delta_T, Real temp_old,
                                     bool print)
{
  Real logT, logT_start, d_logT, logT_l, logT_r;
  logT       = log(0.5 * (T + temp_old));
  logT_start = log(Chem_H.Temp_start);
  logT       = fmax(logT_start, logT);
  logT       = fmin(log(Chem_H.Temp_end), logT);
  d_logT     = (log(Chem_H.Temp_end) - logT_start) / (Chem_H.N_Temp_bins - 1);
  temp_indx  = (int)floor((logT - logT_start) / d_logT);
  temp_indx  = max(0, temp_indx);
  temp_indx  = min(Chem_H.N_Temp_bins - 2, temp_indx);
  logT_l     = logT_start + temp_indx * d_logT;
  logT_r     = logT_start + (temp_indx + 1) * d_logT;
  delta_T    = (logT - logT_l) / (logT_r - logT_l);
  // if (print) printf(" logT_start: %f  logT_end: %f  d_logT: %f   \n",
  // logT_start, log( Chem_H.Temp_end ), d_logT ); if (print) printf(" logT: %f
  // logT_l: %f  logT_r: %f   \n", logT, logT_l, logT_r );
}

__device__ Real interpolate_rate(Real *rate_table, int indx, Real delta)
{
  Real rate_val;
  rate_val = rate_table[indx];
  rate_val = rate_val + delta * (rate_table[indx + 1] - rate_val);
  return rate_val;
}

__device__ Real Get_Cooling_Rates(Thermal_State &TS, Chemistry_Header &Chem_H, Real dens_number_conv, Real current_z,
                                  Real temp_prev, float photo_h_HI, float photo_h_HeI, float photo_h_HeII, bool print)
{
  int temp_indx;
  Real temp, delta_T, U_dot;
  temp = TS.get_temperature(Chem_H.gamma);
  get_temperature_indx(temp, Chem_H, temp_indx, delta_T, temp_prev, print);
  if (print) printf("mu: %f  temp: %f  temp_indx: %d  delta_T: %f  \n", TS.get_MMW(), temp, temp_indx, delta_T);
  U_dot = 0.0;

  // Collisional excitation cooling
  Real cool_ceHI, cool_ceHeI, cool_ceHeII;
  cool_ceHI = interpolate_rate(Chem_H.cool_ceHI_d, temp_indx, delta_T) * TS.d_HI * TS.d_e;
  cool_ceHeI =
      interpolate_rate(Chem_H.cool_ceHeI_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e * TS.d_e * dens_number_conv / 4.0;
  cool_ceHeII = interpolate_rate(Chem_H.cool_ceHeII_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  U_dot -= cool_ceHI + cool_ceHeI + cool_ceHeII;

  // Collisional excitation cooling
  Real cool_ciHI, cool_ciHeI, cool_ciHeII, cool_ciHeIS;
  cool_ciHI   = interpolate_rate(Chem_H.cool_ciHI_d, temp_indx, delta_T) * TS.d_HI * TS.d_e;
  cool_ciHeI  = interpolate_rate(Chem_H.cool_ciHeI_d, temp_indx, delta_T) * TS.d_HeI * TS.d_e / 4.0;
  cool_ciHeII = interpolate_rate(Chem_H.cool_ciHeII_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  cool_ciHeIS =
      interpolate_rate(Chem_H.cool_ciHeIS_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e * TS.d_e * dens_number_conv / 4.0;
  U_dot -= cool_ciHI + cool_ciHeI + cool_ciHeII + cool_ciHeIS;

  // Recombination cooling
  Real cool_reHII, cool_reHeII1, cool_reHeII2, cool_reHeIII;
  cool_reHII   = interpolate_rate(Chem_H.cool_reHII_d, temp_indx, delta_T) * TS.d_HII * TS.d_e;
  cool_reHeII1 = interpolate_rate(Chem_H.cool_reHeII_1_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  cool_reHeII2 = interpolate_rate(Chem_H.cool_reHeII_2_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  cool_reHeIII = interpolate_rate(Chem_H.cool_reHeIII_d, temp_indx, delta_T) * TS.d_HeIII * TS.d_e / 4.0;
  U_dot -= cool_reHII + cool_reHeII1 + cool_reHeII2 + cool_reHeIII;

  // Bremsstrahlung cooling
  Real cool_brem;
  cool_brem =
      interpolate_rate(Chem_H.cool_brem_d, temp_indx, delta_T) * (TS.d_HII + TS.d_HeII / 4.0 + TS.d_HeIII) * TS.d_e;
  U_dot -= cool_brem;

  // Compton cooling or heating
  Real cool_compton, temp_cmb;
  temp_cmb     = 2.73 * (1.0 + current_z);
  cool_compton = Chem_H.cool_compton * pow(1.0 + current_z, 4) * (temp - temp_cmb) * TS.d_e / dens_number_conv;
  U_dot -= cool_compton;

  // Phothoheating
  Real photo_heat;
  photo_heat = (photo_h_HI * TS.d_HI + 0.25 * (photo_h_HeI * TS.d_HeI + photo_h_HeII * TS.d_HeII)) / dens_number_conv;
  U_dot += photo_heat;

  if (temp <= 1.01 * Chem_H.Temp_start && fabs(U_dot) < 0) U_dot = tiny;
  if (fabs(U_dot) < tiny) U_dot = tiny;

  if (print) printf("HI: %e  \n", TS.d_HI);
  if (print) printf("HII: %e  \n", TS.d_HII);
  if (print) printf("HeI: %e  \n", TS.d_HeI);
  if (print) printf("HeII: %e  \n", TS.d_HeII);
  if (print) printf("HeIII: %e  \n", TS.d_HeIII);
  if (print) printf("de: %e  \n", TS.d_e);
  if (print) printf("Cooling ceHI: %e  \n", cool_ceHI);
  if (print) printf("Cooling ceHeI: %e   \n", cool_ceHeI);
  if (print) printf("Cooling ceHeII: %e   \n", cool_ceHeII);
  if (print) printf("Cooling ciHI: %e  \n", cool_ciHI);
  if (print) printf("Cooling ciHeI: %e  \n", cool_ciHeI);
  if (print) printf("Cooling ciHeII: %e  \n", cool_ciHeII);
  if (print) printf("Cooling ciHeIS: %e  \n", cool_ciHeIS);
  if (print) printf("Cooling reHII: %e  \n", cool_reHII);
  if (print) printf("Cooling reHeII1: %e  \n", cool_reHeII1);
  if (print) printf("Cooling reHeII2: %e  \n", cool_reHeII2);
  if (print) printf("Cooling reHeIII: %e  \n", cool_reHeIII);
  if (print) printf("Cooling brem: %e  \n", cool_brem);
  if (print) printf("Cooling piHI: %e   rate: %e \n", photo_h_HI, photo_h_HI * TS.d_HI / dens_number_conv);
  if (print) printf("Cooling piHeI: %e  rate: %e \n", photo_h_HeI, photo_h_HeI * TS.d_HeI / dens_number_conv * 0.25);
  if (print) printf("Cooling piHeII: %e rate: %e \n", photo_h_HeII, photo_h_HeII * TS.d_HeII / dens_number_conv * 0.25);
  if (print) printf("Cooling DOM: %e  \n", dens_number_conv);
  if (print) printf("Cooling compton: %e  \n", cool_compton);
  if (print) printf("Cooling U_dot: %e  \n", U_dot);

  return U_dot;
}

__device__ void Get_Reaction_Rates(Thermal_State &TS, Chemistry_Header &Chem_H, Real &k_coll_i_HI, Real &k_coll_i_HeI,
                                   Real &k_coll_i_HeII, Real &k_coll_i_HI_HI, Real &k_coll_i_HI_HeI, Real &k_recomb_HII,
                                   Real &k_recomb_HeII, Real &k_recomb_HeIII, bool print)
{
  int temp_indx;
  Real temp, delta_T;
  temp = TS.get_temperature(Chem_H.gamma);
  get_temperature_indx(temp, Chem_H, temp_indx, delta_T, temp, print);

  k_coll_i_HI   = interpolate_rate(Chem_H.k_coll_i_HI_d, temp_indx, delta_T);
  k_coll_i_HeI  = interpolate_rate(Chem_H.k_coll_i_HeI_d, temp_indx, delta_T);
  k_coll_i_HeII = interpolate_rate(Chem_H.k_coll_i_HeII_d, temp_indx, delta_T);

  k_coll_i_HI_HI  = interpolate_rate(Chem_H.k_coll_i_HI_HI_d, temp_indx, delta_T);
  k_coll_i_HI_HeI = interpolate_rate(Chem_H.k_coll_i_HI_HeI_d, temp_indx, delta_T);

  k_recomb_HII   = interpolate_rate(Chem_H.k_recomb_HII_d, temp_indx, delta_T);
  k_recomb_HeII  = interpolate_rate(Chem_H.k_recomb_HeII_d, temp_indx, delta_T);
  k_recomb_HeIII = interpolate_rate(Chem_H.k_recomb_HeIII_d, temp_indx, delta_T);

  if (print) printf("logT: %f   temp_indx: %d\n", log(temp), temp_indx);
  if (print) printf("k_coll_i_HI: %e \n", k_coll_i_HI);
  if (print) printf("k_coll_i_HeI: %e \n", k_coll_i_HeI);
  if (print) printf("k_coll_i_HeII: %e \n", k_coll_i_HeII);
  if (print) printf("k_coll_i_HI_HI: %e \n", k_coll_i_HI_HI);
  if (print) printf("k_coll_i_HI_HeI: %e \n", k_coll_i_HI_HeI);
  if (print) printf("k_recomb_HII: %e \n", k_recomb_HII);
  if (print) printf("k_recomb_HeII: %e \n", k_recomb_HeII);
  if (print) printf("k_recomb_HeIII: %e \n", k_recomb_HeIII);
}

__device__ int Binary_Search(int N, Real val, float *data, int indx_l, int indx_r)
{
  int n, indx;
  n    = indx_r - indx_l;
  indx = indx_l + n / 2;
  if (val >= data[N - 1]) return indx_r;
  if (val <= data[0]) return indx_l;
  if (indx_r == indx_l + 1) return indx_l;
  if (data[indx] <= val)
    indx_l = indx;
  else
    indx_r = indx;
  return Binary_Search(N, val, data, indx_l, indx_r);
}

__device__ Real linear_interpolation(Real delta_x, int indx_l, int indx_r, float *array)
{
  float v_l, v_r;
  Real v;
  v_l = array[indx_l];
  v_r = array[indx_r];
  v   = delta_x * (v_r - v_l) + v_l;
  return v;
}

__device__ void Get_Current_UVB_Rates(Real current_z, Chemistry_Header &Chem_H, float &photo_i_HI, float &photo_i_HeI,
                                      float &photo_i_HeII, float &photo_h_HI, float &photo_h_HeI, float &photo_h_HeII,
                                      bool print)
{
  if (current_z > Chem_H.uvb_rates_redshift_d[Chem_H.n_uvb_rates_samples - 1]) {
    photo_h_HI   = 0;
    photo_h_HeI  = 0;
    photo_h_HeII = 0;
    photo_i_HI   = 0;
    photo_i_HeI  = 0;
    photo_i_HeII = 0;
    return;
  }
  // Find closest value of z in rates_z such that z<=current_z
  int indx_l;
  Real z_l, z_r, delta_x;
  indx_l  = Binary_Search(Chem_H.n_uvb_rates_samples, current_z, Chem_H.uvb_rates_redshift_d, 0,
                          Chem_H.n_uvb_rates_samples - 1);
  z_l     = Chem_H.uvb_rates_redshift_d[indx_l];
  z_r     = Chem_H.uvb_rates_redshift_d[indx_l + 1];
  delta_x = (current_z - z_l) / (z_r - z_l);

  photo_i_HI   = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_ion_HI_rate_d);
  photo_i_HeI  = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_ion_HeI_rate_d);
  photo_i_HeII = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_ion_HeII_rate_d);
  photo_h_HI   = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_heat_HI_rate_d);
  photo_h_HeI  = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_heat_HeI_rate_d);
  photo_h_HeII = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_heat_HeII_rate_d);
}

__device__ Real Get_Chemistry_dt(Thermal_State &TS, Chemistry_Header &Chem_H, Real &HI_dot, Real &e_dot, Real U_dot,
                                 Real k_coll_i_HI, Real k_coll_i_HeI, Real k_coll_i_HeII, Real k_coll_i_HI_HI,
                                 Real k_coll_i_HI_HeI, Real k_recomb_HII, Real k_recomb_HeII, Real k_recomb_HeIII,
                                 float photo_i_HI, float photo_i_HeI, float photo_i_HeII, int n_iter, Real HI_dot_prev,
                                 Real e_dot_prev, Real t_chem, Real dt_hydro, bool print)
{
  Real dt, energy;
  // Rate of change of HI
  HI_dot = k_recomb_HII * TS.d_HII * TS.d_e - k_coll_i_HI * TS.d_HI * TS.d_e - k_coll_i_HI_HI * TS.d_HI * TS.d_HI -
           k_coll_i_HI_HeI * TS.d_HI * TS.d_HeI / 4.0 - photo_i_HI * TS.d_HI;

  // Rate of change of electron
  e_dot = k_coll_i_HI * TS.d_HI * TS.d_e + k_coll_i_HeI * TS.d_HeI / 4.0 * TS.d_e +
          k_coll_i_HeII * TS.d_HeII / 4.0 * TS.d_e + k_coll_i_HI_HI * TS.d_HI * TS.d_HI +
          +k_coll_i_HI_HeI * TS.d_HI * TS.d_HeI / 4.0 - k_recomb_HII * TS.d_HII * TS.d_e -
          k_recomb_HeII * TS.d_HeII / 4.0 * TS.d_e - k_recomb_HeIII * TS.d_HeIII / 4.0 * TS.d_e + photo_i_HI * TS.d_HI +
          photo_i_HeI * TS.d_HeI / 4.0 + photo_i_HeII * TS.d_HeII / 4.0;

  // Bound from below to prevent numerical errors
  if (fabs(HI_dot) < tiny) HI_dot = fmin(tiny, TS.d_HI);
  if (fabs(e_dot) < tiny) e_dot = fmin(tiny, TS.d_e);

  // If the net rate is almost perfectly balanced then set
  // it to zero (since it is zero to available precision)
  if (fmin(fabs(k_coll_i_HI * TS.d_HI * TS.d_e), fabs(k_recomb_HII * TS.d_HII * TS.d_e)) /
          fmax(fabs(HI_dot), fabs(e_dot)) >
      1e6) {
    HI_dot = tiny;
    e_dot  = tiny;
  }

  if (n_iter > 50) {
    HI_dot = fmin(fabs(HI_dot), fabs(HI_dot_prev));
    e_dot  = fmin(fabs(e_dot), fabs(e_dot_prev));
  }

  if (TS.d * Chem_H.dens_number_conv > 1e8 && U_dot > 0) {
    printf("#### Equlibrium  \n");
  }

  #ifdef TEMPERATURE_FLOOR
  if (TS.get_temperature(Chem_H.gamma) < Chem_H.temperature_floor) {
    TS.U = TS.compute_U(Chem_H.temperature_floor, Chem_H.gamma);
  }
  #endif

  energy = fmax(TS.U * TS.d, tiny);
  dt     = fmin(fabs(0.1 * TS.d_HI / HI_dot), fabs(0.1 * TS.d_e / e_dot));
  dt     = fmin(fabs(0.1 * energy / U_dot), dt);
  dt     = fmin(0.5 * dt_hydro, dt);
  dt     = fmin(dt_hydro - t_chem, dt);

  if (n_iter == Chem_H.max_iter - 1) {
    printf(
        "##### Chem_GPU: dt_hydro: %e   t_chem: %e   dens: %e   temp: %e  GE: "
        "%e  U_dot: %e   dt_HI: %e   dt_e: %e   dt_U: %e \n",
        dt_hydro, t_chem, TS.d, TS.get_temperature(Chem_H.gamma), energy, U_dot, fabs(0.1 * TS.d_HI / HI_dot),
        fabs(0.1 * TS.d_e / e_dot), fabs(0.1 * TS.U * TS.d / U_dot));
  }

  if (print) printf("HIdot: %e\n", HI_dot);
  if (print) printf("edot: %e\n", e_dot);
  if (print) printf("energy: %e\n", TS.U * TS.d);
  if (print) printf("Udot: %e\n", U_dot);
  if (print) printf("dt_hydro: %e\n", dt_hydro);
  if (print) printf("dt: %e\n", dt);

  return dt;
}

__device__ void Update_Step(Thermal_State &TS, Chemistry_Header &Chem_H, Real dt, Real U_dot, Real k_coll_i_HI,
                            Real k_coll_i_HeI, Real k_coll_i_HeII, Real k_coll_i_HI_HI, Real k_coll_i_HI_HeI,
                            Real k_recomb_HII, Real k_recomb_HeII, Real k_recomb_HeIII, float photo_i_HI,
                            float photo_i_HeI, float photo_i_HeII, Real &HI_dot_prev, Real &e_dot_prev, Real &temp_prev,
                            bool print)
{
  Real d_HI_p, d_HII_p, d_HeI_p, d_HeII_p, d_HeIII_p, d_e_p;
  Real s_coef, a_coef;

  // Update HI
  s_coef = k_recomb_HII * TS.d_HII * TS.d_e;
  a_coef = k_coll_i_HI * TS.d_e + k_coll_i_HI_HI * TS.d_HI + k_coll_i_HI_HeI * TS.d_HeI / 4.0 + photo_i_HI;
  d_HI_p = (dt * s_coef + TS.d_HI) / (1.0 + dt * a_coef);
  if (print) printf("Update HI  s_coef: %e    a_coef: %e   HIp: %e \n", s_coef, a_coef, d_HI_p);

  // Update HII
  s_coef = k_coll_i_HI * d_HI_p * TS.d_e + k_coll_i_HI_HI * d_HI_p * d_HI_p +
           k_coll_i_HI_HeI * d_HI_p * TS.d_HeI / 4.0 + photo_i_HI * d_HI_p;
  a_coef  = k_recomb_HII * TS.d_e;
  d_HII_p = (dt * s_coef + TS.d_HII) / (1.0 + dt * a_coef);
  if (print) printf("Update HII  s_coef: %e    a_coef: %e   HIIp: %e \n", s_coef, a_coef, d_HII_p);

  // Update electron
  s_coef = k_coll_i_HI_HI * d_HI_p * d_HI_p + k_coll_i_HI_HeI * d_HI_p * TS.d_HeI / 4.0 + photo_i_HI * TS.d_HI +
           photo_i_HeI * TS.d_HeI / 4.0 + photo_i_HeII * TS.d_HeII / 4.0;
  a_coef = -k_coll_i_HI * TS.d_HI + k_recomb_HII * TS.d_HII - k_coll_i_HeI * TS.d_HeI / 4.0 +
           k_recomb_HeII * TS.d_HeII / 4.0 - k_coll_i_HeII * TS.d_HeII / 4.0 + k_recomb_HeIII * TS.d_HeIII / 4.0;
  d_e_p = (dt * s_coef + TS.d_e) / (1.0 + dt * a_coef);
  if (print) printf("Update e  s_coef: %e    a_coef: %e   ep: %e \n", s_coef, a_coef, d_e_p);

  // Update HeI
  s_coef  = k_recomb_HeII * TS.d_HeII * TS.d_e;
  a_coef  = k_coll_i_HeI * TS.d_e + photo_i_HeI;
  d_HeI_p = (dt * s_coef + TS.d_HeI) / (1.0 + dt * a_coef);
  if (print) printf("Update HeI  s_coef: %e    a_coef: %e   HeIp: %e \n", s_coef, a_coef, d_HeI_p);

  // Update HeII
  s_coef   = k_coll_i_HeI * d_HeI_p * TS.d_e + k_recomb_HeIII * TS.d_HeIII * TS.d_e + photo_i_HeI * d_HeI_p;
  a_coef   = k_recomb_HeII * TS.d_e + k_coll_i_HeII * TS.d_e + photo_i_HeII;
  d_HeII_p = (dt * s_coef + TS.d_HeII) / (1.0 + dt * a_coef);
  if (print) printf("Update HeII  s_coef: %e    a_coef: %e   HeIIp: %e \n", s_coef, a_coef, d_HeII_p);

  // Update HeIII
  s_coef    = k_coll_i_HeII * d_HeII_p * TS.d_e + photo_i_HeII * d_HeII_p;
  a_coef    = k_recomb_HeIII * TS.d_e;
  d_HeIII_p = (dt * s_coef + TS.d_HeIII) / (1.0 + dt * a_coef);
  if (print) printf("Update HeIII  s_coef: %e    a_coef: %e   HeIIIp: %e \n", s_coef, a_coef, d_HeIII_p);

  // Record the temperature for the next step
  temp_prev = TS.get_temperature(Chem_H.gamma);

  HI_dot_prev = fabs(TS.d_HI - d_HI_p) / fmax(dt, tiny);
  TS.d_HI     = fmax(d_HI_p, tiny);
  TS.d_HII    = fmax(d_HII_p, tiny);
  TS.d_HeI    = fmax(d_HeI_p, tiny);
  TS.d_HeII   = fmax(d_HeII_p, tiny);
  TS.d_HeIII  = fmax(d_HeIII_p, 1e-5 * tiny);

  // Use charge conservation to determine electron fraction
  e_dot_prev = TS.d_e;
  TS.d_e     = TS.d_HII + TS.d_HeII / 4.0 + TS.d_HeIII / 2.0;
  e_dot_prev = fabs(TS.d_e - e_dot_prev) / fmax(dt, tiny);

  // Update internal energy
  TS.U += U_dot / TS.d * dt;
  #ifdef TEMPERATURE_FLOOR
  if (TS.get_temperature(Chem_H.gamma) < Chem_H.temperature_floor) {
    TS.U = TS.compute_U(Chem_H.temperature_floor, Chem_H.gamma);
  }
  #endif
  if (print) printf("Updated U: %e \n", TS.U);
}

__global__ void Update_Chemistry_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields,
                                        Real dt_hydro, Chemistry_Header Chem_H)
{
  int id, xid, yid, zid, n_cells, n_iter;
  Real d, d_inv, vx, vy, vz;
  Real GE, E_kin, dt_chem, t_chem;
  Real current_a, a3, a2;

  Real current_z, density_conv, energy_conv;
  current_z    = Chem_H.current_z;
  density_conv = Chem_H.density_conversion;
  energy_conv  = Chem_H.energy_conversion;

  Real U_dot, HI_dot, e_dot, HI_dot_prev, e_dot_prev, temp_prev;
  Real k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI, k_coll_i_HI_HeI;
  Real k_recomb_HII, k_recomb_HeII, k_recomb_HeIII;
  float photo_i_HI, photo_i_HeI, photo_i_HeII;
  float photo_h_HI, photo_h_HeI, photo_h_HeII;
  Real correct_H, correct_He;

  n_cells = nx * ny * nz;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx * ny);
  yid = (id - zid * nx * ny) / nx;
  xid = id - zid * nx * ny - yid * nx;
  bool print;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    E_kin = 0.5 * d * (vx * vx + vy * vy + vz * vz);
  #ifdef DE
    GE = dev_conserved[(n_fields - 1) * n_cells + id];
  #else
    GE = dev_conserved[4 * n_cells + id] - E_kin;
  #endif

    print = false;
    // if ( xid == n_ghost && yid == n_ghost && zid == n_ghost ) print = true;

    // Convert to cgs units
    current_a = 1 / (current_z + 1);
    a2        = current_a * current_a;
    a3        = a2 * current_a;
    d *= density_conv / a3;
    GE *= energy_conv / a2;
    dt_hydro = dt_hydro / Chem_H.time_units;

  #ifdef COSMOLOGY
    dt_hydro *= current_a * current_a / Chem_H.H0 * 1000 * KPC;
  #endif  // COSMOLOGY
    // dt_hydro = dt_hydro * current_a * current_a / Chem_H.H0 *
    // 1000 * KPC / Chem_H.time_units;
    //  delta_a = Chem_H.H0 * sqrt( Chem_H.Omega_M/current_a +
    //  Chem_H.Omega_L*pow(current_a, 2) ) / ( 1000 * KPC ) *
    //  dt_hydro * Chem_H.time_units;

    // Initialize the thermal state
    Thermal_State TS;
    TS.d       = dev_conserved[id] / a3;
    TS.d_HI    = dev_conserved[id + n_cells * grid_enum::HI_density] / a3;
    TS.d_HII   = dev_conserved[id + n_cells * grid_enum::HII_density] / a3;
    TS.d_HeI   = dev_conserved[id + n_cells * grid_enum::HeI_density] / a3;
    TS.d_HeII  = dev_conserved[id + n_cells * grid_enum::HeII_density] / a3;
    TS.d_HeIII = dev_conserved[id + n_cells * grid_enum::HeIII_density] / a3;
    TS.d_e     = dev_conserved[id + n_cells * grid_enum::e_density] / a3;
    TS.U       = GE * d_inv * 1e-10;

    // Ceiling species
    TS.d_HI    = fmax(TS.d_HI, tiny);
    TS.d_HII   = fmax(TS.d_HII, tiny);
    TS.d_HeI   = fmax(TS.d_HeI, tiny);
    TS.d_HeII  = fmax(TS.d_HeII, tiny);
    TS.d_HeIII = fmax(TS.d_HeIII, 1e-5 * tiny);
    TS.d_e     = fmax(TS.d_e, tiny);

    // Compute temperature at first iteration
    temp_prev = TS.get_temperature(Chem_H.gamma);

    // if (print){
    //   printf("current_z: %f\n", current_z );
    //   printf("density_units: %e\n", Chem_H.density_units );
    //   printf("lenght_units: %e\n", Chem_H.length_units );
    //   printf("velocity_units: %e\n", Chem_H.velocity_units );
    //   printf("time_units: %e\n", Chem_H.time_units );
    //   printf("dom: %e \n", dens_number_conv );
    //   printf("density: %e \n",         TS.d );
    //   printf("HI_density: %e \n",      TS.d_HI );
    //   printf("HII_density: %e \n",     TS.d_HII );
    //   printf("HeI_density: %e \n",     TS.d_HeI );
    //   printf("HeII_density: %e \n",    TS.d_HeII );
    //   printf("HeIII_density: %e \n",   TS.d_HeIII );
    //   printf("e_density: %e \n",       TS.d_e );
    //   printf("internal_energy: %e \n", TS.U );
    //   printf("energy: %e \n", TS.U*TS.d );
    //   printf("dt_hydro: %e \n", dt_hydro / Chem_H.time_units );
    // }

    // Get the photoheating and photoionization rates at z=current_z
    Get_Current_UVB_Rates(current_z, Chem_H, photo_i_HI, photo_i_HeI, photo_i_HeII, photo_h_HI, photo_h_HeI,
                          photo_h_HeII, print);

    HI_dot_prev = 0;
    e_dot_prev  = 0;
    n_iter      = 0;
    t_chem      = 0;
    while (t_chem < dt_hydro) {
      if (print) printf("########################################## Iter %d \n", n_iter);

      U_dot = Get_Cooling_Rates(TS, Chem_H, Chem_H.dens_number_conv, current_z, temp_prev, photo_h_HI, photo_h_HeI,
                                photo_h_HeII, print);

      Get_Reaction_Rates(TS, Chem_H, k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI, k_coll_i_HI_HeI,
                         k_recomb_HII, k_recomb_HeII, k_recomb_HeIII, print);

      dt_chem =
          Get_Chemistry_dt(TS, Chem_H, HI_dot, e_dot, U_dot, k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI,
                           k_coll_i_HI_HeI, k_recomb_HII, k_recomb_HeII, k_recomb_HeIII, photo_i_HI, photo_i_HeI,
                           photo_i_HeII, n_iter, HI_dot_prev, e_dot_prev, t_chem, dt_hydro, print);

      Update_Step(TS, Chem_H, dt_chem, U_dot, k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI, k_coll_i_HI_HeI,
                  k_recomb_HII, k_recomb_HeII, k_recomb_HeIII, photo_i_HI, photo_i_HeI, photo_i_HeII, HI_dot_prev,
                  e_dot_prev, temp_prev, print);

      t_chem += dt_chem;
      n_iter += 1;
      if (n_iter == Chem_H.max_iter) break;
    }
    if (print) printf("Chem_GPU: N Iter:  %d\n", n_iter);

    // Make consistent abundances with the H and He density
    correct_H  = Chem_H.H_fraction * TS.d / (TS.d_HI + TS.d_HII);
    correct_He = (1.0 - Chem_H.H_fraction) * TS.d / (TS.d_HeI + TS.d_HeII + TS.d_HeIII);
    TS.d_HI *= correct_H;
    TS.d_HII *= correct_H;
    TS.d_HeI *= correct_He;
    TS.d_HeII *= correct_He;
    TS.d_HeIII *= correct_He;

    // Use charge conservation to determine electron fractioan
    TS.d_e = TS.d_HII + TS.d_HeII / 4.0 + TS.d_HeIII / 2.0;

    // Write the Updated Thermal State
    dev_conserved[id + n_cells * grid_enum::HI_density]    = TS.d_HI * a3;
    dev_conserved[id + n_cells * grid_enum::HII_density]   = TS.d_HII * a3;
    dev_conserved[id + n_cells * grid_enum::HeI_density]   = TS.d_HeI * a3;
    dev_conserved[id + n_cells * grid_enum::HeII_density]  = TS.d_HeII * a3;
    dev_conserved[id + n_cells * grid_enum::HeIII_density] = TS.d_HeIII * a3;
    dev_conserved[id + n_cells * grid_enum::e_density]     = TS.d_e * a3;
    d                                                      = d / density_conv * a3;
    GE                                                     = TS.U / d_inv / energy_conv * a2 / 1e-10;
    dev_conserved[4 * n_cells + id]                        = GE + E_kin;
  #ifdef DE
    dev_conserved[(n_fields - 1) * n_cells + id] = GE;
  #endif

    if (print) printf("###########################################\n");
    if (print) printf("Updated HI:  %e\n", TS.d_HI * a3);
    if (print) printf("Updated HII:  %e\n", TS.d_HII * a3);
    if (print) printf("Updated HeI:  %e\n", TS.d_HeI * a3);
    if (print) printf("Updated HeII:  %e\n", TS.d_HeII * a3);
    if (print) printf("Updated HeIII:  %e\n", TS.d_HeIII * a3);
    if (print) printf("Updated e:  %e\n", TS.d_e * a3);
    if (print) printf("Updated GE:  %e\n", dev_conserved[(n_fields - 1) * n_cells + id]);
    if (print) printf("Updated E:   %e\n", dev_conserved[4 * n_cells + id]);
  }
}

void Do_Chemistry_Update(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt,
                         Chemistry_Header &Chem_H)
{
  float time;
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  cudaEventRecord(start, 0);

  int ngrid = (nx * ny * nz - 1) / TPB_CHEM + 1;
  dim3 dim1dGrid(ngrid, 1, 1);
  dim3 dim1dBlock(TPB_CHEM, 1, 1);
  hipLaunchKernelGGL(Update_Chemistry_kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields,
                     dt, Chem_H);

  GPU_Error_Check();
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&time, start, stop);
  Chem_H.runtime_chemistry_step = (Real)time / 1000;  // (Convert ms to secs )
}

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // Reaction and cooling rates from Grackle

  // Kelvin to eV conversion factor
  #ifndef tevk
    #define tevk 1.1605e4
  #endif
  // Comparison value
  #ifndef dhuge
    #define dhuge 1.0e30
  #endif
  // Small value
  #ifndef tiny
    #define tiny 1.0e-20
  #endif
  // Boltzmann's constant
  #ifndef kboltz
    #define kboltz 1.3806504e-16  // Boltzmann's constant [cm2gs-2K-1] or [ergK-1]
  #endif

// Calculation of k1 (HI + e --> HII + 2e)
// k1_rate
__device__ Real coll_i_HI_rate(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);

  Real k1 = exp(-32.71396786375 + 13.53655609057 * logT_ev - 5.739328757388 * pow(logT_ev, 2) +
                1.563154982022 * pow(logT_ev, 3) - 0.2877056004391 * pow(logT_ev, 4) +
                0.03482559773736999 * pow(logT_ev, 5) - 0.00263197617559 * pow(logT_ev, 6) +
                0.0001119543953861 * pow(logT_ev, 7) - 2.039149852002e-6 * pow(logT_ev, 8)) /
            units;
  if (T_ev <= 0.8) {
    k1 = fmax(tiny, k1);
  }
  return k1;
}

// Calculation of k3 (HeI + e --> HeII + 2e)
//  k3_rate
__device__ Real coll_i_HeI_rate(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);

  if (T_ev > 0.8) {
    return exp(-44.09864886561001 + 23.91596563469 * logT_ev - 10.75323019821 * pow(logT_ev, 2) +
               3.058038757198 * pow(logT_ev, 3) - 0.5685118909884001 * pow(logT_ev, 4) +
               0.06795391233790001 * pow(logT_ev, 5) - 0.005009056101857001 * pow(logT_ev, 6) +
               0.0002067236157507 * pow(logT_ev, 7) - 3.649161410833e-6 * pow(logT_ev, 8)) /
           units;
  } else {
    return tiny;
  }
}

// Calculation of k4 (HeII + e --> HeI + photon)
//  k4_rate
__device__ Real recomb_HeII_rate(Real T, Real units, bool use_case_B)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);
  // If case B recombination on.
  if (use_case_B) {
    return 1.26e-14 * pow(5.7067e5 / T, 0.75) / units;
  }

  // If case B recombination off.
  if (T_ev > 0.8) {
    return (1.54e-9 * (1.0 + 0.3 / exp(8.099328789667 / T_ev)) / (exp(40.49664394833662 / T_ev) * pow(T_ev, 1.5)) +
            3.92e-13 / pow(T_ev, 0.6353)) /
           units;
  } else {
    return 3.92e-13 / pow(T_ev, 0.6353) / units;
  }
}
// k4_rate Case A
__device__ Real recomb_HeII_rate_case_A(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);
  if (T_ev > 0.8) {
    return (1.54e-9 * (1.0 + 0.3 / exp(8.099328789667 / T_ev)) / (exp(40.49664394833662 / T_ev) * pow(T_ev, 1.5)) +
            3.92e-13 / pow(T_ev, 0.6353)) /
           units;
  } else {
    return 3.92e-13 / pow(T_ev, 0.6353) / units;
  }
}
// k4_rate Case B
__device__ Real recomb_HeII_rate_case_B(Real T, Real units)
{
  // If case B recombination on.
  return 1.26e-14 * pow(5.7067e5 / T, 0.75) / units;
}

// Calculation of k2 (HII + e --> HI + photon)
//  k2_rate
__device__ Real recomb_HII_rate(Real T, Real units, bool use_case_B)
{
  if (use_case_B) {
    if (T < 1.0e9) {
      return 4.881357e-6 * pow(T, -1.5) * pow((1.0 + 1.14813e2 * pow(T, -0.407)), -2.242) / units;
    } else {
      return tiny;
    }
  } else {
    if (T > 5500) {
      // Convert temperature to appropriate form.
      Real T_ev    = T / tevk;
      Real logT_ev = log(T_ev);

      return exp(-28.61303380689232 - 0.7241125657826851 * logT_ev - 0.02026044731984691 * pow(logT_ev, 2) -
                 0.002380861877349834 * pow(logT_ev, 3) - 0.0003212605213188796 * pow(logT_ev, 4) -
                 0.00001421502914054107 * pow(logT_ev, 5) + 4.989108920299513e-6 * pow(logT_ev, 6) +
                 5.755614137575758e-7 * pow(logT_ev, 7) - 1.856767039775261e-8 * pow(logT_ev, 8) -
                 3.071135243196595e-9 * pow(logT_ev, 9)) /
             units;
    } else {
      return recomb_HeII_rate(T, units, use_case_B);
    }
  }
}
// k2_rate Case A
__device__ Real recomb_HII_rate_case_A(Real T, Real units)
{
  if (T > 5500) {
    // Convert temperature to appropriate form.
    Real T_ev    = T / tevk;
    Real logT_ev = log(T_ev);

    return exp(-28.61303380689232 - 0.7241125657826851 * logT_ev - 0.02026044731984691 * pow(logT_ev, 2) -
               0.002380861877349834 * pow(logT_ev, 3) - 0.0003212605213188796 * pow(logT_ev, 4) -
               0.00001421502914054107 * pow(logT_ev, 5) + 4.989108920299513e-6 * pow(logT_ev, 6) +
               5.755614137575758e-7 * pow(logT_ev, 7) - 1.856767039775261e-8 * pow(logT_ev, 8) -
               3.071135243196595e-9 * pow(logT_ev, 9)) /
           units;
  } else {
    return recomb_HeII_rate_case_A(T, units);
  }
}

// k2_rate Case B
__device__ Real recomb_HII_rate_case_B(Real T, Real units)
{
  if (T < 1.0e9) {
    return 4.881357e-6 * pow(T, -1.5) * pow((1.0 + 1.14813e2 * pow(T, -0.407)), -2.242) / units;
  } else {
    return tiny;
  }
}

// Calculation of k5 (HeII + e --> HeIII + 2e)
//  k5_rate
__device__ Real coll_i_HeII_rate(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);

  Real k5;
  if (T_ev > 0.8) {
    k5 = exp(-68.71040990212001 + 43.93347632635 * logT_ev - 18.48066993568 * pow(logT_ev, 2) +
             4.701626486759002 * pow(logT_ev, 3) - 0.7692466334492 * pow(logT_ev, 4) +
             0.08113042097303 * pow(logT_ev, 5) - 0.005324020628287001 * pow(logT_ev, 6) +
             0.0001975705312221 * pow(logT_ev, 7) - 3.165581065665e-6 * pow(logT_ev, 8)) /
         units;
  } else {
    k5 = tiny;
  }
  return k5;
}

// Calculation of k6 (HeIII + e --> HeII + photon)
//  k6_rate
__device__ Real recomb_HeIII_rate(Real T, Real units, bool use_case_B)
{
  Real k6;
  // Has case B recombination setting.
  if (use_case_B) {
    if (T < 1.0e9) {
      k6 = 7.8155e-5 * pow(T, -1.5) * pow((1.0 + 2.0189e2 * pow(T, -0.407)), -2.242) / units;
    } else {
      k6 = tiny;
    }
  } else {
    k6 = 3.36e-10 / sqrt(T) / pow(T / 1.0e3, 0.2) / (1.0 + pow(T / 1.0e6, 0.7)) / units;
  }
  return k6;
}
// k6_rate Case A
__device__ Real recomb_HeIII_rate_case_A(Real T, Real units)
{
  Real k6;
  // Has case B recombination setting.
  k6 = 3.36e-10 / sqrt(T) / pow(T / 1.0e3, 0.2) / (1.0 + pow(T / 1.0e6, 0.7)) / units;
  return k6;
}
// k6_rate Case B
__device__ Real recomb_HeIII_rate_case_B(Real T, Real units)
{
  Real k6;
  // Has case B recombination setting.
  if (T < 1.0e9) {
    k6 = 7.8155e-5 * pow(T, -1.5) * pow((1.0 + 2.0189e2 * pow(T, -0.407)), -2.242) / units;
  } else {
    k6 = tiny;
  }
  return k6;
}

// Calculation of k57 (HI + HI --> HII + HI + e)
//  k57_rate
__device__ Real coll_i_HI_HI_rate(Real T, Real units)
{
  // These rate coefficients are from Lenzuni, Chernoff & Salpeter (1991).
  // k57 value based on experimental cross-sections from Gealy & van Zyl (1987).
  if (T > 3.0e3) {
    return 1.2e-17 * pow(T, 1.2) * exp(-1.578e5 / T) / units;
  } else {
    return tiny;
  }
}

// Calculation of k58 (HI + HeI --> HII + HeI + e)
//  k58_rate
__device__ Real coll_i_HI_HeI_rate(Real T, Real units)
{
  // These rate coefficients are from Lenzuni, Chernoff & Salpeter (1991).
  // k58 value based on cross-sections from van Zyl, Le & Amme (1981).
  if (T > 3.0e3) {
    return 1.75e-17 * pow(T, 1.3) * exp(-1.578e5 / T) / units;
  } else {
    return tiny;
  }
}

// Calculation of ceHI.
//  Cooling collisional excitation HI
__host__ __device__ Real cool_ceHI_rate(Real T, Real units)
{
  return 7.5e-19 * exp(-fmin(log(dhuge), 118348.0 / T)) / (1.0 + sqrt(T / 1.0e5)) / units;
}

// Calculation of ceHeI.
//  Cooling collisional ionization HeI
__host__ __device__ Real cool_ceHeI_rate(Real T, Real units)
{
  return 9.1e-27 * exp(-fmin(log(dhuge), 13179.0 / T)) * pow(T, -0.1687) / (1.0 + sqrt(T / 1.0e5)) / units;
}

// Calculation of ceHeII.
//  Cooling collisional excitation HeII
__host__ __device__ Real cool_ceHeII_rate(Real T, Real units)
{
  return 5.54e-17 * exp(-fmin(log(dhuge), 473638.0 / T)) * pow(T, -0.3970) / (1.0 + sqrt(T / 1.0e5)) / units;
}

// Calculation of ciHeIS.
//  Cooling collisional ionization HeIS
__host__ __device__ Real cool_ciHeIS_rate(Real T, Real units)
{
  return 5.01e-27 * pow(T, -0.1687) / (1.0 + sqrt(T / 1.0e5)) * exp(-fmin(log(dhuge), 55338.0 / T)) / units;
}

// Calculation of ciHI.
//  Cooling collisional ionization HI
__host__ __device__ Real cool_ciHI_rate(Real T, Real units)
{
  // Collisional ionization. Polynomial fit from Tom Abel.
  return 2.18e-11 * coll_i_HI_rate(T, 1) / units;
}

// Calculation of ciHeI.
//  Cooling collisional ionization HeI
__host__ __device__ Real cool_ciHeI_rate(Real T, Real units)
{
  // Collisional ionization. Polynomial fit from Tom Abel.
  return 3.94e-11 * coll_i_HeI_rate(T, 1) / units;
}

// Calculation of ciHeII.
//  Cooling collisional ionization HeII
__host__ __device__ Real cool_ciHeII_rate(Real T, Real units)
{
  // Collisional ionization. Polynomial fit from Tom Abel.
  return 8.72e-11 * coll_i_HeII_rate(T, 1) / units;
}

// Calculation of reHII.
//  Cooling recombination HII
__host__ __device__ Real cool_reHII_rate(Real T, Real units, bool use_case_B)
{
  Real lambdaHI = 2.0 * 157807.0 / T;
  if (use_case_B) {
    return 3.435e-30 * T * pow(lambdaHI, 1.970) / pow(1.0 + pow(lambdaHI / 2.25, 0.376), 3.720) / units;
  } else {
    return 1.778e-29 * T * pow(lambdaHI, 1.965) / pow(1.0 + pow(lambdaHI / 0.541, 0.502), 2.697) / units;
  }
}

// Calculation of reHII.
//  Cooling recombination HII Case A
__host__ __device__ Real cool_reHII_rate_case_A(Real T, Real units)
{
  Real lambdaHI = 2.0 * 157807.0 / T;
  return 1.778e-29 * T * pow(lambdaHI, 1.965) / pow(1.0 + pow(lambdaHI / 0.541, 0.502), 2.697) / units;
}

// Calculation of reHII.
//  Cooling recombination HII Case B
__host__ __device__ Real cool_reHII_rate_case_B(Real T, Real units)
{
  Real lambdaHI = 2.0 * 157807.0 / T;
  return 3.435e-30 * T * pow(lambdaHI, 1.970) / pow(1.0 + pow(lambdaHI / 2.25, 0.376), 3.720) / units;
}

// Calculation of reHII.
//  Cooling recombination HeII
__host__ __device__ Real cool_reHeII1_rate(Real T, Real units, bool use_case_B)
{
  Real lambdaHeII = 2.0 * 285335.0 / T;
  if (use_case_B) {
    return 1.26e-14 * kboltz * T * pow(lambdaHeII, 0.75) / units;
  } else {
    return 3e-14 * kboltz * T * pow(lambdaHeII, 0.654) / units;
  }
}

// Calculation of reHII.
//  Cooling recombination HeII Case A
__host__ __device__ Real cool_reHeII1_rate_case_A(Real T, Real units)
{
  Real lambdaHeII = 2.0 * 285335.0 / T;
  return 3e-14 * kboltz * T * pow(lambdaHeII, 0.654) / units;
}

// Calculation of reHII.
//  Cooling recombination HeII Case B
__host__ __device__ Real cool_reHeII1_rate_case_B(Real T, Real units)
{
  Real lambdaHeII = 2.0 * 285335.0 / T;
  return 1.26e-14 * kboltz * T * pow(lambdaHeII, 0.75) / units;
}

// Calculation of reHII2.
//  Cooling recombination HeII Dielectronic
__host__ __device__ Real cool_reHeII2_rate(Real T, Real units)
{
  // Dielectronic recombination (Cen, 1992).
  return 1.24e-13 * pow(T, -1.5) * exp(-fmin(log(dhuge), 470000.0 / T)) *
         (1.0 + 0.3 * exp(-fmin(log(dhuge), 94000.0 / T))) / units;
}

// Calculation of reHIII.
//  Cooling recombination HeIII
__host__ __device__ Real cool_reHeIII_rate(Real T, Real units, bool use_case_B)
{
  Real lambdaHeIII = 2.0 * 631515.0 / T;
  if (use_case_B) {
    return 8.0 * 3.435e-30 * T * pow(lambdaHeIII, 1.970) / pow(1.0 + pow(lambdaHeIII / 2.25, 0.376), 3.720) / units;
  } else {
    return 8.0 * 1.778e-29 * T * pow(lambdaHeIII, 1.965) / pow(1.0 + pow(lambdaHeIII / 0.541, 0.502), 2.697) / units;
  }
}

// Calculation of reHIII.
//  Cooling recombination HeIII Case A
__host__ __device__ Real cool_reHeIII_rate_case_A(Real T, Real units)
{
  Real lambdaHeIII = 2.0 * 631515.0 / T;
  return 8.0 * 1.778e-29 * T * pow(lambdaHeIII, 1.965) / pow(1.0 + pow(lambdaHeIII / 0.541, 0.502), 2.697) / units;
}

// Calculation of reHIII.
//  Cooling recombination HeIII Case B
__host__ __device__ Real cool_reHeIII_rate_case_B(Real T, Real units)
{
  Real lambdaHeIII = 2.0 * 631515.0 / T;
  return 8.0 * 3.435e-30 * T * pow(lambdaHeIII, 1.970) / pow(1.0 + pow(lambdaHeIII / 2.25, 0.376), 3.720) / units;
}
// Calculation of brem.
//  Cooling Bremsstrahlung
__host__ __device__ Real cool_brem_rate(Real T, Real units)
{
  return 1.43e-27 * sqrt(T) * (1.1 + 0.34 * exp(-pow(5.5 - log10(T), 2) / 3.0)) / units;
}

#endif
