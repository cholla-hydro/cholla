
#include <hip/hip_runtime.h>
/*! \file hydro_cuda.cu
 *  \brief Definitions of functions used in all cuda integration algorithms. */
#ifdef CUDA

  #include <float.h>
  #include <math.h>
  #include <stdio.h>

  #include <limits>

  #include "../global/global.h"
  #include "../global/global_cuda.h"
  #include "../gravity/static_grav.h"
  #include "../hydro/hydro_cuda.h"
  #include "../utils/DeviceVector.h"
  #include "../utils/cuda_utilities.h"
  #include "../utils/gpu.hpp"
  #include "../utils/hydro_utilities.h"
  #include "../utils/reduction_utilities.h"

__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int x_off, int n_ghost,
                                              Real dx, Real xbound, Real dt, Real gamma, int n_fields)
{
  int id;
  #ifdef STATIC_GRAV
  Real d, d_inv, vx;
  Real gx, d_n, d_inv_n, vx_n;
  gx = 0.0;
  #endif

  Real dtodx = dt / dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;

  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells - n_ghost) {
  #ifdef STATIC_GRAV
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
  #endif

    // update the conserved variable array
    dev_conserved[id] += dtodx * (dev_F[id - 1] - dev_F[id]);
    dev_conserved[n_cells + id] += dtodx * (dev_F[n_cells + id - 1] - dev_F[n_cells + id]);
    dev_conserved[2 * n_cells + id] += dtodx * (dev_F[2 * n_cells + id - 1] - dev_F[2 * n_cells + id]);
    dev_conserved[3 * n_cells + id] += dtodx * (dev_F[3 * n_cells + id - 1] - dev_F[3 * n_cells + id]);
    dev_conserved[4 * n_cells + id] += dtodx * (dev_F[4 * n_cells + id - 1] - dev_F[4 * n_cells + id]);
  #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      dev_conserved[(5 + i) * n_cells + id] +=
          dtodx * (dev_F[(5 + i) * n_cells + id - 1] - dev_F[(5 + i) * n_cells + id]);
    }
  #endif
  #ifdef DE
    dev_conserved[(n_fields - 1) * n_cells + id] +=
        dtodx * (dev_F[(n_fields - 1) * n_cells + id - 1] - dev_F[(n_fields - 1) * n_cells + id]);
  #endif
  #ifdef STATIC_GRAV  // add gravitational source terms, time averaged from n to
                      // n+1
    calc_g_1D(id, x_off, n_ghost, dx, xbound, &gx);
    d_n     = dev_conserved[id];
    d_inv_n = 1.0 / d_n;
    vx_n    = dev_conserved[1 * n_cells + id] * d_inv_n;
    dev_conserved[n_cells + id] += 0.5 * dt * gx * (d + d_n);
    dev_conserved[4 * n_cells + id] += 0.25 * dt * gx * (d + d_n) * (vx + vx_n);
  #endif
    if (dev_conserved[id] != dev_conserved[id]) {
      printf("%3d Thread crashed in final update. %f\n", id, dev_conserved[id]);
    }
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) *
    (gamma - 1.0); if (P < 0.0) printf("%d Negative pressure after final
    update.\n", id);
    */
  }
}

__global__ void Update_Conserved_Variables_2D(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y, int nx, int ny,
                                              int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound,
                                              Real ybound, Real dt, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy;
  Real gx, gy, d_n, d_inv_n, vx_n, vy_n;
  gx = 0.0;
  gy = 0.0;
  #endif

  Real dtodx = dt / dx;
  Real dtody = dt / dy;

  n_cells = nx * ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  id          = threadIdx.x + blockId * blockDim.x;
  yid         = id / nx;
  xid         = id - yid * nx;
  imo         = xid - 1 + yid * nx;
  jmo         = xid + (yid - 1) * nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost) {
  #ifdef STATIC_GRAV
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
  #endif
    // update the conserved variable array
    dev_conserved[id] += dtodx * (dev_F_x[imo] - dev_F_x[id]) + dtody * (dev_F_y[jmo] - dev_F_y[id]);
    dev_conserved[n_cells + id] += dtodx * (dev_F_x[n_cells + imo] - dev_F_x[n_cells + id]) +
                                   dtody * (dev_F_y[n_cells + jmo] - dev_F_y[n_cells + id]);
    dev_conserved[2 * n_cells + id] += dtodx * (dev_F_x[2 * n_cells + imo] - dev_F_x[2 * n_cells + id]) +
                                       dtody * (dev_F_y[2 * n_cells + jmo] - dev_F_y[2 * n_cells + id]);
    dev_conserved[3 * n_cells + id] += dtodx * (dev_F_x[3 * n_cells + imo] - dev_F_x[3 * n_cells + id]) +
                                       dtody * (dev_F_y[3 * n_cells + jmo] - dev_F_y[3 * n_cells + id]);
    dev_conserved[4 * n_cells + id] += dtodx * (dev_F_x[4 * n_cells + imo] - dev_F_x[4 * n_cells + id]) +
                                       dtody * (dev_F_y[4 * n_cells + jmo] - dev_F_y[4 * n_cells + id]);
  #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      dev_conserved[(5 + i) * n_cells + id] +=
          dtodx * (dev_F_x[(5 + i) * n_cells + imo] - dev_F_x[(5 + i) * n_cells + id]) +
          dtody * (dev_F_y[(5 + i) * n_cells + jmo] - dev_F_y[(5 + i) * n_cells + id]);
    }
  #endif
  #ifdef DE
    dev_conserved[(n_fields - 1) * n_cells + id] +=
        dtodx * (dev_F_x[(n_fields - 1) * n_cells + imo] - dev_F_x[(n_fields - 1) * n_cells + id]) +
        dtody * (dev_F_y[(n_fields - 1) * n_cells + jmo] - dev_F_y[(n_fields - 1) * n_cells + id]);
  #endif
  #ifdef STATIC_GRAV
    // calculate the gravitational acceleration as a function of x & y position
    calc_g_2D(xid, yid, x_off, y_off, n_ghost, dx, dy, xbound, ybound, &gx, &gy);
    // add gravitational source terms, time averaged from n to n+1
    d_n     = dev_conserved[id];
    d_inv_n = 1.0 / d_n;
    vx_n    = dev_conserved[1 * n_cells + id] * d_inv_n;
    vy_n    = dev_conserved[2 * n_cells + id] * d_inv_n;
    dev_conserved[n_cells + id] += 0.5 * dt * gx * (d + d_n);
    dev_conserved[2 * n_cells + id] += 0.5 * dt * gy * (d + d_n);
    dev_conserved[4 * n_cells + id] +=
        0.25 * dt * gx * (d + d_n) * (vx + vx_n) + 0.25 * dt * gy * (d + d_n) * (vy + vy_n);
  #endif
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id]) {
      printf("%3d %3d Thread crashed in final update. %f %f %f\n", xid, yid, dtodx * (dev_F_x[imo] - dev_F_x[id]),
             dtody * (dev_F_y[jmo] - dev_F_y[id]), dev_conserved[id]);
    }
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) *
    (gamma - 1.0); if (P < 0.0) printf("%3d %3d Negative pressure after final
    update. %f %f %f %f\n", xid, yid, dev_conserved[4*n_cells + id],
    0.5*d*vx*vx, 0.5*d*vy*vy, P);
    */
  }
}

__global__ void Update_Conserved_Variables_3D(Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry,
                                              Real *Q_Lz, Real *Q_Rz, Real *dev_F_x, Real *dev_F_y, Real *dev_F_z,
                                              int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost,
                                              Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt,
                                              Real gamma, int n_fields, Real density_floor, Real *dev_potential)
{
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;

  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy, vz;
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;
  #endif

  #ifdef DENSITY_FLOOR
  Real dens_0;
  #endif

  #ifdef GRAVITY
  Real d, d_inv, vx, vy, vz;
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  Real pot_l, pot_r;
  int id_l, id_r;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;

    #ifdef GRAVITY_5_POINTS_GRADIENT
  int id_ll, id_rr;
  Real pot_ll, pot_rr;
    #endif

  #endif  // GRAVITY

  Real dtodx = dt / dx;
  Real dtody = dt / dy;
  Real dtodz = dt / dz;
  n_cells    = nx * ny * nz;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx * ny);
  yid = (id - zid * nx * ny) / nx;
  xid = id - zid * nx * ny - yid * nx;
  imo = xid - 1 + yid * nx + zid * nx * ny;
  jmo = xid + (yid - 1) * nx + zid * nx * ny;
  kmo = xid + yid * nx + (zid - 1) * nx * ny;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
  #if defined(STATIC_GRAV) || defined(GRAVITY)
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
  #endif

    // update the conserved variable array
    dev_conserved[id] += dtodx * (dev_F_x[imo] - dev_F_x[id]) + dtody * (dev_F_y[jmo] - dev_F_y[id]) +
                         dtodz * (dev_F_z[kmo] - dev_F_z[id]);
    dev_conserved[n_cells + id] += dtodx * (dev_F_x[n_cells + imo] - dev_F_x[n_cells + id]) +
                                   dtody * (dev_F_y[n_cells + jmo] - dev_F_y[n_cells + id]) +
                                   dtodz * (dev_F_z[n_cells + kmo] - dev_F_z[n_cells + id]);
    dev_conserved[2 * n_cells + id] += dtodx * (dev_F_x[2 * n_cells + imo] - dev_F_x[2 * n_cells + id]) +
                                       dtody * (dev_F_y[2 * n_cells + jmo] - dev_F_y[2 * n_cells + id]) +
                                       dtodz * (dev_F_z[2 * n_cells + kmo] - dev_F_z[2 * n_cells + id]);
    dev_conserved[3 * n_cells + id] += dtodx * (dev_F_x[3 * n_cells + imo] - dev_F_x[3 * n_cells + id]) +
                                       dtody * (dev_F_y[3 * n_cells + jmo] - dev_F_y[3 * n_cells + id]) +
                                       dtodz * (dev_F_z[3 * n_cells + kmo] - dev_F_z[3 * n_cells + id]);
    dev_conserved[4 * n_cells + id] += dtodx * (dev_F_x[4 * n_cells + imo] - dev_F_x[4 * n_cells + id]) +
                                       dtody * (dev_F_y[4 * n_cells + jmo] - dev_F_y[4 * n_cells + id]) +
                                       dtodz * (dev_F_z[4 * n_cells + kmo] - dev_F_z[4 * n_cells + id]);
  #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      dev_conserved[(5 + i) * n_cells + id] +=
          dtodx * (dev_F_x[(5 + i) * n_cells + imo] - dev_F_x[(5 + i) * n_cells + id]) +
          dtody * (dev_F_y[(5 + i) * n_cells + jmo] - dev_F_y[(5 + i) * n_cells + id]) +
          dtodz * (dev_F_z[(5 + i) * n_cells + kmo] - dev_F_z[(5 + i) * n_cells + id]);
    #ifdef COOLING_GRACKLE
      // If the updated value is negative, then revert to the value before the
      // update
      if (dev_conserved[(5 + i) * n_cells + id] < 0) {
        dev_conserved[(5 + i) * n_cells + id] -=
            dtodx * (dev_F_x[(5 + i) * n_cells + imo] - dev_F_x[(5 + i) * n_cells + id]) +
            dtody * (dev_F_y[(5 + i) * n_cells + jmo] - dev_F_y[(5 + i) * n_cells + id]) +
            dtodz * (dev_F_z[(5 + i) * n_cells + kmo] - dev_F_z[(5 + i) * n_cells + id]);
      }
    #endif
    }
  #endif
  #ifdef DE
    dev_conserved[(n_fields - 1) * n_cells + id] +=
        dtodx * (dev_F_x[(n_fields - 1) * n_cells + imo] - dev_F_x[(n_fields - 1) * n_cells + id]) +
        dtody * (dev_F_y[(n_fields - 1) * n_cells + jmo] - dev_F_y[(n_fields - 1) * n_cells + id]) +
        dtodz * (dev_F_z[(n_fields - 1) * n_cells + kmo] - dev_F_z[(n_fields - 1) * n_cells + id]);
      // +  0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) +
      // dtodz*(vz_kmo-vz_kpo));
      // Note: this term is added in a separate kernel to avoid synchronization
      // issues
  #endif

  #ifdef DENSITY_FLOOR
    if (dev_conserved[id] < density_floor) {
      if (dev_conserved[id] > 0) {
        dens_0 = dev_conserved[id];
        // Set the density to the density floor
        dev_conserved[id] = density_floor;
        // Scale the conserved values to the new density
        dev_conserved[1 * n_cells + id] *= (density_floor / dens_0);
        dev_conserved[2 * n_cells + id] *= (density_floor / dens_0);
        dev_conserved[3 * n_cells + id] *= (density_floor / dens_0);
        dev_conserved[4 * n_cells + id] *= (density_floor / dens_0);
    #ifdef DE
        dev_conserved[(n_fields - 1) * n_cells + id] *= (density_floor / dens_0);
    #endif
      } else {
        // If the density is negative: average the density on that cell
        dens_0 = dev_conserved[id];
        Average_Cell_Single_Field(0, xid, yid, zid, nx, ny, nz, n_cells, dev_conserved);
      }
    }
  #endif  // DENSITY_FLOOR

  #ifdef STATIC_GRAV
    calc_g_3D(xid, yid, zid, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, &gx, &gy, &gz);
    d_n     = dev_conserved[id];
    d_inv_n = 1.0 / d_n;
    vx_n    = dev_conserved[1 * n_cells + id] * d_inv_n;
    vy_n    = dev_conserved[2 * n_cells + id] * d_inv_n;
    vz_n    = dev_conserved[3 * n_cells + id] * d_inv_n;
    dev_conserved[n_cells + id] += 0.5 * dt * gx * (d + d_n);
    dev_conserved[2 * n_cells + id] += 0.5 * dt * gy * (d + d_n);
    dev_conserved[3 * n_cells + id] += 0.5 * dt * gz * (d + d_n);
    dev_conserved[4 * n_cells + id] += 0.25 * dt * gx * (d + d_n) * (vx + vx_n) +
                                       0.25 * dt * gy * (d + d_n) * (vy + vy_n) +
                                       0.25 * dt * gz * (d + d_n) * (vz + vz_n);
  #endif

  #ifdef GRAVITY
    d_n     = dev_conserved[id];
    d_inv_n = 1.0 / d_n;
    vx_n    = dev_conserved[1 * n_cells + id] * d_inv_n;
    vy_n    = dev_conserved[2 * n_cells + id] * d_inv_n;
    vz_n    = dev_conserved[3 * n_cells + id] * d_inv_n;

    // Calculate the -gradient of potential
    // Get X componet of gravity field
    id_l  = (xid - 1) + (yid)*nx + (zid)*nx * ny;
    id_r  = (xid + 1) + (yid)*nx + (zid)*nx * ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll  = (xid - 2) + (yid)*nx + (zid)*nx * ny;
    id_rr  = (xid + 2) + (yid)*nx + (zid)*nx * ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gx     = -1 * (-pot_rr + 8 * pot_r - 8 * pot_l + pot_ll) / (12 * dx);
    #else
    gx = -0.5 * (pot_r - pot_l) / dx;
    #endif

    // Get Y componet of gravity field
    id_l  = (xid) + (yid - 1) * nx + (zid)*nx * ny;
    id_r  = (xid) + (yid + 1) * nx + (zid)*nx * ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll  = (xid) + (yid - 2) * nx + (zid)*nx * ny;
    id_rr  = (xid) + (yid + 2) * nx + (zid)*nx * ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gy     = -1 * (-pot_rr + 8 * pot_r - 8 * pot_l + pot_ll) / (12 * dx);
    #else
    gy = -0.5 * (pot_r - pot_l) / dy;
    #endif
    // Get Z componet of gravity field
    id_l  = (xid) + (yid)*nx + (zid - 1) * nx * ny;
    id_r  = (xid) + (yid)*nx + (zid + 1) * nx * ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll  = (xid) + (yid)*nx + (zid - 2) * nx * ny;
    id_rr  = (xid) + (yid)*nx + (zid + 2) * nx * ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gz     = -1 * (-pot_rr + 8 * pot_r - 8 * pot_l + pot_ll) / (12 * dx);
    #else
    gz = -0.5 * (pot_r - pot_l) / dz;
    #endif

    // Add gravity term to Momentum
    dev_conserved[n_cells + id] += 0.5 * dt * gx * (d + d_n);
    dev_conserved[2 * n_cells + id] += 0.5 * dt * gy * (d + d_n);
    dev_conserved[3 * n_cells + id] += 0.5 * dt * gz * (d + d_n);

    // Add gravity term to Total Energy
    // Add the work done by the gravitational force
    dev_conserved[4 * n_cells + id] +=
        0.5 * dt * (gx * (d * vx + d_n * vx_n) + gy * (d * vy + d_n * vy_n) + gz * (d * vz + d_n * vz_n));

  #endif  // GRAVITY

  #if !(defined(DENSITY_FLOOR) && defined(TEMPERATURE_FLOOR))
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id] || dev_conserved[4 * n_cells + id] < 0.0 ||
        dev_conserved[4 * n_cells + id] != dev_conserved[4 * n_cells + id]) {
      printf("%3d %3d %3d Thread crashed in final update. %e %e %e %e %e\n", xid + x_off, yid + y_off, zid + z_off,
             dev_conserved[id], dtodx * (dev_F_x[imo] - dev_F_x[id]), dtody * (dev_F_y[jmo] - dev_F_y[id]),
             dtodz * (dev_F_z[kmo] - dev_F_z[id]), dev_conserved[4 * n_cells + id]);
      Average_Cell_All_Fields(xid, yid, zid, nx, ny, nz, n_cells, n_fields, gamma, dev_conserved);
    }
  #endif  // DENSITY_FLOOR
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) *
    (gamma - 1.0); if (P < 0.0) printf("%3d %3d %3d Negative pressure after
    final update. %f %f %f %f %f\n", xid, yid, zid, dev_conserved[4*n_cells +
    id], 0.5*d*vx*vx, 0.5*d*vy*vy, 0.5*d*vz*vz, P);
    */
  }
}

__device__ __host__ Real hydroInverseCrossingTime(Real const &E, Real const &d, Real const &d_inv, Real const &vx,
                                                  Real const &vy, Real const &vz, Real const &dx, Real const &dy,
                                                  Real const &dz, Real const &gamma)
{
  // Compute pressure and sound speed
  Real P  = (E - 0.5 * d * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0);
  Real cs = sqrt(d_inv * gamma * P);

  // Find maximum inverse crossing time in the cell (i.e. minimum crossing time)
  Real cellMaxInverseDt = fmax((fabs(vx) + cs) / dx, (fabs(vy) + cs) / dy);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, (fabs(vz) + cs) / dz);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, 0.0);

  return cellMaxInverseDt;
}

__device__ __host__ Real mhdInverseCrossingTime(Real const &E, Real const &d, Real const &d_inv, Real const &vx,
                                                Real const &vy, Real const &vz, Real const &avgBx, Real const &avgBy,
                                                Real const &avgBz, Real const &dx, Real const &dy, Real const &dz,
                                                Real const &gamma)
{
  // Compute the gas pressure and fast magnetosonic speed
  Real gasP = mhd::utils::computeGasPressure(E, d, vx * d, vy * d, vz * d, avgBx, avgBy, avgBz, gamma);
  Real cf   = mhd::utils::fastMagnetosonicSpeed(d, gasP, avgBx, avgBy, avgBz, gamma);

  // Find maximum inverse crossing time in the cell (i.e. minimum crossing time)
  Real cellMaxInverseDt = fmax((fabs(vx) + cf) / dx, (fabs(vy) + cf) / dy);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, (fabs(vz) + cf) / dz);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, 0.0);

  return cellMaxInverseDt;
}

__global__ void Calc_dt_1D(Real *dev_conserved, Real *dev_dti, Real gamma, int n_ghost, int nx, Real dx)
{
  Real max_dti = -DBL_MAX;

  Real d, d_inv, vx, vy, vz, P, cs;
  int n_cells = nx;

  // Grid stride loop to perform as much of the reduction as possible. The
  // fact that `id` has type `size_t` is important. I'm not totally sure why
  // but setting it to int results in some kind of silent over/underflow issue
  // even though we're not hitting those kinds of numbers. Setting it to type
  // uint or size_t fixes them
  for (size_t id = threadIdx.x + blockIdx.x * blockDim.x; id < n_cells; id += blockDim.x * gridDim.x) {
    // threads corresponding to real cells do the calculation
    if (id > n_ghost - 1 && id < n_cells - n_ghost) {
      // start timestep calculation here
      // every thread collects the conserved variables it needs from global
      // memory
      d     = dev_conserved[id];
      d_inv = 1.0 / d;
      vx    = dev_conserved[1 * n_cells + id] * d_inv;
      vy    = dev_conserved[2 * n_cells + id] * d_inv;
      vz    = dev_conserved[3 * n_cells + id] * d_inv;
      P     = (dev_conserved[4 * n_cells + id] - 0.5 * d * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0);
      P     = fmax(P, (Real)TINY_NUMBER);
      // find the max wavespeed in that cell, use it to calculate the inverse
      // timestep
      cs      = sqrt(d_inv * gamma * P);
      max_dti = fmax(max_dti, (fabs(vx) + cs) / dx);
    }
  }

  // do the grid wide reduction (find the max inverse timestep in the grid)
  reduction_utilities::gridReduceMax(max_dti, dev_dti);
}

__global__ void Calc_dt_2D(Real *dev_conserved, Real *dev_dti, Real gamma, int n_ghost, int nx, int ny, Real dx,
                           Real dy)
{
  Real max_dti = -DBL_MAX;

  Real d, d_inv, vx, vy, vz, P, cs;
  int xid, yid, n_cells;
  n_cells = nx * ny;

  // Grid stride loop to perform as much of the reduction as possible. The
  // fact that `id` has type `size_t` is important. I'm not totally sure why
  // but setting it to int results in some kind of silent over/underflow issue
  // even though we're not hitting those kinds of numbers. Setting it to type
  // uint or size_t fixes them
  for (size_t id = threadIdx.x + blockIdx.x * blockDim.x; id < n_cells; id += blockDim.x * gridDim.x) {
    // get a global thread ID
    yid = id / nx;
    xid = id - yid * nx;

    // threads corresponding to real cells do the calculation
    if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost) {
      // every thread collects the conserved variables it needs from global
      // memory
      d     = dev_conserved[id];
      d_inv = 1.0 / d;
      vx    = dev_conserved[1 * n_cells + id] * d_inv;
      vy    = dev_conserved[2 * n_cells + id] * d_inv;
      vz    = dev_conserved[3 * n_cells + id] * d_inv;
      P     = (dev_conserved[4 * n_cells + id] - 0.5 * d * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0);
      P     = fmax(P, (Real)1.0e-20);
      // find the max wavespeed in that cell, use it to calculate the inverse
      // timestep
      cs      = sqrt(d_inv * gamma * P);
      max_dti = fmax(max_dti, fmax((fabs(vx) + cs) / dx, (fabs(vy) + cs) / dy));
    }
  }

  // do the grid wide reduction (find the max inverse timestep in the grid)
  reduction_utilities::gridReduceMax(max_dti, dev_dti);
}

__global__ void Calc_dt_3D(Real *dev_conserved, Real *dev_dti, Real gamma, int n_ghost, int n_fields, int nx, int ny,
                           int nz, Real dx, Real dy, Real dz)
{
  Real max_dti = -DBL_MAX;

  Real d, d_inv, vx, vy, vz, E;
  int xid, yid, zid, n_cells;

  n_cells = nx * ny * nz;

  // Grid stride loop to perform as much of the reduction as possible. The
  // fact that `id` has type `size_t` is important. I'm not totally sure why
  // but setting it to int results in some kind of silent over/underflow issue
  // even though we're not hitting those kinds of numbers. Setting it to type
  // uint or size_t fixes them
  for (size_t id = threadIdx.x + blockIdx.x * blockDim.x; id < n_cells; id += blockDim.x * gridDim.x) {
    // get a global thread ID
    cuda_utilities::compute3DIndices(id, nx, ny, xid, yid, zid);

    // threads corresponding to real cells do the calculation
    if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
        zid < nz - n_ghost) {
      // every thread collects the conserved variables it needs from global
      // memory
      d     = dev_conserved[id];
      d_inv = 1.0 / d;
      vx    = dev_conserved[1 * n_cells + id] * d_inv;
      vy    = dev_conserved[2 * n_cells + id] * d_inv;
      vz    = dev_conserved[3 * n_cells + id] * d_inv;
      E     = dev_conserved[4 * n_cells + id];

  // Compute the maximum inverse crossing time in the cell
  #ifdef MHD
      // Compute the cell centered magnetic field using a straight average of
      // the faces
      auto const [avgBx, avgBy, avgBz] =
          mhd::utils::cellCenteredMagneticFields(dev_conserved, id, xid, yid, zid, n_cells, nx, ny);
      max_dti = fmax(max_dti, mhdInverseCrossingTime(E, d, d_inv, vx, vy, vz, avgBx, avgBy, avgBz, dx, dy, dz, gamma));
  #else   // not MHD
      max_dti = fmax(max_dti, hydroInverseCrossingTime(E, d, d_inv, vx, vy, vz, dx, dy, dz, gamma));
  #endif  // MHD
    }
  }

  // do the grid wide reduction (find the max inverse timestep in the grid)
  reduction_utilities::gridReduceMax(max_dti, dev_dti);
}

Real Calc_dt_GPU(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dx, Real dy, Real dz,
                 Real gamma)
{
  // Allocate the device memory
  cuda_utilities::DeviceVector<Real> static dev_dti(1);

  // Set the device side inverse time step to the smallest possible double so
  // that the reduction isn't using the maximum value of the previous iteration
  dev_dti.assign(std::numeric_limits<double>::lowest());

  // compute dt and store in dev_dti
  if (nx > 1 && ny == 1 && nz == 1)  // 1D
  {
    // set launch parameters for GPU kernels.
    cuda_utilities::AutomaticLaunchParams static const launchParams(Calc_dt_1D);
    hipLaunchKernelGGL(Calc_dt_1D, launchParams.numBlocks, launchParams.threadsPerBlock, 0, 0, dev_conserved,
                       dev_dti.data(), gamma, n_ghost, nx, dx);
  } else if (nx > 1 && ny > 1 && nz == 1)  // 2D
  {
    // set launch parameters for GPU kernels.
    cuda_utilities::AutomaticLaunchParams static const launchParams(Calc_dt_2D);
    hipLaunchKernelGGL(Calc_dt_2D, launchParams.numBlocks, launchParams.threadsPerBlock, 0, 0, dev_conserved,
                       dev_dti.data(), gamma, n_ghost, nx, ny, dx, dy);
  } else if (nx > 1 && ny > 1 && nz > 1)  // 3D
  {
    // set launch parameters for GPU kernels.
    cuda_utilities::AutomaticLaunchParams static const launchParams(Calc_dt_3D);
    hipLaunchKernelGGL(Calc_dt_3D, launchParams.numBlocks, launchParams.threadsPerBlock, 0, 0, dev_conserved,
                       dev_dti.data(), gamma, n_ghost, n_fields, nx, ny, nz, dx, dy, dz);
  }
  CudaCheckError();

  // Note: dev_dti[0] is DeviceVector syntactic sugar for returning a value via
  // cudaMemcpy
  return dev_dti[0];
}

  #ifdef VELOCITY_CEILING

__global__ void Velocity_Ceiling_Kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields,
                                        Real gamma, Real V_ceiling, int *counter)
{
  const int id      = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_cells = nx * ny * nz;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(id, nx, ny, xid, yid, zid);
  const bool real_cell = (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost &&
                          zid > n_ghost - 1 && zid < nz - n_ghost);
  if (!real_cell) return;

  const Real d            = dev_conserved[id];
  const Real max_momentum = d * V_ceiling;

  const Real d_inv = 1.0 / d;
  /*
  const Real vx    = dev_conserved[1 * n_cells + id] * d_inv;
  const Real vy    = dev_conserved[2 * n_cells + id] * d_inv;
  const Real vz    = dev_conserved[3 * n_cells + id] * d_inv;
  const Real E     = dev_conserved[4 * n_cells + id];
  */
  for (int momentum_index = 1; momentum_index <= 3; momentum_index++) {
    // Reduce momentum if velocity is too large

    const Real momentum = dev_conserved[momentum_index * n_cells + id];
    if (abs(momentum) > max_momentum) {
      const Real new_momentum = max_momentum * momentum / abs(momentum);
      const Real diff_energy  = 0.5 * d_inv * ((momentum * momentum) - (max_momentum * max_momentum));
      // Write in the new momentum
      dev_conserved[momentum_index * n_cells + id] = new_momentum;
    // Thermalize the energy
    #ifdef DE
      dev_conserved[(n_fields - 1) * n_cells + id] += diff_energy;
    #endif  // DE
      atomicAdd(counter, 1);
    }
  }
}

void Velocity_Ceiling(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real gamma,
                      Real V_ceiling)
{
  int n_cells = nx * ny * nz;
  int ngrid   = (n_cells + TPB - 1) / TPB;
  dim3 dim1dGrid(ngrid, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);

  cuda_utilities::DeviceVector<int> counter(1, true);
  int *dev_counter = counter.data();

  if (nx > 1 && ny > 1 && nz > 1) {  // 3D
    hipLaunchKernelGGL(Velocity_Ceiling_Kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost,
                       n_fields, gamma, V_ceiling, dev_counter);
  }
  int host_counter = counter[0];
  if (host_counter > 0) {
    printf("HYDRO WARNING: Velocity Ceiling applied to num_cells: %d \n", host_counter);
  }
}

  #endif

  #ifdef TEMPERATURE_CEILING

__global__ void Temperature_Ceiling_Kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields,
                                           Real gamma, Real T_ceiling, int *counter)
{
  const int id      = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_cells = nx * ny * nz;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(id, nx, ny, xid, yid, zid);
  const bool real_cell = (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost &&
                          zid > n_ghost - 1 && zid < nz - n_ghost);
  if (!real_cell) return;

  const Real d     = dev_conserved[id];
  const Real d_inv = 1.0 / d;
  const Real vx    = dev_conserved[1 * n_cells + id] * d_inv;
  const Real vy    = dev_conserved[2 * n_cells + id] * d_inv;
  const Real vz    = dev_conserved[3 * n_cells + id] * d_inv;
  const Real E     = dev_conserved[4 * n_cells + id];

  const Real temperature_Kelvin =
      (gamma - 1) * (E - 0.5 * (vx * vx + vy * vy + vz * vz) * d) * ENERGY_UNIT / (d * DENSITY_UNIT / 0.6 / MP) / KB;

  if (temperature_Kelvin <= T_ceiling) return;

  const Real factor = T_ceiling / temperature_Kelvin;

  dev_conserved[4 * n_cells + id] *= factor;
    #ifdef DE
  dev_conserved[(n_fields - 1) * n_cells + id] *= factor;
    #endif  // DE
  atomicAdd(counter, 1);
}

void Temperature_Ceiling(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real gamma,
                         Real T_ceiling)
{
  int n_cells = nx * ny * nz;
  int ngrid   = (n_cells + TPB - 1) / TPB;
  dim3 dim1dGrid(ngrid, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);

  cuda_utilities::DeviceVector<int> counter(1, true);
  int *dev_counter = counter.data();

  if (nx > 1 && ny > 1 && nz > 1) {  // 3D
    hipLaunchKernelGGL(Temperature_Ceiling_Kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost,
                       n_fields, gamma, T_ceiling, dev_counter);
  }
  int host_counter = counter[0];
  if (host_counter > 0) {
    printf("HYDRO WARNING: Temperature Ceiling applied to num_cells: %d \n", host_counter);
  }
}

  #endif  // TEMPERATURE_CEILING

  #ifdef AVERAGE_SLOW_CELLS

void Average_Slow_Cells(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dx, Real dy,
                        Real dz, Real gamma, Real max_dti_slow, Real xbound, Real ybound, Real zbound, int nx_offset,
                        int ny_offset, int nz_offset)
{
  // set values for GPU kernels
  int n_cells = nx * ny * nz;
  int ngrid   = (n_cells + TPB - 1) / TPB;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB, 1, 1);

  if (nx > 1 && ny > 1 && nz > 1) {  // 3D
    hipLaunchKernelGGL(Average_Slow_Cells_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields,
                       dx, dy, dz, gamma, max_dti_slow, xbound, ybound, zbound, nx_offset, ny_offset, nz_offset);
  }
}

__global__ void Average_Slow_Cells_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dx,
                                      Real dy, Real dz, Real gamma, Real max_dti_slow, Real xbound, Real ybound,
                                      Real zbound, int nx_offset, int ny_offset, int nz_offset)
{
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, max_dti;
  Real speed, temp, P, cs;

  // get a global thread ID
  id      = threadIdx.x + blockIdx.x * blockDim.x;
  n_cells = nx * ny * nz;

  cuda_utilities::compute3DIndices(id, nx, ny, xid, yid, zid);

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    E     = dev_conserved[4 * n_cells + id];

    // Compute the maximum inverse crossing time in the cell
    max_dti = hydroInverseCrossingTime(E, d, d_inv, vx, vy, vz, dx, dy, dz, gamma);

    if (max_dti > max_dti_slow) {
      speed  = sqrt(vx * vx + vy * vy + vz * vz);
      temp   = (gamma - 1) * (E - 0.5 * (speed * speed) * d) * ENERGY_UNIT / (d * DENSITY_UNIT / 0.6 / MP) / KB;
      P      = (E - 0.5 * d * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0);
      cs     = sqrt(d_inv * gamma * P) * VELOCITY_UNIT * 1e-5;
      Real x = xbound + (nx_offset + xid - n_ghost + 0.5) * dx;
      Real y = ybound + (ny_offset + yid - n_ghost + 0.5) * dy;
      Real z = zbound + (nz_offset + zid - n_ghost + 0.5) * dz;
      // Average this cell
      kernel_printf(
          " Average Slow Cell [ %.5e %.5e %.5e ] -> dt_cell=%f    dt_min=%f, n=%.3e, "
          "T=%.3e, v=%.3e (%.3e, %.3e, %.3e), cs=%.3e\n",
          x, y, z, 1. / max_dti, 1. / max_dti_slow, dev_conserved[id] * DENSITY_UNIT / 0.6 / MP, temp,
          speed * VELOCITY_UNIT * 1e-5, vx * VELOCITY_UNIT * 1e-5, vy * VELOCITY_UNIT * 1e-5, vz * VELOCITY_UNIT * 1e-5,
          cs);
      Average_Cell_All_Fields(xid, yid, zid, nx, ny, nz, n_cells, n_fields, gamma, dev_conserved);
    }
  }
}
  #endif  // AVERAGE_SLOW_CELLS

  #ifdef DE
__global__ void Partial_Update_Advected_Internal_Energy_1D(Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, int nx,
                                                           int n_ghost, Real dx, Real dt, Real gamma, int n_fields)
{
  int id, xid, n_cells;
  int imo, ipo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo;
  Real P, E, E_kin, GE;

  Real dtodx = dt / dx;
  n_cells    = nx;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost) {
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    // PRESSURE_DE
    E     = dev_conserved[4 * n_cells + id];
    GE    = dev_conserved[(n_fields - 1) * n_cells + id];
    E_kin = 0.5 * d * (vx * vx + vy * vy + vz * vz);
    P     = hydro_utilities::Get_Pressure_From_DE(E, E - E_kin, GE, gamma);
    P     = fmax(P, (Real)TINY_NUMBER);

    imo = xid - 1;
    ipo = xid + 1;

    vx_imo = dev_conserved[1 * n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1 * n_cells + ipo] / dev_conserved[ipo];

    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields - 1) * n_cells + id] += 0.5 * P * (dtodx * (vx_imo - vx_ipo));
  }
}

__global__ void Partial_Update_Advected_Internal_Energy_2D(Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly,
                                                           Real *Q_Ry, int nx, int ny, int n_ghost, Real dx, Real dy,
                                                           Real dt, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  int imo, jmo;
  int ipo, jpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo;
  Real P, E, E_kin, GE;

  Real dtodx = dt / dx;
  Real dtody = dt / dy;
  n_cells    = nx * ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  id          = threadIdx.x + blockId * blockDim.x;
  yid         = id / nx;
  xid         = id - yid * nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost) {
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    // PRESSURE_DE
    E     = dev_conserved[4 * n_cells + id];
    GE    = dev_conserved[(n_fields - 1) * n_cells + id];
    E_kin = 0.5 * d * (vx * vx + vy * vy + vz * vz);
    P     = hydro_utilities::Get_Pressure_From_DE(E, E - E_kin, GE, gamma);
    P     = fmax(P, (Real)TINY_NUMBER);

    imo = xid - 1 + yid * nx;
    ipo = xid + 1 + yid * nx;
    jmo = xid + (yid - 1) * nx;
    jpo = xid + (yid + 1) * nx;

    vx_imo = dev_conserved[1 * n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1 * n_cells + ipo] / dev_conserved[ipo];
    vy_jmo = dev_conserved[2 * n_cells + jmo] / dev_conserved[jmo];
    vy_jpo = dev_conserved[2 * n_cells + jpo] / dev_conserved[jpo];

    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields - 1) * n_cells + id] += 0.5 * P * (dtodx * (vx_imo - vx_ipo) + dtody * (vy_jmo - vy_jpo));
  }
}

__global__ void Partial_Update_Advected_Internal_Energy_3D(Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly,
                                                           Real *Q_Ry, Real *Q_Lz, Real *Q_Rz, int nx, int ny, int nz,
                                                           int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma,
                                                           int n_fields)
{
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;
  int ipo, jpo, kpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo;
  Real P, E, E_kin, GE;
  // Real vx_L, vx_R, vy_L, vy_R, vz_L, vz_R;

  Real dtodx = dt / dx;
  Real dtody = dt / dy;
  Real dtodz = dt / dz;
  n_cells    = nx * ny * nz;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx * ny);
  yid = (id - zid * nx * ny) / nx;
  xid = id - zid * nx * ny - yid * nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    // PRESSURE_DE
    E     = dev_conserved[4 * n_cells + id];
    GE    = dev_conserved[(n_fields - 1) * n_cells + id];
    E_kin = hydro_utilities::Calc_Kinetic_Energy_From_Velocity(d, vx, vy, vz);
    #ifdef MHD
    // Add the magnetic energy
    auto [centeredBx, centeredBy, centeredBz] = mhd::utils::cellCenteredMagneticFields(
        dev_conserved, id, xid, yid, zid, n_cells, nx, ny) E_kin += mhd::utils::computeMagneticEnergy(magX, magY, magZ);
    #endif  // MHD
    P = hydro_utilities::Get_Pressure_From_DE(E, E - E_kin, GE, gamma);
    P = fmax(P, (Real)TINY_NUMBER);

    imo = xid - 1 + yid * nx + zid * nx * ny;
    jmo = xid + (yid - 1) * nx + zid * nx * ny;
    kmo = xid + yid * nx + (zid - 1) * nx * ny;

    ipo = xid + 1 + yid * nx + zid * nx * ny;
    jpo = xid + (yid + 1) * nx + zid * nx * ny;
    kpo = xid + yid * nx + (zid + 1) * nx * ny;

    vx_imo = dev_conserved[1 * n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1 * n_cells + ipo] / dev_conserved[ipo];
    vy_jmo = dev_conserved[2 * n_cells + jmo] / dev_conserved[jmo];
    vy_jpo = dev_conserved[2 * n_cells + jpo] / dev_conserved[jpo];
    vz_kmo = dev_conserved[3 * n_cells + kmo] / dev_conserved[kmo];
    vz_kpo = dev_conserved[3 * n_cells + kpo] / dev_conserved[kpo];

    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields - 1) * n_cells + id] +=
        0.5 * P * (dtodx * (vx_imo - vx_ipo) + dtody * (vy_jmo - vy_jpo) + dtodz * (vz_kmo - vz_kpo));

    // OPTION 2: Use the reconstructed velocities to compute the velocity
    // gradient
    // Use the reconstructed Velocities instead of neighbor cells centered
    // values
    // vx_R = Q_Lx[1*n_cells + id]  / Q_Lx[id];
    // vx_L = Q_Rx[1*n_cells + imo] / Q_Rx[imo];
    // vy_R = Q_Ly[2*n_cells + id]  / Q_Ly[id];
    // vy_L = Q_Ry[2*n_cells + jmo] / Q_Ry[jmo];
    // vz_R = Q_Lz[3*n_cells + id]  / Q_Lz[id];
    // vz_L = Q_Rz[3*n_cells + kmo] / Q_Rz[kmo];

    // Use the reconstructed Velocities instead of neighbor cells centered
    // values
    //  dev_conserved[(n_fields-1)*n_cells + id] +=  P * ( dtodx * ( vx_L - vx_R
    //  ) + dtody * ( vy_L - vy_R ) + dtodz * ( vz_L - vz_R ) );
  }
}

__global__ void Select_Internal_Energy_1D(Real *dev_conserved, int nx, int n_ghost, int n_fields)
{
  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo;
  n_cells = nx;

  Real eta_1 = DE_ETA_1;
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;

  imo = max(xid - 1, n_ghost);
  ipo = min(xid + 1, nx - n_ghost - 1);

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost) {
    // every thread collects the conserved variables it needs from global memory
    d          = dev_conserved[id];
    d_inv      = 1.0 / d;
    vx         = dev_conserved[1 * n_cells + id] * d_inv;
    vy         = dev_conserved[2 * n_cells + id] * d_inv;
    vz         = dev_conserved[3 * n_cells + id] * d_inv;
    E          = dev_conserved[4 * n_cells + id];
    U_advected = dev_conserved[(n_fields - 1) * n_cells + id];
    U_total    = E - 0.5 * d * (vx * vx + vy * vy + vz * vz);

    // find the max nearby total energy
    Emax = fmax(dev_conserved[4 * n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + ipo]);

    // We only use the "advected" internal energy if both:
    // - the thermal energy divided by total energy is a small fraction (smaller than eta_1)
    // - AND we aren't masking shock heating (details controlled by Emax & eta_2)
    if ((U_total / E > eta_1) or (U_total / Emax > eta_2)) {
      U = U_total;
    } else {
      U = U_advected;
    }

    // Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real)TINY_NUMBER);

    // Write Selected internal energy to the GasEnergy array ONLY
    // to avoid mixing updated and non-updated values of E
    // since the Dual Energy condition depends on the neighbor cells
    dev_conserved[(n_fields - 1) * n_cells + id] = U;
  }
}

__global__ void Select_Internal_Energy_2D(Real *dev_conserved, int nx, int ny, int n_ghost, int n_fields)
{
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo;
  n_cells = nx * ny;

  Real eta_1 = DE_ETA_1;
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  id          = threadIdx.x + blockId * blockDim.x;
  yid         = id / nx;
  xid         = id - yid * nx;

  imo = max(xid - 1, n_ghost);
  imo = imo + yid * nx;
  ipo = min(xid + 1, nx - n_ghost - 1);
  ipo = ipo + yid * nx;
  jmo = max(yid - 1, n_ghost);
  jmo = xid + jmo * nx;
  jpo = min(yid + 1, ny - n_ghost - 1);
  jpo = xid + jpo * nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost) {
    // every thread collects the conserved variables it needs from global memory
    d          = dev_conserved[id];
    d_inv      = 1.0 / d;
    vx         = dev_conserved[1 * n_cells + id] * d_inv;
    vy         = dev_conserved[2 * n_cells + id] * d_inv;
    vz         = dev_conserved[3 * n_cells + id] * d_inv;
    E          = dev_conserved[4 * n_cells + id];
    U_advected = dev_conserved[(n_fields - 1) * n_cells + id];
    U_total    = E - 0.5 * d * (vx * vx + vy * vy + vz * vz);

    // find the max nearby total energy
    Emax = fmax(dev_conserved[4 * n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + jpo]);

    // We only use the "advected" internal energy if both:
    // - the thermal energy divided by total energy is a small fraction (smaller than eta_1)
    // - AND we aren't masking shock heating (details controlled by Emax & eta_2)
    if ((U_total / E > eta_1) or (U_total / Emax > eta_2)) {
      U = U_total;
    } else {
      U = U_advected;
    }

    // Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real)TINY_NUMBER);

    // Write Selected internal energy to the GasEnergy array ONLY
    // to avoid mixing updated and non-updated values of E
    // since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields - 1) * n_cells + id] = U;
  }
}

__global__ void Select_Internal_Energy_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields)
{
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo, kmo, kpo;
  n_cells = nx * ny * nz;

  Real eta_1 = DE_ETA_1;
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx * ny);
  yid = (id - zid * nx * ny) / nx;
  xid = id - zid * nx * ny - yid * nx;

  imo = max(xid - 1, n_ghost);
  imo = imo + yid * nx + zid * nx * ny;
  ipo = min(xid + 1, nx - n_ghost - 1);
  ipo = ipo + yid * nx + zid * nx * ny;
  jmo = max(yid - 1, n_ghost);
  jmo = xid + jmo * nx + zid * nx * ny;
  jpo = min(yid + 1, ny - n_ghost - 1);
  jpo = xid + jpo * nx + zid * nx * ny;
  kmo = max(zid - 1, n_ghost);
  kmo = xid + yid * nx + kmo * nx * ny;
  kpo = min(zid + 1, nz - n_ghost - 1);
  kpo = xid + yid * nx + kpo * nx * ny;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
    // every thread collects the conserved variables it needs from global memory
    d          = dev_conserved[id];
    d_inv      = 1.0 / d;
    vx         = dev_conserved[1 * n_cells + id] * d_inv;
    vy         = dev_conserved[2 * n_cells + id] * d_inv;
    vz         = dev_conserved[3 * n_cells + id] * d_inv;
    E          = dev_conserved[4 * n_cells + id];
    U_advected = dev_conserved[(n_fields - 1) * n_cells + id];
    U_total    = E - 0.5 * d * (vx * vx + vy * vy + vz * vz);

    // find the max nearby total energy
    Emax = fmax(dev_conserved[4 * n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + jpo]);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + kmo]);
    Emax = fmax(Emax, dev_conserved[4 * n_cells + kpo]);

    // We only use the "advected" internal energy if both:
    // - the thermal energy divided by total energy is a small fraction (smaller than eta_1)
    // - AND we aren't masking shock heating (details controlled by Emax & eta_2)
    if ((U_total / E > eta_1) or (U_total / Emax > eta_2)) {
      U = U_total;
    } else {
      U = U_advected;
    }

    // Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real)TINY_NUMBER);

    // Write Selected internal energy to the GasEnergy array ONLY
    // to avoid mixing updated and non-updated values of E
    // since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields - 1) * n_cells + id] = U;
  }
}

__global__ void Sync_Energies_1D(Real *dev_conserved, int nx, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost) {
    // every thread collects the conserved variables it needs from global memory
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    U     = dev_conserved[(n_fields - 1) * n_cells + id];

    // Use the previously selected Internal Energy to update the total energy
    dev_conserved[4 * n_cells + id] = 0.5 * d * (vx * vx + vy * vy + vz * vz) + U;
  }
}

__global__ void Sync_Energies_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx * ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  id          = threadIdx.x + blockId * blockDim.x;
  yid         = id / nx;
  xid         = id - yid * nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost) {
    // every thread collects the conserved variables it needs from global memory
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    U     = dev_conserved[(n_fields - 1) * n_cells + id];

    // Use the previously selected Internal Energy to update the total energy
    dev_conserved[4 * n_cells + id] = 0.5 * d * (vx * vx + vy * vy + vz * vz) + U;
  }
}

__global__ void Sync_Energies_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real gamma, int n_fields)
{
  // Called in a separate kernel to avoid interfering with energy selection in
  // Select_Internal_Energy

  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx * ny * nz;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx * ny);
  yid = (id - zid * nx * ny) / nx;
  xid = id - zid * nx * ny - yid * nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
    // every thread collects the conserved variables it needs from global memory
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    U     = dev_conserved[(n_fields - 1) * n_cells + id];

    // Use the previously selected Internal Energy to update the total energy
    dev_conserved[4 * n_cells + id] = 0.5 * d * (vx * vx + vy * vy + vz * vz) + U;
  }
}

  #endif  // DE

  #ifdef TEMPERATURE_FLOOR
__global__ void Apply_Temperature_Floor(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields,
                                        Real U_floor)
{
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, Ekin, U;
  n_cells = nx * ny * nz;

  // get a global thread ID
  id  = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx * ny);
  yid = (id - zid * nx * ny) / nx;
  xid = id - zid * nx * ny - yid * nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    E     = dev_conserved[4 * n_cells + id];
    Ekin  = 0.5 * d * (vx * vx + vy * vy + vz * vz);

    U = (E - Ekin) / d;
    if (U < U_floor) {
      dev_conserved[4 * n_cells + id] = Ekin + d * U_floor;
    }

    #ifdef DE
    U = dev_conserved[(n_fields - 1) * n_cells + id] / d;
    if (U < U_floor) {
      dev_conserved[(n_fields - 1) * n_cells + id] = d * U_floor;
    }
    #endif
  }
}
  #endif  // TEMPERATURE_FLOOR

__device__ Real Average_Cell_Single_Field(int field_indx, int i, int j, int k, int nx, int ny, int nz, int ncells,
                                          Real *conserved)
{
  Real v_l, v_r, v_d, v_u, v_b, v_t, v_avrg;
  int id;

  id                                  = (i - 1) + (j)*nx + (k)*nx * ny;
  v_l                                 = conserved[field_indx * ncells + id];
  id                                  = (i + 1) + (j)*nx + (k)*nx * ny;
  v_r                                 = conserved[field_indx * ncells + id];
  id                                  = (i) + (j - 1) * nx + (k)*nx * ny;
  v_d                                 = conserved[field_indx * ncells + id];
  id                                  = (i) + (j + 1) * nx + (k)*nx * ny;
  v_u                                 = conserved[field_indx * ncells + id];
  id                                  = (i) + (j)*nx + (k - 1) * nx * ny;
  v_b                                 = conserved[field_indx * ncells + id];
  id                                  = (i) + (j)*nx + (k + 1) * nx * ny;
  v_t                                 = conserved[field_indx * ncells + id];
  v_avrg                              = (v_l + v_r + v_d + v_u + v_b + v_t) / 6;
  id                                  = (i) + (j)*nx + (k)*nx * ny;
  conserved[field_indx * ncells + id] = v_avrg;
  return v_avrg;
}

__device__ void Average_Cell_All_Fields(int i, int j, int k, int nx, int ny, int nz, int ncells, int n_fields,
                                        Real gamma, Real *conserved)
{
  int id = i + (j)*nx + (k)*nx * ny;

  Real d, mx, my, mz, E, P;
  d  = conserved[grid_enum::density * ncells + id];
  mx = conserved[grid_enum::momentum_x * ncells + id];
  my = conserved[grid_enum::momentum_y * ncells + id];
  mz = conserved[grid_enum::momentum_z * ncells + id];
  E  = conserved[grid_enum::Energy * ncells + id];
  P  = (E - (0.5 / d) * (mx * mx + my * my + mz * mz)) * (gamma - 1.0);

  printf("%3d %3d %3d BC: d: %e  E:%e  P:%e  vx:%e  vy:%e  vz:%e\n", i, j, k, d, E, P, mx / d, my / d, mz / d);

  int idn;
  int N = 0;
  Real d_av, vx_av, vy_av, vz_av, P_av;
  d_av = vx_av = vy_av = vz_av = P_av = 0.0;
  #ifdef SCALAR
  Real scalar[NSCALARS], scalar_av[NSCALARS];
  for (int n = 0; n < NSCALARS; n++) {
    scalar_av[n] = 0.0;
  }
  #endif

  for (int kk = k - 1; kk <= k + 1; kk++) {
    for (int jj = j - 1; jj <= j + 1; jj++) {
      for (int ii = i - 1; ii <= i + 1; ii++) {
        idn = ii + jj * nx + kk * nx * ny;
        d   = conserved[grid_enum::density * ncells + idn];
        mx  = conserved[grid_enum::momentum_x * ncells + idn];
        my  = conserved[grid_enum::momentum_y * ncells + idn];
        mz  = conserved[grid_enum::momentum_z * ncells + idn];
        P   = (conserved[grid_enum::Energy * ncells + idn] - (0.5 / d) * (mx * mx + my * my + mz * mz)) * (gamma - 1.0);
  #ifdef SCALAR
        for (int n = 0; n < NSCALARS; n++) {
          scalar[n] = conserved[grid_enum::scalar * ncells + idn];
        }
  #endif
        if (d > 0.0 && P > 0.0) {
          d_av += d;
          vx_av += mx;
          vy_av += my;
          vz_av += mz;
          P_av += P / (gamma - 1.0);
  #ifdef SCALAR
          for (int n = 0; n < NSCALARS; n++) {
            scalar_av[n] += scalar[n];
          }
  #endif
          N++;
        }
      }
    }
  }

  P_av  = P_av / N;
  vx_av = vx_av / d_av;
  vy_av = vy_av / d_av;
  vz_av = vz_av / d_av;
  #ifdef SCALAR
  for (int n = 0; n < NSCALARS; n++) {
    scalar_av[n] = scalar_av[n] / d_av;
  }
  #endif
  d_av = d_av / N;

  // replace cell values with new averaged values
  conserved[id + ncells * grid_enum::density]    = d_av;
  conserved[id + ncells * grid_enum::momentum_x] = d_av * vx_av;
  conserved[id + ncells * grid_enum::momentum_y] = d_av * vy_av;
  conserved[id + ncells * grid_enum::momentum_z] = d_av * vz_av;
  conserved[id + ncells * grid_enum::Energy] =
      P_av / (gamma - 1.0) + 0.5 * d_av * (vx_av * vx_av + vy_av * vy_av + vz_av * vz_av);
  #ifdef DE
  conserved[id + ncells * grid_enum::GasEnergy] = P_av / (gamma - 1.0);
  #endif
  #ifdef SCALAR
  for (int n = 0; n < NSCALARS; n++) {
    conserved[id + ncells * grid_enum::scalar] = d_av * scalar_av[n];
  }
  #endif

  d = d_av;
  E = P_av / (gamma - 1.0) + 0.5 * d_av * (vx_av * vx_av + vy_av * vy_av + vz_av * vz_av);
  P = P_av;

  printf("%3d %3d %3d FC: d: %e  E:%e  P:%e  vx:%e  vy:%e  vz:%e\n", i, j, k, d, E, P, vx_av, vy_av, vz_av);
}

#endif  // CUDA
