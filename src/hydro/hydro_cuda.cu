
#include <hip/hip_runtime.h>
/*! \file hydro_cuda.cu
 *  \brief Definitions of functions used in all cuda integration algorithms. */
#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include"gpu.hpp"
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"gravity_cuda.h"


__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int x_off, int n_ghost, Real dx, Real xbound, Real dt, Real gamma, int n_fields)
{
  int id;
  #ifdef STATIC_GRAV
  Real d, d_inv, vx;  
  Real gx, d_n, d_inv_n, vx_n;
  gx = 0.0;
  #endif
  
  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    #ifdef STATIC_GRAV
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    #endif
  
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F[            id-1] - dev_F[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F[  n_cells + id-1] - dev_F[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F[2*n_cells + id-1] - dev_F[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F[3*n_cells + id-1] - dev_F[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F[4*n_cells + id-1] - dev_F[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F[(5+i)*n_cells + id-1] - dev_F[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F[(n_fields-1)*n_cells + id-1] - dev_F[(n_fields-1)*n_cells + id]);
    #endif
    #ifdef STATIC_GRAV // add gravitational source terms, time averaged from n to n+1
    calc_g_1D(id, x_off, n_ghost, dx, xbound, &gx);    
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n);
    #endif    
    if (dev_conserved[id] != dev_conserved[id]) printf("%3d Thread crashed in final update. %f\n", id, dev_conserved[id]);
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%d Negative pressure after final update.\n", id);
    */
  }


}


__global__ void Update_Conserved_Variables_2D(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy;
  Real gx, gy, d_n, d_inv_n, vx_n, vy_n;
  gx = 0.0;
  gy = 0.0;
  #endif

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  imo = xid-1 + yid*nx;
  jmo = xid + (yid-1)*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    #ifdef STATIC_GRAV
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    #endif
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]) 
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]) 
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]); 
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                        +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id]);
    #endif
    #ifdef STATIC_GRAV 
    // calculate the gravitational acceleration as a function of x & y position
    calc_g_2D(xid, yid, x_off, y_off, n_ghost, dx, dy, xbound, ybound, &gx, &gy);
    // add gravitational source terms, time averaged from n to n+1                                 
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n);
    #endif
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id]) {
      printf("%3d %3d Thread crashed in final update. %f %f %f\n", xid, yid, dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dev_conserved[id]);
    }   
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0)
      printf("%3d %3d Negative pressure after final update. %f %f %f %f\n", xid, yid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, P);    
    */
  }

}



__global__ void Update_Conserved_Variables_3D(Real *dev_conserved,
                                              Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, Real *Q_Lz, Real *Q_Rz,                                              
                                              Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z,
                                              int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, 
                                              Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt,
                                              Real gamma, int n_fields, Real density_floor, Real *dev_potential )
{
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;
  
  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy, vz;
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;
  #endif
  
  #ifdef DENSITY_FLOOR
  Real dens_0;
  #endif
  
  #ifdef GRAVITY
  Real d, d_inv, vx, vy, vz;
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  Real pot_l, pot_r;
  int id_l, id_r;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;
  
  #ifdef GRAVITY_5_POINTS_GRADIENT
  int id_ll, id_rr;
  Real pot_ll, pot_rr;
  #endif
  
  #ifdef COUPLE_DELTA_E_KINETIC
  Real Ekin_0, Ekin_1;
  #endif//COUPLE_DELTA_E_KINETIC
  #endif //GRAVTY

  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  imo = xid-1 + yid*nx + zid*nx*ny;
  jmo = xid + (yid-1)*nx + zid*nx*ny;
  kmo = xid + yid*nx + (zid-1)*nx*ny;
  
  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    #if defined(STATIC_GRAV) ||  defined(GRAVITY) 
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    #endif

    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                  +  dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                  +  dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                  +  dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                  +  dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                  +  dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                    +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                    +  dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
      #ifdef COOLING_GRACKLE
      // If the updated value is negative, then revert to the value before the update
      if ( dev_conserved[(5+i)*n_cells + id] < 0 ){
        dev_conserved[(5+i)*n_cells + id] -= dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                      +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                      +  dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
      } 
      #endif
    }                              
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                  +  dtodz * (dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id]);
                                  // +  0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
                                  //Note: this term is added in a separate kernel to avoid syncronization issues
    #endif
    
    #ifdef DENSITY_FLOOR
    if ( dev_conserved[            id] < density_floor ){
      if (dev_conserved[            id] > 0){  
        dens_0 = dev_conserved[            id];
        // Set the density to the density floor
        dev_conserved[            id] = density_floor;
        // Scale the conserved values to the new density
        dev_conserved[1*n_cells + id] *= (density_floor / dens_0);
        dev_conserved[2*n_cells + id] *= (density_floor / dens_0);
        dev_conserved[3*n_cells + id] *= (density_floor / dens_0);
        dev_conserved[4*n_cells + id] *= (density_floor / dens_0);
        #ifdef DE
        dev_conserved[(n_fields-1)*n_cells + id] *= (density_floor / dens_0);
        #endif
      }
      else{
        // If the density is negative: average the density on that cell
        dens_0 = dev_conserved[            id];
        Average_Cell_Single_Field( 0, xid, yid, zid, nx, ny, nz, n_cells, dev_conserved );    
      }
    }
    #endif//DENSITY_FLOOR

    #ifdef STATIC_GRAV 
    calc_g_3D(xid, yid, zid, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, &gx, &gy, &gz);
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    vz_n =  dev_conserved[3*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[3*n_cells + id] += 0.5*dt*gz*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n)
                                  +  0.25*dt*gz*(d + d_n)*(vz + vz_n);
    #endif
    
    #ifdef GRAVITY 
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    vz_n =  dev_conserved[3*n_cells + id] * d_inv_n;
    
    #ifdef COUPLE_DELTA_E_KINETIC
    //The Kinetic Energy before adding the gravity term to the Momentum
    Ekin_0 = 0.5 * d_n * ( vx_n*vx_n + vy_n*vy_n + vz_n*vz_n );
    #endif
    
    // Calculate the -gradient of potential
    // Get X componet of gravity field
    id_l = (xid-1) + (yid)*nx + (zid)*nx*ny;
    id_r = (xid+1) + (yid)*nx + (zid)*nx*ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll = (xid-2) + (yid)*nx + (zid)*nx*ny;
    id_rr = (xid+2) + (yid)*nx + (zid)*nx*ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gx = -1 * ( -pot_rr + 8*pot_r - 8*pot_l + pot_ll) / (12*dx);
    #else
    gx = -0.5*( pot_r - pot_l ) / dx;
    #endif
    
    //Get Y componet of gravity field
    id_l = (xid) + (yid-1)*nx + (zid)*nx*ny;
    id_r = (xid) + (yid+1)*nx + (zid)*nx*ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll = (xid) + (yid-2)*nx + (zid)*nx*ny;
    id_rr = (xid) + (yid+2)*nx + (zid)*nx*ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gy = -1 * ( -pot_rr + 8*pot_r - 8*pot_l + pot_ll) / (12*dx);
    #else
    gy = -0.5*( pot_r - pot_l ) / dy;
    #endif
    //Get Z componet of gravity field
    id_l = (xid) + (yid)*nx + (zid-1)*nx*ny;
    id_r = (xid) + (yid)*nx + (zid+1)*nx*ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll = (xid) + (yid)*nx + (zid-2)*nx*ny;
    id_rr = (xid) + (yid)*nx + (zid+2)*nx*ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gz = -1 * ( -pot_rr + 8*pot_r - 8*pot_l + pot_ll) / (12*dx);
    #else
    gz = -0.5*( pot_r - pot_l ) / dz;
    #endif
    
    //Add gravity term to Momentum
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[3*n_cells + id] += 0.5*dt*gz*(d + d_n);
    
    //Add gravity term to Total Energy
    #ifdef COUPLE_GRAVITATIONAL_WORK
    //Add the work done by the gravitational force 
    dev_conserved[4*n_cells + id] += 0.5* dt * ( gx*(d*vx + d_n*vx_n) +  gy*(d*vy + d_n*vy_n) +  gz*(d*vz + d_n*vz_n) );
    #endif
    
    #ifdef COUPLE_DELTA_E_KINETIC
    //Add the the exact change in kinetic energy due to the gravity term added to the Momentum
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    vz_n =  dev_conserved[3*n_cells + id] * d_inv_n;
    Ekin_1 = 0.5 * d_n * ( vx_n*vx_n + vy_n*vy_n + vz_n*vz_n );
    dev_conserved[4*n_cells + id] += Ekin_1 - Ekin_0;
    #endif
    
    
    #endif
    
    
    #if !( defined(DENSITY_FLOOR) && defined(TEMPERATURE_FLOOR) )   
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id] || dev_conserved[4*n_cells + id] < 0.0 || dev_conserved[4*n_cells+id] != dev_conserved[4*n_cells+id]) {
      printf("%3d %3d %3d Thread crashed in final update. %e %e %e %e %e\n", xid+x_off, yid+y_off, zid+z_off, dev_conserved[id], dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dtodz*(dev_F_z[kmo]-dev_F_z[id]), dev_conserved[4*n_cells+id]);
    }
    #endif//DENSITY_FLOOR
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%3d %3d %3d Negative pressure after final update. %f %f %f %f %f\n", xid, yid, zid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, 0.5*d*vz*vz, P);
    */
  }

}





__global__ void Calc_dt_1D(Real *dev_conserved, int n_cells, int n_ghost, Real dx, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // start timestep calculation here
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) TINY_NUMBER);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = (fabs(vx)+cs)/dx;
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];


}



__global__ void Calc_dt_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real dx, Real dy, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid, xid, yid, n_cells;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) 1.0e-20);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockId] = max_dti[0];

}


__global__ void Calc_dt_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real *dti_array, Real gamma, Real max_dti_slow)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, E, P, cs;
  int id, xid, yid, zid, n_cells;
  int tid;

  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  // and a thread id within the block  
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  = dev_conserved[4*n_cells + id];
    P  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
    max_dti[tid] = fmax(max_dti[tid], (fabs(vz)+cs)/dz);
    max_dti[tid] = fmax(max_dti[tid], 0.0);
    
    #ifdef AVERAGE_SLOW_CELLS
    // If the cell delta_t is smaller than the min_delta_t, then the cell is averaged over its neighbours
    if (max_dti[tid] > max_dti_slow){
      // Average this cell
      printf(" Average Slow Cell [ %d %d %d ] -> dt_cell=%f    dt_min=%f\n", xid, yid, zid, 1./max_dti[tid],  1./max_dti_slow );
      Average_Cell_All_Fields( xid, yid, zid, nx, ny, nz, n_cells, dev_conserved );
      
      // Recompute max_dti for this cell
      d  =  dev_conserved[            id];
      d_inv = 1.0 / d;
      vx =  dev_conserved[1*n_cells + id] * d_inv;
      vy =  dev_conserved[2*n_cells + id] * d_inv;
      vz =  dev_conserved[3*n_cells + id] * d_inv;
      E  = dev_conserved[4*n_cells + id];
      P  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
      cs = sqrt(d_inv * gamma * P);
      max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
      max_dti[tid] = fmax(max_dti[tid], (fabs(vz)+cs)/dz);
      max_dti[tid] = fmax(max_dti[tid], 0.0);
    }
    
    #endif
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];

}

#ifdef DE
__global__ void Partial_Update_Advected_Internal_Energy_1D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, int nx, int n_ghost, Real dx, Real dt, Real gamma, int n_fields ){
  
  int id, xid, n_cells;
  int imo, ipo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo;
  Real  P, E, E_kin, GE;
  
  
  Real dtodx = dt/dx;
  n_cells = nx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;

  
  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = Get_Pressure_From_DE( E, E - E_kin, GE, gamma );  
    P  = fmax(P, (Real) TINY_NUMBER); 

    imo = xid-1;
    ipo = xid+1;
    
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    
    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo));
 
  }  
}


__global__ void Partial_Update_Advected_Internal_Energy_2D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, Real gamma, int n_fields ){
  
  int id, xid, yid, n_cells;
  int imo, jmo;
  int ipo, jpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo;
  Real  P, E, E_kin, GE;
  
  
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  
  
  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = Get_Pressure_From_DE( E, E - E_kin, GE, gamma );  
    P  = fmax(P, (Real) TINY_NUMBER); 

    imo = xid-1 + yid*nx;
    ipo = xid+1 + yid*nx;
    jmo = xid + (yid-1)*nx;
    jpo = xid + (yid+1)*nx;
    
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    
    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo));
 
  }  
}

__global__ void Partial_Update_Advected_Internal_Energy_3D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, Real *Q_Lz, Real *Q_Rz, int nx, int ny, int nz,  int n_ghost, Real dx, Real dy, Real dz,  Real dt, Real gamma, int n_fields ){
  
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;
  int ipo, jpo, kpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo;
  Real  P, E, E_kin, GE;
  // Real vx_L, vx_R, vy_L, vy_R, vz_L, vz_R;
  
  
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  
  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = Get_Pressure_From_DE( E, E - E_kin, GE, gamma );  
    P  = fmax(P, (Real) TINY_NUMBER); 

    imo = xid-1 + yid*nx + zid*nx*ny;
    jmo = xid + (yid-1)*nx + zid*nx*ny;
    kmo = xid + yid*nx + (zid-1)*nx*ny;
    
    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;
    
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo]; 
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo];
    
    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
 
    // OPTION 2: Use the reconstrcted velocities to compute the velocity gradient
    //Use the reconstructed Velocities instead of neighbor cells centered values 
    // vx_R = Q_Lx[1*n_cells + id]  / Q_Lx[id]; 
    // vx_L = Q_Rx[1*n_cells + imo] / Q_Rx[imo]; 
    // vy_R = Q_Ly[2*n_cells + id]  / Q_Ly[id]; 
    // vy_L = Q_Ry[2*n_cells + jmo] / Q_Ry[jmo];
    // vz_R = Q_Lz[3*n_cells + id]  / Q_Lz[id]; 
    // vz_L = Q_Rz[3*n_cells + kmo] / Q_Rz[kmo]; 
    
    //Use the reconstructed Velocities instead of neighbor cells centered values
    // dev_conserved[(n_fields-1)*n_cells + id] +=  P * ( dtodx * ( vx_L - vx_R ) + dtody * ( vy_L - vy_R ) + dtodz * ( vz_L - vz_R ) );

    
  }  
}


__global__ void Select_Internal_Energy_1D( Real *dev_conserved, int nx, int n_ghost, int n_fields ){
  
  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo;
  n_cells = nx;
  
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;
  
  imo = max(xid-1, n_ghost);
  ipo = min(xid+1, nx-n_ghost-1);


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );
    
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
  
    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;
    
    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);
    
    //Write Selected internal energy to the GasEnergy array ONLY 
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;
  
  }
}


__global__ void Select_Internal_Energy_2D( Real *dev_conserved, int nx, int ny, int n_ghost, int n_fields ){
  
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo;
  n_cells = nx*ny;
  
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  
  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );
    
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);
  
    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;
    
    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);
    
    //Write Selected internal energy to the GasEnergy array ONLY 
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;
  
  }
}


__global__ void Select_Internal_Energy_3D( Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields ){
  
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo, kmo, kpo;
  n_cells = nx*ny*nz;
  
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  
  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx + zid*nx*ny;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx + zid*nx*ny;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx + zid*nx*ny;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx + zid*nx*ny;
  kmo = max(zid-1, n_ghost);
  kmo = xid + yid*nx + kmo*nx*ny;
  kpo = min(zid+1, nz-n_ghost-1);
  kpo = xid + yid*nx + kpo*nx*ny;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );
    
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kpo]);
  
    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;
    
    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);
    
    //Write Selected internal energy to the GasEnergy array ONLY 
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;
  
  }
}

__global__ void Sync_Energies_1D(Real *dev_conserved, int nx, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previusly selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }

}


__global__ void Sync_Energies_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previusly selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }

}


__global__ void Sync_Energies_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real gamma, int n_fields)
{
  //Called in a separate kernel to avoid interfering with energy selection in Select_Internal_Energy

  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previusly selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }
}


#endif //DE

#ifdef TEMPERATURE_FLOOR
__global__ void Apply_Temperature_Floor(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields,  Real U_floor )
{
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, Ekin, U;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];    
    Ekin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    
    U = ( E - Ekin ) / d;
    if ( U < U_floor ) dev_conserved[4*n_cells + id] = Ekin + d*U_floor;
      
    #ifdef DE
    U = dev_conserved[(n_fields-1)*n_cells + id] / d ;
    if ( U < U_floor ) dev_conserved[(n_fields-1)*n_cells + id] = d*U_floor ;
    #endif
  }
}
#endif //TEMPERATURE_FLOOR


__device__ Real Average_Cell_Single_Field( int field_indx, int i, int j, int k, int nx, int ny, int nz, int ncells, Real *conserved ){
  Real v_l, v_r, v_d, v_u, v_b, v_t, v_avrg;
  int id;

  id = (i-1) + (j)*nx + (k)*nx*ny;
  v_l = conserved[ field_indx*ncells + id ];
  id = (i+1) + (j)*nx + (k)*nx*ny;
  v_r = conserved[ field_indx*ncells + id ];
  id = (i) + (j-1)*nx + (k)*nx*ny;
  v_d = conserved[ field_indx*ncells + id ];
  id = (i) + (j+1)*nx + (k)*nx*ny;
  v_u = conserved[ field_indx*ncells + id ];
  id = (i) + (j)*nx + (k-1)*nx*ny;
  v_b = conserved[ field_indx*ncells + id ];
  id = (i) + (j)*nx + (k+1)*nx*ny;
  v_t = conserved[ field_indx*ncells + id ];
  v_avrg = ( v_l + v_r + v_d + v_u + v_b + v_t ) / 6;
  id = (i) + (j)*nx + (k)*nx*ny;
  conserved[ field_indx*ncells + id ] = v_avrg;
  return v_avrg;

}

__device__ void Average_Cell_All_Fields( int i, int j, int k, int nx, int ny, int nz, int ncells, Real *conserved ){
  
  // Average Density
  Average_Cell_Single_Field( 0, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Momentum_x
  Average_Cell_Single_Field( 1, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Momentum_y
  Average_Cell_Single_Field( 2, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Momentum_z
  Average_Cell_Single_Field( 3, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Energy
  Average_Cell_Single_Field( 4, i, j, k, nx, ny, nz, ncells, conserved );
  #ifdef DE
  // Average GasEnergy
  Average_Cell_Single_Field( 5, i, j, k, nx, ny, nz, ncells, conserved );
  #endif
}


#endif //CUDA
