
#include <hip/hip_runtime.h>
/*! \file VL_1D_cuda.cu
 *  \brief Definitions of the cuda VL algorithm functions. */

#ifdef CUDA
  #ifdef VL

    #include <math.h>
    #include <stdio.h>
    #include <stdlib.h>

    #include "../global/global.h"
    #include "../global/global_cuda.h"
    #include "../hydro/hydro_cuda.h"
    #include "../integrators/VL_1D_cuda.h"
    #include "../io/io.h"
    #include "../reconstruction/pcm_cuda.h"
    #include "../reconstruction/plmc_cuda.h"
    #include "../reconstruction/plmp_cuda.h"
    #include "../reconstruction/ppmc_cuda.h"
    #include "../reconstruction/ppmp_cuda.h"
    #include "../riemann_solvers/exact_cuda.h"
    #include "../riemann_solvers/hllc_cuda.h"
    #include "../riemann_solvers/roe_cuda.h"
    #include "../utils/error_handling.h"
    #include "../utils/gpu.hpp"

__global__ void Update_Conserved_Variables_1D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F,
                                                   int n_cells, int n_ghost, Real dx, Real dt, Real gamma,
                                                   int n_fields);

void VL_Algorithm_1D_CUDA(Real *d_conserved, int nx, int x_off, int n_ghost, Real dx, Real xbound, Real dt,
                          int n_fields, int custom_grav)
{
  // Here, *dev_conserved contains the entire
  // set of conserved variables on the grid

  int n_cells             = nx;
  [[maybe_unused]] int ny = 1;
  [[maybe_unused]] int nz = 1;
  int ngrid               = (n_cells + TPB - 1) / TPB;

  // set the dimensions of the cuda grid
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  if (!memory_allocated) {
    // allocate memory on the GPU
    dev_conserved = d_conserved;
    // GPU_Error_Check( cudaMalloc((void**)&dev_conserved,
    // n_fields*n_cells*sizeof(Real)) );
    GPU_Error_Check(cudaMalloc((void **)&dev_conserved_half, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&Q_Lx, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&Q_Rx, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&F_x, n_fields * n_cells * sizeof(Real)));

    // If memory is single allocated: memory_allocated becomes true and
    // successive timesteps won't allocate memory. If the memory is not single
    // allocated: memory_allocated remains Null and memory is allocated every
    // timestep.
    memory_allocated = true;
  }

  // Step 1: Use PCM reconstruction to put conserved variables into interface
  // arrays
  hipLaunchKernelGGL(PCM_Reconstruction_1D, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, gama,
                     n_fields);
  GPU_Error_Check();

    // Step 2: Calculate first-order upwind fluxes
    #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama,
                     0, n_fields);
    #endif
    #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0,
                     n_fields);
    #endif
    #ifdef HLLC
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0,
                     n_fields);
    #endif
  GPU_Error_Check();

  // Step 3: Update the conserved variables half a timestep
  hipLaunchKernelGGL(Update_Conserved_Variables_1D_half, dimGrid, dimBlock, 0, 0, dev_conserved, dev_conserved_half,
                     F_x, n_cells, n_ghost, dx, 0.5 * dt, gama, n_fields);
  GPU_Error_Check();

    // Step 4: Construct left and right interface values using updated conserved
    // variables
    #ifdef PCM
  hipLaunchKernelGGL(PCM_Reconstruction_1D, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, n_ghost, gama,
                     n_fields);
    #endif
    #ifdef PLMC
  hipLaunchKernelGGL(PLMC_cuda, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, dx, dt, gama, 0,
                     n_fields);
    #endif
    #ifdef PLMP
  hipLaunchKernelGGL(PLMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt,
                     gama, 0, n_fields);
    #endif
    #ifdef PPMP
  hipLaunchKernelGGL(PPMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt,
                     gama, 0, n_fields);
    #endif
    #ifdef PPMC
  hipLaunchKernelGGL(PPMC_VL, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, gama, 0);
    #endif
  GPU_Error_Check();

    // Step 5: Calculate the fluxes again
    #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama,
                     0, n_fields);
    #endif
    #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0,
                     n_fields);
    #endif
    #ifdef HLLC
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0,
                     n_fields);
    #endif
  GPU_Error_Check();

    #ifdef DE
  // Compute the divergence of velocity before updating the conserved array,
  // this solves synchronization issues when adding this term on
  // Update_Conserved_Variables
  hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_1D, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx,
                     n_ghost, dx, dt, gama, n_fields);
    #endif

  // Step 6: Update the conserved variable array
  hipLaunchKernelGGL(Update_Conserved_Variables_1D, dimGrid, dimBlock, 0, 0, dev_conserved, F_x, n_cells, x_off,
                     n_ghost, dx, xbound, dt, gama, n_fields, custom_grav);
  GPU_Error_Check();

    #ifdef DE
  hipLaunchKernelGGL(Select_Internal_Energy_1D, dimGrid, dimBlock, 0, 0, dev_conserved, nx, n_ghost, n_fields);
  hipLaunchKernelGGL(Sync_Energies_1D, dimGrid, dimBlock, 0, 0, dev_conserved, nx, n_ghost, gama, n_fields);
  GPU_Error_Check();
    #endif

  return;
}

void Free_Memory_VL_1D()
{
  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(F_x);
}

__global__ void Update_Conserved_Variables_1D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F,
                                                   int n_cells, int n_ghost, Real dx, Real dt, Real gamma, int n_fields)
{
  int id, imo;
  Real dtodx = dt / dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;

    #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, P;
  int ipo;
    #endif

  // threads corresponding all cells except outer ring of ghost cells do the
  // calculation
  if (id > 0 && id < n_cells - 1) {
    imo = id - 1;
    #ifdef DE
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    P     = (dev_conserved[4 * n_cells + id] - 0.5 * d * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0);
    // if (d < 0.0 || d != d) printf("Negative density before half step
    // update.\n"); if (P < 0.0) printf("%d Negative pressure before half step
    // update.\n", id);
    ipo    = id + 1;
    vx_imo = dev_conserved[1 * n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1 * n_cells + ipo] / dev_conserved[ipo];
    #endif
    // update the conserved variable array
    dev_conserved_half[id] = dev_conserved[id] + dtodx * (dev_F[imo] - dev_F[id]);
    dev_conserved_half[n_cells + id] =
        dev_conserved[n_cells + id] + dtodx * (dev_F[n_cells + imo] - dev_F[n_cells + id]);
    dev_conserved_half[2 * n_cells + id] =
        dev_conserved[2 * n_cells + id] + dtodx * (dev_F[2 * n_cells + imo] - dev_F[2 * n_cells + id]);
    dev_conserved_half[3 * n_cells + id] =
        dev_conserved[3 * n_cells + id] + dtodx * (dev_F[3 * n_cells + imo] - dev_F[3 * n_cells + id]);
    dev_conserved_half[4 * n_cells + id] =
        dev_conserved[4 * n_cells + id] + dtodx * (dev_F[4 * n_cells + imo] - dev_F[4 * n_cells + id]);
    #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      dev_conserved_half[(5 + i) * n_cells + id] =
          dev_conserved[(5 + i) * n_cells + id] +
          dtodx * (dev_F[(5 + i) * n_cells + imo] - dev_F[(5 + i) * n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved_half[(n_fields - 1) * n_cells + id] =
        dev_conserved[(n_fields - 1) * n_cells + id] +
        dtodx * (dev_F[(n_fields - 1) * n_cells + imo] - dev_F[(n_fields - 1) * n_cells + id]) +
        0.5 * P * (dtodx * (vx_imo - vx_ipo));
    #endif
  }
}

  #endif  // VL
#endif    // CUDA
