
#include <hip/hip_runtime.h>
/*! \file VL_1D_cuda.cu
 *  \brief Definitions of the cuda VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"gpu.hpp"
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_1D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"cooling_cuda.h"
#include"error_handling.h"
#include"io.h"


__global__ void Update_Conserved_Variables_1D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F, 
                                                   int n_cells, int n_ghost, Real dx, Real dt, Real gamma, int n_fields);



Real VL_Algorithm_1D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int x_off, int n_ghost, Real dx, Real xbound, Real dt, int n_fields)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //host_conserved0 contains the values at time n
  //host_conserved1 will contain the values at time n+1

  // Initialize dt values
  Real max_dti = 0;
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  #endif  

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  // set the dimensions of the cuda grid
  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  if ( !memory_allocated ) {

    // allocate an array on the CPU to hold max_dti returned from each thread block
    CudaSafeCall( cudaHostAlloc(&host_dti_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaHostAlloc(&host_dt_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #endif
  
    // allocate memory on the GPU
    CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif  

    #ifndef DYNAMIC_GPU_ALLOC 
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif 
  }

  // copy the conserved variable array onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, host_conserved0, n_fields*n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
  CudaCheckError();

  // Step 1: Use PCM reconstruction to put conserved variables into interface arrays
  hipLaunchKernelGGL(PCM_Reconstruction_1D, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, gama, n_fields);
  CudaCheckError();

  // Step 2: Calculate first-order upwind fluxes 
  #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef HLLC 
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();


  // Step 3: Update the conserved variables half a timestep 
  hipLaunchKernelGGL(Update_Conserved_Variables_1D_half, dimGrid, dimBlock, 0, 0, dev_conserved, dev_conserved_half, F_x, n_cells, n_ghost, dx, 0.5*dt, gama, n_fields);
  CudaCheckError();


  // Step 4: Construct left and right interface values using updated conserved variables
  #ifdef PCM
  hipLaunchKernelGGL(PCM_Reconstruction_1D, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, n_ghost, gama, n_fields);
  #endif
  #ifdef PLMC
  hipLaunchKernelGGL(PLMC_cuda, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif  
  #ifdef PLMP
  hipLaunchKernelGGL(PLMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif
  #ifdef PPMP
  hipLaunchKernelGGL(PPMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif
  #ifdef PPMC
  hipLaunchKernelGGL(PPMC_cuda, dimGrid, dimBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif
  CudaCheckError();


  // Step 5: Calculate the fluxes again
  #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef HLLC 
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();

  #ifdef DE
  // Compute the divergence of velocity before updating the conserved array, this solves syncronization issues when adding this term on Update_Conserved_Variables
  hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_1D, dimGrid, dimBlock, 0, 0,  dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, dx, dt, gama, n_fields );
  #endif


  // Step 6: Update the conserved variable array
  hipLaunchKernelGGL(Update_Conserved_Variables_1D, dimGrid, dimBlock, 0, 0, dev_conserved, F_x, n_cells, x_off, n_ghost, dx, xbound, dt, gama, n_fields);
  CudaCheckError();
   

  #ifdef DE
  hipLaunchKernelGGL(Select_Internal_Energy_1D, dimGrid, dimBlock, 0, 0, dev_conserved, nx, n_ghost, n_fields);
  hipLaunchKernelGGL(Sync_Energies_1D, dimGrid, dimBlock, 0, 0, dev_conserved, nx, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif    


  // Apply cooling
  #ifdef COOLING_GPU
  hipLaunchKernelGGL(cooling_kernel, dimGrid, dimBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields, dt, gama, dev_dt_array);
  CudaCheckError();
  #endif


  // Step 7: Calculate the next timestep
  hipLaunchKernelGGL(Calc_dt_1D, dimGrid, dimBlock, 0, 0, dev_conserved, n_cells, n_ghost, dx, dev_dti_array, gama);
  CudaCheckError();


  // copy the conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(host_conserved1, dev_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );

  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }
  #ifdef COOLING_GPU
  // copy the dt array from cooling onto the CPU
  CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // find maximum inverse timestep from cooling time
  for (int i=0; i<ngrid; i++) {
    min_dt = fmin(min_dt, host_dt_array[i]);
  }  
  if (min_dt < C_cfl/max_dti) {
    max_dti = C_cfl/min_dt;
  }
  #endif

  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_VL_1D();
  #endif


  // return the maximum inverse timestep
  return max_dti;


}

void Free_Memory_VL_1D() {

  // free the CPU memory
  CudaSafeCall( cudaFreeHost(host_dti_array) );
  #ifdef COOLING_GPU
  CudaSafeCall( cudaFreeHost(host_dt_array) );  
  #endif  

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(F_x);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

}

__global__ void Update_Conserved_Variables_1D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F, int n_cells, int n_ghost, Real dx, Real dt, Real gamma, int n_fields)
{
  int id, imo;
  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  
  #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, P;
  int ipo;
  #endif

  // threads corresponding all cells except outer ring of ghost cells do the calculation
  if (id > 0 && id < n_cells-1)
  {
    imo = id-1;
    #ifdef DE
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (d < 0.0 || d != d) printf("Negative density before half step update.\n");
    //if (P < 0.0) printf("%d Negative pressure before half step update.\n", id);
    ipo = id+1;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    #endif
    // update the conserved variable array
    dev_conserved_half[            id] = dev_conserved[            id] + dtodx * (dev_F[            imo] - dev_F[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] + dtodx * (dev_F[  n_cells + imo] - dev_F[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] + dtodx * (dev_F[2*n_cells + imo] - dev_F[2*n_cells + id]);
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] + dtodx * (dev_F[3*n_cells + imo] - dev_F[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] + dtodx * (dev_F[4*n_cells + imo] - dev_F[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved_half[(5+i)*n_cells + id] = dev_conserved[(5+i)*n_cells + id] + dtodx * (dev_F[(5+i)*n_cells + imo] - dev_F[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved_half[(n_fields-1)*n_cells + id] = dev_conserved[(n_fields-1)*n_cells + id] 
                                       + dtodx * (dev_F[(n_fields-1)*n_cells + imo] - dev_F[(n_fields-1)*n_cells + id])
                                       + 0.5*P*(dtodx*(vx_imo-vx_ipo));
    #endif    
  }


}





#endif //VL
#endif //CUDA
