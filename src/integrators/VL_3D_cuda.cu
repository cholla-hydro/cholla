
#include <hip/hip_runtime.h>
/*! \file VL_3D_cuda.cu
 *  \brief Definitions of the cuda 3D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"gpu.hpp"
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_3D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_3D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_3D.h"
#include"io.h"
#include"hll_cuda.h"

__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma, int n_fields, Real density_floor);



Real VL_Algorithm_3D_CUDA(Real *host_conserved0, Real *host_conserved1, 
    Real *d_conserved, Real *d_grav_potential, int nx, int ny, int nz, int x_off, int y_off, 
    int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, 
    Real ybound, Real zbound, Real dt, int n_fields, Real density_floor, 
    Real U_floor, Real *host_grav_potential, Real max_dti_slow)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array
  //host_conserved0 contains the values at time n,
  //host_conserved1 will contain the values at time n+1

  // Initialize dt values 
  Real max_dti = 0;
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  #endif  


  if ( !block_size ) {
    // calculate the dimensions for the subgrid blocks
    sub_dimensions_3D(nx, ny, nz, n_ghost, &nx_s, &ny_s, &nz_s, &block1_tot, &block2_tot, &block3_tot, &remainder1, &remainder2, &remainder3, n_fields);
    //printf("Subgrid dimensions set: %d %d %d %d %d %d %d %d %d\n", nx_s, ny_s, nz_s, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3);
    //fflush(stdout);
    block_tot = block1_tot*block2_tot*block3_tot;
    // number of cells in one subgrid block
    BLOCK_VOL = nx_s*ny_s*nz_s;
    // dimensions for the 1D GPU grid
    ngrid = (BLOCK_VOL + TPB - 1) / TPB;
    #ifndef DYNAMIC_GPU_ALLOC
    block_size = true;
    #endif
  }
  // set values for GPU kernels
  // number of blocks per 1D grid  
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block   
  dim3 dim1dBlock(TPB, 1, 1);

  // Set up pointers for the location to copy from and to
  if (block_tot == 1) {
    tmp1 = host_conserved0;
    tmp2 = host_conserved1;
    //host_grav_potential is NULL if not using GRAVITY
    temp_potential = host_grav_potential;
  }

  if ( !memory_allocated ){

    // allocate buffer to copy conserved variable blocks to/from
    if (block_tot > 1) {
      if ( cudaSuccess != cudaHostAlloc(&buffer, n_fields*BLOCK_VOL*sizeof(Real), cudaHostAllocDefault) ) {
        printf("Failed to allocate CPU buffer.\n");
      }
      tmp1 = buffer;
      tmp2 = buffer;
      
      #if defined( GRAVITY )
      if ( cudaSuccess != cudaHostAlloc(&buffer_potential, BLOCK_VOL*sizeof(Real), cudaHostAllocDefault) ) {
        printf("Failed to allocate CPU Grav_Potential buffer.\n");
      }
      #else
      buffer_potential = NULL;
      #endif
      temp_potential = buffer_potential;
    }
    // allocate an array on the CPU to hold max_dti returned from each thread block
    CudaSafeCall( cudaHostAlloc(&host_dti_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaHostAlloc(&host_dt_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #endif  

    // allocate memory on the GPU
    //CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
    dev_conserved = d_conserved;
    CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif 
    
    #if defined( GRAVITY ) 
    // CudaSafeCall( cudaMalloc((void**)&dev_grav_potential, BLOCK_VOL*sizeof(Real)) );
    dev_grav_potential = d_grav_potential;
    #else
    dev_grav_potential = NULL;
    #endif
    
    #ifndef DYNAMIC_GPU_ALLOC 
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif
    //d_conserved = dev_conserved;
  }  

  // counter for which block we're on
  int block = 0;
  

  // START LOOP OVER SUBGRID BLOCKS
  while (block < block_tot) {

    // copy the conserved variable block to the buffer
    host_copy_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved0, buffer, n_fields, host_grav_potential, buffer_potential);

    // calculate the global x, y, and z offsets of this subgrid block
    get_offsets_3D(nx_s, ny_s, nz_s, n_ghost, x_off, y_off, z_off, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, &x_off_s, &y_off_s, &z_off_s);

    // copy the conserved variables onto the GPU
    #ifndef HYDRO_GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );
    #endif
    
    #if defined( GRAVITY ) && !defined( GRAVITY_GPU )
    CudaSafeCall( cudaMemcpy(dev_grav_potential, temp_potential, BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );
    #endif
 

    // Step 1: Use PCM reconstruction to put primitive variables into interface arrays
    hipLaunchKernelGGL(PCM_Reconstruction_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    CudaCheckError();


    // Step 2: Calculate first-order upwind fluxes 
    #ifdef EXACT
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //ROE
    #ifdef HLLC 
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    #ifdef HLL 
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    CudaCheckError();


    // Step 3: Update the conserved variables half a timestep 
    hipLaunchKernelGGL(Update_Conserved_Variables_3D_half, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, dev_conserved_half, F_x, F_y, F_z, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, 0.5*dt, gama, n_fields, density_floor );
    CudaCheckError();


    // Step 4: Construct left and right interface values using updated conserved variables
    #ifdef PCM
    hipLaunchKernelGGL(PCM_Reconstruction_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    #endif
    #ifdef PLMP
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PLMP 
    #ifdef PLMC
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);  
    #endif
    #ifdef PPMP
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMP
    #ifdef PPMC
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMC
    CudaCheckError();
    

    // Step 5: Calculate the fluxes again
    #ifdef EXACT
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //ROE
    #ifdef HLLC 
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    #ifdef HLL 
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    CudaCheckError();
    
    #ifdef DE
    // Compute the divergence of Vel before updating the conserved array, this solves syncronization issues when adding this term on Update_Conserved_Variables_3D
    hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0,  dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz,  dt, gama, n_fields );
    CudaCheckError();
    #endif


    // Step 6: Update the conserved variable array
    hipLaunchKernelGGL(Update_Conserved_Variables_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, F_x, F_y, F_z, nx_s, ny_s, nz_s, x_off_s, y_off_s, z_off_s, n_ghost, dx, dy, dz, xbound, ybound, zbound, dt, gama, n_fields, density_floor, dev_grav_potential);
    CudaCheckError();

    #ifdef DE
    hipLaunchKernelGGL(Select_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields);
    hipLaunchKernelGGL(Sync_Energies_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    CudaCheckError();
    #endif
    
    #ifdef TEMPERATURE_FLOOR
    hipLaunchKernelGGL(Apply_Temperature_Floor, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, U_floor );
    CudaCheckError();
    #endif //TEMPERATURE_FLOOR

    // Apply cooling
    #ifdef COOLING_GPU
    hipLaunchKernelGGL(cooling_kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, dt, gama, dev_dt_array);  
    CudaCheckError();
    #endif
 
    // Step 7: Calculate the next time step
    hipLaunchKernelGGL(Calc_dt_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dev_dti_array, gama, max_dti_slow );
    CudaCheckError();

    // copy the updated conserved variable array back to the CPU
    #ifndef HYDRO_GPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );
    #endif

    // copy the updated conserved variable array from the buffer into the host_conserved array on the CPU
    host_return_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved1, buffer, n_fields);

    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // find maximum inverse timestep from CFL condition
    for (int i=0; i<ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }
    #ifdef COOLING_GPU
    // copy the dt array from cooling onto the CPU
    CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // find maximum inverse timestep from cooling time
    for (int i=0; i<ngrid; i++) {
      min_dt = fmin(min_dt, host_dt_array[i]);
    }  
    if (min_dt < C_cfl/max_dti) {
      max_dti = C_cfl/min_dt;
    }
    #endif

    // add one to the counter
    block++;

  }

  
  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_VL_3D();
  #endif


  // return the maximum inverse timestep
  return max_dti;

}


void Free_Memory_VL_3D(){
  
  // free CPU memory
  if (block_tot > 1) CudaSafeCall( cudaFreeHost(buffer) );
  CudaSafeCall( cudaFreeHost(host_dti_array) );  
  #ifdef COOLING_GPU
  CudaSafeCall( cudaFreeHost(host_dt_array) );  
  #endif  
  
  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif
  #if defined( GRAVITY ) 
  cudaFree(dev_grav_potential);
  if (block_tot > 1) CudaSafeCall( cudaFreeHost(buffer_potential) );
  #endif

}

__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma, int n_fields, Real density_floor )
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  int n_cells = nx*ny*nz;

  // get a global thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;
  int id = xid + yid*nx + zid*nx*ny;

  int imo = xid-1 + yid*nx + zid*nx*ny;
  int jmo = xid + (yid-1)*nx + zid*nx*ny;
  int kmo = xid + yid*nx + (zid-1)*nx*ny;

  #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo, P, E, E_kin, GE;
  int ipo, jpo, kpo;
  #endif
  
  #ifdef DENSITY_FLOOR
  Real dens_0;
  #endif

  // threads corresponding to all cells except outer ring of ghost cells do the calculation
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1 && zid > 0 && zid < nz-1)
  {
    #ifdef DE
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = Get_Pressure_From_DE( E, E - E_kin, GE, gamma );  
    P  = fmax(P, (Real) TINY_NUMBER); 
    // P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (d < 0.0 || d != d) printf("Negative density before half step update.\n");
    //if (P < 0.0) printf("%d Negative pressure before half step update.\n", id);
    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo]; 
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo]; 
    #endif
  
    // update the conserved variable array
    dev_conserved_half[            id] = dev_conserved[            id]
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                       + dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] 
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                       + dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] 
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                       + dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] 
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                       + dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] 
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                       + dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR                                   
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved_half[(5+i)*n_cells + id] = dev_conserved[(5+i)*n_cells + id] 
                                         + dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                         + dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                         + dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
    }                                   
    #endif
    #ifdef DE
    dev_conserved_half[(n_fields-1)*n_cells + id] = dev_conserved[(n_fields-1)*n_cells + id] 
                                       + dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                       + dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                       + dtodz * (dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id])
                                       + 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
    #endif
    
    #ifdef DENSITY_FLOOR
    if ( dev_conserved_half[            id] < density_floor ){
      dens_0 = dev_conserved_half[            id];
      printf("###Thread density change  %f -> %f \n", dens_0, density_floor );
      dev_conserved_half[            id] = density_floor;
      // Scale the conserved values to the new density
      dev_conserved_half[1*n_cells + id] *= (density_floor / dens_0);
      dev_conserved_half[2*n_cells + id] *= (density_floor / dens_0);
      dev_conserved_half[3*n_cells + id] *= (density_floor / dens_0);
      dev_conserved_half[4*n_cells + id] *= (density_floor / dens_0);
      #ifdef DE
      dev_conserved_half[(n_fields-1)*n_cells + id] *= (density_floor / dens_0);
      #endif
    }
    #endif
    //if (dev_conserved_half[id] < 0.0 || dev_conserved_half[id] != dev_conserved_half[id] || dev_conserved_half[4*n_cells+id] < 0.0 || dev_conserved_half[4*n_cells+id] != dev_conserved_half[4*n_cells+id]) {
      //printf("%3d %3d %3d Thread crashed in half step update. d: %e E: %e\n", xid, yid, zid, dev_conserved_half[id], dev_conserved_half[4*n_cells+id]);
    //}    

  }

}




#endif //VL
#endif //CUDA
