
#include <hip/hip_runtime.h>
/*! \file simple_3D_cuda.cu
 *  \brief Definitions of the cuda 3D simple algorithm functions. */

#ifdef CUDA
  #ifdef SIMPLE

    #include <math.h>
    #include <stdio.h>
    #include <stdlib.h>

    #include "../global/global.h"
    #include "../global/global_cuda.h"
    #include "../hydro/hydro_cuda.h"
    #include "../integrators/simple_3D_cuda.h"
    #include "../io/io.h"
    #include "../reconstruction/pcm_cuda.h"
    #include "../reconstruction/plmc_cuda.h"
    #include "../reconstruction/plmp_cuda.h"
    #include "../reconstruction/ppmc_cuda.h"
    #include "../reconstruction/ppmp_cuda.h"
    #include "../riemann_solvers/exact_cuda.h"
    #include "../riemann_solvers/hll_cuda.h"
    #include "../riemann_solvers/hllc_cuda.h"
    #include "../riemann_solvers/roe_cuda.h"
    #include "../utils/gpu.hpp"

void Simple_Algorithm_3D_CUDA(Real *d_conserved, Real *d_grav_potential, int nx, int ny, int nz, int x_off, int y_off,
                              int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound,
                              Real dt, int n_fields, int custom_grav, Real density_floor, Real U_floor,
                              Real *host_grav_potential)
{
  // Here, *dev_conserved contains the entire
  // set of conserved variables on the grid
  // concatenated into a 1-d array
  int n_cells = nx * ny * nz;
  int ngrid   = (n_cells + TPB - 1) / TPB;

  // set values for GPU kernels
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB, 1, 1);

  // host_grav_potential is NULL if not using GRAVITY
  temp_potential = host_grav_potential;

  if (!memory_allocated) {
    size_t global_free, global_total;
    GPU_Error_Check(cudaMemGetInfo(&global_free, &global_total));

    // allocate memory on the GPU
    chprintf(
        " Allocating Hydro Memory: nfields: %d   n_cells: %d   nx: %d  ny: %d  "
        "nz: %d \n",
        n_fields, n_cells, nx, ny, nz);
    chprintf(" Memory needed: %f GB    Free: %f GB    Total:  %f GB  \n", n_fields * n_cells * sizeof(Real) / 1e9,
             global_free / 1e9, global_total / 1e9);
    dev_conserved = d_conserved;
    GPU_Error_Check(cudaMalloc((void **)&Q_Lx, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&Q_Rx, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&Q_Ly, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&Q_Ry, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&Q_Lz, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&Q_Rz, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&F_x, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&F_y, n_fields * n_cells * sizeof(Real)));
    GPU_Error_Check(cudaMalloc((void **)&F_z, n_fields * n_cells * sizeof(Real)));

    #if defined(GRAVITY)
    // GPU_Error_Check( cudaMalloc((void**)&dev_grav_potential,
    // n_cells*sizeof(Real)) );
    dev_grav_potential = d_grav_potential;
    #else
    dev_grav_potential = NULL;
    #endif

    // If memory is single allocated: memory_allocated becomes true and
    // successive timesteps won't allocate memory. If the memory is not single
    // allocated: memory_allocated remains Null and memory is allocated every
    // timestep.
    memory_allocated = true;
    chprintf(" Memory allocated \n");
  }

    #if defined(GRAVITY) && !defined(GRAVITY_GPU)
  GPU_Error_Check(cudaMemcpy(dev_grav_potential, temp_potential, n_cells * sizeof(Real), cudaMemcpyHostToDevice));
    #endif

    // Step 1: Construct left and right interface values using updated conserved
    // variables
    #ifdef PCM
  hipLaunchKernelGGL(PCM_Reconstruction_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz,
                     Q_Rz, nx, ny, nz, n_ghost, gama, n_fields);
    #endif
    #ifdef PLMP
  hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt,
                     gama, 0, n_fields);
  hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt,
                     gama, 1, n_fields);
  hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt,
                     gama, 2, n_fields);
    #endif  // PLMP
    #ifdef PLMC
  hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, dx, dt, gama, 0,
                     n_fields);
  hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, dy, dt, gama, 1,
                     n_fields);
  hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, dz, dt, gama, 2,
                     n_fields);
    #endif
    #ifdef PPMP
  hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt,
                     gama, 0, n_fields);
  hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt,
                     gama, 1, n_fields);
  hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt,
                     gama, 2, n_fields);
    #endif  // PPMP
    #ifdef PPMC
  hipLaunchKernelGGL(PPMC_CTU, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, dx, dt, gama, 0);
  hipLaunchKernelGGL(PPMC_CTU, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, dy, dt, gama, 1);
  hipLaunchKernelGGL(PPMC_CTU, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, dz, dt, gama, 2);
  GPU_Error_Check();
    #endif  // PPMC

    // Step 2: Calculate the fluxes
    #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost,
                     gama, 0, n_fields);
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost,
                     gama, 1, n_fields);
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost,
                     gama, 2, n_fields);
    #endif  // EXACT
    #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama,
                     0, n_fields);
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama,
                     1, n_fields);
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama,
                     2, n_fields);
    #endif  // ROE
    #ifdef HLLC
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost,
                     gama, 0, n_fields);
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost,
                     gama, 1, n_fields);
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost,
                     gama, 2, n_fields);
    #endif  // HLLC
    #ifdef HLL
  hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama,
                     0, n_fields);
  hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama,
                     1, n_fields);
  hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama,
                     2, n_fields);
    #endif  // HLL
  GPU_Error_Check();

    #ifdef DE
  // Compute the divergence of Vel before updating the conserved array, this
  // solves synchronization issues when adding this term on
  // Update_Conserved_Variables_3D
  hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx,
                     Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dx, dy, dz, dt, gama, n_fields);
  GPU_Error_Check();
    #endif

  // Step 3: Update the conserved variable array
  hipLaunchKernelGGL(Update_Conserved_Variables_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry,
                     Q_Lz, Q_Rz, F_x, F_y, F_z, nx, ny, nz, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound,
                     zbound, dt, gama, n_fields, custom_grav, density_floor, dev_grav_potential);
  GPU_Error_Check();

    #ifdef DE
  hipLaunchKernelGGL(Select_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost,
                     n_fields);
  hipLaunchKernelGGL(Sync_Energies_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, gama, n_fields);
  GPU_Error_Check();
    #endif

    #ifdef TEMPERATURE_FLOOR
  hipLaunchKernelGGL(Apply_Temperature_Floor, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields,
                     U_floor);
  GPU_Error_Check();
    #endif  // TEMPERATURE_FLOOR

  return;
}

void Free_Memory_Simple_3D()
{
  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);
}

  #endif  // SIMPLE
#endif    // CUDA
