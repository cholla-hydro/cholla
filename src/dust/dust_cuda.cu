
#include <hip/hip_runtime.h>
/*!
 * \file dust_cuda.cu
 * \author Helena Richie (helenarichie@gmail.com)
 * \brief Contains code that updates the dust density scalar field. The dust_kernel function determines the rate of
 * change of dust density, which is controlled by the sputtering timescale. The sputtering timescale is from the
 * McKinnon et al. (2017) model of dust sputtering, which depends on the cell's gas density and temperature.
 */

#ifdef DUST

  // STL includes
  #include <stdio.h>

  #include <cstdio>
  #include <fstream>
  #include <vector>

  // Local includes
  #include "../dust/dust_cuda.h"
  #include "../global/global.h"
  #include "../global/global_cuda.h"
  #include "../grid/grid3D.h"
  #include "../grid/grid_enum.h"
  #include "../utils/cuda_utilities.h"
  #include "../utils/gpu.hpp"
  #include "../utils/hydro_utilities.h"

void Dust_Update(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt, Real gamma)
{
  int n_cells = nx * ny * nz;
  int ngrid   = (n_cells + TPB - 1) / TPB;
  dim3 dim1dGrid(ngrid, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);
  hipLaunchKernelGGL(Dust_Kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields, dt, gamma);
  GPU_Error_Check();
}

__global__ void Dust_Kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt, Real gamma)
{
  // get grid indices
  int n_cells = nx * ny * nz;
  int is, ie, js, je, ks, ke;
  cuda_utilities::Get_Real_Indices(n_ghost, nx, ny, nz, is, ie, js, je, ks, ke);
  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int id      = threadIdx.x + blockId * blockDim.x;
  int id_z    = id / (nx * ny);
  int id_y    = (id - id_z * nx * ny) / nx;
  int id_x    = id - id_z * nx * ny - id_y * nx;

  // define physics variables
  Real density_gas, density_dust;           // fluid mass densities
  Real number_density;                      // gas number density
  Real mu = 0.6;                            // mean molecular weight
  Real temperature, energy, pressure;       // temperature, energy, pressure
  Real velocity_x, velocity_y, velocity_z;  // velocities
  #ifdef DE
  Real energy_gas;
  #endif  // DE

  // define integration variables
  Real dd_dt;          // instantaneous rate of change in dust density
  Real dd;             // change in dust density at current timestep
  Real dd_max = 0.01;  // allowable percentage of dust density increase
  Real dt_sub;         // refined timestep

  if (id_x >= is && id_x < ie && id_y >= js && id_y < je && id_z >= ks && id_z < ke) {
    // get conserved quanitites
    density_gas  = dev_conserved[id + n_cells * grid_enum::density];
    density_dust = dev_conserved[id + n_cells * grid_enum::dust_density];
    energy       = dev_conserved[id + n_cells * grid_enum::Energy];

    // convert mass density to number density
    number_density = density_gas * DENSITY_UNIT / (mu * MP);

    if (energy < 0.0 || energy != energy) {
      return;
    }

    // get conserved quanitites
    velocity_x = dev_conserved[id + n_cells * grid_enum::momentum_x] / density_gas;
    velocity_y = dev_conserved[id + n_cells * grid_enum::momentum_y] / density_gas;
    velocity_z = dev_conserved[id + n_cells * grid_enum::momentum_z] / density_gas;
  #ifdef DE
    energy_gas = dev_conserved[id + n_cells * grid_enum::GasEnergy] / density_gas;
    energy_gas = fmax(ge, (Real)TINY_NUMBER);
  #endif  // DE

    // calculate physical quantities
    pressure = hydro_utilities::Calc_Pressure_Primitive(energy, density_gas, velocity_x, velocity_y, velocity_z, gamma);

    Real temperature_init;
    temperature_init = hydro_utilities::Calc_Temp(pressure, number_density);

  #ifdef DE
    temperature_init = hydro_utilities::Calc_Temp_DE(density_gas, energy_gas, gamma, number_density);
  #endif  // DE

    // if dual energy is turned on use temp from total internal energy
    temperature = temperature_init;

    Real tau_sp =
        Calc_Sputtering_Timescale(number_density, temperature) / TIME_UNIT;  // sputtering timescale, kyr (sim units)

    dd_dt = Calc_dd_dt(density_dust, tau_sp);  // rate of change in dust density at current timestep
    dd    = dd_dt * dt;                        // change in dust density at current timestep

    // ensure that dust density is not changing too rapidly
    while (dd / density_dust > dd_max) {
      dt_sub = dd_max * density_dust / dd_dt;
      density_dust += dt_sub * dd_dt;
      dt -= dt_sub;
      dd_dt = Calc_dd_dt(density_dust, tau_sp);
      dd    = dt * dd_dt;
    }

    // update dust density
    density_dust += dd;

    dev_conserved[id + n_cells * grid_enum::dust_density] = density_dust;

  #ifdef DE
    dev_conserved[id + n_cells * grid_enum::GasEnergy] = density_dust * energy_gas;
  #endif
  }
}

// McKinnon et al. (2017) sputtering timescale
__device__ __host__ Real Calc_Sputtering_Timescale(Real number_density, Real temperature)
{
  Real grain_radius  = 1;          // dust grain size in units of 0.1 micrometers
  Real temperature_0 = 2e6;        // temp above which the sputtering rate is ~constant in K
  Real omega         = 2.5;        // controls the low-temperature scaling of the sputtering rate
  Real A             = 5.3618e15;  // 0.17 Gyr in s

  number_density /= (6e-4);  // gas number density in units of 10^-27 g/cm^3

  // sputtering timescale, s
  Real tau_sp = A * (grain_radius / number_density) * (pow(temperature_0 / temperature, omega) + 1);

  return tau_sp;
}

// McKinnon et al. (2017) sputtering model
__device__ __host__ Real Calc_dd_dt(Real density_dust, Real tau_sp) { return -density_dust / (tau_sp / 3); }

#endif  // DUST
