#include "hip/hip_runtime.h"
/*!
 * \file mhd_utilities_tests.cpp
 * \author Robert 'Bob' Caddy (rvc@pitt.edu)
 * \brief Tests for the contents of mhd_utilities.h and mhd_utilities.cpp
 *
 */

// STL Includes
#include <cmath>
#include <iostream>
#include <numeric>
#include <random>
#include <string>
#include <vector>

// External Includes
#include <gtest/gtest.h>  // Include GoogleTest and related libraries/headers

// Local Includes
#include "../global/global.h"
#include "../grid/grid3D.h"
#include "../utils/mhd_utilities.h"
#include "../utils/testing_utilities.h"

// =============================================================================
// Local helper functions
namespace
{
struct TestParams {
  double gamma = 5. / 3.;
  std::vector<double> density{8.4087201154e-100, 1.6756968986e2, 5.4882403847e100};
  std::vector<double> velocityX{7.0378624601e-100, 7.0829278656e2, 1.8800514112e100};
  std::vector<double> velocityY{7.3583469014e-100, 5.9283073464e2, 5.2725717864e100};
  std::vector<double> velocityZ{1.7182972216e-100, 8.8417748226e2, 1.5855352639e100};
  std::vector<double> momentumX{8.2340416681e-100, 8.1019429453e2, 5.5062596954e100};
  std::vector<double> momentumY{4.9924582299e-100, 7.1254780684e2, 6.5939640992e100};
  std::vector<double> momentumZ{3.6703192739e-100, 7.5676716066e2, 7.2115881803e100};
  std::vector<double> energy{3.0342082433e-100, 7.6976906577e2, 1.9487120853e100};
  std::vector<double> pressureGas{2.2244082909e-100, 8.6772951021e2, 6.7261085663e100};
  std::vector<double> pressureTotal{8.1704748693e-100, 2.6084125198e2, 1.8242151369e100};
  std::vector<double> magneticX{2.8568843801e-100, 9.2400807786e2, 2.1621115264e100};
  std::vector<double> magneticY{9.2900880344e-100, 8.0382409757e2, 6.6499532343e100};
  std::vector<double> magneticZ{9.5795678229e-100, 3.3284839263e2, 9.2337456649e100};
  std::vector<std::string> names{"Small number case", "Medium number case", "Large number case"};
};
}  // namespace
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::computeThermalEnergy function
// =============================================================================
/*!
 * \brief Test the mhd::utils::computeThermalEnergy function with the standard
 * set of parameters.
 *
 */
TEST(tMHDComputeThermalEnergy, CorrectInputExpectCorrectOutput)
{
  TestParams parameters;
  std::vector<double> energyMultiplier{1.0E85, 1.0E4, 1.0E105};
  std::vector<double> fiducialGasPressures{3.0342082433e-15, 6887152.1495634327, 1.9480412919836246e+205};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testGasPressure = mhd::utils::computeThermalEnergy(
        energyMultiplier.at(i) * parameters.energy.at(i), parameters.density.at(i), parameters.momentumX.at(i),
        parameters.momentumY.at(i), parameters.momentumZ.at(i), parameters.magneticX.at(i), parameters.magneticY.at(i),
        parameters.magneticZ.at(i), parameters.gamma);

    testing_utilities::Check_Results(fiducialGasPressures.at(i), testGasPressure, parameters.names.at(i));
  }
}
// =============================================================================
// End of tests for the mhd::utils::computeThermalEnergy function
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::computeMagneticEnergy function
// =============================================================================
/*!
 * \brief Test the mhd::utils::computeMagneticEnergy function with the standard
 * set of parameters.
 *
 */
TEST(tMHDcomputeMagneticEnergy, CorrectInputExpectCorrectOutput)
{
  TestParams parameters;
  std::vector<double> energyMultiplier{1.0E85, 1.0E4, 1.0E105};
  std::vector<double> fiducialEnergy{0.0, 805356.08013056568, 6.7079331637514162e+201};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testMagneticEnergy = mhd::utils::computeMagneticEnergy(parameters.magneticX.at(i), parameters.magneticY.at(i),
                                                                parameters.magneticZ.at(i));

    testing_utilities::Check_Results(fiducialEnergy.at(i), testMagneticEnergy, parameters.names.at(i));
  }
}
// =============================================================================
// End of tests for the mhd::utils::computeMagneticEnergy function
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::computeTotalPressure function
// =============================================================================
/*!
 * \brief Test the mhd::utils::computeTotalPressure function with the standard
 * set of parameters.
 *
 */
TEST(tMHDComputeTotalPressure, CorrectInputExpectCorrectOutput)
{
  TestParams parameters;
  std::vector<double> fiducialTotalPressures{9.9999999999999995e-21, 806223.80964077567, 6.7079331637514151e+201};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testTotalPressure = mhd::utils::computeTotalPressure(parameters.pressureGas.at(i), parameters.magneticX.at(i),
                                                              parameters.magneticY.at(i), parameters.magneticZ.at(i));

    testing_utilities::Check_Results(fiducialTotalPressures.at(i), testTotalPressure, parameters.names.at(i));
  }
}

/*!
 * \brief Test the mhd::utils::computeTotalPressure function with a the standard
 * set of parameters. Gas pressure has been multiplied and made negative to
 * generate negative total pressures
 *
 */
TEST(tMHDComputeTotalPressure, NegativePressureExpectAutomaticFix)
{
  TestParams parameters;
  std::vector<double> pressureMultiplier{1.0, -1.0e4, -1.0e105};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testTotalPressure = mhd::utils::computeTotalPressure(pressureMultiplier.at(i) * parameters.pressureGas.at(i),
                                                              parameters.magneticX.at(i), parameters.magneticY.at(i),
                                                              parameters.magneticZ.at(i));

    // I'm using the binary equality assertion here since in the case of
    // negative pressure the function should return exactly TINY_NUMBER
    EXPECT_EQ(TINY_NUMBER, testTotalPressure) << "Difference in " << parameters.names.at(i) << std::endl;
  }
}
// =============================================================================
// End of tests for the mhd::utils::computeTotalPressure function
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::fastMagnetosonicSpeed function
// =============================================================================
/*!
 * \brief Test the mhd::utils::fastMagnetosonicSpeed function with the standard
 * set of parameters. All values are reduced by 1e-25 in the large number case
 * to avoid overflow
 *
 */
TEST(tMHDFastMagnetosonicSpeed, CorrectInputExpectCorrectOutput)
{
  TestParams parameters;
  std::vector<double> fiducialFastMagnetosonicSpeed{1.9254472601190615e-40, 98.062482309387562, 1.5634816865472293e+38};
  std::vector<double> coef{1.0, 1.0, 1.0e-25};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testFastMagnetosonicSpeed = mhd::utils::fastMagnetosonicSpeed(
        coef.at(i) * parameters.density.at(i), coef.at(i) * parameters.pressureGas.at(i),
        coef.at(i) * parameters.magneticX.at(i), coef.at(i) * parameters.magneticY.at(i),
        coef.at(i) * parameters.magneticZ.at(i), parameters.gamma);

    testing_utilities::Check_Results(fiducialFastMagnetosonicSpeed.at(i), testFastMagnetosonicSpeed,
                                     parameters.names.at(i));
  }
}

/*!
 * \brief Test the mhd::utils::fastMagnetosonicSpeed function with the standard
 * set of parameters, density is negative. All values are reduced by 1e-25 in
 * the large number case to avoid overflow.
 *
 */
TEST(tMHDFastMagnetosonicSpeed, NegativeDensityExpectAutomaticFix)
{
  TestParams parameters;
  std::vector<double> fiducialFastMagnetosonicSpeed{1.9254472601190615e-40, 12694062010603.15, 1.1582688085027081e+86};
  std::vector<double> coef{1.0, 1.0, 1.0e-25};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testFastMagnetosonicSpeed = mhd::utils::fastMagnetosonicSpeed(
        -coef.at(i) * parameters.density.at(i), coef.at(i) * parameters.pressureGas.at(i),
        coef.at(i) * parameters.magneticX.at(i), coef.at(i) * parameters.magneticY.at(i),
        coef.at(i) * parameters.magneticZ.at(i), parameters.gamma);

    testing_utilities::Check_Results(fiducialFastMagnetosonicSpeed.at(i), testFastMagnetosonicSpeed,
                                     parameters.names.at(i));
  }
}
// =============================================================================
// End of tests for the mhd::utils::fastMagnetosonicSpeed function
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::slowMagnetosonicSpeed function
// =============================================================================
/*!
 * \brief Test the mhd::utils::slowMagnetosonicSpeed function with the standard
 * set of parameters. All values are reduced by 1e-25 in the large number case
 * to avoid overflow
 *
 */
TEST(tMHDSlowMagnetosonicSpeed, CorrectInputExpectCorrectOutput)
{
  TestParams parameters;
  std::vector<double> fiducialSlowMagnetosonicSpeed{0.0, 2.138424778167535, 0.26678309355540852};
  // Coefficient to make sure the output is well defined and not nan or inf
  double const coef = 1E-95;

  for (size_t i = 2; i < parameters.names.size(); i++) {
    Real testSlowMagnetosonicSpeed = mhd::utils::slowMagnetosonicSpeed(
        parameters.density.at(i) * coef, parameters.pressureGas.at(i) * coef, parameters.magneticX.at(i) * coef,
        parameters.magneticY.at(i) * coef, parameters.magneticZ.at(i) * coef, parameters.gamma);

    testing_utilities::Check_Results(fiducialSlowMagnetosonicSpeed.at(i), testSlowMagnetosonicSpeed,
                                     parameters.names.at(i));
  }
}

/*!
 * \brief Test the mhd::utils::slowMagnetosonicSpeed function with the standard
 * set of parameters, density is negative. All values are reduced by 1e-25 in
 * the large number case to avoid overflow.
 *
 */
TEST(tMHDSlowMagnetosonicSpeed, NegativeDensityExpectAutomaticFix)
{
  TestParams parameters;
  std::vector<double> fiducialSlowMagnetosonicSpeed{0.0, 276816332809.37604, 1976400098318.3574};
  // Coefficient to make sure the output is well defined and not nan or inf
  double const coef = 1E-95;

  for (size_t i = 2; i < parameters.names.size(); i++) {
    Real testSlowMagnetosonicSpeed = mhd::utils::slowMagnetosonicSpeed(
        -parameters.density.at(i) * coef, parameters.pressureGas.at(i) * coef, parameters.magneticX.at(i) * coef,
        parameters.magneticY.at(i) * coef, parameters.magneticZ.at(i) * coef, parameters.gamma);

    testing_utilities::Check_Results(fiducialSlowMagnetosonicSpeed.at(i), testSlowMagnetosonicSpeed,
                                     parameters.names.at(i));
  }
}
// =============================================================================
// End of tests for the mhd::utils::slowMagnetosonicSpeed function
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::alfvenSpeed function
// =============================================================================
/*!
 * \brief Test the mhd::utils::alfvenSpeed function with the standard set of
 * parameters.
 *
 */
TEST(tMHDAlfvenSpeed, CorrectInputExpectCorrectOutput)
{
  TestParams parameters;
  std::vector<double> fiducialAlfvenSpeed{2.8568843800999998e-90, 71.380245120271113, 9.2291462785524423e+49};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testAlfvenSpeed = mhd::utils::alfvenSpeed(parameters.magneticX.at(i), parameters.density.at(i));

    testing_utilities::Check_Results(fiducialAlfvenSpeed.at(i), testAlfvenSpeed, parameters.names.at(i));
  }
}

/*!
 * \brief Test the mhd::utils::alfvenSpeed function with the standard set of
 * parameters except density is negative
 *
 */
TEST(tMHDAlfvenSpeed, NegativeDensityExpectAutomaticFix)
{
  TestParams parameters;
  std::vector<double> fiducialAlfvenSpeed{2.8568843800999998e-90, 9240080778600, 2.1621115263999998e+110};

  for (size_t i = 0; i < parameters.names.size(); i++) {
    Real testAlfvenSpeed = mhd::utils::alfvenSpeed(parameters.magneticX.at(i), -parameters.density.at(i));

    testing_utilities::Check_Results(fiducialAlfvenSpeed.at(i), testAlfvenSpeed, parameters.names.at(i));
  }
}
// =============================================================================
// End of tests for the mhd::utils::alfvenSpeed function
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::cellCenteredMagneticFields function
// =============================================================================
#ifdef MHD
TEST(tMHDCellCenteredMagneticFields, CorrectInputExpectCorrectOutput)
{
  // Initialize the test grid and other state variables
  size_t const nx = 3, ny = nx;
  size_t const xid = std::floor(nx / 2), yid = xid, zid = xid;
  size_t const id = xid + yid * nx + zid * nx * ny;

  size_t const n_cells = std::pow(5, 3);
  // Make sure the vector is large enough that the locations where the
  // magnetic field would be in the real grid are filled
  std::vector<double> testGrid(n_cells * (grid_enum::num_fields));
  // Populate the grid with values where testGrid.at(i) = double(i). The
  // values chosen aren't that important, just that every cell has a unique
  // value
  std::iota(std::begin(testGrid), std::end(testGrid), 0.);

  // Fiducial and test variables
  double const fiducialAvgBx = 637.5, fiducialAvgBy = 761.5, fiducialAvgBz = 883.5;

  // Call the function to test
  auto [testAvgBx, testAvgBy, testAvgBz] =
      mhd::utils::cellCenteredMagneticFields(testGrid.data(), id, xid, yid, zid, n_cells, nx, ny);

  // Check the results
  testing_utilities::Check_Results(fiducialAvgBx, testAvgBx, "cell centered Bx value");
  testing_utilities::Check_Results(fiducialAvgBy, testAvgBy, "cell centered By value");
  testing_utilities::Check_Results(fiducialAvgBz, testAvgBz, "cell centered Bz value");
}
#endif  // MHD
// =============================================================================
// End of tests for the mhd::utils::cellCenteredMagneticFields function
// =============================================================================

// =============================================================================
// Tests for the mhd::utils::Init_Magnetic_Field_With_Vector_Potential function
// =============================================================================
#ifdef MHD
TEST(tMHDInitMagneticFieldWithVectorPotential, CorrectInputExpectCorrectOutput)
{
  // Mock up Header and Conserved structs
  Header H;
  Grid3D::Conserved C;

  H.nx      = 2;
  H.ny      = 2;
  H.nz      = 2;
  H.n_cells = H.nx * H.ny * H.nz;
  H.dx      = 0.2;
  H.dy      = 0.2;
  H.dz      = 0.2;

  double const default_fiducial = -999;
  std::vector<double> conserved_vector(H.n_cells * grid_enum::num_fields, default_fiducial);
  C.host       = conserved_vector.data();
  C.density    = &(C.host[grid_enum::density * H.n_cells]);
  C.momentum_x = &(C.host[grid_enum::momentum_x * H.n_cells]);
  C.momentum_y = &(C.host[grid_enum::momentum_y * H.n_cells]);
  C.momentum_z = &(C.host[grid_enum::momentum_z * H.n_cells]);
  C.Energy     = &(C.host[grid_enum::Energy * H.n_cells]);
  C.magnetic_x = &(C.host[grid_enum::magnetic_x * H.n_cells]);
  C.magnetic_y = &(C.host[grid_enum::magnetic_y * H.n_cells]);
  C.magnetic_z = &(C.host[grid_enum::magnetic_z * H.n_cells]);

  // Mock up vector potential
  std::vector<double> vector_potential(H.n_cells * 3, 0);
  std::iota(vector_potential.begin(), vector_potential.end(), 0);

  // Run the function
  mhd::utils::Init_Magnetic_Field_With_Vector_Potential(H, C, vector_potential);

  // Check the results
  double const bx_fiducial = -10.0;
  double const by_fiducial = 15.0;
  double const bz_fiducial = -5.0;

  for (size_t i = 0; i < conserved_vector.size(); i++) {
    if (i == 47) {
      testing_utilities::Check_Results(bx_fiducial, conserved_vector.at(i), "value at i = " + std::to_string(i));
    } else if (i == 55) {
      testing_utilities::Check_Results(by_fiducial, conserved_vector.at(i), "value at i = " + std::to_string(i));
    } else if (i == 63) {
      testing_utilities::Check_Results(bz_fiducial, conserved_vector.at(i), "value at i = " + std::to_string(i));
    } else {
      testing_utilities::Check_Results(default_fiducial, conserved_vector.at(i), "value at i = " + std::to_string(i));
    }
  }
}
#endif  // MHD
// =============================================================================
// End of tests for the mhd::utils::Init_Magnetic_Field_With_Vector_Potential function
// =============================================================================
