
#include <hip/hip_runtime.h>
/*! \file global_cuda.cu
 *  \brief Declarations of the cuda global variables. */

#ifdef CUDA

  #include "../global/global.h"

// Declare global variables
bool memory_allocated;
Real *dev_conserved, *dev_conserved_half;
Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *Q_Lz, *Q_Rz, *F_x, *F_y, *F_z;
Real *ctElectricFields;
Real *eta_x, *eta_y, *eta_z, *etah_x, *etah_y, *etah_z;

// Arrays for potential in GPU: Will be set to NULL if not using GRAVITY
Real *dev_grav_potential;
Real *temp_potential;
Real *buffer_potential;

#endif  // CUDA
