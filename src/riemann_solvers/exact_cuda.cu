#include "hip/hip_runtime.h"
/*! \file exact_cuda.cu
 *  \brief Function definitions for the cuda exact Riemann solver.*/

#include <math.h>
#include <stdio.h>

#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../reconstruction/reconstruction.h"
#include "../riemann_solvers/exact_cuda.h"
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"

/*! \fn Calculate_Exact_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real
 * *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir, int
 * n_fields) \brief Exact Riemann solver based on the Fortran code given in
 * Sec. 4.9 of Toro (1999). */
__global__ void Calculate_Exact_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny,
                                            int nz, int n_ghost, Real gamma, int dir, int n_fields)
{
  // get a thread index
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int tid     = threadIdx.x + blockId * blockDim.x;
  int zid     = tid / (nx * ny);
  int yid     = (tid - zid * nx * ny) / nx;
  int xid     = tid - zid * nx * ny - yid * nx;

  int n_cells = nx * ny * nz;
  int o1, o2, o3;
  if (dir == 0) {
    o1 = 1;
    o2 = 2;
    o3 = 3;
  }
  if (dir == 1) {
    o1 = 2;
    o2 = 3;
    o3 = 1;
  }
  if (dir == 2) {
    o1 = 3;
    o2 = 1;
    o3 = 2;
  }

  reconstruction::InterfaceState left_state, right_state;
  Real cl, cr;          // sound speed (left, right)
  Real ds, vs, ps, Es;  // sample_CUDAd density, velocity, pressure, total
                        // energy
  Real vm, pm;          // velocity and pressure in the star region

#ifdef DE
  Real E_kin, E, dge;
#endif

  // Each thread executes the solver independently
  // if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid < ny && zid < nz)
  if (xid < nx && yid < ny && zid < nz) {
    // retrieve primitive variables
    left_state.density    = dev_bounds_L[tid];
    left_state.velocity.x = dev_bounds_L[o1 * n_cells + tid] / left_state.density;
    left_state.velocity.y = dev_bounds_L[o2 * n_cells + tid] / left_state.density;
    left_state.velocity.z = dev_bounds_L[o3 * n_cells + tid] / left_state.density;
#ifdef DE  // PRESSURE_DE
    E     = dev_bounds_L[4 * n_cells + tid];
    E_kin = 0.5 * left_state.density *
            (left_state.velocity.x * left_state.velocity.x + left_state.velocity.y * left_state.velocity.y +
             left_state.velocity.z * left_state.velocity.z);
    dge                 = dev_bounds_L[(n_fields - 1) * n_cells + tid];
    left_state.pressure = hydro_utilities::Get_Pressure_From_DE(E, E - E_kin, dge, gamma);
#else
    left_state.pressure = (dev_bounds_L[4 * n_cells + tid] - 0.5 * left_state.density *
                                                                 (left_state.velocity.x * left_state.velocity.x +
                                                                  left_state.velocity.y * left_state.velocity.y +
                                                                  left_state.velocity.z * left_state.velocity.z)) *
                          (gamma - 1.0);
#endif  // PRESSURE_DE
    left_state.pressure = fmax(left_state.pressure, (Real)TINY_NUMBER);
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      left_state.scalar_specific[i] = dev_bounds_L[(5 + i) * n_cells + tid] / left_state.density;
    }
#endif
#ifdef DE
    left_state.gas_energy_specific = dge / left_state.density;
#endif
    right_state.density    = dev_bounds_R[tid];
    right_state.velocity.x = dev_bounds_R[o1 * n_cells + tid] / right_state.density;
    right_state.velocity.y = dev_bounds_R[o2 * n_cells + tid] / right_state.density;
    right_state.velocity.z = dev_bounds_R[o3 * n_cells + tid] / right_state.density;
#ifdef DE  // PRESSURE_DE
    E     = dev_bounds_R[4 * n_cells + tid];
    E_kin = 0.5 * right_state.density *
            (right_state.velocity.x * right_state.velocity.x + right_state.velocity.y * right_state.velocity.y +
             right_state.velocity.z * right_state.velocity.z);
    dge                  = dev_bounds_R[(n_fields - 1) * n_cells + tid];
    right_state.pressure = hydro_utilities::Get_Pressure_From_DE(E, E - E_kin, dge, gamma);
#else
    right_state.pressure = (dev_bounds_R[4 * n_cells + tid] - 0.5 * right_state.density *
                                                                  (right_state.velocity.x * right_state.velocity.x +
                                                                   right_state.velocity.y * right_state.velocity.y +
                                                                   right_state.velocity.z * right_state.velocity.z)) *
                           (gamma - 1.0);
#endif  // PRESSURE_DE
    right_state.pressure = fmax(right_state.pressure, (Real)TINY_NUMBER);
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      right_state.scalar_specific[i] = dev_bounds_R[(5 + i) * n_cells + tid] / right_state.density;
    }
#endif
#ifdef DE
    right_state.gas_energy_specific = dge / right_state.density;
#endif

    // compute sounds speeds in left and right regions
    cl = sqrt(gamma * left_state.pressure / left_state.density);
    cr = sqrt(gamma * right_state.pressure / right_state.density);

    // test for the pressure positivity condition
    if ((2.0 / (gamma - 1.0)) * (cl + cr) <= (right_state.velocity.x - left_state.velocity.x)) {
      // the initial data is such that vacuum is generated
      printf("Vacuum is generated by the initial data.\n");
      printf("%f %f %f %f %f %f\n", left_state.density, left_state.velocity.x, left_state.pressure, right_state.density,
             right_state.velocity.x, right_state.pressure);
    }

    // Find the exact solution for pressure and velocity in the star region
    starpv_CUDA(&pm, &vm, left_state.density, left_state.velocity.x, left_state.pressure, cl, right_state.density,
                right_state.velocity.x, right_state.pressure, cr, gamma);

    // sample_CUDA the solution at the cell interface
    sample_CUDA(pm, vm, &ds, &vs, &ps, left_state.density, left_state.velocity.x, left_state.pressure, cl,
                right_state.density, right_state.velocity.x, right_state.pressure, cr, gamma);

    // calculate the fluxes through the cell interface
    dev_flux[tid]                = ds * vs;
    dev_flux[o1 * n_cells + tid] = ds * vs * vs + ps;
    if (vs >= 0) {
      dev_flux[o2 * n_cells + tid] = ds * vs * left_state.velocity.y;
      dev_flux[o3 * n_cells + tid] = ds * vs * left_state.velocity.z;
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        dev_flux[(5 + i) * n_cells + tid] = ds * vs * left_state.scalar_specific[i];
      }
#endif
#ifdef DE
      dev_flux[(n_fields - 1) * n_cells + tid] = ds * vs * left_state.gas_energy_specific;
#endif
      Es = (ps / (gamma - 1.0)) + 0.5 * ds *
                                      (vs * vs + left_state.velocity.y * left_state.velocity.y +
                                       left_state.velocity.z * left_state.velocity.z);
    } else {
      dev_flux[o2 * n_cells + tid] = ds * vs * right_state.velocity.y;
      dev_flux[o3 * n_cells + tid] = ds * vs * right_state.velocity.z;
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        dev_flux[(5 + i) * n_cells + tid] = ds * vs * right_state.scalar_specific[i];
      }
#endif
#ifdef DE
      dev_flux[(n_fields - 1) * n_cells + tid] = ds * vs * right_state.gas_energy_specific;
#endif
      Es = (ps / (gamma - 1.0)) + 0.5 * ds *
                                      (vs * vs + right_state.velocity.y * right_state.velocity.y +
                                       right_state.velocity.z * right_state.velocity.z);
    }
    dev_flux[4 * n_cells + tid] = (Es + ps) * vs;
  }
}

__device__ Real guessp_CUDA(Real dl, Real vxl, Real pl, Real cl, Real dr, Real vxr, Real pr, Real cr, Real gamma)
{
  // purpose:  to provide a guessed value for pressure
  //    pm in the Star Region. The choice is made
  //    according to adaptive Riemann solver using
  //    the PVRS and TSRS approximate Riemann
  //    solvers. See Sect. 9.5 of Toro (1999)

  Real gl, gr, ppv, pm;
  const Real TOL = 1.0e-6;

  // compute guess pressure from PVRS Riemann solver
  ppv = 0.5 * (pl + pr) + 0.125 * (vxl - vxr) * (dl + dr) * (cl + cr);

  if (ppv < 0.0) {
    ppv = 0.0;
  }
  // Two-Shock Riemann solver with PVRS as estimate
  gl = sqrt((2.0 / ((gamma + 1.0) * dl)) / (((gamma - 1.0) / (gamma + 1.0)) * pl + ppv));
  gr = sqrt((2.0 / ((gamma + 1.0) * dr)) / (((gamma - 1.0) / (gamma + 1.0)) * pr + ppv));
  pm = (gl * pl + gr * pr - (vxr - vxl)) / (gl + gr);

  if (pm < 0.0) {
    pm = TOL;
  }

  return pm;
}

__device__ void prefun_CUDA(Real *f, Real *fd, Real p, Real dk, Real pk, Real ck, Real gamma)
{
  // purpose:  to evaluate the pressure functions
  // fl and fr in the exact Riemann solver
  // and their first derivatives

  Real qrt;

  if (p <= pk) {
    // rarefaction wave
    *f  = (2.0 / (gamma - 1.0)) * ck * (powf(p / pk, (gamma - 1.0) / (2.0 * gamma)) - 1.0);
    *fd = (1.0 / (dk * ck)) * powf((p / pk), -((gamma + 1.0) / (2.0 * gamma)));
  } else {
    // shock wave
    qrt = sqrt(((2.0 / (gamma + 1.0)) / dk) / ((((gamma - 1.0) / (gamma + 1.0)) * pk) + p));
    *f  = (p - pk) * qrt;
    *fd = (1.0 - 0.5 * (p - pk) / ((((gamma - 1.0) / (gamma + 1.0)) * pk) + p)) * qrt;
  }
}

__device__ void starpv_CUDA(Real *p, Real *v, Real dl, Real vxl, Real pl, Real cl, Real dr, Real vxr, Real pr, Real cr,
                            Real gamma)
{
  // purpose:  Uses Newton-Raphson iteration
  // to compute the solution for pressure and
  // velocity in the Star Region

  const int nriter = 20;
  const Real TOL   = 1.0e-6;
  Real change, fl, fld, fr, frd, pold, pstart;

  // guessed value pstart is computed
  pstart = guessp_CUDA(dl, vxl, pl, cl, dr, vxr, pr, cr, gamma);
  pold   = pstart;

  int i = 0;
  for (i = 0; i <= nriter; i++) {
    prefun_CUDA(&fl, &fld, pold, dl, pl, cl, gamma);
    prefun_CUDA(&fr, &frd, pold, dr, pr, cr, gamma);
    *p     = pold - (fl + fr + vxr - vxl) / (fld + frd);
    change = 2.0 * fabs((*p - pold) / (*p + pold));

    if (change <= TOL) {
      break;
    }
    if (*p < 0.0) {
      *p = TOL;
    }
    pold = *p;
  }
  if (i > nriter) {
    // printf("Divergence in Newton-Raphson iteration. p = %e\n", *p);
  }

  // compute velocity in star region
  *v = 0.5 * (vxl + vxr + fr - fl);
}

__device__ void sample_CUDA(const Real pm, const Real vm, Real *d, Real *v, Real *p, Real dl, Real vxl, Real pl,
                            Real cl, Real dr, Real vxr, Real pr, Real cr, Real gamma)
{
  // purpose:  to sample the solution throughout the wave
  //   pattern. Pressure pm and velocity vm in the
  //   star region are known. Sampled
  //   values are d, v, p.

  Real c, sl, sr;

  if (vm >= 0)  // sampling point lies to the left of the contact discontinuity
  {
    if (pm <= pl)  // left rarefaction
    {
      if (vxl - cl >= 0)  // sampled point is in left data state
      {
        *d = dl;
        *v = vxl;
        *p = pl;
      } else {
        if (vm - cl * powf(pm / pl, (gamma - 1.0) / (2.0 * gamma)) < 0)  // sampled point is in star left state
        {
          *d = dl * powf(pm / pl, 1.0 / gamma);
          *v = vm;
          *p = pm;
        } else  // sampled point is inside left fan
        {
          c  = (2.0 / (gamma + 1.0)) * (cl + ((gamma - 1.0) / 2.0) * vxl);
          *v = c;
          *d = dl * powf(c / cl, 2.0 / (gamma - 1.0));
          *p = pl * powf(c / cl, 2.0 * gamma / (gamma - 1.0));
        }
      }
    } else  // left shock
    {
      sl = vxl - cl * sqrt(((gamma + 1.0) / (2.0 * gamma)) * (pm / pl) + ((gamma - 1.0) / (2.0 * gamma)));
      if (sl >= 0)  // sampled point is in left data state
      {
        *d = dl;
        *v = vxl;
        *p = pl;
      } else  // sampled point is in star left state
      {
        *d = dl * (pm / pl + ((gamma - 1.0) / (gamma + 1.0))) / ((pm / pl) * ((gamma - 1.0) / (gamma + 1.0)) + 1.0);
        *v = vm;
        *p = pm;
      }
    }
  } else  // sampling point lies to the right of the contact discontinuity
  {
    if (pm > pr)  // right shock
    {
      sr = vxr + cr * sqrt(((gamma + 1.0) / (2.0 * gamma)) * (pm / pr) + ((gamma - 1.0) / (2.0 * gamma)));
      if (sr <= 0)  // sampled point is in right data state
      {
        *d = dr;
        *v = vxr;
        *p = pr;
      } else  // sampled point is in star right state
      {
        *d = dr * (pm / pr + ((gamma - 1.0) / (gamma + 1.0))) / ((pm / pr) * ((gamma - 1.0) / (gamma + 1.0)) + 1.0);
        *v = vm;
        *p = pm;
      }
    } else  // right rarefaction
    {
      if (vxr + cr <= 0)  // sampled point is in right data state
      {
        *d = dr;
        *v = vxr;
        *p = pr;
      } else {
        if (vm + cr * powf(pm / pr, (gamma - 1.0) / (2.0 * gamma)) >= 0)  // sampled point is in star right state
        {
          *d = dr * powf(pm / pr, (1.0 / gamma));
          *v = vm;
          *p = pm;
        } else  // sampled point is inside right fan
        {
          c  = (2.0 / (gamma + 1.0)) * (cr - ((gamma - 1.0) / 2.0) * vxr);
          *v = -c;
          *d = dr * powf(c / cr, 2.0 / (gamma - 1.0));
          *p = pr * powf(c / cr, 2.0 * gamma / (gamma - 1.0));
        }
      }
    }
  }
}
