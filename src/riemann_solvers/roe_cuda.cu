#include "hip/hip_runtime.h"
/*! \file roe_cuda.cu
 *  \brief Function definitions for the cuda Roe Riemann solver.*/

#include <math.h>

#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../reconstruction/reconstruction.h"
#include "../riemann_solvers/roe_cuda.h"
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"

template <int reconstruction, uint direction>
__global__ void Calculate_Roe_Fluxes_CUDA(Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R,
                                          Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells,
                                          Real const gamma, Real const dx, Real const dt, int const n_fields)
{
  // get a thread index
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int tid     = threadIdx.x + blockId * blockDim.x;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(tid, nx, ny, xid, yid, zid);

  reconstruction::InterfaceState left_state, right_state;

  Real etah = 0.0;
  Real g1   = gamma - 1.0;
  Real Hl, Hr;
  Real sqrtdl, sqrtdr, vx, vy, vz, H;
  Real vsq, asq, a;
  Real lambda_m, lambda_0, lambda_p;
  Real f_d_l, f_mx_l, f_my_l, f_mz_l, f_E_l;
  Real f_d_r, f_mx_r, f_my_r, f_mz_r, f_E_r;
  Real del_d, del_mx, del_my, del_mz, del_E;
  Real a0, a1, a2, a3, a4;
  a0 = a1 = a2 = a3 = a4 = 0.0;
  Real sum_0, sum_1, sum_2, sum_3, sum_4;
  sum_0 = sum_1 = sum_2 = sum_3 = sum_4 = 0.0;
  Real test0, test1, test2, test3, test4;
  int hlle_flag = 0;

#ifdef SCALAR
  Real dscalarl[NSCALARS], dscalarr[NSCALARS], f_scalar_l[NSCALARS], f_scalar_r[NSCALARS];
#endif

  int o1, o2, o3;
  if constexpr (direction == 0) {
    o1 = 1;
    o2 = 2;
    o3 = 3;
  }
  if constexpr (direction == 1) {
    o1 = 2;
    o2 = 3;
    o3 = 1;
  }
  if constexpr (direction == 2) {
    o1 = 3;
    o2 = 1;
    o3 = 2;
  }

  // Thread guard to avoid overrun
  if (not reconstruction::Riemann_Thread_Guard<reconstruction>(nx, ny, nz, xid, yid, zid)) {
    // =========================
    // Load the interface states
    // =========================

    // Check if the reconstruction chosen is implemented as a device function yet
    if constexpr (reconstruction == reconstruction::Kind::pcm) {
      reconstruction::Reconstruct_Interface_States<reconstruction, direction>(
          dev_conserved, xid, yid, zid, nx, ny, n_cells, gamma, dx, dt, left_state, right_state);
    } else {
      // retrieve conserved variables
      left_state.density    = dev_bounds_L[tid];
      left_state.momentum.x = dev_bounds_L[o1 * n_cells + tid];
      left_state.momentum.y = dev_bounds_L[o2 * n_cells + tid];
      left_state.momentum.z = dev_bounds_L[o3 * n_cells + tid];
      left_state.energy     = dev_bounds_L[4 * n_cells + tid];
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        dscalarl[i] = dev_bounds_L[(5 + i) * n_cells + tid];
      }
#endif
#ifdef DE
      Real gas_energy_left = dev_bounds_L[(n_fields - 1) * n_cells + tid];
#endif

      right_state.density    = dev_bounds_R[tid];
      right_state.momentum.x = dev_bounds_R[o1 * n_cells + tid];
      right_state.momentum.y = dev_bounds_R[o2 * n_cells + tid];
      right_state.momentum.z = dev_bounds_R[o3 * n_cells + tid];
      right_state.energy     = dev_bounds_R[4 * n_cells + tid];
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        dscalarr[i] = dev_bounds_R[(5 + i) * n_cells + tid];
      }
#endif
#ifdef DE
      Real gas_energy_right = dev_bounds_R[(n_fields - 1) * n_cells + tid];
#endif

      // calculate primitive variables
      left_state.velocity.x = left_state.momentum.x / left_state.density;
      left_state.velocity.y = left_state.momentum.y / left_state.density;
      left_state.velocity.z = left_state.momentum.z / left_state.density;
#ifdef DE  // PRESSURE_DE
      Real E_kin = 0.5 * left_state.density *
                   (left_state.velocity.x * left_state.velocity.x + left_state.velocity.y * left_state.velocity.y +
                    left_state.velocity.z * left_state.velocity.z);
      left_state.pressure =
          hydro_utilities::Get_Pressure_From_DE(left_state.energy, left_state.energy - E_kin, gas_energy_left, gamma);
#else
      left_state.pressure = (left_state.energy - 0.5 * left_state.density *
                                                     (left_state.velocity.x * left_state.velocity.x +
                                                      left_state.velocity.y * left_state.velocity.y +
                                                      left_state.velocity.z * left_state.velocity.z)) *
                            (gamma - 1.0);
#endif  // PRESSURE_DE
      left_state.pressure = fmax(left_state.pressure, (Real)TINY_NUMBER);
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        left_state.scalar_specific[i] = dscalarl[i] / left_state.density;
      }
#endif
#ifdef DE
      left_state.gas_energy_specific = gas_energy_left / left_state.density;
#endif
      right_state.velocity.x = right_state.momentum.x / right_state.density;
      right_state.velocity.y = right_state.momentum.y / right_state.density;
      right_state.velocity.z = right_state.momentum.z / right_state.density;
#ifdef DE  // PRESSURE_DE
      E_kin = 0.5 * right_state.density *
              (right_state.velocity.x * right_state.velocity.x + right_state.velocity.y * right_state.velocity.y +
               right_state.velocity.z * right_state.velocity.z);
      right_state.pressure = hydro_utilities::Get_Pressure_From_DE(right_state.energy, right_state.energy - E_kin,
                                                                   gas_energy_right, gamma);
#else
      right_state.pressure = (right_state.energy - 0.5 * right_state.density *
                                                       (right_state.velocity.x * right_state.velocity.x +
                                                        right_state.velocity.y * right_state.velocity.y +
                                                        right_state.velocity.z * right_state.velocity.z)) *
                             (gamma - 1.0);
#endif  // PRESSURE_DE
      right_state.pressure = fmax(right_state.pressure, (Real)TINY_NUMBER);
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        right_state.scalar_specific[i] = dscalarr[i] / right_state.density;
      }
#endif
#ifdef DE
      right_state.gas_energy_specific = gas_energy_right / right_state.density;
#endif
    }
    // calculate the enthalpy in each cell
    Hl = (left_state.energy + left_state.pressure) / left_state.density;
    Hr = (right_state.energy + right_state.pressure) / right_state.density;

    // calculate averages of the variables needed for the Roe Jacobian
    // (see Stone et al., 2008, Eqn 65, or Toro 2009, 11.118)
    sqrtdl = sqrt(left_state.density);
    sqrtdr = sqrt(right_state.density);
    vx     = (sqrtdl * left_state.velocity.x + sqrtdr * right_state.velocity.x) / (sqrtdl + sqrtdr);
    vy     = (sqrtdl * left_state.velocity.y + sqrtdr * right_state.velocity.y) / (sqrtdl + sqrtdr);
    vz     = (sqrtdl * left_state.velocity.z + sqrtdr * right_state.velocity.z) / (sqrtdl + sqrtdr);
    H      = (sqrtdl * Hl + sqrtdr * Hr) / (sqrtdl + sqrtdr);

    // calculate the sound speed squared (Stone B2)
    vsq = (vx * vx + vy * vy + vz * vz);
    asq = g1 * fmax((H - 0.5 * vsq), TINY_NUMBER);
    a   = sqrt(asq);

    // calculate the averaged eigenvectors of the Roe matrix (Stone Eqn B2,
    // Toro 11.107)
    lambda_m = vx - a;
    lambda_0 = vx;
    lambda_p = vx + a;

    // calculate the fluxes for the left and right input states,
    // based on the average values in either cell
    f_d_l  = left_state.momentum.x;
    f_mx_l = left_state.momentum.x * left_state.velocity.x + left_state.pressure;
    f_my_l = left_state.momentum.x * left_state.velocity.y;
    f_mz_l = left_state.momentum.x * left_state.velocity.z;
    f_E_l  = (left_state.energy + left_state.pressure) * left_state.velocity.x;
#ifdef DE
    Real f_ge_l = left_state.momentum.x * left_state.gas_energy_specific;
#endif
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      f_scalar_l[i] = left_state.momentum.x * left_state.scalar_specific[i];
    }
#endif

    f_d_r  = right_state.momentum.x;
    f_mx_r = right_state.momentum.x * right_state.velocity.x + right_state.pressure;
    f_my_r = right_state.momentum.x * right_state.velocity.y;
    f_mz_r = right_state.momentum.x * right_state.velocity.z;
    f_E_r  = (right_state.energy + right_state.pressure) * right_state.velocity.x;
#ifdef DE
    Real f_ge_r = right_state.momentum.x * right_state.gas_energy_specific;
#endif
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      f_scalar_r[i] = right_state.momentum.x * right_state.scalar_specific[i];
    }
#endif

    // return upwind flux if flow is supersonic
    if (lambda_m >= 0.0) {
      dev_flux[tid]                = f_d_l;
      dev_flux[o1 * n_cells + tid] = f_mx_l;
      dev_flux[o2 * n_cells + tid] = f_my_l;
      dev_flux[o3 * n_cells + tid] = f_mz_l;
      dev_flux[4 * n_cells + tid]  = f_E_l;
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        dev_flux[(5 + i) * n_cells + tid] = f_scalar_l[i];
      }
#endif
#ifdef DE
      dev_flux[(n_fields - 1) * n_cells + tid] = f_ge_l;
#endif
      return;
    } else if (lambda_p <= 0.0) {
      dev_flux[tid]                = f_d_r;
      dev_flux[o1 * n_cells + tid] = f_mx_r;
      dev_flux[o2 * n_cells + tid] = f_my_r;
      dev_flux[o3 * n_cells + tid] = f_mz_r;
      dev_flux[4 * n_cells + tid]  = f_E_r;
#ifdef SCALAR
      for (int i = 0; i < NSCALARS; i++) {
        dev_flux[(5 + i) * n_cells + tid] = f_scalar_r[i];
      }
#endif
#ifdef DE
      dev_flux[(n_fields - 1) * n_cells + tid] = f_ge_r;
#endif
      return;
    }
    // otherwise calculate the Roe fluxes
    else {
      // calculate the difference in conserved variables across the cell
      // interface Stone Eqn 68
      del_d  = right_state.density - left_state.density;
      del_mx = right_state.momentum.x - left_state.momentum.x;
      del_my = right_state.momentum.y - left_state.momentum.y;
      del_mz = right_state.momentum.z - left_state.momentum.z;
      del_E  = right_state.energy - left_state.energy;

      // evaluate the flux function (Stone Eqn 66 & 67, Toro Eqn 11.29)

      Real Na    = 0.5 / asq;
      Real coeff = 0.0;

      // left eigenvector [0] * del_q
      a0 = del_d * Na * (0.5 * g1 * vsq + vx * a) - del_mx * Na * (g1 * vx + a) - del_my * Na * g1 * vy -
           del_mz * Na * g1 * vz + del_E * Na * g1;
      coeff = a0 * fmax(fabs(lambda_m), etah);
      sum_0 += coeff;
      sum_1 += coeff * (vx - a);
      sum_2 += coeff * vy;
      sum_3 += coeff * vz;
      sum_4 += coeff * (H - vx * a);
      // left eigenvector [1] * del_q
      a1    = -del_d * vy + del_my;
      coeff = a1 * fmax(fabs(lambda_0), etah);
      sum_2 += coeff;
      sum_4 += coeff * vy;
      // left eigenvector [2] * del_q
      a2    = -del_d * vz + del_mz;
      coeff = a2 * fmax(fabs(lambda_0), etah);
      sum_3 += coeff;
      sum_4 += coeff * vz;
      // left eigenvector [3] * del_q
      a3 = del_d * (1.0 - Na * g1 * vsq) + del_mx * g1 * vx / asq + del_my * g1 * vy / asq + del_mz * g1 * vz / asq -
           del_E * g1 / asq;
      coeff = a3 * fmax(fabs(lambda_0), etah);
      sum_0 += coeff;
      sum_1 += coeff * vx;
      sum_2 += coeff * vy;
      sum_3 += coeff * vz;
      sum_4 += coeff * 0.5 * vsq;
      // left eigenvector [4] * del_q
      a4 = del_d * Na * (0.5 * g1 * vsq - vx * a) - del_mx * Na * (g1 * vx - a) - del_my * Na * g1 * vy -
           del_mz * Na * g1 * vz + del_E * Na * g1;
      coeff = a4 * fmax(fabs(lambda_p), etah);
      sum_0 += coeff;
      sum_1 += coeff * (vx + a);
      sum_2 += coeff * vy;
      sum_3 += coeff * vz;
      sum_4 += coeff * (H + vx * a);

      // if density or pressure is negative, compute the HLLE fluxes
      // test intermediate states
      test0 = left_state.density + a0;
      test1 = left_state.momentum.x + a0 * (vx - a);
      test2 = left_state.momentum.y + a0 * vy;
      test3 = left_state.momentum.z + a0 * vz;
      test4 = left_state.energy + a0 * (H - vx * a);

      if (lambda_0 > lambda_m) {
        if (test0 <= 0.0) {
          hlle_flag = 1;
        }
        if (test4 - 0.5 * (test1 * test1 + test2 * test2 + test3 * test3) / test0 < 0.0) {
          hlle_flag = 2;
        }
      }

      test0 += a3 + a4;
      test1 += a3 * vx;
      test2 += a1 + a3 * vy;
      test3 += a2 + a3 * vz;
      test4 += a1 * vy + a2 * vz + a3 * 0.5 * vsq;

      if (lambda_p > lambda_0) {
        if (test0 <= 0.0) {
          hlle_flag = 1;
        }
        if (test4 - 0.5 * (test1 * test1 + test2 * test2 + test3 * test3) / test0 < 0.0) {
          hlle_flag = 2;
        }
      }

      // if pressure or density is negative, and we have not already returned
      // the supersonic fluxes, return the HLLE fluxes
      if (hlle_flag != 0) {
        Real cfl, cfr, bm, bp, tmp;

        // compute max and fmin wave speeds
        cfl = sqrt(gamma * left_state.pressure / left_state.density);    // sound speed in left state
        cfr = sqrt(gamma * right_state.pressure / right_state.density);  // sound speed in right state

        // take max/fmin of Roe eigenvalues and left and right sound speeds
        bm = fmin(fmin(lambda_m, left_state.velocity.x - cfl), (Real)0.0);
        bp = fmax(fmax(lambda_p, right_state.velocity.x + cfr), (Real)0.0);

        // compute left and right fluxes
        f_d_l = left_state.momentum.x - bm * left_state.density;
        f_d_r = right_state.momentum.x - bp * right_state.density;

        f_mx_l = left_state.momentum.x * (left_state.velocity.x - bm) + left_state.pressure;
        f_mx_r = right_state.momentum.x * (right_state.velocity.x - bp) + right_state.pressure;

        f_my_l = left_state.momentum.y * (left_state.velocity.x - bm);
        f_my_r = right_state.momentum.y * (right_state.velocity.x - bp);

        f_mz_l = left_state.momentum.z * (left_state.velocity.x - bm);
        f_mz_r = right_state.momentum.z * (right_state.velocity.x - bp);

        f_E_l = left_state.energy * (left_state.velocity.x - bm) + left_state.pressure * left_state.velocity.x;
        f_E_r = right_state.energy * (right_state.velocity.x - bp) + right_state.pressure * right_state.velocity.x;

#ifdef DE
        f_ge_l = left_state.gas_energy_specific * left_state.density * (left_state.velocity.x - bm);
        f_ge_r = right_state.gas_energy_specific * right_state.density * (right_state.velocity.x - bp);
#endif

#ifdef SCALAR
        for (int i = 0; i < NSCALARS; i++) {
          f_scalar_l[i] = dscalarl[i] * (left_state.velocity.x - bm);
          f_scalar_r[i] = dscalarr[i] * (right_state.velocity.x - bp);
        }
#endif

        // compute the HLLE flux at the interface
        tmp = 0.5 * (bp + bm) / (bp - bm);

        dev_flux[tid]                = 0.5 * (f_d_l + f_d_r) + (f_d_l - f_d_r) * tmp;
        dev_flux[o1 * n_cells + tid] = 0.5 * (f_mx_l + f_mx_r) + (f_mx_l - f_mx_r) * tmp;
        dev_flux[o2 * n_cells + tid] = 0.5 * (f_my_l + f_my_r) + (f_my_l - f_my_r) * tmp;
        dev_flux[o3 * n_cells + tid] = 0.5 * (f_mz_l + f_mz_r) + (f_mz_l - f_mz_r) * tmp;
        dev_flux[4 * n_cells + tid]  = 0.5 * (f_E_l + f_E_r) + (f_E_l - f_E_r) * tmp;
#ifdef SCALAR
        for (int i = 0; i < NSCALARS; i++) {
          dev_flux[(5 + i) * n_cells + tid] =
              0.5 * (f_scalar_l[i] + f_scalar_r[i]) + (f_scalar_l[i] - f_scalar_r[i]) * tmp;
        }
#endif
#ifdef DE
        dev_flux[(n_fields - 1) * n_cells + tid] = 0.5 * (f_ge_l + f_ge_r) + (f_ge_l - f_ge_r) * tmp;
#endif
        return;
      }
      // otherwise return the roe fluxes
      else {
        dev_flux[tid]                = 0.5 * (f_d_l + f_d_r - sum_0);
        dev_flux[o1 * n_cells + tid] = 0.5 * (f_mx_l + f_mx_r - sum_1);
        dev_flux[o2 * n_cells + tid] = 0.5 * (f_my_l + f_my_r - sum_2);
        dev_flux[o3 * n_cells + tid] = 0.5 * (f_mz_l + f_mz_r - sum_3);
        dev_flux[4 * n_cells + tid]  = 0.5 * (f_E_l + f_E_r - sum_4);
#ifdef SCALAR
        for (int i = 0; i < NSCALARS; i++) {
          if (dev_flux[tid] >= 0.0) {
            dev_flux[(5 + i) * n_cells + tid] = dev_flux[tid] * left_state.scalar_specific[i];
          } else {
            dev_flux[(5 + i) * n_cells + tid] = dev_flux[tid] * right_state.scalar_specific[i];
          }
        }
#endif
#ifdef DE
        if (dev_flux[tid] >= 0.0) {
          dev_flux[(n_fields - 1) * n_cells + tid] = dev_flux[tid] * left_state.gas_energy_specific;
        } else {
          dev_flux[(n_fields - 1) * n_cells + tid] = dev_flux[tid] * right_state.gas_energy_specific;
        }
#endif
      }
    }
  }
}

// Instantiate the templates we need
int const threads_per_block = 256;

template __global__ __launch_bounds__(threads_per_block) void Calculate_Roe_Fluxes_CUDA<reconstruction::Kind::pcm, 0>(
    Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real *dev_flux, int const nx,
    int const ny, int const nz, int const n_cells, Real const gamma, Real const dx, Real const dt, int const n_fields);
template __global__ __launch_bounds__(threads_per_block) void Calculate_Roe_Fluxes_CUDA<reconstruction::Kind::pcm, 1>(
    Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real *dev_flux, int const nx,
    int const ny, int const nz, int const n_cells, Real const gamma, Real const dx, Real const dt, int const n_fields);
template __global__ __launch_bounds__(threads_per_block) void Calculate_Roe_Fluxes_CUDA<reconstruction::Kind::pcm, 2>(
    Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real *dev_flux, int const nx,
    int const ny, int const nz, int const n_cells, Real const gamma, Real const dx, Real const dt, int const n_fields);

#ifndef PCM
template __global__ __launch_bounds__(threads_per_block) void Calculate_Roe_Fluxes_CUDA<
    reconstruction::Kind::chosen, 0>(Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R,
                                     Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells,
                                     Real const gamma, Real const dx, Real const dt, int const n_fields);
template __global__ __launch_bounds__(threads_per_block) void Calculate_Roe_Fluxes_CUDA<
    reconstruction::Kind::chosen, 1>(Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R,
                                     Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells,
                                     Real const gamma, Real const dx, Real const dt, int const n_fields);
template __global__ __launch_bounds__(threads_per_block) void Calculate_Roe_Fluxes_CUDA<
    reconstruction::Kind::chosen, 2>(Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R,
                                     Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells,
                                     Real const gamma, Real const dx, Real const dt, int const n_fields);
#endif  // PCM