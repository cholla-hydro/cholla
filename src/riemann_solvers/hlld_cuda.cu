#include "hip/hip_runtime.h"
/*!
 * \file hlld_cuda.cu
 * \author Robert 'Bob' Caddy (rvc@pitt.edu)
 * \brief Contains the implementation of the HLLD solver from Miyoshi & Kusano
 * 2005 "A multi-state HLL approximate Riemann solver for ideal
 * magnetohydrodynamics", hereafter referred to as M&K 2005
 *
 */

// External Includes

// Local Includes
#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../grid/grid_enum.h"
#include "../riemann_solvers/hlld_cuda.h"
#include "../utils/cuda_utilities.h"
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"
#include "../utils/math_utilities.h"
#include "../utils/mhd_utilities.h"

#ifdef DE  // PRESSURE_DE
  #include "../utils/hydro_utilities.h"
#endif  // DE

#ifdef MHD
namespace mhd
{
// =========================================================================
template <int reconstruction, uint direction>
__global__ void Calculate_HLLD_Fluxes_CUDA(Real const *dev_conserved, Real const *dev_bounds_L,
                                           Real const *dev_bounds_R, Real const *dev_magnetic_face, Real *dev_flux,
                                           int const nx, int const ny, int const nz, int const n_cells,
                                           Real const gamma, Real const dx, Real const dt, int const n_fields)
{
  // get a thread index
  int const threadId = threadIdx.x + blockIdx.x * blockDim.x;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(threadId, nx, ny, xid, yid, zid);

  // Thread guard to avoid overrun
  if (reconstruction::Riemann_Thread_Guard<reconstruction>(nx, ny, nz, xid, yid, zid)) {
    return;
  }

  // Offsets & indices
  int o1, o2, o3;
  if constexpr (direction == 0) {
    o1 = grid_enum::momentum_x;
    o2 = grid_enum::momentum_y;
    o3 = grid_enum::momentum_z;
  } else if constexpr (direction == 1) {
    o1 = grid_enum::momentum_y;
    o2 = grid_enum::momentum_z;
    o3 = grid_enum::momentum_x;
  } else if constexpr (direction == 2) {
    o1 = grid_enum::momentum_z;
    o2 = grid_enum::momentum_x;
    o3 = grid_enum::momentum_y;
  }

  // ============================
  // Retrieve state variables
  // ============================
  // The magnetic field in the X-direction
  Real const magneticX = dev_magnetic_face[threadId];

  reconstruction::InterfaceState stateL, stateR;
  // Check if the reconstruction chosen is implemented as a device function yet
  if constexpr (reconstruction == reconstruction::Kind::pcm or reconstruction == reconstruction::Kind::plmc) {
    reconstruction::Reconstruct_Interface_States<reconstruction, direction>(
        dev_conserved, xid, yid, zid, nx, ny, n_cells, gamma, dx, dt, stateL, stateR, magneticX);
  } else {
    stateL = mhd::internal::loadState(dev_bounds_L, magneticX, gamma, threadId, n_cells, o1, o2, o3);
    stateR = mhd::internal::loadState(dev_bounds_R, magneticX, gamma, threadId, n_cells, o1, o2, o3);
  }

  // Compute the approximate Left and Right wave speeds
  mhd::internal::Speeds speed = mhd::internal::approximateLRWaveSpeeds(stateL, stateR, magneticX, gamma);

  // =================================================================
  // Compute the fluxes in the non-star states
  // =================================================================
  // Left state
  mhd::internal::Flux fluxL = mhd::internal::nonStarFluxes(stateL, magneticX);

  // If we're in the L state then assign fluxes and return.
  // In this state the flow is supersonic
  // M&K 2005 equation 66
  if (speed.L > 0.0) {
    mhd::internal::returnFluxes(threadId, o1, o2, o3, n_cells, dev_flux, fluxL, stateL);
    return;
  }
  // Right state
  mhd::internal::Flux fluxR = mhd::internal::nonStarFluxes(stateR, magneticX);

  // If we're in the R state then assign fluxes and return.
  // In this state the flow is supersonic
  // M&K 2005 equation 66
  if (speed.R < 0.0) {
    mhd::internal::returnFluxes(threadId, o1, o2, o3, n_cells, dev_flux, fluxR, stateR);
    return;
  }

  // =================================================================
  // Compute the fluxes in the star states
  // =================================================================
  // Shared quantities:
  // - velocityStarX = speedM
  // - totalPrssureStar is the same on both sides
  speed.M                      = approximateMiddleWaveSpeed(stateL, stateR, speed);
  Real const totalPressureStar = mhd::internal::starTotalPressure(stateL, stateR, speed);

  // Left star state
  mhd::internal::StarState const starStateL =
      mhd::internal::computeStarState(stateL, speed, speed.L, magneticX, totalPressureStar);

  // Left star speed
  speed.LStar = mhd::internal::approximateStarWaveSpeed(starStateL, speed, magneticX, -1);

  // If we're in the L* state then assign fluxes and return.
  // In this state the flow is subsonic
  // M&K 2005 equation 66
  if (speed.LStar > 0.0 and speed.L <= 0.0) {
    fluxL = mhd::internal::starFluxes(starStateL, stateL, fluxL, speed, speed.L);
    mhd::internal::returnFluxes(threadId, o1, o2, o3, n_cells, dev_flux, fluxL, stateL);
    return;
  }

  // Right star state
  mhd::internal::StarState const starStateR =
      mhd::internal::computeStarState(stateR, speed, speed.R, magneticX, totalPressureStar);

  // Right star speed
  speed.RStar = mhd::internal::approximateStarWaveSpeed(starStateR, speed, magneticX, 1);

  // If we're in the R* state then assign fluxes and return.
  // In this state the flow is subsonic
  // M&K 2005 equation 66
  if (speed.RStar <= 0.0 and speed.R >= 0.0) {
    fluxR = mhd::internal::starFluxes(starStateR, stateR, fluxR, speed, speed.R);
    mhd::internal::returnFluxes(threadId, o1, o2, o3, n_cells, dev_flux, fluxR, stateR);
    return;
  }

  // =================================================================
  // Compute the fluxes in the double star states
  // =================================================================
  mhd::internal::DoubleStarState const doubleStarState =
      mhd::internal::computeDoubleStarState(starStateL, starStateR, magneticX, totalPressureStar, speed);

  // Compute and return L** fluxes
  // M&K 2005 equation 66
  if (speed.M > 0.0 and speed.LStar <= 0.0) {
    fluxL = mhd::internal::computeDoubleStarFluxes(doubleStarState, doubleStarState.energyL, starStateL, stateL, fluxL,
                                                   speed, speed.L, speed.LStar);
    mhd::internal::returnFluxes(threadId, o1, o2, o3, n_cells, dev_flux, fluxL, stateL);
    return;
  }
  // Compute and return R** fluxes
  // M&K 2005 equation 66
  if (speed.RStar > 0.0 and speed.M <= 0.0) {
    fluxR = mhd::internal::computeDoubleStarFluxes(doubleStarState, doubleStarState.energyR, starStateR, stateR, fluxR,
                                                   speed, speed.R, speed.RStar);
    mhd::internal::returnFluxes(threadId, o1, o2, o3, n_cells, dev_flux, fluxR, stateR);
    return;
  }
}
// =========================================================================

namespace internal
{
// =====================================================================
__device__ __host__ reconstruction::InterfaceState loadState(Real const *interfaceArr, Real const &magneticX,
                                                             Real const &gamma, int const &threadId, int const &n_cells,
                                                             int const &o1, int const &o2, int const &o3)
{
  reconstruction::InterfaceState state;
  state.density    = interfaceArr[threadId + n_cells * grid_enum::density];
  state.density    = fmax(state.density, (Real)TINY_NUMBER);
  state.velocity.x = interfaceArr[threadId + n_cells * o1] / state.density;
  state.velocity.y = interfaceArr[threadId + n_cells * o2] / state.density;
  state.velocity.z = interfaceArr[threadId + n_cells * o3] / state.density;
  state.energy     = interfaceArr[threadId + n_cells * grid_enum::Energy];
  state.energy     = fmax(state.energy, (Real)TINY_NUMBER);
  state.magnetic.y = interfaceArr[threadId + n_cells * grid_enum::Q_x_magnetic_y];
  state.magnetic.z = interfaceArr[threadId + n_cells * grid_enum::Q_x_magnetic_z];

  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    state.scalar_specific[i] = interfaceArr[threadId + n_cells * (grid_enum::scalar + i)] / state.density;
  }
  #endif  // SCALAR
  #ifdef DE
  state.gas_energy_specific = interfaceArr[threadId + n_cells * grid_enum::GasEnergy] / state.density;

  Real energyNonThermal = hydro_utilities::Calc_Kinetic_Energy_From_Velocity(state.density, state.velocity.x,
                                                                             state.velocity.y, state.velocity.z) +
                          mhd::utils::computeMagneticEnergy(magneticX, state.magnetic.y, state.magnetic.z);

  state.pressure = fmax(hydro_utilities::Get_Pressure_From_DE(state.energy, state.energy - energyNonThermal,
                                                              state.gas_energy_specific * state.density, gamma),
                        (Real)TINY_NUMBER);
  #else
  // Note that this function does the positive pressure check
  // internally
  state.pressure = mhd::internal::Calc_Pressure_Primitive(state, magneticX, gamma);
  #endif  // DE

  state.total_pressure =
      mhd::utils::computeTotalPressure(state.pressure, magneticX, state.magnetic.y, state.magnetic.z);

  return state;
}
// =====================================================================

// =====================================================================
__device__ __host__ mhd::internal::Speeds approximateLRWaveSpeeds(reconstruction::InterfaceState const &stateL,
                                                                  reconstruction::InterfaceState const &stateR,
                                                                  Real const &magneticX, Real const &gamma)
{
  // Get the fast magnetosonic wave speeds
  Real magSonicL = mhd::utils::fastMagnetosonicSpeed(stateL.density, stateL.pressure, magneticX, stateL.magnetic.y,
                                                     stateL.magnetic.z, gamma);
  Real magSonicR = mhd::utils::fastMagnetosonicSpeed(stateR.density, stateR.pressure, magneticX, stateR.magnetic.y,
                                                     stateR.magnetic.z, gamma);

  // Compute the S_L and S_R wave speeds.
  // Version suggested by Miyoshi & Kusano 2005 and used in Athena
  // M&K 2005 equation 67
  Real magSonicMax = fmax(magSonicL, magSonicR);
  mhd::internal::Speeds speed;
  speed.L = fmin(stateL.velocity.x, stateR.velocity.x) - magSonicMax;
  speed.R = fmax(stateL.velocity.x, stateR.velocity.x) + magSonicMax;

  return speed;
}
// =====================================================================

// =====================================================================
__device__ __host__ Real approximateMiddleWaveSpeed(reconstruction::InterfaceState const &stateL,
                                                    reconstruction::InterfaceState const &stateR,
                                                    mhd::internal::Speeds const &speed)
{
  // Compute the S_M wave speed
  // M&K 2005 equation 38
  Real const speed_r_diff = speed.R - stateR.velocity.x;
  Real const speed_l_diff = speed.L - stateL.velocity.x;

  return  // Numerator
      (speed_r_diff * stateR.density * stateR.velocity.x - speed_l_diff * stateL.density * stateL.velocity.x -
       stateR.total_pressure + stateL.total_pressure) /
      // Denominator
      (speed_r_diff * stateR.density - speed_l_diff * stateL.density);
}
// =====================================================================

// =====================================================================
__device__ __host__ Real approximateStarWaveSpeed(mhd::internal::StarState const &starState,
                                                  mhd::internal::Speeds const &speed, Real const &magneticX,
                                                  Real const &side)
{
  // Compute the S_L^* and S_R^* wave speeds
  // M&K 2005 equation 51
  return speed.M + side * mhd::utils::alfvenSpeed(magneticX, starState.density);
}
// =====================================================================

// =====================================================================
__device__ __host__ mhd::internal::Flux nonStarFluxes(reconstruction::InterfaceState const &state,
                                                      Real const &magneticX)
{
  mhd::internal::Flux flux;
  // M&K 2005 equation 2
  flux.density = state.density * state.velocity.x;

  flux.momentumX = flux.density * state.velocity.x + state.total_pressure - magneticX * magneticX;
  flux.momentumY = flux.density * state.velocity.y - magneticX * state.magnetic.y;
  flux.momentumZ = flux.density * state.velocity.z - magneticX * state.magnetic.z;

  flux.magneticY = state.magnetic.y * state.velocity.x - magneticX * state.velocity.y;
  flux.magneticZ = state.magnetic.z * state.velocity.x - magneticX * state.velocity.z;

  // Group transverse terms for FP associative symmetry
  flux.energy = state.velocity.x * (state.energy + state.total_pressure) -
                magneticX * (state.velocity.x * magneticX +
                             ((state.velocity.y * state.magnetic.y) + (state.velocity.z * state.magnetic.z)));

  return flux;
}
// =====================================================================

// =====================================================================
__device__ __host__ void returnFluxes(int const &threadId, int const &o1, int const &o2, int const &o3,
                                      int const &n_cells, Real *dev_flux, mhd::internal::Flux const &flux,
                                      reconstruction::InterfaceState const &state)
{
  // Note that the direction of the grid_enum::fluxX_magnetic_DIR is the
  // direction of the electric field that the magnetic flux is, not the magnetic
  // flux
  dev_flux[threadId + n_cells * grid_enum::density]          = flux.density;
  dev_flux[threadId + n_cells * o1]                          = flux.momentumX;
  dev_flux[threadId + n_cells * o2]                          = flux.momentumY;
  dev_flux[threadId + n_cells * o3]                          = flux.momentumZ;
  dev_flux[threadId + n_cells * grid_enum::Energy]           = flux.energy;
  dev_flux[threadId + n_cells * grid_enum::fluxX_magnetic_z] = flux.magneticY;
  dev_flux[threadId + n_cells * grid_enum::fluxX_magnetic_y] = flux.magneticZ;

  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    dev_flux[threadId + n_cells * (grid_enum::scalar + i)] = state.scalar_specific[i] * flux.density;
  }
  #endif  // SCALAR
  #ifdef DE
  dev_flux[threadId + n_cells * grid_enum::GasEnergy] = state.gas_energy_specific * flux.density;
  #endif  // DE
}
// =====================================================================

// =====================================================================
__device__ __host__ Real starTotalPressure(reconstruction::InterfaceState const &stateL,
                                           reconstruction::InterfaceState const &stateR,
                                           mhd::internal::Speeds const &speed)
{
  // M&K 2005 equation 41
  return  // Numerator
      (stateR.density * stateL.total_pressure * (speed.R - stateR.velocity.x) -
       stateL.density * stateR.total_pressure * (speed.L - stateL.velocity.x) +
       stateL.density * stateR.density * (speed.R - stateR.velocity.x) * (speed.L - stateL.velocity.x) *
           (stateR.velocity.x - stateL.velocity.x)) /
      // Denominator
      (stateR.density * (speed.R - stateR.velocity.x) - stateL.density * (speed.L - stateL.velocity.x));
}
// =====================================================================

// =====================================================================
__device__ __host__ mhd::internal::StarState computeStarState(reconstruction::InterfaceState const &state,
                                                              mhd::internal::Speeds const &speed, Real const &speedSide,
                                                              Real const &magneticX, Real const &totalPressureStar)
{
  mhd::internal::StarState starState;

  // Compute the densities in the star state
  // M&K 2005 equation 43
  starState.density = state.density * (speedSide - state.velocity.x) / (speedSide - speed.M);

  // Check for and handle the degenerate case
  // Explained at the top of page 326 in M&K 2005
  if (fabs(state.density * (speedSide - state.velocity.x) * (speedSide - speed.M) - (magneticX * magneticX)) <
      totalPressureStar * mhd::internal::_hlldSmallNumber) {
    starState.velocityY = state.velocity.y;
    starState.velocityZ = state.velocity.z;
    starState.magneticY = state.magnetic.y;
    starState.magneticZ = state.magnetic.z;
  } else {
    // Denominator for M&K 2005 equations 44-47
    Real const denom = state.density * (speedSide - state.velocity.x) * (speedSide - speed.M) - (magneticX * magneticX);

    // Compute the velocity and magnetic field in the star state
    // M&K 2005 equations 44 & 46
    Real coef           = magneticX * (speed.M - state.velocity.x) / denom;
    starState.velocityY = state.velocity.y - state.magnetic.y * coef;
    starState.velocityZ = state.velocity.z - state.magnetic.z * coef;

    // M&K 2005 equations 45 & 47
    Real tmpPower       = (speedSide - state.velocity.x);
    tmpPower            = tmpPower * tmpPower;
    coef                = (state.density * tmpPower - (magneticX * magneticX)) / denom;
    starState.magneticY = state.magnetic.y * coef;
    starState.magneticZ = state.magnetic.z * coef;
  }

  // M&K 2005 equation 48
  starState.energy = (state.energy * (speedSide - state.velocity.x) - state.total_pressure * state.velocity.x +
                      totalPressureStar * speed.M +
                      magneticX * (math_utils::dotProduct(state.velocity.x, state.velocity.y, state.velocity.z,
                                                          magneticX, state.magnetic.y, state.magnetic.z) -
                                   math_utils::dotProduct(speed.M, starState.velocityY, starState.velocityZ, magneticX,
                                                          starState.magneticY, starState.magneticZ))) /
                     (speedSide - speed.M);

  return starState;
}
// =====================================================================

// =====================================================================
__device__ __host__ mhd::internal::Flux starFluxes(mhd::internal::StarState const &starState,
                                                   reconstruction::InterfaceState const &state,
                                                   mhd::internal::Flux const &flux, mhd::internal::Speeds const &speed,
                                                   Real const &speedSide)
{
  mhd::internal::Flux starFlux;

  // Now compute the star state fluxes
  // M&K 2005 equations 64
  starFlux.density   = flux.density + speedSide * (starState.density - state.density);
  starFlux.momentumX = flux.momentumX + speedSide * (starState.density * speed.M - state.density * state.velocity.x);
  starFlux.momentumY =
      flux.momentumY + speedSide * (starState.density * starState.velocityY - state.density * state.velocity.y);
  starFlux.momentumZ =
      flux.momentumZ + speedSide * (starState.density * starState.velocityZ - state.density * state.velocity.z);
  starFlux.energy    = flux.energy + speedSide * (starState.energy - state.energy);
  starFlux.magneticY = flux.magneticY + speedSide * (starState.magneticY - state.magnetic.y);
  starFlux.magneticZ = flux.magneticZ + speedSide * (starState.magneticZ - state.magnetic.z);

  return starFlux;
}
// =====================================================================

// =====================================================================
__device__ __host__ mhd::internal::DoubleStarState computeDoubleStarState(mhd::internal::StarState const &starStateL,
                                                                          mhd::internal::StarState const &starStateR,
                                                                          Real const &magneticX,
                                                                          Real const &totalPressureStar,
                                                                          mhd::internal::Speeds const &speed)
{
  mhd::internal::DoubleStarState doubleStarState;

  // if Bx is zero then just return the star state
  // Explained at the top of page 328 in M&K 2005. Essentially when
  // magneticX is 0 this reduces to the HLLC solver
  if (0.5 * (magneticX * magneticX) < mhd::internal::_hlldSmallNumber * totalPressureStar) {
    if (speed.M >= 0.0) {
      // We're in the L** state but Bx=0 so return L* state
      doubleStarState.velocityY = starStateL.velocityY;
      doubleStarState.velocityZ = starStateL.velocityZ;
      doubleStarState.magneticY = starStateL.magneticY;
      doubleStarState.magneticZ = starStateL.magneticZ;
      doubleStarState.energyL   = starStateL.energy;
    } else {
      // We're in the L** state but Bx=0 so return L* state
      doubleStarState.velocityY = starStateR.velocityY;
      doubleStarState.velocityZ = starStateR.velocityZ;
      doubleStarState.magneticY = starStateR.magneticY;
      doubleStarState.magneticZ = starStateR.magneticZ;
      doubleStarState.energyR   = starStateR.energy;
    }
  } else {
    // Setup some variables we'll need later
    Real sqrtDL           = sqrt(starStateL.density);
    Real sqrtDR           = sqrt(starStateR.density);
    Real inverseDensities = 1.0 / (sqrtDL + sqrtDR);
    Real magXSign         = copysign(1.0, magneticX);

    // All we need to do now is compute the transverse velocities
    // and magnetic fields along with the energy

    // Double Star velocities
    // M&K 2005 equations 59 & 60
    doubleStarState.velocityY = inverseDensities * (sqrtDL * starStateL.velocityY + sqrtDR * starStateR.velocityY +
                                                    magXSign * (starStateR.magneticY - starStateL.magneticY));
    doubleStarState.velocityZ = inverseDensities * (sqrtDL * starStateL.velocityZ + sqrtDR * starStateR.velocityZ +
                                                    magXSign * (starStateR.magneticZ - starStateL.magneticZ));

    // Double star magnetic fields
    // M&K 2005 equations 61 & 62
    doubleStarState.magneticY =
        inverseDensities * (sqrtDL * starStateR.magneticY + sqrtDR * starStateL.magneticY +
                            magXSign * (sqrtDL * sqrtDR) * (starStateR.velocityY - starStateL.velocityY));
    doubleStarState.magneticZ =
        inverseDensities * (sqrtDL * starStateR.magneticZ + sqrtDR * starStateL.magneticZ +
                            magXSign * (sqrtDL * sqrtDR) * (starStateR.velocityZ - starStateL.velocityZ));

    // Double star energy
    Real velDblStarDotMagDblStar =
        math_utils::dotProduct(speed.M, doubleStarState.velocityY, doubleStarState.velocityZ, magneticX,
                               doubleStarState.magneticY, doubleStarState.magneticZ);
    // M&K 2005 equation 63
    doubleStarState.energyL =
        starStateL.energy - sqrtDL * magXSign *
                                (math_utils::dotProduct(speed.M, starStateL.velocityY, starStateL.velocityZ, magneticX,
                                                        starStateL.magneticY, starStateL.magneticZ) -
                                 velDblStarDotMagDblStar);
    doubleStarState.energyR =
        starStateR.energy + sqrtDR * magXSign *
                                (math_utils::dotProduct(speed.M, starStateR.velocityY, starStateR.velocityZ, magneticX,
                                                        starStateR.magneticY, starStateR.magneticZ) -
                                 velDblStarDotMagDblStar);
  }

  return doubleStarState;
}
// =====================================================================

// =====================================================================
__device__ __host__ mhd::internal::Flux computeDoubleStarFluxes(mhd::internal::DoubleStarState const &doubleStarState,
                                                                Real const &doubleStarStateEnergy,
                                                                mhd::internal::StarState const &starState,
                                                                reconstruction::InterfaceState const &state,
                                                                mhd::internal::Flux const &flux,
                                                                mhd::internal::Speeds const &speed,
                                                                Real const &speedSide, Real const &speedSideStar)
{
  mhd::internal::Flux doubleStarFlux;

  Real const speed_diff = speedSideStar - speedSide;

  // M&K 2005 equation 65
  doubleStarFlux.density =
      flux.density - speedSide * state.density - speed_diff * starState.density + speedSideStar * starState.density;

  doubleStarFlux.momentumX = flux.momentumX - speedSide * (state.density * state.velocity.x) -
                             speed_diff * (starState.density * speed.M) + speedSideStar * (starState.density * speed.M);
  doubleStarFlux.momentumY = flux.momentumY - speedSide * (state.density * state.velocity.y) -
                             speed_diff * (starState.density * starState.velocityY) +
                             speedSideStar * (starState.density * doubleStarState.velocityY);
  doubleStarFlux.momentumZ = flux.momentumZ - speedSide * (state.density * state.velocity.z) -
                             speed_diff * (starState.density * starState.velocityZ) +
                             speedSideStar * (starState.density * doubleStarState.velocityZ);
  doubleStarFlux.energy =
      flux.energy - speedSide * state.energy - speed_diff * starState.energy + speedSideStar * doubleStarStateEnergy;
  doubleStarFlux.magneticY = flux.magneticY - speedSide * state.magnetic.y - speed_diff * starState.magneticY +
                             speedSideStar * doubleStarState.magneticY;
  doubleStarFlux.magneticZ = flux.magneticZ - speedSide * state.magnetic.z - speed_diff * starState.magneticZ +
                             speedSideStar * doubleStarState.magneticZ;

  return doubleStarFlux;
}
// =====================================================================

}  // namespace internal
}  // end namespace mhd

// Instantiate the templates we need
int const tpb_pcm = 512;

template __global__ __launch_bounds__(tpb_pcm) void mhd::Calculate_HLLD_Fluxes_CUDA<reconstruction::Kind::pcm, 0>(
    Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real const *dev_magnetic_face,
    Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells, Real const gamma, Real const dx,
    Real const dt, int const n_fields);
template __global__ __launch_bounds__(tpb_pcm) void mhd::Calculate_HLLD_Fluxes_CUDA<reconstruction::Kind::pcm, 1>(
    Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real const *dev_magnetic_face,
    Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells, Real const gamma, Real const dx,
    Real const dt, int const n_fields);
template __global__ __launch_bounds__(tpb_pcm) void mhd::Calculate_HLLD_Fluxes_CUDA<reconstruction::Kind::pcm, 2>(
    Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real const *dev_magnetic_face,
    Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells, Real const gamma, Real const dx,
    Real const dt, int const n_fields);

  #ifndef PCM
int const tpb_higher_order = 256;
template __global__
    __launch_bounds__(tpb_higher_order) void mhd::Calculate_HLLD_Fluxes_CUDA<reconstruction::Kind::chosen, 0>(
        Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real const *dev_magnetic_face,
        Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells, Real const gamma, Real const dx,
        Real const dt, int const n_fields);
template __global__
    __launch_bounds__(tpb_higher_order) void mhd::Calculate_HLLD_Fluxes_CUDA<reconstruction::Kind::chosen, 1>(
        Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real const *dev_magnetic_face,
        Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells, Real const gamma, Real const dx,
        Real const dt, int const n_fields);
template __global__
    __launch_bounds__(tpb_higher_order) void mhd::Calculate_HLLD_Fluxes_CUDA<reconstruction::Kind::chosen, 2>(
        Real const *dev_conserved, Real const *dev_bounds_L, Real const *dev_bounds_R, Real const *dev_magnetic_face,
        Real *dev_flux, int const nx, int const ny, int const nz, int const n_cells, Real const gamma, Real const dx,
        Real const dt, int const n_fields);
  #endif  // PCM
#endif    // MHD