
#include <hip/hip_runtime.h>
/*! \file cooling_cuda.cu
 *  \brief Functions to calculate cooling rate for a given rho, P, dt. */

#ifdef CUDA
#ifdef COOLING_GPU

#include<cuda.h>
#include<math.h>
#include<io.h>
#include"global.h"
#include"cooling_cuda.h"



/*! \fn void cooling_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dt, Real gamma)
 *  \brief When passed an array of conserved variables and a timestep, adjust the value
           of the total energy for each cell according to the specified cooling function. */
__global__ void cooling_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dt, Real gamma)
{
  int n_cells = nx*ny*nz;
  
  Real d, vx, vy, vz, p, E, E_old;
  Real n, T, T_init, T_init_p;
  Real del_T, dt_sub;
  Real cool; //cooling rate per volume, erg/s/cm^3
  #ifdef DE
  Real ge, T_init_ge;
  #endif

  // get a thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int id;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost) {
  //if (xid < nx && yid < ny && zid < nz) {

    // load values of density and pressure
    id = xid + yid*nx + zid*nx*ny;
    d  =  dev_conserved[            id];
    vx =  dev_conserved[1*n_cells + id] / d;
    vy =  dev_conserved[2*n_cells + id] / d;
    vz =  dev_conserved[3*n_cells + id] / d;
    E  =  dev_conserved[4*n_cells + id];
    p  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    p  = fmax(p, (Real) TINY_NUMBER);
    #ifdef DE
    ge = dev_conserved[5*n_cells + id] / d;
    p  = d * ge * (gamma - 1.0);
    #endif
    
    // calculate the number density of the gas (in cgs)
    n = d*DENSITY_UNIT / MP;

    // calculate the temperature of the gas
    T_init_p = p*PRESSURE_UNIT/ (n*KB);
    T_init = T_init_p;
    #ifdef DE
    T_init_ge = ge*(gamma-1.0)*SP_ENERGY_UNIT*MP/KB;
    T_init = T_init_ge;
    #endif
    //if (xid == 130 && yid == 6 && zid ==81) printf("%f %f\n", T_init_p, T_init_ge);

    // only allow cooling above 10^4 K
    if (T_init > 1e4 && T_init < 1e9) {
    // calculate cooling rate per volume
    T = T_init;

    if (T > 1e4) {
      cool = Schure_cool(n, T); 
    } else {
      cool = KI_cool(n, T); 
    }
    

    // calculate change in temperature given dt
    del_T = cool*dt*TIME_UNIT*(gamma-1.0)/(n*KB);
    //printf("%d %f %f %f\n", tid, cool, del_T, del_T/T);

    // limit change in temperature to 5%
    while (del_T/T > 0.05) {
      // what dt gives del_T = 0.1*T?
      dt_sub = 0.05*T*n*KB/(cool*TIME_UNIT*(gamma-1.0));
      // apply that dt
      T -= cool*dt_sub*TIME_UNIT*(gamma-1.0)/(n*KB);
      // how much time is left from the original timestep?
      dt -= dt_sub;
      // calculate cooling again
      if (T > 1e4) {
        cool = Schure_cool(n, T);
      } else {
        cool = KI_cool(n, T);
      }
      // calculate new change in temperature
      del_T = cool*dt*TIME_UNIT*(gamma-1.0)/(n*KB);
    }

    // calculate final temperature
    T -= del_T;

    // adjust value of energy based on total change in temperature
    del_T = T_init - T; // total change in T
    E_old = E;
    E -= n*KB*del_T / ((gamma-1.0)*ENERGY_UNIT);
    if (E < 0.0) printf("%3d %3d %3d Negative E after cooling. %f %f %f %f %f\n", xid, yid, zid, del_T, T_init, E_old, n, E);
    #ifdef DE
    ge -= KB*del_T / (MP*(gamma-1.0)*SP_ENERGY_UNIT);
    if (ge < 0.0) printf("%3d %3d %3d Negative ge after cooling. %f %f %f %f %f %f\n", xid, yid, zid, dev_conserved[4*n_cells + id], d*dev_conserved[5*n_cells + id], n, T_init_p, T_init_ge, del_T);
    #endif

    // and send back from kernel
    dev_conserved[4*n_cells + id] = E;
    #ifdef DE
    dev_conserved[5*n_cells + id] = d*ge;
    #endif
    }

  }

}


/* \fn __device__ Real test_cool(Real n, Real T)
 * \brief Cooling function from Creasey 2011. */
__device__ Real test_cool(int tid, Real n, Real T)
{
  Real T0, T1, lambda, cool;
  T0 = 10000.0;
  T1 = 20*T0;
  cool = 0.0;
  //lambda = 5.0e-24; //cooling coefficient, 5e-24 erg cm^3 s^-1
  lambda = 5.0e-20; //cooling coefficient, 5e-24 erg cm^3 s^-1

  // constant cooling rate 
  //cool = n*n*lambda;

  // Creasey cooling function
  if (T >= T0 && T <= 0.5*(T1+T0)) {
    cool = n*n*lambda*(T - T0) / T0;
  }
  if (T >= 0.5*(T1+T0) && T <= T1) {
    cool = n*n*lambda*(T1 - T) / T0;
  }
 

  //printf("%d %f %f\n", tid, T, cool);
  return cool;

}


/* \fn __device__ Real primordial_cool(Real n, Real T)
 * \brief Primordial hydrogen/helium cooling curve 
          derived according to Katz et al. 1996. */
__device__ Real primordial_cool(Real n, Real T)
{
  Real n_h, Y, y, g_ff, cool;
  Real n_h0, n_hp, n_he0, n_hep, n_hepp, n_e, n_e_old; 
  Real alpha_hp, alpha_hep, alpha_d, alpha_hepp, gamma_eh0, gamma_ehe0, gamma_ehep;
  Real le_h0, le_hep, li_h0, li_he0, li_hep, lr_hp, lr_hep, lr_hepp, ld_hep, l_ff;
  Real gamma_lh0, gamma_lhe0, gamma_lhep, e_h0, e_he0, e_hep, H;
  int heat_flag, n_iter;
  Real diff, tol;

  // set flag to 1 for photoionization & heating
  heat_flag = 0;

  //Real X = 0.76; //hydrogen abundance by mass
  Y = 0.24; //helium abundance by mass
  y = Y/(4 - 4*Y);  

  // set the hydrogen number density 
  n_h = n; 

  // calculate the recombination and collisional ionziation rates
  // (Table 2 from Katz 1996)
  alpha_hp   = (8.4e-11) * (1.0/sqrt(T)) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7))));
  alpha_hep  = (1.5e-10) * (pow(T,(-0.6353)));
  alpha_d    = (1.9e-3)  * (pow(T,(-1.5))) * exp(-470000.0/T) * (1.0 + 0.3*exp(-94000.0/T));
  alpha_hepp = (3.36e-10)* (1.0/sqrt(T)) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7))));
  gamma_eh0  = (5.85e-11)* sqrt(T) * exp(-157809.1/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  gamma_ehe0 = (2.38e-11)* sqrt(T) * exp(-285335.4/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  gamma_ehep = (5.68e-12)* sqrt(T) * exp(-631515.0/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  // externally evaluated integrals for photoionziation rates
  // assumed J(nu) = 10^-22 (nu_L/nu)
  gamma_lh0 = 3.19851e-13;
  gamma_lhe0 = 3.13029e-13;
  gamma_lhep = 2.00541e-14; 
  // externally evaluated integrals for heating rates
  e_h0 = 2.4796e-24;
  e_he0 = 6.86167e-24;
  e_hep = 6.21868e-25; 
  

  // assuming no photoionization, solve equations for number density of
  // each species
  n_e = n_h; //as a first guess, use the hydrogen number density
  n_iter = 20;
  diff = 1.0;
  tol = 1.0e-6;
  if (heat_flag) { 
    for (int i=0; i<n_iter; i++) {
      n_e_old = n_e;
      n_h0   = n_h*alpha_hp / (alpha_hp + gamma_eh0 + gamma_lh0/n_e);
      n_hp   = n_h - n_h0;
      n_hep  = y*n_h / (1.0 + (alpha_hep + alpha_d)/(gamma_ehe0 + gamma_lhe0/n_e) + (gamma_ehep + gamma_lhep/n_e)/alpha_hepp );
      n_he0  = n_hep*(alpha_hep + alpha_d) / (gamma_ehe0 + gamma_lhe0/n_e);
      n_hepp = n_hep*(gamma_ehep + gamma_lhep/n_e)/alpha_hepp;
      n_e    = n_hp + n_hep + 2*n_hepp;
      diff = fabs(n_e_old - n_e);
      if (diff < tol) break;
    }
  }  
  else {
    n_h0   = n_h*alpha_hp / (alpha_hp + gamma_eh0);
    n_hp   = n_h - n_h0;
    n_hep  = y*n_h / (1.0 + (alpha_hep + alpha_d)/(gamma_ehe0) + (gamma_ehep)/alpha_hepp );
    n_he0  = n_hep*(alpha_hep + alpha_d) / (gamma_ehe0);
    n_hepp = n_hep*(gamma_ehep)/alpha_hepp;
    n_e    = n_hp + n_hep + 2*n_hepp;
  }

  // using number densities, calculate cooling rates for
  // various processes (Table 1 from Katz 1996)
  le_h0 = (7.50e-19) * exp(-118348.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_h0;
  le_hep = (5.54e-17) * pow(T,(-0.397)) * exp(-473638.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_hep;
  li_h0 = (1.27e-21) * sqrt(T) * exp(-157809.1/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_h0;
  li_he0 = (9.38e-22) * sqrt(T) * exp(-285335.4/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_he0;
  li_hep = (4.95e-22) * sqrt(T) * exp(-631515.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_hep;
  lr_hp = (8.70e-27) * sqrt(T) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7)))) * n_e * n_hp;
  lr_hep = (1.55e-26) * pow(T,(0.3647)) * n_e * n_hep;
  lr_hepp = (3.48e-26) * sqrt(T) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7)))) * n_e * n_hepp;
  ld_hep = (1.24e-13) * pow(T,(-1.5)) * exp(-470000.0/T) * (1.0 + 0.3*exp(-94000.0/T)) * n_e * n_hep;
  g_ff = 1.1 + 0.34*exp(-(5.5-log(T))*(5.5-log(T))/3.0); // Gaunt factor
  l_ff = (1.42e-27) * g_ff * sqrt(T) * (n_hp + n_hep + 4*n_hepp) * n_e;

  // calculate total cooling rate (erg s^-1 cm^-3)
  cool = le_h0 + le_hep + li_h0 + li_he0 + li_hep + lr_hp + lr_hep + lr_hepp + ld_hep + l_ff;

  // calculate total photoionization heating rate
  H = 0.0;
  if (heat_flag) {
    H = n_h0*e_h0 + n_he0*e_he0 + n_hep*e_hep; 
  }
  
  cool -= H;

  return cool;

}


/* \fn __device__ Real KI_cool(Real n, Real T)
 * \brief Analytic fit to solar metallicity ISM cooling curve 
          defined in Koyama & Inutsuka, 2002. */
__device__ Real KI_cool(Real n, Real T)
{
  Real heat = 2.0e-26; //heating rate, erg s^-1 
  Real lambda = 0.0; //cooling rate, erg s^-1 cm^3
  Real cool = 0.0; //cooling per unit volume, erg /s / cm^3

  // KI cooling function 
  lambda = heat * (1.0e7 * exp(-1.14800e5/(T + 1000.)) + 14. * sqrt(T) * exp(-92./T));

  if (T > 10) {
    cool = n*n*lambda;
  }

  return cool;

}


/* \fn __device__ Real Schure_cool(Real n, Real T)
 * \brief Analytic fit to the solar metallicity CIE cooling curve 
          defined in Schure et al., 2009. */
__device__ Real Schure_cool(Real n, Real T)
{
  Real lambda = 0.0; //cooling rate, erg s^-1 cm^3
  Real cool = 0.0; //cooling per unit volume, erg /s / cm^3
  
  // fit to Schure cooling function 
  if (log10(T) > 5.36) {
    lambda = pow(10.0, (0.38 * (log10(T) -7.5) * (log10(T) - 7.5) - 22.6));
  }
  else if (log10(T) < 4.0) {
    lambda = 0.0;
  }
  else {
    lambda = pow(10.0, (-2.5 * (log10(T) - 5.1) * (log10(T) - 5.1) - 20.7));
  }

  // cooling rate per unit volume
  cool = n*n*lambda;

  return cool;

}


#endif //COOLING_GPU
#endif //CUDA
