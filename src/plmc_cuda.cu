
#include <hip/hip_runtime.h>
/*! \file plmc_cuda.cu
 *  \brief Definitions of the piecewise linear reconstruction functions with 
           limiting applied in the characteristic variables, as described
           in Stone et al., 2008. */
#ifdef CUDA
#ifdef PLMC

#include"gpu.hpp"
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"plmc_cuda.h"

#ifdef DE //PRESSURE_DE
#include"hydro_cuda.h"
#endif


/*! \fn __global__ void PLMC_cuda(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx, int ny, int nz, int n_ghost, Real dx, Real dt, Real gamma, int dir)
 *  \brief When passed a stencil of conserved variables, returns the left and right 
           boundary values for the interface calculated using plm. */
__global__ void PLMC_cuda(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx, int ny, int nz, int n_ghost, Real dx, Real dt, Real gamma, int dir, int n_fields)
{
  int n_cells = nx*ny*nz;
  int o1, o2, o3;
  if (dir == 0) {
    o1 = 1; o2 = 2; o3 = 3;
  }
  if (dir == 1) {
    o1 = 2; o2 = 3; o3 = 1;
  }
  if (dir == 2) {
    o1 = 3; o2 = 1; o3 = 2;
  }

  // declare primative variables for each stencil
  // these will be placed into registers for each thread
  Real d_i, vx_i, vy_i, vz_i, p_i;
  Real d_imo, vx_imo, vy_imo, vz_imo, p_imo; 
  Real d_ipo, vx_ipo, vy_ipo, vz_ipo, p_ipo;

  // declare other variables to be used
  Real a_i;
  Real del_d_L, del_vx_L, del_vy_L, del_vz_L, del_p_L;
  Real del_d_R, del_vx_R, del_vy_R, del_vz_R, del_p_R;
  Real del_d_C, del_vx_C, del_vy_C, del_vz_C, del_p_C;
  Real del_d_G, del_vx_G, del_vy_G, del_vz_G, del_p_G;
  Real del_a_0_L, del_a_1_L, del_a_2_L, del_a_3_L, del_a_4_L;
  Real del_a_0_R, del_a_1_R, del_a_2_R, del_a_3_R, del_a_4_R;
  Real del_a_0_C, del_a_1_C, del_a_2_C, del_a_3_C, del_a_4_C;
  Real del_a_0_G, del_a_1_G, del_a_2_G, del_a_3_G, del_a_4_G;
  Real del_a_0_m, del_a_1_m, del_a_2_m, del_a_3_m, del_a_4_m;
  Real lim_slope_a, lim_slope_b;
  Real del_d_m_i, del_vx_m_i, del_vy_m_i, del_vz_m_i, del_p_m_i;
  Real d_L_iph, vx_L_iph, vy_L_iph, vz_L_iph, p_L_iph;
  Real d_R_imh, vx_R_imh, vy_R_imh, vz_R_imh, p_R_imh;
  Real C;
  // #ifdef CTU
  #ifndef VL
  Real dtodx = dt/dx;
  Real lambda_m, lambda_0, lambda_p;
  Real qx;
  Real lamdiff;
  Real sum_0, sum_1, sum_2, sum_3, sum_4;  
  #endif //CTU
  #ifdef DE
  Real ge_i, ge_imo, ge_ipo;
  Real del_ge_L, del_ge_R, del_ge_C, del_ge_G;
  Real del_ge_m_i;
  Real ge_L_iph, ge_R_imh;
  Real E, E_kin, dge;
  #ifndef VL
  Real sum_ge;
  #endif //CTU 
  #endif
  #ifdef SCALAR 
  Real scalar_i[NSCALARS], scalar_imo[NSCALARS], scalar_ipo[NSCALARS];
  Real del_scalar_L[NSCALARS], del_scalar_R[NSCALARS], del_scalar_C[NSCALARS], del_scalar_G[NSCALARS];
  Real del_scalar_m_i[NSCALARS];
  Real scalar_L_iph[NSCALARS], scalar_R_imh[NSCALARS];
  // #ifdef CTU
  #ifndef VL
  Real sum_scalar[NSCALARS];
  #endif //CTU
  #endif

  // get a thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId*blockDim.x;
  int id;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;

  int xs, xe, ys, ye, zs, ze;
  if (dir == 0) {
    xs = 1; xe = nx-2;
    ys = 0; ye = ny;
    zs = 0; ze = nz;
  }
  if (dir == 1) {
    xs = 0; xe = nx;
    ys = 1; ye = ny-2;
    zs = 0; ze = nz;
  }
  if (dir == 2) {
    xs = 0; xe = nx;
    ys = 0; ye = ny;
    zs = 1; ze = nz-2;
  }


  if (xid >= xs && xid < xe && yid >= ys && yid < ye && zid >= zs && zid < ze)
  {
    // load the 3-cell stencil into registers
    // cell i
    id = xid + yid*nx + zid*nx*ny;
    d_i  =  dev_conserved[            id];
    vx_i =  dev_conserved[o1*n_cells + id] / d_i;
    vy_i =  dev_conserved[o2*n_cells + id] / d_i;
    vz_i =  dev_conserved[o3*n_cells + id] / d_i;
    #ifdef DE //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    E_kin = 0.5 * d_i * ( vx_i*vx_i + vy_i*vy_i + vz_i*vz_i );
    dge = dev_conserved[(n_fields-1)*n_cells + id];
    p_i = Get_Pressure_From_DE( E, E - E_kin, dge, gamma ); 
    #else
    p_i  = (dev_conserved[4*n_cells + id] - 0.5*d_i*(vx_i*vx_i + vy_i*vy_i + vz_i*vz_i)) * (gamma - 1.0);
    #endif //PRESSURE_DE
    p_i  = fmax(p_i, (Real) TINY_NUMBER);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalar_i[i] = dev_conserved[(5+i)*n_cells + id] / d_i;
    }
    #endif
    #ifdef DE
    ge_i =  dge / d_i;
    #endif
    // cell i-1
    if (dir == 0) id = xid-1 + yid*nx + zid*nx*ny;
    if (dir == 1) id = xid + (yid-1)*nx + zid*nx*ny;
    if (dir == 2) id = xid + yid*nx + (zid-1)*nx*ny;
    d_imo  =  dev_conserved[            id];
    vx_imo =  dev_conserved[o1*n_cells + id] / d_imo;
    vy_imo =  dev_conserved[o2*n_cells + id] / d_imo;
    vz_imo =  dev_conserved[o3*n_cells + id] / d_imo;
    #ifdef DE //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    E_kin = 0.5 * d_imo * ( vx_imo*vx_imo + vy_imo*vy_imo + vz_imo*vz_imo );
    dge = dev_conserved[(n_fields-1)*n_cells + id];
    p_imo = Get_Pressure_From_DE( E, E - E_kin, dge, gamma ); 
    #else
    p_imo  = (dev_conserved[4*n_cells + id] - 0.5*d_imo*(vx_imo*vx_imo + vy_imo*vy_imo + vz_imo*vz_imo)) * (gamma - 1.0);
    #endif //PRESSURE_DE
    p_imo  = fmax(p_imo, (Real) TINY_NUMBER);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalar_imo[i] = dev_conserved[(5+i)*n_cells + id] / d_imo;
    }
    #endif
    #ifdef DE
    ge_imo =  dge / d_imo;
    #endif
    // cell i+1
    if (dir == 0) id = xid+1 + yid*nx + zid*nx*ny;
    if (dir == 1) id = xid + (yid+1)*nx + zid*nx*ny;
    if (dir == 2) id = xid + yid*nx + (zid+1)*nx*ny;
    d_ipo  =  dev_conserved[            id];
    vx_ipo =  dev_conserved[o1*n_cells + id] / d_ipo;
    vy_ipo =  dev_conserved[o2*n_cells + id] / d_ipo;
    vz_ipo =  dev_conserved[o3*n_cells + id] / d_ipo;
    #ifdef DE //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    E_kin = 0.5 * d_ipo * ( vx_ipo*vx_ipo + vy_ipo*vy_ipo + vz_ipo*vz_ipo );
    dge = dev_conserved[(n_fields-1)*n_cells + id];
    p_ipo = Get_Pressure_From_DE( E, E - E_kin, dge, gamma ); 
    #else
    p_ipo  = (dev_conserved[4*n_cells + id] - 0.5*d_ipo*(vx_ipo*vx_ipo + vy_ipo*vy_ipo + vz_ipo*vz_ipo)) * (gamma - 1.0);
    #endif //PRESSURE_DE
    p_ipo  = fmax(p_ipo, (Real) TINY_NUMBER);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalar_ipo[i] = dev_conserved[(5+i)*n_cells + id] / d_ipo;
    }
    #endif
    #ifdef DE
    ge_ipo =  dge / d_ipo;
    #endif


    // calculate the adiabatic sound speed in cell i
    a_i   = sqrt(gamma*p_i/d_i);


    // Compute the eigenvalues of the linearized equations in the
    // primative variables using the cell-centered primative variables
    // #ifdef CTU
    #ifndef VL
    lambda_m = vx_i-a_i;
    lambda_0 = vx_i;
    lambda_p = vx_i+a_i; 
    #endif

    // Compute the left, right, centered, and van Leer differences of the primative variables
    // Note that here L and R refer to locations relative to the cell center

    // left
    del_d_L  = d_i - d_imo;
    del_vx_L = vx_i - vx_imo;
    del_vy_L = vy_i - vy_imo;
    del_vz_L = vz_i - vz_imo;
    del_p_L  = p_i  - p_imo;

    // right
    del_d_R  = d_ipo  - d_i;
    del_vx_R = vx_ipo - vx_i;
    del_vy_R = vy_ipo - vy_i;
    del_vz_R = vz_ipo - vz_i;
    del_p_R  = p_ipo  - p_i;

    // centered
    del_d_C  = 0.5*(d_ipo - d_imo);
    del_vx_C = 0.5*(vx_ipo - vx_imo);
    del_vy_C = 0.5*(vy_ipo - vy_imo);
    del_vz_C = 0.5*(vz_ipo - vz_imo);
    del_p_C  = 0.5*(p_ipo - p_imo);

    // Van Leer
    if (del_d_L*del_d_R > 0.0) { del_d_G = 2.0*del_d_L*del_d_R / (del_d_L+del_d_R); }
    else { del_d_G = 0.0; }
    if (del_vx_L*del_vx_R > 0.0) { del_vx_G = 2.0*del_vx_L*del_vx_R / (del_vx_L+del_vx_R); }
    else { del_vx_G = 0.0; } 
    if (del_vy_L*del_vy_R > 0.0) { del_vy_G = 2.0*del_vy_L*del_vy_R / (del_vy_L+del_vy_R); }
    else { del_vy_G = 0.0; } 
    if (del_vz_L*del_vz_R > 0.0) { del_vz_G = 2.0*del_vz_L*del_vz_R / (del_vz_L+del_vz_R); }
    else { del_vz_G = 0.0; } 
    if (del_p_L*del_p_R > 0.0) { del_p_G = 2.0*del_p_L*del_p_R / (del_p_L+del_p_R); }
    else { del_p_G = 0.0; } 

    #ifdef DE
    del_ge_L = ge_i - ge_imo;
    del_ge_R = ge_ipo - ge_i;
    del_ge_C = 0.5*(ge_ipo - ge_imo);
    if (del_ge_L*del_ge_R > 0.0) { del_ge_G = 2.0*del_ge_L*del_ge_R / (del_ge_L+del_ge_R); }
    else { del_ge_G = 0.0; } 
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      del_scalar_L[i] = scalar_i[i] - scalar_imo[i];
      del_scalar_R[i] = scalar_ipo[i] - scalar_i[i];
      del_scalar_C[i] = 0.5*(scalar_ipo[i] - scalar_imo[i]);
      if (del_scalar_L[i]*del_scalar_R[i] > 0.0) { del_scalar_G[i] = 2.0*del_scalar_L[i]*del_scalar_R[i] / (del_scalar_L[i]+del_scalar_R[i]); }
      else { del_scalar_G[i] = 0.0; } 
    }
    #endif


    // Project the left, right, centered and van Leer differences onto the characteristic variables
    // Stone Eqn 37 (del_a are differences in characteristic variables, see Stone for notation)
    // Use the eigenvectors given in Stone 2008, Appendix A
    del_a_0_L = -d_i * del_vx_L / (2*a_i) + del_p_L / (2*a_i*a_i);
    del_a_1_L = del_d_L - del_p_L / (a_i*a_i);
    del_a_2_L = del_vy_L;
    del_a_3_L = del_vz_L;
    del_a_4_L = d_i * del_vx_L / (2*a_i) + del_p_L / (2*a_i*a_i);

    del_a_0_R = -d_i * del_vx_R / (2*a_i) + del_p_R / (2*a_i*a_i);
    del_a_1_R = del_d_R - del_p_R / (a_i*a_i);
    del_a_2_R = del_vy_R;
    del_a_3_R = del_vz_R;
    del_a_4_R = d_i * del_vx_R / (2*a_i) + del_p_R / (2*a_i*a_i);

    del_a_0_C = -d_i * del_vx_C / (2*a_i) + del_p_C / (2*a_i*a_i);
    del_a_1_C = del_d_C - del_p_C / (a_i*a_i);
    del_a_2_C = del_vy_C;
    del_a_3_C = del_vz_C;
    del_a_4_C = d_i * del_vx_C / (2*a_i) + del_p_C / (2*a_i*a_i);

    del_a_0_G = -d_i * del_vx_G / (2*a_i) + del_p_G / (2*a_i*a_i);
    del_a_1_G = del_d_G - del_p_G / (a_i*a_i);
    del_a_2_G = del_vy_G;
    del_a_3_G = del_vz_G;
    del_a_4_G = d_i * del_vx_G / (2*a_i) + del_p_G / (2*a_i*a_i); 


    // Apply monotonicity constraints to the differences in the characteristic variables

    del_a_0_m = del_a_1_m = del_a_2_m = del_a_3_m = del_a_4_m = 0.0;
  
    if (del_a_0_L*del_a_0_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_0_L), fabs(del_a_0_R));
      lim_slope_b = fmin(fabs(del_a_0_C), fabs(del_a_0_G));
      del_a_0_m = sgn_CUDA(del_a_0_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_1_L*del_a_1_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_1_L), fabs(del_a_1_R));
      lim_slope_b = fmin(fabs(del_a_1_C), fabs(del_a_1_G));
      del_a_1_m = sgn_CUDA(del_a_1_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_2_L*del_a_2_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_2_L), fabs(del_a_2_R));
      lim_slope_b = fmin(fabs(del_a_2_C), fabs(del_a_2_G));
      del_a_2_m = sgn_CUDA(del_a_2_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_3_L*del_a_3_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_3_L), fabs(del_a_3_R));
      lim_slope_b = fmin(fabs(del_a_3_C), fabs(del_a_3_G));
      del_a_3_m = sgn_CUDA(del_a_3_C) * fmin(2.0*lim_slope_a, lim_slope_b);
    }
    if (del_a_4_L*del_a_4_R > 0.0) {
      lim_slope_a = fmin(fabs(del_a_4_L), fabs(del_a_4_R));
      lim_slope_b = fmin(fabs(del_a_4_C), fabs(del_a_4_G));
      del_a_4_m = sgn_CUDA(del_a_4_C) * fmin(2.0*lim_slope_a, lim_slope_b); 
    }
    #ifdef DE
    del_ge_m_i = 0.0;
    if (del_ge_L*del_ge_R > 0.0) {
      lim_slope_a = fmin(fabs(del_ge_L), fabs(del_ge_R));
      lim_slope_b = fmin(fabs(del_ge_C), fabs(del_ge_G));
      del_ge_m_i = sgn_CUDA(del_ge_C) * fmin(2.0*lim_slope_a, lim_slope_b); 
    }
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      del_scalar_m_i[i] = 0.0;
      if (del_scalar_L[i]*del_scalar_R[i] > 0.0) {
        lim_slope_a = fmin(fabs(del_scalar_L[i]), fabs(del_scalar_R[i]));
        lim_slope_b = fmin(fabs(del_scalar_C[i]), fabs(del_scalar_G[i]));
        del_scalar_m_i[i] = sgn_CUDA(del_scalar_C[i]) * fmin(2.0*lim_slope_a, lim_slope_b); 
      }
    }
    #endif
    


    // Project the monotonized difference in the characteristic variables back onto the 
    // primative variables
    // Stone Eqn 39
    del_d_m_i  = del_a_0_m + del_a_1_m + del_a_4_m;
    del_vx_m_i = -a_i*del_a_0_m / d_i + a_i* del_a_4_m / d_i;
    del_vy_m_i = del_a_2_m;
    del_vz_m_i = del_a_3_m;
    del_p_m_i  = a_i*a_i*del_a_0_m + a_i*a_i*del_a_4_m;  


    // Compute the left and right interface values using the monotonized difference in the
    // primative variables

    d_R_imh  = d_i  - 0.5*del_d_m_i; 
    vx_R_imh = vx_i - 0.5*del_vx_m_i;
    vy_R_imh = vy_i - 0.5*del_vy_m_i;
    vz_R_imh = vz_i - 0.5*del_vz_m_i;
    p_R_imh  = p_i  - 0.5*del_p_m_i;
 
    d_L_iph  = d_i  + 0.5*del_d_m_i; 
    vx_L_iph = vx_i + 0.5*del_vx_m_i;
    vy_L_iph = vy_i + 0.5*del_vy_m_i;
    vz_L_iph = vz_i + 0.5*del_vz_m_i;
    p_L_iph  = p_i  + 0.5*del_p_m_i; 

    #ifdef DE
    ge_R_imh = ge_i - 0.5*del_ge_m_i;
    ge_L_iph = ge_i + 0.5*del_ge_m_i;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalar_R_imh[i] = scalar_i[i] - 0.5*del_scalar_m_i[i];
      scalar_L_iph[i] = scalar_i[i] + 0.5*del_scalar_m_i[i];
    }
    #endif


    C = d_R_imh + d_L_iph;
    d_R_imh = fmax( fmin(d_i, d_imo), d_R_imh );
    d_R_imh = fmin( fmax(d_i, d_imo), d_R_imh );
    d_L_iph = C - d_R_imh;
    d_L_iph = fmax( fmin(d_i, d_ipo), d_L_iph );
    d_L_iph = fmin( fmax(d_i, d_ipo), d_L_iph );
    d_R_imh = C - d_L_iph;

    C = vx_R_imh + vx_L_iph;
    vx_R_imh = fmax( fmin(vx_i, vx_imo), vx_R_imh );
    vx_R_imh = fmin( fmax(vx_i, vx_imo), vx_R_imh );
    vx_L_iph = C - vx_R_imh;
    vx_L_iph = fmax( fmin(vx_i, vx_ipo), vx_L_iph );
    vx_L_iph = fmin( fmax(vx_i, vx_ipo), vx_L_iph );
    vx_R_imh = C - vx_L_iph;  

    C = vy_R_imh + vy_L_iph;
    vy_R_imh = fmax( fmin(vy_i, vy_imo), vy_R_imh );
    vy_R_imh = fmin( fmax(vy_i, vy_imo), vy_R_imh );
    vy_L_iph = C - vy_R_imh;
    vy_L_iph = fmax( fmin(vy_i, vy_ipo), vy_L_iph );
    vy_L_iph = fmin( fmax(vy_i, vy_ipo), vy_L_iph );
    vy_R_imh = C - vy_L_iph;
 
    C = vz_R_imh + vz_L_iph;
    vz_R_imh = fmax( fmin(vz_i, vz_imo), vz_R_imh );
    vz_R_imh = fmin( fmax(vz_i, vz_imo), vz_R_imh );
    vz_L_iph = C - vz_R_imh; 
    vz_L_iph = fmax( fmin(vz_i, vz_ipo), vz_L_iph );
    vz_L_iph = fmin( fmax(vz_i, vz_ipo), vz_L_iph );
    vz_R_imh = C - vz_L_iph;

    C = p_R_imh + p_L_iph;
    p_R_imh = fmax( fmin(p_i, p_imo), p_R_imh );
    p_R_imh = fmin( fmax(p_i, p_imo), p_R_imh );
    p_L_iph = C - p_R_imh;
    p_L_iph = fmax( fmin(p_i, p_ipo), p_L_iph );
    p_L_iph = fmin( fmax(p_i, p_ipo), p_L_iph );
    p_R_imh = C - p_L_iph;

    del_d_m_i  = d_L_iph  - d_R_imh;
    del_vx_m_i = vx_L_iph - vx_R_imh;
    del_vy_m_i = vy_L_iph - vy_R_imh;
    del_vz_m_i = vz_L_iph - vz_R_imh;
    del_p_m_i  = p_L_iph  - p_R_imh;

    #ifdef DE
    C = ge_R_imh + ge_L_iph;
    ge_R_imh = fmax( fmin(ge_i, ge_imo), ge_R_imh );
    ge_R_imh = fmin( fmax(ge_i, ge_imo), ge_R_imh );
    ge_L_iph = C - ge_R_imh; 
    ge_L_iph = fmax( fmin(ge_i, ge_ipo), ge_L_iph );
    ge_L_iph = fmin( fmax(ge_i, ge_ipo), ge_L_iph );
    ge_R_imh = C - ge_L_iph;    
    del_ge_m_i = ge_L_iph - ge_R_imh;
    #endif

    #ifdef SCALAR 
    for (int i=0; i<NSCALARS; i++) {
      C = scalar_R_imh[i] + scalar_L_iph[i];
      scalar_R_imh[i] = fmax( fmin(scalar_i[i], scalar_imo[i]), scalar_R_imh[i] );
      scalar_R_imh[i] = fmin( fmax(scalar_i[i], scalar_imo[i]), scalar_R_imh[i] );
      scalar_L_iph[i] = C - scalar_R_imh[i]; 
      scalar_L_iph[i] = fmax( fmin(scalar_i[i], scalar_ipo[i]), scalar_L_iph[i] );
      scalar_L_iph[i] = fmin( fmax(scalar_i[i], scalar_ipo[i]), scalar_L_iph[i] );
      scalar_R_imh[i] = C - scalar_L_iph[i];    
      del_scalar_m_i[i] = scalar_L_iph[i] - scalar_R_imh[i];
    }
    #endif


    // #ifdef CTU
    #ifndef VL
    // Integrate linear interpolation function over domain of dependence
    // defined by max(min) eigenvalue
    qx = -0.5*fmin(lambda_m, 0.0)*dtodx;
    d_R_imh  = d_R_imh  + qx * del_d_m_i;
    vx_R_imh = vx_R_imh + qx * del_vx_m_i;
    vy_R_imh = vy_R_imh + qx * del_vy_m_i;
    vz_R_imh = vz_R_imh + qx * del_vz_m_i;
    p_R_imh  = p_R_imh  + qx * del_p_m_i;

    qx = 0.5*fmax(lambda_p, 0.0)*dtodx;
    d_L_iph  = d_L_iph  - qx * del_d_m_i;
    vx_L_iph = vx_L_iph - qx * del_vx_m_i;
    vy_L_iph = vy_L_iph - qx * del_vy_m_i;
    vz_L_iph = vz_L_iph - qx * del_vz_m_i;
    p_L_iph  = p_L_iph  - qx * del_p_m_i;

    #ifdef DE
    ge_R_imh = ge_R_imh + qx * del_ge_m_i;
    ge_L_iph = ge_L_iph - qx * del_ge_m_i;
    #endif

    #ifdef  SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalar_R_imh[i] = scalar_R_imh[i] + qx * del_scalar_m_i[i];
      scalar_L_iph[i] = scalar_L_iph[i] - qx * del_scalar_m_i[i];
    }
    #endif


    // Perform the characteristic tracing
    // Stone Eqns 42 & 43

    // left-hand interface value, i+1/2
    sum_0 = sum_1 = sum_2 = sum_3 = sum_4 = 0;
    #ifdef DE
    sum_ge = 0;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      sum_scalar[i] = 0.0;
    }
    #endif
    if (lambda_m >= 0)
    {
      lamdiff = lambda_p - lambda_m;

      sum_0 += lamdiff * (-d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 - del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (-d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }
    if (lambda_0 >= 0)
    {
      lamdiff = lambda_p - lambda_0;
  
      sum_0 += lamdiff * (del_d_m_i - del_p_m_i/(a_i*a_i));
      sum_2 += lamdiff * del_vy_m_i;
      sum_3 += lamdiff * del_vz_m_i;
      #ifdef DE
      sum_ge += lamdiff * del_ge_m_i;
      #endif
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        sum_scalar[i] += lamdiff * del_scalar_m_i[i];
      }
      #endif
    }
    if (lambda_p >= 0)
    {
      lamdiff = lambda_p - lambda_p;

      sum_0 += lamdiff * (d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 + del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }

    // add the corrections to the initial guesses for the interface values
    d_L_iph  += 0.5*dtodx*sum_0;
    vx_L_iph += 0.5*dtodx*sum_1;
    vy_L_iph += 0.5*dtodx*sum_2;
    vz_L_iph += 0.5*dtodx*sum_3;
    p_L_iph  += 0.5*dtodx*sum_4;
    #ifdef DE
    ge_L_iph += 0.5*dtodx*sum_ge;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalar_L_iph[i] += 0.5*dtodx*sum_scalar[i];
    }
    #endif


    // right-hand interface value, i-1/2
    sum_0 = sum_1 = sum_2 = sum_3 = sum_4 = 0;
    #ifdef DE
    sum_ge = 0;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      sum_scalar[i] = 0;
    }
    #endif
    if (lambda_m <= 0)
    {
      lamdiff = lambda_m - lambda_m; 

      sum_0 += lamdiff * (-d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 - del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (-d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }
    if (lambda_0 <= 0)
    {
      lamdiff = lambda_m - lambda_0;
  
      sum_0 += lamdiff * (del_d_m_i - del_p_m_i/(a_i*a_i));
      sum_2 += lamdiff * del_vy_m_i;
      sum_3 += lamdiff * del_vz_m_i;
      #ifdef DE
      sum_ge += lamdiff * del_ge_m_i;
      #endif
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        sum_scalar[i] += lamdiff * del_scalar_m_i[i];
      }
      #endif
    }
    if (lambda_p <= 0)
    {
      lamdiff = lambda_m - lambda_p;

      sum_0 += lamdiff * (d_i*del_vx_m_i/(2*a_i) + del_p_m_i/(2*a_i*a_i));
      sum_1 += lamdiff * (del_vx_m_i/2.0 + del_p_m_i/(2*a_i*d_i));
      sum_4 += lamdiff * (d_i*del_vx_m_i*a_i/2.0 + del_p_m_i/2.0);
    }

    // add the corrections
    d_R_imh  += 0.5*dtodx*sum_0;
    vx_R_imh += 0.5*dtodx*sum_1;
    vy_R_imh += 0.5*dtodx*sum_2;
    vz_R_imh += 0.5*dtodx*sum_3;
    p_R_imh  += 0.5*dtodx*sum_4;
    #ifdef DE
    ge_R_imh += 0.5*dtodx*sum_ge;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalar_R_imh[i] += 0.5*dtodx*sum_scalar[i];
    }
    #endif
    #endif //CTU

    // apply minimum constraints
    d_R_imh = fmax(d_R_imh, (Real) TINY_NUMBER);
    d_L_iph = fmax(d_L_iph, (Real) TINY_NUMBER);
    p_R_imh = fmax(p_R_imh, (Real) TINY_NUMBER);
    p_L_iph = fmax(p_L_iph, (Real) TINY_NUMBER);

    // Convert the left and right states in the primitive to the conserved variables
    // send final values back from kernel
    // bounds_R refers to the right side of the i-1/2 interface
    if (dir == 0) id = xid-1 + yid*nx + zid*nx*ny;
    if (dir == 1) id = xid + (yid-1)*nx + zid*nx*ny;
    if (dir == 2) id = xid + yid*nx + (zid-1)*nx*ny;
    dev_bounds_R[            id] = d_R_imh;
    dev_bounds_R[o1*n_cells + id] = d_R_imh*vx_R_imh;
    dev_bounds_R[o2*n_cells + id] = d_R_imh*vy_R_imh;
    dev_bounds_R[o3*n_cells + id] = d_R_imh*vz_R_imh;
    dev_bounds_R[4*n_cells + id] = (p_R_imh/(gamma-1.0)) + 0.5*d_R_imh*(vx_R_imh*vx_R_imh + vy_R_imh*vy_R_imh + vz_R_imh*vz_R_imh);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_bounds_R[(5+i)*n_cells + id] = d_R_imh*scalar_R_imh[i];
    }
    #endif
    #ifdef DE
    dev_bounds_R[(n_fields-1)*n_cells + id] = d_R_imh*ge_R_imh;
    #endif
    // bounds_L refers to the left side of the i+1/2 interface
    id = xid + yid*nx + zid*nx*ny;
    dev_bounds_L[            id] = d_L_iph;
    dev_bounds_L[o1*n_cells + id] = d_L_iph*vx_L_iph;
    dev_bounds_L[o2*n_cells + id] = d_L_iph*vy_L_iph;
    dev_bounds_L[o3*n_cells + id] = d_L_iph*vz_L_iph;
    dev_bounds_L[4*n_cells + id] = (p_L_iph/(gamma-1.0)) + 0.5*d_L_iph*(vx_L_iph*vx_L_iph + vy_L_iph*vy_L_iph + vz_L_iph*vz_L_iph);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_bounds_L[(5+i)*n_cells + id] = d_L_iph*scalar_L_iph[i];
    }
    #endif
    #ifdef DE
    dev_bounds_L[(n_fields-1)*n_cells + id] = d_L_iph*ge_L_iph;
    #endif

  }
}
    


#endif //PLMC
#endif //CUDA
