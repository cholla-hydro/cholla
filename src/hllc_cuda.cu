
#include <hip/hip_runtime.h>
/*! \file hllc_cuda.cu
 *  \brief Function definitions for the cuda HLLC Riemann solver.*/

#ifdef CUDA

#include<cuda.h>
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"roe_cuda.h"



/*! \fn Calculate_HLLC_Fluxes(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir)
 *  \brief HLLC Riemann solver based on the version described in Toro (2006), Sec. 10.4. */
__global__ void Calculate_HLLC_Fluxes(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir)
{
  // get a thread index
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;

  int n_cells = nx*ny*nz;

  Real dl, vxl, mxl, vyl, myl, vzl, mzl, pl, El;
  Real dr, vxr, mxr, vyr, myr, vzr, mzr, pr, Er;

  Real g1 = gamma - 1.0; 
  Real Hl, Hr;
  Real sqrtdl, sqrtdr, vx, vy, vz, H;
  Real vsq, asq, a;
  Real lambda_m, lambda_p;
  Real f_d_l, f_mx_l, f_my_l, f_mz_l, f_E_l;
  Real f_d_r, f_mx_r, f_my_r, f_mz_r, f_E_r;
  Real dls, drs, mxls, mxrs, myls, myrs, mzls, mzrs, Els, Ers;
  Real f_d, f_mx, f_my, f_mz, f_E;
  Real Sl, Sr, Sm, cfl, cfr, ps;
  #ifdef DE
  Real gel, ger, f_gel, f_ger, f_ge;
  #endif

  int o1, o2, o3;
  if (dir==0) {
    o1 = 1; o2 = 2; o3 = 3;
  }
  if (dir==1) {
    o1 = 2; o2 = 3; o3 = 1;
  }
  if (dir==2) {
    o1 = 3; o2 = 1; o3 = 2;
  }

  // Each thread executes the solver independently
  //if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid < ny && zid < nz) 
  if (xid < nx && yid < ny && zid < nz) 
  {
    // retrieve conserved variables
    dl  = dev_bounds_L[             tid];
    mxl = dev_bounds_L[o1*n_cells + tid];
    myl = dev_bounds_L[o2*n_cells + tid];
    mzl = dev_bounds_L[o3*n_cells + tid];
    El  = dev_bounds_L[4*n_cells + tid];
    #ifdef DE
    gel = dev_bounds_L[5*n_cells + tid] / dl;
    #endif

    dr  = dev_bounds_R[            tid];
    mxr = dev_bounds_R[o1*n_cells + tid];
    myr = dev_bounds_R[o2*n_cells + tid];
    mzr = dev_bounds_R[o3*n_cells + tid];
    Er  = dev_bounds_R[4*n_cells + tid]; 
    #ifdef DE
    ger = dev_bounds_R[5*n_cells + tid] / dr;
    #endif

    // calculate primative variables
    vxl = mxl / dl;
    vyl = myl / dl;
    vzl = mzl / dl;
    pl  = (El - 0.5*dl*(vxl*vxl + vyl*vyl + vzl*vzl)) * (gamma - 1.0);
    pl  = fmax(pl, (Real) TINY_NUMBER);
    vxr = mxr / dr;
    vyr = myr / dr;
    vzr = mzr / dr;
    pr  = (Er - 0.5*dr*(vxr*vxr + vyr*vyr + vzr*vzr)) * (gamma - 1.0);
    pr  = fmax(pr, (Real) TINY_NUMBER);    

    // calculate the enthalpy in each cell
    Hl = (El + pl) / dl;
    Hr = (Er + pr) / dr;

    // calculate averages of the variables needed for the Roe Jacobian 
    // (see Stone et al., 2008, Eqn 65, or Toro 2009, 11.118)
    sqrtdl = sqrt(dl);
    sqrtdr = sqrt(dr);
    vx = (sqrtdl*vxl + sqrtdr*vxr) / (sqrtdl + sqrtdr);
    vy = (sqrtdl*vyl + sqrtdr*vyr) / (sqrtdl + sqrtdr);
    vz = (sqrtdl*vzl + sqrtdr*vzr) / (sqrtdl + sqrtdr);
    H  = (sqrtdl*Hl  + sqrtdr*Hr)  / (sqrtdl + sqrtdr); 

    // calculate the sound speed squared (Stone B2)
    vsq = (vx*vx + vy*vy + vz*vz);
    asq = g1*(H - 0.5*vsq);
    a = sqrt(asq);

    // calculate the averaged eigenvectors of the Roe matrix (Stone Eqn B2, Toro 11.107)
    lambda_m = vx - a; 
    lambda_p = vx + a;

    // compute max and min wave speeds
    cfl = sqrt(gamma*pl/dl);  // sound speed in left state
    cfr = sqrt(gamma*pr/dr);  // sound speed in right state

    // for signal speeds, take max/min of Roe eigenvalues and left and right sound speeds
    // Batten eqn. 48
    Sl = fmin(lambda_m, vxl - cfl);
    Sr = fmax(lambda_p, vxr + cfr);
 
    // left and right fluxes 
    f_d_l  = mxl;
    f_mx_l = mxl*vxl + pl;
    f_my_l = myl*vxl;
    f_mz_l = mzl*vxl;
    f_E_l  = (El + pl)*vxl;

    f_d_r  = mxr;
    f_mx_r = mxr*vxr + pr;
    f_my_r = myr*vxr;
    f_mz_r = mzr*vxr;
    f_E_r  = (Er + pr)*vxr;

    // return upwind flux if flow is supersonic 
    if (Sl > 0.0) {
      dev_flux[           tid] = f_d_l;
      dev_flux[o1*n_cells+tid] = f_mx_l;
      dev_flux[o2*n_cells+tid] = f_my_l;
      dev_flux[o3*n_cells+tid] = f_mz_l;
      dev_flux[4*n_cells+tid]  = f_E_l;
      #ifdef DE
      dev_flux[5*n_cells+tid]  = f_ge_l;
      #endif
      return;
    }
    else if (Sr < 0.0) {
      dev_flux[           tid] = f_d_r;
      dev_flux[o1*n_cells+tid] = f_mx_r;
      dev_flux[o2*n_cells+tid] = f_my_r;
      dev_flux[o3*n_cells+tid] = f_mz_r;
      dev_flux[4*n_cells+tid]  = f_E_r;
      #ifdef DE
      dev_flux[5*n_cells+tid]  = f_ge_r;
      #endif
      return;
    }
    // otherwise compute subsonic flux
    else { 

      // compute contact wave speed and pressure in star region (Batten eqns 34 & 36)
      Sm = (dr*vxr*(Sr - vxr) - dl*vxl*(Sl - vxl) + pl - pr) / (dr*(Sr - vxr) - dl*(Sl - vxl));
      ps = dl*(vxl - Sl)*(vxl - Sm) + pl;

      // conserved variables in the left star state (Batten eqns 35 - 40)
      dls = dl * (Sl - vxl) / (Sl - Sm);
      mxls = (mxl*(Sl - vxl) + ps - pl) / (Sl - Sm);
      myls = myl*(Sl - vxl) / (Sl - Sm);
      mzls = mzl*(Sl - vxl) / (Sl - Sm);
      Els = (El*(Sl - vxl) - pl*vxl + ps*Sm) / (Sl - Sm);

      // conserved variables in the right star state
      drs = dr * (Sr - vxr) / (Sr - Sm);
      mxrs = (mxr*(Sr - vxr) + ps - pr) / (Sr - Sm);
      myrs = myr*(Sr - vxr) / (Sr - Sm);
      mzrs = mzr*(Sr - vxr) / (Sr - Sm);
      Ers = (Er*(Sr - vxr) - pr*vxr + ps*Sm) / (Sr - Sm);


      // compute the hllc flux (Batten eqn 27)
      f_d  = 0.5*(f_d_l  + f_d_r  + (Sr - fabs(Sm))*drs  + (Sl + fabs(Sm))*dls  - Sl*dl  - Sr*dr);
      f_mx = 0.5*(f_mx_l + f_mx_r + (Sr - fabs(Sm))*mxrs + (Sl + fabs(Sm))*mxls - Sl*mxl - Sr*mxr);
      f_my = 0.5*(f_my_l + f_my_r + (Sr - fabs(Sm))*myrs + (Sl + fabs(Sm))*myls - Sl*myl - Sr*myr);
      f_mz = 0.5*(f_mz_l + f_mz_r + (Sr - fabs(Sm))*mzrs + (Sl + fabs(Sm))*mzls - Sl*mzl - Sr*mzr);
      f_E  = 0.5*(f_E_l  + f_E_r  + (Sr - fabs(Sm))*Ers  + (Sl + fabs(Sm))*Els  - Sl*El  - Sr*Er);


      // return the hllc fluxes
      dev_flux[           tid] = f_d;
      dev_flux[o1*n_cells+tid] = f_mx;
      dev_flux[o2*n_cells+tid] = f_my;
      dev_flux[o3*n_cells+tid] = f_mz;
      dev_flux[4*n_cells+tid]  = f_E;
      #ifdef DE
      dev_flux[5*n_cells+tid]  = f_ge;
      #endif

    }
  }

}


#endif //CUDA
