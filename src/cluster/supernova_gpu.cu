
#include <hip/hip_runtime.h>
#ifdef SUPERNOVA
  #include <math.h>

  #include "../cooling/cooling_cuda.h"
  #include "../global/global_cuda.h"  //includes gpu.hpp
  #include "../io/io.h"               //defines chprintf
  #include "supernova.h"

// texture<float, 1, cudaReadModeElementType> mdotTexObj;
// texture<float, 1, cudaReadModeElementType> edotTexObj;
namespace Supernova
{
Real *d_cluster_array;
Real *d_omega_array;
bool *d_flags_array;
Real *d_mdot;        // table data
Real *d_edot;        // table data
Real *d_mdot_array;  // holds m_dot(cluster)[time]
Real *d_edot_array;  // holds e_dot(cluster)[time]
Real *d_dti;
Real *d_cooling_weight;

Real *d_tracker;
Real h_tracker[] = {0, 0, 0, 0, 0};
int n_tracker    = 5;

int n_cluster;

}  // namespace Supernova

/*
__device__ double atomicMax(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(fmax(val, __longlong_as_double(assumed)))
                    );
  } while (assumed != old);
  return __longlong_as_double(old);
}
*/

__device__ Real Calc_Timestep(Real *hydro_dev, int gidx, int n_cells, Real gamma, Real dx, Real dy, Real dz)
{
  Real density = fmax(hydro_dev[gidx], TINY_NUMBER);
  Real d_inv   = 1.0 / density;
  Real vx      = d_inv * hydro_dev[gidx + n_cells];
  Real vy      = d_inv * hydro_dev[gidx + 2 * n_cells];
  Real vz      = d_inv * hydro_dev[gidx + 3 * n_cells];
  Real P       = fmax((hydro_dev[gidx + 4 * n_cells] - 0.5 * density * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0),
                      TINY_NUMBER);
  Real cs      = sqrt(d_inv * gamma * P);
  return fmax(fmax((fabs(vx) + cs) / dx, (fabs(vy) + cs) / dy), (fabs(vz) + cs) / dz);
}

void Supernova::Initialize_GPU(void)
{
  #include "cluster_list_MW.data"
  // Defines cluster_data in local scope so it is deleted
  n_cluster = sizeof(cluster_data) / sizeof(cluster_data[0]) / 5;
  GPU_Error_Check(cudaMalloc(&d_cluster_array, 5 * n_cluster * sizeof(Real)));
  cudaMemcpy(d_cluster_array, cluster_data, 5 * n_cluster * sizeof(Real), cudaMemcpyHostToDevice);
  GPU_Error_Check(cudaMalloc(&d_omega_array, n_cluster * sizeof(Real)));
  GPU_Error_Check(cudaMalloc(&d_flags_array, n_cluster * sizeof(bool)));
  GPU_Error_Check(cudaMalloc(&d_tracker, n_tracker * sizeof(Real)));
  cudaMemcpy(d_tracker, h_tracker, n_tracker * sizeof(Real), cudaMemcpyHostToDevice);

  #ifdef COOLING_GPU
  GPU_Error_Check(cudaMalloc(&d_cooling_weight, n_cells * sizeof(Real)));
  #endif

  Calc_Omega();
  InitializeS99();
}

void Supernova::InitializeS99(void)
{
  #include "S99_table.data"
  int n_entries = sizeof(s99_data) / sizeof(s99_data[0]) / 3;
  Real M_dot[n_entries];
  Real E_dot[n_entries];
  for (int i = 0; i < n_entries; i++) {
    M_dot[i] = s99_data[3 * i + 1];
    E_dot[i] = s99_data[3 * i + 2];
  }
  // Allocate M_dot and E_dot arrays on cuda
  GPU_Error_Check(cudaMalloc(&d_mdot, n_entries * sizeof(Real)));
  GPU_Error_Check(cudaMalloc(&d_edot, n_entries * sizeof(Real)));
  cudaMemcpy(d_mdot, M_dot, n_entries * sizeof(Real), cudaMemcpyHostToDevice);
  cudaMemcpy(d_edot, E_dot, n_entries * sizeof(Real), cudaMemcpyHostToDevice);
  GPU_Error_Check(cudaMalloc(&d_mdot_array, n_cluster * sizeof(Real)));
  GPU_Error_Check(cudaMalloc(&d_edot_array, n_cluster * sizeof(Real)));
}

__device__ Real distance(double x, double y, double z) { return x * x + y * y + z * z; }

__device__ Real Supernova_Helper(Real *hydro_dev, Real pos_x, Real pos_y, Real pos_z, Real dx, Real dy, Real dz,
                                 int local_i, int local_j, int local_k, int n_cells, int n_fields, Real R_cl,
                                 Real density, Real energy, int gidx)
{
  // Compute the effect on hydro_dev fields of a supernova at pos_x,y,z on grid index local_i,j,k for spacing dx,y,z

  // pos_x, pos_y, pos_z is supernova position relative to local grid
  // dx,dy,dz grid spacing
  // local_i, local_j, local_k indices relative to local grid

  // cell center distances to supernova
  Real xc = fabs((local_i + 0.5) * dx - pos_x);
  Real yc = fabs((local_j + 0.5) * dy - pos_y);
  Real zc = fabs((local_k + 0.5) * dz - pos_z);
  // cell corner distances to supernova
  Real rl = distance(fmax(xc - 0.5 * dx, 0.0), fmax(yc - 0.5 * dy, 0.0), fmax(zc - 0.5 * dz, 0.0));
  // fmax because if the distance becomes negative we should still inject
  Real rr    = distance(xc + 0.5 * dx, yc + 0.5 * dy, zc + 0.5 * dz);
  Real R_cl2 = R_cl * R_cl;
  Real weight;

  if (rl >= R_cl2) {
    // closest edge is outside, return
    return 0.0;
  }

  if (rr <= R_cl2) {
    // furthest edge is inside, entire cell is inside
    weight = 1.0;
  } else {
    // fraction of cell is inside, calculate fraction
    // Minor impact on kernel runtime
    int count = 0;

  #ifdef SUPERD
    for (int i = 0; i < 10; i++) {
      for (int j = 0; j < 10; j++) {
        for (int k = 0; k < 10; k++) {
          if (distance(xc + (0.1 * i - 0.45) * dx, yc + (0.1 * j - 0.45) * dy, zc + (0.1 * k - 0.45) * dz) < R_cl2) {
            count++;
          }
        }
      }
    }

    weight = count / 1000.0;
  #else
    for (int i = 0; i < 20; i++) {
      for (int j = 0; j < 20; j++) {
        for (int k = 0; k < 20; k++) {
          if (distance(xc + (0.05 * i - 0.5 + 0.025) * dx, yc + (0.05 * j - 0.5 + 0.025) * dy,
                       zc + (0.05 * k - 0.5 + 0.025) * dz) < R_cl2) {
            count++;
          }
        }
      }
    }
    weight = count / 8000.0;
  #endif  // SUPERD
  }       // endif

  // Add values to hydro_dev
  atomicAdd(&hydro_dev[gidx], weight * density);
  atomicAdd(&hydro_dev[gidx + 4 * n_cells], weight * energy);
  #ifdef BASIC_SCALAR
  atomicAdd(&hydro_dev[gidx + 5 * n_cells], 5.0 * weight * density);
  #endif
  #ifdef DE
  atomicAdd(&hydro_dev[gidx + (n_fields - 1) * n_cells], weight * energy);
  #endif

  return weight;
}

__global__ void Particle_Feedback_Kernel(Real *hydro_dev, Real *d_tracker, Real *pos_x_dev, Real *pos_y_dev,
                                         Real *pos_z_dev, Real xMin, Real yMin, Real zMin, Real dx, Real dy, Real dz,
                                         int nx, int ny, int nz, int n_cells, int n_fields, int n_ghost, Real R_cl,
                                         Real density, Real energy)
{
  // Assume x,y,z Min and Max are edges of grid[i][j][k]
  // nx,ny,nz are grid sizes

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute sizes based on R_cl
  int pnx     = (int)ceil(R_cl / dx);
  int pny     = (int)ceil(R_cl / dy);
  int pnz     = (int)ceil(R_cl / dz);
  int isize   = 1 + 2 * pnx;
  int jsize   = 1 + 2 * pny;
  int ksize   = 1 + 2 * pnz;
  int ijsize  = isize * jsize;
  int ijksize = ijsize * ksize;

  // Determine Particle
  int pre_pid = tid / ijksize;
  // TODO: calculate which particle by looping through flags until tid/ncells is satisfied
  int pid    = pre_pid;
  Real pos_x = pos_x_dev[pid] - xMin;
  Real pos_y = pos_y_dev[pid] - yMin;
  Real pos_z = pos_z_dev[pid] - zMin;

  // i,j,k of the block
  int rel_k = (tid - pre_pid * ijksize) / ijsize;
  int rel_j = (tid - pre_pid * ijksize - rel_k * ijsize) / isize;
  int rel_i = tid - pre_pid * ijksize - rel_k * ijsize - rel_j * isize;

  // particle cell location in grid - pni is the left corner of our kernel block
  // local_i is index of cell in grid index coordinates
  int local_i = (int)floor(pos_x / dx) - pnx + rel_i;
  int local_j = (int)floor(pos_y / dy) - pny + rel_j;
  int local_k = (int)floor(pos_z / dz) - pnz + rel_k;

  // Check if local cell is inside Grid
  if (local_i < 0 || local_j < 0 || local_k < 0) {
    return;
  }

  if (local_i >= nx || local_j >= ny || local_k >= nz) {
    return;
  }
  // int gidx = local_i + (local_j + local_k * ny) * nx;

  // Supernova_Helper(hydro_dev, d_tracker, pos_x, pos_y, pos_z, dx, dy, dz, local_i, local_j, local_k, n_cells,
  // n_fields, R_cl, density, energy, gidx);
}

// TODO Make version of Kernel which launches per-particle kernels

__global__ void Calc_Omega_Kernel(Real *cluster_array, Real *omega_array, int n_cluster)
{
  // cluster_array and omega_array should be on device
  // n_cluster is the total number
  Real r_pos;
  Real z_pos;
  Real r_sph, a, v;

  // properties of halo and disk
  Real a_disk_r, a_halo, a_halo_r;
  Real M_vir, M_d, R_vir, R_d, z_d, R_h, M_h, c_vir, phi_0_h, x;

  // GN is defined by global.h

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n_cluster) {
    return;
  }
  // Get r_pos and z_pos from cluster array
  //
  r_pos = cluster_array[5 * tid + 2];
  z_pos = cluster_array[5 * tid + 4];

  // for halo component, calculate spherical r
  r_sph = sqrt(r_pos * r_pos + z_pos * z_pos);

  #ifdef MW_MODEL
  // MW model
  M_d   = 6.5e10;     // virial mass in M_sun
  M_vir = 1.0e12;     // virial mass in M_sun
  R_d   = 3.5;        // disk scale length in kpc
  R_vir = 261.;       // virial radius in kpc
  z_d   = 3.5 / 5.0;  // disk scale height in kpc
  c_vir = 20.0;       // halo concentration

  #else
  // M82 model
  M_d   = 1.0e10;       // mass of disk in M_sun
  M_vir = 5.0e10;       // virial mass in M_sun
  R_d   = 0.8;          // disk scale length in kpc
  R_vir = R_d / 0.015;  // virial radius in kpc
  z_d   = 0.15;         // disk scale height in kpc
  c_vir = 10.0;         // halo concentration
  #endif  // MW_MODEL

  M_h     = M_vir - M_d;    // halo mass in M_sun
  R_h     = R_vir / c_vir;  // halo scale length in kpc
  phi_0_h = GN * M_h / (log(1.0 + c_vir) - c_vir / (1.0 + c_vir));
  x       = r_sph / R_h;

  // calculate acceleration due to NFW halo & Miyamoto-Nagai disk
  a_halo   = -phi_0_h * (log(1 + x) - x / (1 + x)) / (r_sph * r_sph);
  a_halo_r = a_halo * (r_pos / r_sph);
  a_disk_r = -GN * M_d * r_pos * pow(r_pos * r_pos + pow(R_d + sqrt(z_pos * z_pos + z_d * z_d), 2), -1.5);
  // total acceleration is the sum of the halo + disk components
  a = fabs(a_halo_r) + fabs(a_disk_r);
  // radial velocity
  v = sqrt(r_pos * a);
  // how far has the cluster gone?
  // omega = v/r_pos;
  omega_array[tid] = v / r_pos;
}

void Supernova::Calc_Omega(void)
{
  dim3 dim1dGrid((n_cluster + TPB - 1) / TPB, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);
  hipLaunchKernelGGL(Calc_Omega_Kernel, dim1dGrid, dim1dBlock, 0, 0, d_cluster_array, d_omega_array, n_cluster);
}

__global__ void Calc_Flag_Kernel(Real *cluster_array, Real *omega_array, bool *flag_array, Real *d_mdot, Real *d_edot,
                                 Real *d_mdot_array, Real *d_edot_array, int n_cluster, Real time, Real xMin, Real yMin,
                                 Real zMin, Real xMax, Real yMax, Real zMax, Real R_cl, Real SFR)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n_cluster) {
    return;
  }
  // Check if it is time for this cluster to be active
  // SF_cl/20000 < t < SF_cl/20000 + 40000
  Real total_SF     = cluster_array[5 * tid + 1];
  Real convert_time = (((time - total_SF / SFR) * 1e3) - 1e4) * 1e-5;
  int table_index   = __double2int_rd(convert_time);
  // int table_index = (int)floor(convert_time);

  if (table_index < 0) {
    flag_array[tid] = false;
    return;
  }
  // SB99 table goes up to 9e7 yr = 9e4 kyr (code time)
  // but we'll cut off at 80 Myr
  if (time > total_SF / SFR + 8e4) {
    flag_array[tid] = false;
    return;
  }
  // Check if this cluster can affect the domain
  // Z position Check
  Real pos_z = cluster_array[5 * tid + 4];
  if (pos_z > zMax + R_cl) {
    flag_array[tid] = false;
    return;
  }
  if (pos_z < zMin - R_cl) {
    flag_array[tid] = false;
    return;
  }
  // XY position checks
  Real pos_phi = cluster_array[5 * tid + 3] + omega_array[tid] * time;
  Real pos_r   = cluster_array[5 * tid + 2];
  Real pos_y   = pos_r * sin(pos_phi);
  Real pos_x   = pos_r * cos(pos_phi);

  if (pos_y > yMax + R_cl) {
    flag_array[tid] = false;
    return;
  }
  if (pos_y < yMin - R_cl) {
    flag_array[tid] = false;
    return;
  }
  if (pos_x > xMax + R_cl) {
    flag_array[tid] = false;
    return;
  }
  if (pos_x < xMin - R_cl) {
    flag_array[tid] = false;
    return;
  }

  /*
  if (pos_y > yMax + R_cl || pos_y < yMin - R_cl || pos_x > xMax + R_cl || pos_x < xMin - R_cl){
    flag_array[tid] = false;
    return;
  }
  */

  flag_array[tid] = true;

  // Use table to set arrays
  // 1e3 is KYR conversion
  // SB99 table starts at 1e4
  // 1e5 is SB99 table spacing
  // Real convert_time = ((time - total_SF/SFR)*1e3-1e4)*1e-5;
  // int table_index = (int)floor(convert_time);

  // If we got this far, then table_index will be a valid index for this array

  Real volume_cl      = (4. / 3.) * M_PI * R_cl * R_cl * R_cl;
  Real table_fraction = convert_time - table_index;
  Real f              = (cluster_array[5 * tid] * 1e-6) / volume_cl;
  Real M_slope        = d_mdot[table_index + 1] - d_mdot[table_index];
  Real E_slope        = d_edot[table_index + 1] - d_edot[table_index];
  d_mdot_array[tid]   = f * (d_mdot[table_index] + table_fraction * M_slope);
  d_edot_array[tid]   = f * pow(10, (d_edot[table_index] + table_fraction * E_slope)) * TIME_UNIT /
                      (MASS_UNIT * VELOCITY_UNIT * VELOCITY_UNIT);

  return;
}

void Supernova::Calc_Flags(Real time)
{
  GPU_Error_Check(cudaDeviceSynchronize());
  // double start_time = get_time();
  dim3 dim1dGrid((n_cluster + TPB - 1) / TPB, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);
  hipLaunchKernelGGL(Calc_Flag_Kernel, dim1dGrid, dim1dBlock, 0, 0, d_cluster_array, d_omega_array, d_flags_array,
                     d_mdot, d_edot, d_mdot_array, d_edot_array, n_cluster, time, xMin, yMin, zMin, xMax, yMax, zMax,
                     R_cl, SFR);
  GPU_Error_Check(cudaDeviceSynchronize());
  // double end_time = get_time();
  // chprintf("Supernova Calc Flags: %9.4f \n",1000*(end_time-start_time));
}

// Then make a kernel based on this flag kernel thing

// Lastly start doing some cuda timing tests on the flag + supernova step

__global__ void Supernova_Feedback_Kernel(Real *hydro_dev, Real *cluster_array, Real *omega_array, bool *flags_array,
                                          Real *d_mdot_array, Real *d_edot_array, Real *d_dti, Real *d_tracker,
                                          Real *d_cooling_weight, Real xMin, Real yMin, Real zMin, Real dx, Real dy,
                                          Real dz, int nx, int ny, int nz, int pnx, int pny, int pnz, int n_cells,
                                          int n_fields, int n_ghost, Real R_cl, Real density, Real gamma, Real time,
                                          Real dt, int max_pid, int supernova_e)
{
  // Assume x,y,z Min and Max are edges of grid[i][j][k]
  // nx,ny,nz are grid sizes

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute sizes based on R_cl
  /*
  int pnx = (int)ceil(R_cl/dx);
  int pny = (int)ceil(R_cl/dy);
  int pnz = (int)ceil(R_cl/dz);
  */
  int isize   = 1 + 2 * pnx;
  int jsize   = 1 + 2 * pny;
  int ksize   = 1 + 2 * pnz;
  int ijsize  = isize * jsize;
  int ijksize = ijsize * ksize;

  // Determine Particle
  int pre_pid = tid / ijksize;
  if (pre_pid >= max_pid) {
    return;
  }
  // TODO?: calculate which particle by looping through flags until tid/ncells is satisfied Not necessary, since
  // launching a kernel that gets to this point has almost no cost.
  int pid = pre_pid;
  if (!flags_array[pid]) {
    return;
  }
  Real pos_r   = cluster_array[5 * pid + 2];
  Real pos_phi = cluster_array[5 * pid + 3] + omega_array[pid] * time;
  Real pos_x   = pos_r * cos(pos_phi) - xMin;
  Real pos_y   = pos_r * sin(pos_phi) - yMin;
  Real pos_z   = cluster_array[5 * pid + 4] - zMin;

  // i,j,k of the block
  int rel_k = (tid - pre_pid * ijksize) / ijsize;
  int rel_j = (tid - pre_pid * ijksize - rel_k * ijsize) / isize;
  int rel_i = tid - pre_pid * ijksize - rel_k * ijsize - rel_j * isize;

  // particle cell location in grid - pni is the left corner of our kernel block
  // local_i is index of cell in grid index coordinates
  int local_i = (int)floor(pos_x / dx) - pnx + rel_i;
  int local_j = (int)floor(pos_y / dy) - pny + rel_j;
  int local_k = (int)floor(pos_z / dz) - pnz + rel_k;

  // Check if local cell is inside Grid
  if (local_i < 0 || local_j < 0 || local_k < 0) {
    return;
  }

  if (local_i >= nx || local_j >= ny || local_k >= nz) {
    return;
  }
  int gidx       = local_i + (local_j + local_k * ny) * nx;
  Real a_density = dt * d_mdot_array[pid];
  Real a_energy  = supernova_e * dt * d_edot_array[pid];

  // Supernova_Helper(hydro_dev, pos_x, pos_y, pos_z, dx, dy, dz, local_i, local_j, local_k, n_cells, n_fields, R_cl,
  // density, energy, gidx);
  Real weight = Supernova_Helper(hydro_dev, pos_x, pos_y, pos_z, dx, dy, dz, local_i, local_j, local_k, n_cells,
                                 n_fields, R_cl, a_density, a_energy, gidx);
  #ifdef COOLING_GPU
  if (weight > 0.5) {
    d_cooling_weight[gidx] = 1.0;
  }
  #endif
  if (weight > 0.0 && dt > 0.0) {
    Real dti = Calc_Timestep(hydro_dev, gidx, n_cells, gamma, dx, dy, dz);
    atomicMax(d_dti, dti);
  }

  // Tracker Code to track quantities

  if (weight <= 0.0) {
    return;
  }

  // Real cell update
  if (local_i >= n_ghost && local_j >= n_ghost && local_k >= n_ghost && local_i < nx - n_ghost &&
      local_j < ny - n_ghost && local_k < nz - n_ghost) {
    atomicAdd(&d_tracker[0], weight * a_density);
    atomicAdd(&d_tracker[1], weight * a_energy);
    if (dt > 0.0) {
      atomicAdd(&d_tracker[2], weight);
    }
  }

  // Particle is entirely contained within hydro grid
  if (pos_x >= R_cl && pos_y >= R_cl && pos_z >= R_cl && pos_x < nx * dx - R_cl && pos_y < ny * dy - R_cl &&
      pos_z < nz * dz - R_cl) {
    // Track volume in # cells
    atomicAdd(&d_tracker[3], weight);
    if (rel_i == pnx && rel_j == pny && rel_k == pnz) {
      // For central i,j,k add 1
      atomicAdd(&d_tracker[4], 1);
    }
  }

  // Add tracker values
  // Total mass if local_i,j,k not in ghost cell
  // Total energy if local_i,j,k not in ghost cell
  // Total volume if particle is eligible
  // # of if particle is eligible
  // Energy slated to be cooled

  return;
}

Real Supernova::Feedback(Real density, Real energy, Real time, Real dt)
{
  GPU_Error_Check(cudaDeviceSynchronize());

  Real h_dti = 0.0;
  Real *d_dti;
  cudaMalloc(&d_dti, sizeof(Real));
  cudaMemcpy(d_dti, &h_dti, sizeof(Real), cudaMemcpyHostToDevice);

  // Reset weights to 0
  #ifdef COOLING_GPU
  GPU_Error_Check(cudaMemset(d_cooling_weight, 0, n_cells * sizeof(Real)));
  #endif

  // double start_time = get_time();
  int isize = 1 + 2 * pnx;
  int jsize = 1 + 2 * pny;
  int ksize = 1 + 2 * pnz;
  // printf("n_cluster: %d\n",n_cluster);
  dim3 dim1dGrid((n_cluster * isize * jsize * ksize + TPB - 1) / TPB, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);
  hipLaunchKernelGGL(Supernova_Feedback_Kernel, dim1dGrid, dim1dBlock, 0, 0, d_hydro_array, d_cluster_array,
                     d_omega_array, d_flags_array, d_mdot_array, d_edot_array, d_dti, d_tracker, d_cooling_weight, xMin,
                     yMin, zMin, dx, dy, dz, nx, ny, nz, pnx, pny, pnz, n_cells, n_fields, n_ghost, R_cl, density, gama,
                     time, dt, n_cluster, supernova_e);
  GPU_Error_Check(cudaDeviceSynchronize());

  if (dt > 0.0) {
    cudaMemcpy(&h_dti, d_dti, sizeof(Real), cudaMemcpyDeviceToHost);
    cudaFree(d_dti);
  }
  // chprintf("h_dti: %9.4f \n",1./h_dti);
  //  double end_time = get_time();
  //  chprintf("Supernova Feedback Time: %9.4f \n",1000*(end_time-start_time));
  return h_dti;
}

void Supernova::Copy_Tracker()
{
  GPU_Error_Check(cudaDeviceSynchronize());
  GPU_Error_Check(cudaMemcpy(h_tracker, d_tracker, n_tracker * sizeof(Real), cudaMemcpyDeviceToHost));
  return;
}

#endif  // SUPERNOVA
