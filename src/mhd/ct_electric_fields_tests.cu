/*!
 * \file ct_electric_fields_tests.cu
 * \author Robert 'Bob' Caddy (rvc@pitt.edu)
 * \brief Tests for the CT electric fields
 *
 */

// STL Includes
#include <cmath>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>

// External Includes
#include <gtest/gtest.h>  // Include GoogleTest and related libraries/headers

// Local Includes
#include "../global/global.h"
#include "../io/io.h"
#include "../mhd/ct_electric_fields.h"
#include "../utils/testing_utilities.h"

#ifdef MHD
// =============================================================================
// Tests for the mhd::Calculate_CT_Electric_Fields kernel
// =============================================================================

// =============================================================================
/*!
 * \brief Test fixture for tMHDCalculateCTElectricFields test suite
 *
 */
// NOLINTNEXTLINE(readability-identifier-naming)
class tMHDCalculateCTElectricFields : public ::testing::Test
{
 public:
  /*!
   * \brief Initialize and allocate all the various required variables and
   * arrays
   *
   */
  tMHDCalculateCTElectricFields()
      : n_cells(nx * ny * nz),
        fluxX(n_cells * (grid_enum::num_flux_fields)),
        fluxY(n_cells * (grid_enum::num_flux_fields)),
        fluxZ(n_cells * (grid_enum::num_flux_fields)),
        grid(n_cells * (grid_enum::num_fields)),
        testCTElectricFields(n_cells * 3, -999.),
        fiducialData(n_cells * 3, -999.),
        dimGrid((n_cells + TPB - 1) / TPB, 1, 1),
        dimBlock(TPB, 1, 1)
  {
    // Allocate device arrays
    GPU_Error_Check(hipMalloc(&dev_fluxX, fluxX.size() * sizeof(double)));
    GPU_Error_Check(hipMalloc(&dev_fluxY, fluxY.size() * sizeof(double)));
    GPU_Error_Check(hipMalloc(&dev_fluxZ, fluxZ.size() * sizeof(double)));
    GPU_Error_Check(hipMalloc(&dev_grid, grid.size() * sizeof(double)));
    GPU_Error_Check(hipMalloc(&dev_testCTElectricFields, testCTElectricFields.size() * sizeof(double)));

    // Populate the grids with values where vector.at(i) = double(i). The
    // values chosen aren't that important, just that every cell has a unique
    // value
    std::iota(std::begin(fluxX), std::end(fluxX), 0.);
    std::iota(std::begin(fluxY), std::end(fluxY), fluxX.back() + 1);
    std::iota(std::begin(fluxZ), std::end(fluxZ), fluxY.back() + 1);
    std::iota(std::begin(grid), std::end(grid), fluxZ.back() + 1);
  }
  ~tMHDCalculateCTElectricFields() = default;

 protected:
  // Initialize the test grid and other state variables
  size_t const nx = 2, ny = nx, nz = nx;
  size_t const n_cells;

  // Launch Parameters
  dim3 const dimGrid;   // How many blocks in the grid
  dim3 const dimBlock;  // How many threads per block

  // Make sure the vector is large enough that the locations where the
  // magnetic field would be in the real grid are filled
  std::vector<double> fluxX;
  std::vector<double> fluxY;
  std::vector<double> fluxZ;
  std::vector<double> grid;
  std::vector<double> testCTElectricFields;
  std::vector<double> fiducialData;

  // device pointers
  double *dev_fluxX, *dev_fluxY, *dev_fluxZ, *dev_grid, *dev_testCTElectricFields;

  /*!
   * \brief Launch the kernel and check results
   *
   */
  void Run_Test()
  {
    // Copy values to GPU
    GPU_Error_Check(hipMemcpy(dev_fluxX, fluxX.data(), fluxX.size() * sizeof(Real), hipMemcpyHostToDevice));
    GPU_Error_Check(hipMemcpy(dev_fluxY, fluxY.data(), fluxY.size() * sizeof(Real), hipMemcpyHostToDevice));
    GPU_Error_Check(hipMemcpy(dev_fluxZ, fluxZ.data(), fluxZ.size() * sizeof(Real), hipMemcpyHostToDevice));
    GPU_Error_Check(hipMemcpy(dev_grid, grid.data(), grid.size() * sizeof(Real), hipMemcpyHostToDevice));
    GPU_Error_Check(hipMemcpy(dev_testCTElectricFields, testCTElectricFields.data(),
                               testCTElectricFields.size() * sizeof(Real), hipMemcpyHostToDevice));

    // Call the kernel to test
    hipLaunchKernelGGL(mhd::Calculate_CT_Electric_Fields, dimGrid, dimBlock, 0, 0, dev_fluxX, dev_fluxY, dev_fluxZ,
                       dev_grid, dev_testCTElectricFields, nx, ny, nz, n_cells);
    GPU_Error_Check();

    // Copy test data back
    GPU_Error_Check(hipMemcpy(testCTElectricFields.data(), dev_testCTElectricFields,
                               testCTElectricFields.size() * sizeof(Real), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    // Check the results
    for (size_t i = 0; i < fiducialData.size(); i++) {
      int xid, yid, zid;
      testing_utilities::Check_Results(fiducialData.at(i), testCTElectricFields.at(i),
                                       "value at i = " + std::to_string(i) + ", xid  = " + std::to_string(xid) +
                                           ", yid  = " + std::to_string(yid) + ", zid  = " + std::to_string(zid));
    }
  }
};
// =============================================================================

// =============================================================================
TEST_F(tMHDCalculateCTElectricFields, PositiveVelocityExpectCorrectOutput)
{
  // Fiducial values
  fiducialData.at(7)  = 60.951467108788492;
  fiducialData.at(15) = -98.736587665919359;
  fiducialData.at(23) = 61.768055665002557;

  // Launch kernel and check results
  Run_Test();
}
// =============================================================================

// =============================================================================
TEST_F(tMHDCalculateCTElectricFields, NegativeVelocityExpectCorrectOutput)
{
  // Fiducial values
  fiducialData.at(7)  = 59.978246483260179;
  fiducialData.at(15) = -97.279949010457187;
  fiducialData.at(23) = 61.280813140085613;

  // Set the density fluxes to be negative to indicate a negative velocity
  // across the face
  for (size_t i = 0; i < n_cells; i++) {
    fluxX.at(i) = -fluxX.at(i);
    fluxY.at(i) = -fluxY.at(i);
    fluxZ.at(i) = -fluxZ.at(i);
  }

  // Launch kernel and check results
  Run_Test();
}
// =============================================================================

// =============================================================================
TEST_F(tMHDCalculateCTElectricFields, ZeroVelocityExpectCorrectOutput)
{
  // Fiducial values
  fiducialData.at(7)  = 60.464856796024335;
  fiducialData.at(15) = -98.008268338188287;
  fiducialData.at(23) = 61.524434402544081;

  // Set the density fluxes to be negative to indicate a negative velocity
  // across the face
  for (size_t i = 0; i < n_cells; i++) {
    fluxX.at(i) = 0.0;
    fluxY.at(i) = 0.0;
    fluxZ.at(i) = 0.0;
  }

  // Launch kernel and check results
  Run_Test();
}
// =============================================================================

// =============================================================================
TEST(tMHDCTSlope, CorrectInputExpectCorrectOutput)
{
  // Set up the basic parameters
  size_t const nx      = 5;
  size_t const ny      = nx;
  size_t const nz      = nx;
  int const xid        = nx / 2;
  int const yid        = ny / 2;
  int const zid        = nz / 2;
  size_t const n_cells = nx * ny * nz;

  // Set up the grid
  std::vector<double> flux(grid_enum::num_fields * n_cells), conserved(grid_enum::num_fields * n_cells);

  std::mt19937 prng(1);
  std::uniform_real_distribution<double> doubleRand(-5, 5);

  for (double& conserved_data : conserved) {
    conserved_data = doubleRand(prng);
  }
  for (double& flux_data : flux) {
    flux_data = doubleRand(prng);
  }

  // Fiducial data
  std::vector<double> fiducial_data = {
      -6.8725060451062561, -77.056763568617669, 1.4564238051915397,  5.4541656143291437,  -0.83503550003671911,
      -78.091781647940934, -2.6187125848387525, -5.6934594000939542, -16.243259069749971, -59.321631150095314,
      0.99291378610068892, 4.4004574252725384,  -1.6902722376320516, -63.074645759822637, -4.5776373499662899,
      -19.476095152639683, -2.0173881091784471, -74.484407919605786, -7.8184484634991724, -0.23206265131850434,
      0.41622472388590037, -74.479121547383727, -6.9903417764222358, -1.832282425083853};

  // Get test data. Only test the options that will be used
  std::vector<double> test_data;
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 0, 2, -1, 1, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 0, -1, -1, 1, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 0, 1, 2, 1, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 0, 1, -1, 1, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 0, 1, -1, 1, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 0, -1, -1, 2, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 0, 1, 2, 1, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 0, 2, -1, -1, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 1, 2, -1, 0, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 1, -1, -1, 0, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 1, 0, 2, 0, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 1, 0, -1, 0, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 1, 0, -1, 0, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 1, -1, -1, 2, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 1, 0, 2, 0, 2, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 1, 2, -1, 2, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 2, 0, -1, 0, 1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 2, -1, -1, 1, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 2, 0, 1, 0, 1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), 1, 2, 1, -1, 1, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 2, 1, -1, 0, 1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 2, -1, -1, 0, -1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 2, 0, 1, 0, 1, xid, yid, zid, nx, ny, n_cells));
  test_data.emplace_back(
      mhd::internal::_ctSlope(flux.data(), conserved.data(), -1, 2, 0, -1, 0, -1, xid, yid, zid, nx, ny, n_cells));

  // Check the results
  ASSERT_EQ(test_data.size(), fiducial_data.size());

  for (size_t i = 0; i < test_data.size(); i++) {
    testing_utilities::Check_Results(fiducial_data.at(i), test_data.at(i), "");
  }
}
// =============================================================================
#endif  // MHD
