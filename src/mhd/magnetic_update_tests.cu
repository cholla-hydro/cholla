/*!
 * \file magnetic_update_tests.cu
 * \author Robert 'Bob' Caddy (rvc@pitt.edu)
 * \brief Tests for the magnetic update code
 *
 */

// STL Includes
#include <iostream>
#include <numeric>
#include <string>
#include <vector>

// External Includes
#include <gtest/gtest.h>  // Include GoogleTest and related libraries/headers

// Local Includes
#include "../mhd/magnetic_update.h"
#include "../utils/cuda_utilities.h"
#include "../utils/testing_utilities.h"

#ifdef MHD
// =============================================================================
/*!
 * \brief Test fixture for tMHDUpdateMagneticField3D test suite
 *
 */
// NOLINTNEXTLINE(readability-identifier-naming)
class tMHDUpdateMagneticField3D : public ::testing::Test
{
 public:
  /*!
   * \brief Initialize and allocate all the various required variables and
   * arrays
   *
   */
  tMHDUpdateMagneticField3D()
      : n_cells(nx * ny * nz),
        sourceGrid(n_cells * (grid_enum::num_fields)),
        destinationGrid(n_cells * (grid_enum::num_fields), -999.),
        ctElectricFields(n_cells * 3),
        fiducialData(n_cells * (grid_enum::num_fields), -999.),
        dimGrid((n_cells + TPB - 1) / TPB, 1, 1),
        dimBlock(TPB, 1, 1)
  {
    // Allocate device arrays
    GPU_Error_Check(hipMalloc(&dev_sourceGrid, sourceGrid.size() * sizeof(double)));
    GPU_Error_Check(hipMalloc(&dev_destinationGrid, destinationGrid.size() * sizeof(double)));
    GPU_Error_Check(hipMalloc(&dev_ctElectricFields, ctElectricFields.size() * sizeof(double)));

    // Populate the grids with values where vector.at(i) = double(i). The
    // values chosen aren't that important, just that every cell has a unique
    // value
    std::iota(std::begin(sourceGrid), std::end(sourceGrid), 0.);
    std::iota(std::begin(ctElectricFields), std::end(ctElectricFields), sourceGrid.back() + 1);
  }
  ~tMHDUpdateMagneticField3D() = default;

 protected:
  // Initialize the test grid and other state variables
  size_t const nx = 3, ny = nx, nz = nx;
  size_t const n_cells;
  Real const dt = 3.2, dx = 2.5, dy = dx, dz = dx;

  // Launch Parameters
  dim3 const dimGrid;   // How many blocks in the grid
  dim3 const dimBlock;  // How many threads per block

  // Make sure the vector is large enough that the locations where the
  // magnetic field would be in the real grid are filled
  std::vector<double> sourceGrid;
  std::vector<double> destinationGrid;
  std::vector<double> ctElectricFields;
  std::vector<double> fiducialData;

  // device pointers
  double *dev_sourceGrid, *dev_destinationGrid, *dev_ctElectricFields, *dev_fiducialData;

  /*!
   * \brief Launch the kernel and check results
   *
   */
  void Run_Test()
  {
    // Copy values to GPU
    GPU_Error_Check(
        hipMemcpy(dev_sourceGrid, sourceGrid.data(), sourceGrid.size() * sizeof(Real), hipMemcpyHostToDevice));
    GPU_Error_Check(hipMemcpy(dev_destinationGrid, destinationGrid.data(), destinationGrid.size() * sizeof(Real),
                               hipMemcpyHostToDevice));
    GPU_Error_Check(hipMemcpy(dev_ctElectricFields, ctElectricFields.data(), ctElectricFields.size() * sizeof(Real),
                               hipMemcpyHostToDevice));

    // Call the kernel to test
    hipLaunchKernelGGL(mhd::Update_Magnetic_Field_3D, dimGrid, dimBlock, 0, 0, dev_sourceGrid, dev_destinationGrid,
                       dev_ctElectricFields, nx, ny, nz, n_cells, dt, dx, dy, dz);
    GPU_Error_Check();

    // Copy test data back
    GPU_Error_Check(hipMemcpy(destinationGrid.data(), dev_destinationGrid, destinationGrid.size() * sizeof(Real),
                               hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    // Check the results
    for (size_t i = 0; i < fiducialData.size(); i++) {
      int xid, yid, zid;
      cuda_utilities::compute3DIndices(i, nx, ny, xid, yid, zid);
      testing_utilities::Check_Results(fiducialData.at(i), destinationGrid.at(i),
                                       "value at i = " + std::to_string(i) + ", xid  = " + std::to_string(xid) +
                                           ", yid  = " + std::to_string(yid) + ", zid  = " + std::to_string(zid));
    }
  }
};
// =============================================================================

// =============================================================================
TEST_F(tMHDUpdateMagneticField3D, CorrectInputExpectCorrectOutput)
{
  // Fiducial values
  fiducialData.at(148) = 155.68000000000001;
  fiducialData.at(175) = 164.75999999999999;
  fiducialData.at(202) = 204.56;

  // Launch kernel and check results
  Run_Test();
}
// =============================================================================
#endif  // MHD
