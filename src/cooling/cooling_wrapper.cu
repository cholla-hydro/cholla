
#include <hip/hip_runtime.h>
/*! \file cooling_wrapper.cu
 *  \brief Wrapper file for to load CUDA cooling tables. */

#ifdef CUDA
#ifdef CLOUDY_COOL

#include<stdio.h>
#include<stdlib.h>
#include"global.h"
#include"cooling_wrapper.h"
#include"cooling_cuda.h"

texture<float, 2, cudaReadModeElementType> coolTexObj;
texture<float, 2, cudaReadModeElementType> heatTexObj;
cudaArray* cuCoolArray;
cudaArray* cuHeatArray;


/* \fn void Load_Cuda_Textures()
 * \brief Load the Cloudy cooling tables into texture memory on the GPU. */
void Load_Cuda_Textures()
{

  float *cooling_table;
  float *heating_table;
  const int nx = 81;
  const int ny = 121;

  // allocate host arrays to be copied to textures
  // these arrays are declared as external pointers in global.h
  CudaSafeCall( cudaHostAlloc(&cooling_table, nx*ny*sizeof(float), cudaHostAllocDefault) );
  CudaSafeCall( cudaHostAlloc(&heating_table, nx*ny*sizeof(float), cudaHostAllocDefault) );

  // Load cooling tables into the host arrays
  Load_Cooling_Tables(cooling_table, heating_table);

  // Allocate CUDA arrays in device memory
  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatKindFloat);
  cudaMallocArray(&cuCoolArray, &channelDesc, nx, ny);
  cudaMallocArray(&cuHeatArray, &channelDesc, nx, ny);
  // Copy to device memory the cooling and heating arrays
  // in host memory
  cudaMemcpyToArray(cuCoolArray, 0, 0, cooling_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);
  cudaMemcpyToArray(cuHeatArray, 0, 0, heating_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);


  // Specify texture reference parameters (same for both tables)
  coolTexObj.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  coolTexObj.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  coolTexObj.filterMode = cudaFilterModeLinear; // bi-linear interpolation
  coolTexObj.normalized = true;
  heatTexObj.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  heatTexObj.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  heatTexObj.filterMode = cudaFilterModeLinear; // bi-linear interpolation
  heatTexObj.normalized = true;

  cudaBindTextureToArray(coolTexObj, cuCoolArray);
  cudaBindTextureToArray(heatTexObj, cuHeatArray);

  // Free the memory associated with the cooling tables on the host
  CudaSafeCall( cudaFreeHost(cooling_table) );
  CudaSafeCall( cudaFreeHost(heating_table) );

}


/* \fn void Load_Cooling_Tables(float* cooling_table, float* heating_table)
 * \brief Load the Cloudy cooling tables into host (CPU) memory. */
void Load_Cooling_Tables(float* cooling_table, float* heating_table)
{
  double *n_arr;
  double *T_arr;
  double *L_arr;
  double *H_arr;

  int i;
  int nx = 121;
  int ny = 81;

  FILE *infile;
  char buffer[0x1000];
  char * pch;

  // allocate arrays for temperature data
  n_arr = (double *) malloc(nx*ny*sizeof(double));
  T_arr = (double *) malloc(nx*ny*sizeof(double));
  L_arr = (double *) malloc(nx*ny*sizeof(double));
  H_arr = (double *) malloc(nx*ny*sizeof(double));

  // Read in cloudy cooling/heating curve (function of density and temperature)
  i=0;
  infile = fopen("../cooling/cloudy_coolingcurve.txt", "r");
  if (infile == NULL) {
    printf("Unable to open Cloudy file.\n");
    exit(1);
  }
  while (fgets(buffer, sizeof(buffer), infile) != NULL)
  {
    if (buffer[0] == '#') {
      continue;
    }
    else {
      pch = strtok(buffer, "\t");
      n_arr[i] = atof(pch);
      while (pch != NULL)
      {
        pch = strtok(NULL, "\t");
        if (pch != NULL)
          T_arr[i] = atof(pch);
        pch = strtok(NULL, "\t");
        if (pch != NULL)
          L_arr[i] = atof(pch);
        pch = strtok(NULL, "\t");
        if (pch != NULL)
          H_arr[i] = atof(pch);
      }
      i++;
    }
  }
  fclose(infile);

  // copy data from cooling array into the table
  for (i=0; i<nx*ny; i++)
  {
    cooling_table[i] = float(L_arr[i]);
    heating_table[i] = float(H_arr[i]);
  }

  // Free arrays used to read in table data
  free(n_arr);
  free(T_arr);
  free(L_arr);
  free(H_arr);
}


void Free_Cuda_Textures()
{
  // unbind the cuda textures
  cudaUnbindTexture(coolTexObj);
  cudaUnbindTexture(heatTexObj);

  // Free the device memory associated with the cuda arrays
  cudaFreeArray(cuCoolArray);
  cudaFreeArray(cuHeatArray);

}


#endif
#endif
