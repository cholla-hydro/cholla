
#include <hip/hip_runtime.h>
/*! \file cooling_cuda.cu
 *  \brief Functions to calculate cooling rate for a given rho, P, dt. */

#ifdef CUDA
#ifdef COOLING_GPU

#include"gpu.hpp"
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"cooling_cuda.h"

extern texture<float, 2, cudaReadModeElementType> coolTexObj;
extern texture<float, 2, cudaReadModeElementType> heatTexObj;

/*! \fn void cooling_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt, Real gamma)
 *  \brief When passed an array of conserved variables and a timestep, adjust the value
           of the total energy for each cell according to the specified cooling function. */
__global__ void cooling_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt, Real gamma, Real *dt_array)
{
  __shared__ Real min_dt[TPB];

  int n_cells = nx*ny*nz;
  int is, ie, js, je, ks, ke;
  is = n_ghost;
  ie = nx-n_ghost;
  if (ny == 1) {
    js = 0;
    je = 1;
  }
  else {
    js = n_ghost;
    je = ny-n_ghost;
  }
  if (nz == 1) {
    ks = 0;
    ke = 1;
  }
  else {
    ks = n_ghost;
    ke = nz-n_ghost;
  }
 
  Real d, E;
  Real n, T, T_init;
  Real del_T, dt_sub;
  Real mu; // mean molecular weight
  Real cool; //cooling rate per volume, erg/s/cm^3
  //#ifndef DE
  Real vx, vy, vz, p;
  //#endif
  #ifdef DE
  Real ge;
  #endif
  //Real T_min = 1.0e4; // minimum temperature allowed

  mu = 0.6;
  //mu = 1.27;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int id = threadIdx.x + blockId * blockDim.x;
  int zid = id / (nx*ny);
  int yid = (id - zid*nx*ny) / nx;
  int xid = id - zid*nx*ny - yid*nx;
  // and a thread id within the block
  int tid = threadIdx.x;

  // set min dt to a high number
  min_dt[tid] = 1e10;
  __syncthreads();
  
  // only threads corresponding to real cells do the calculation
  if (xid >= is && xid < ie && yid >= js && yid < je && zid >= ks && zid < ke) {

    // load values of density and pressure
    d  =  dev_conserved[            id];
    E  =  dev_conserved[4*n_cells + id];
    // don't apply cooling if this thread crashed
    if (E < 0.0 || E != E) return;
    //#ifndef DE
    vx =  dev_conserved[1*n_cells + id] / d;
    vy =  dev_conserved[2*n_cells + id] / d;
    vz =  dev_conserved[3*n_cells + id] / d;
    p  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    p  = fmax(p, (Real) TINY_NUMBER);
    //#endif
    #ifdef DE
    ge = dev_conserved[(n_fields-1)*n_cells + id] / d;
    ge = fmax(ge, (Real) TINY_NUMBER);
    #endif
    
    // calculate the number density of the gas (in cgs)
    n = d*DENSITY_UNIT / (mu * MP);

    // calculate the temperature of the gas
    //#ifndef DE
    T_init = p*PRESSURE_UNIT/ (n*KB);
    //#endif
    #ifdef DE
    //T_init = ge*(gamma-1.0)*SP_ENERGY_UNIT*mu*MP/KB;
    T_init = d*ge*(gamma-1.0)*PRESSURE_UNIT/(n*KB);
    #endif

    // calculate cooling rate per volume
    T = T_init;
    //if (T > T_max) printf("%3d %3d %3d High T cell. n: %e  T: %e\n", xid, yid, zid, n, T);
    // call the cooling function
    cool = CIE_cool(n, T); 
    //cool = Cloudy_cool(n, T); 
    
    // calculate change in temperature given dt
    del_T = cool*dt*TIME_UNIT*(gamma-1.0)/(n*KB);

    // limit change in temperature to 1%
    while (del_T/T > 0.01) {
      // what dt gives del_T = 0.01*T?
      dt_sub = 0.01*T*n*KB/(cool*TIME_UNIT*(gamma-1.0));
      // apply that dt
      T -= cool*dt_sub*TIME_UNIT*(gamma-1.0)/(n*KB);
      // how much time is left from the original timestep?
      dt -= dt_sub;
      // calculate cooling again
      cool = CIE_cool(n, T);
      //cool = Cloudy_cool(n, T);
      // calculate new change in temperature
      del_T = cool*dt*TIME_UNIT*(gamma-1.0)/(n*KB);
    }

    // calculate final temperature
    T -= del_T;

    // set a temperature floor
    // (don't change this cell if the thread crashed)
    //if (T > 0.0 && E > 0.0) T = fmax(T, T_min);
    // set a temperature ceiling 
    //T = fmin(T, T_max);

    // adjust value of energy based on total change in temperature
    del_T = T_init - T; // total change in T
    E -= n*KB*del_T / ((gamma-1.0)*ENERGY_UNIT);
    #ifdef DE
    ge -= KB*del_T / (mu*MP*(gamma-1.0)*SP_ENERGY_UNIT);
    #endif
    // calculate cooling rate for new T
    cool = CIE_cool(n, T);
    //cool = Cloudy_cool(n, T);
    //printf("%d %d %d %e %e %e\n", xid, yid, zid, n, T, cool);
    // only use good cells in timestep calculation (in case some have crashed)
    if (n > 0 && T > 0 && cool > 0.0) {
      // limit the timestep such that delta_T is 10% 
      min_dt[tid] = 0.1*T*n*KB/(cool*TIME_UNIT*(gamma-1.0));
    }

    // and send back from kernel
    dev_conserved[4*n_cells + id] = E;
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] = d*ge;
    #endif

  }
  __syncthreads();

  // do the reduction in shared memory (find the min timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      min_dt[tid] = fmin(min_dt[tid], min_dt[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dt_array[blockIdx.x] = min_dt[0];
  

}


/* \fn __device__ Real test_cool(Real n, Real T)
 * \brief Cooling function from Creasey 2011. */
__device__ Real test_cool(int tid, Real n, Real T)
{
  Real T0, T1, lambda, cool;
  T0 = 10000.0;
  T1 = 20*T0;
  cool = 0.0;
  //lambda = 5.0e-24; //cooling coefficient, 5e-24 erg cm^3 s^-1
  lambda = 5.0e-20; //cooling coefficient, 5e-24 erg cm^3 s^-1

  // constant cooling rate 
  //cool = n*n*lambda;

  // Creasey cooling function
  if (T >= T0 && T <= 0.5*(T1+T0)) {
    cool = n*n*lambda*(T - T0) / T0;
  }
  if (T >= 0.5*(T1+T0) && T <= T1) {
    cool = n*n*lambda*(T1 - T) / T0;
  }
 

  //printf("%d %f %f\n", tid, T, cool);
  return cool;

}


/* \fn __device__ Real primordial_cool(Real n, Real T)
 * \brief Primordial hydrogen/helium cooling curve 
          derived according to Katz et al. 1996. */
__device__ Real primordial_cool(Real n, Real T)
{
  Real n_h, Y, y, g_ff, cool;
  Real n_h0, n_hp, n_he0, n_hep, n_hepp, n_e, n_e_old; 
  Real alpha_hp, alpha_hep, alpha_d, alpha_hepp, gamma_eh0, gamma_ehe0, gamma_ehep;
  Real le_h0, le_hep, li_h0, li_he0, li_hep, lr_hp, lr_hep, lr_hepp, ld_hep, l_ff;
  Real gamma_lh0, gamma_lhe0, gamma_lhep, e_h0, e_he0, e_hep, H;
  int heat_flag, n_iter;
  Real diff, tol;

  // set flag to 1 for photoionization & heating
  heat_flag = 0;

  //Real X = 0.76; //hydrogen abundance by mass
  Y = 0.24; //helium abundance by mass
  y = Y/(4 - 4*Y);  

  // set the hydrogen number density 
  n_h = n; 

  // calculate the recombination and collisional ionziation rates
  // (Table 2 from Katz 1996)
  alpha_hp   = (8.4e-11) * (1.0/sqrt(T)) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7))));
  alpha_hep  = (1.5e-10) * (pow(T,(-0.6353)));
  alpha_d    = (1.9e-3)  * (pow(T,(-1.5))) * exp(-470000.0/T) * (1.0 + 0.3*exp(-94000.0/T));
  alpha_hepp = (3.36e-10)* (1.0/sqrt(T)) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7))));
  gamma_eh0  = (5.85e-11)* sqrt(T) * exp(-157809.1/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  gamma_ehe0 = (2.38e-11)* sqrt(T) * exp(-285335.4/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  gamma_ehep = (5.68e-12)* sqrt(T) * exp(-631515.0/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  // externally evaluated integrals for photoionziation rates
  // assumed J(nu) = 10^-22 (nu_L/nu)
  gamma_lh0 = 3.19851e-13;
  gamma_lhe0 = 3.13029e-13;
  gamma_lhep = 2.00541e-14; 
  // externally evaluated integrals for heating rates
  e_h0 = 2.4796e-24;
  e_he0 = 6.86167e-24;
  e_hep = 6.21868e-25; 
  

  // assuming no photoionization, solve equations for number density of
  // each species
  n_e = n_h; //as a first guess, use the hydrogen number density
  n_iter = 20;
  diff = 1.0;
  tol = 1.0e-6;
  if (heat_flag) { 
    for (int i=0; i<n_iter; i++) {
      n_e_old = n_e;
      n_h0   = n_h*alpha_hp / (alpha_hp + gamma_eh0 + gamma_lh0/n_e);
      n_hp   = n_h - n_h0;
      n_hep  = y*n_h / (1.0 + (alpha_hep + alpha_d)/(gamma_ehe0 + gamma_lhe0/n_e) + (gamma_ehep + gamma_lhep/n_e)/alpha_hepp );
      n_he0  = n_hep*(alpha_hep + alpha_d) / (gamma_ehe0 + gamma_lhe0/n_e);
      n_hepp = n_hep*(gamma_ehep + gamma_lhep/n_e)/alpha_hepp;
      n_e    = n_hp + n_hep + 2*n_hepp;
      diff = fabs(n_e_old - n_e);
      if (diff < tol) break;
    }
  }  
  else {
    n_h0   = n_h*alpha_hp / (alpha_hp + gamma_eh0);
    n_hp   = n_h - n_h0;
    n_hep  = y*n_h / (1.0 + (alpha_hep + alpha_d)/(gamma_ehe0) + (gamma_ehep)/alpha_hepp );
    n_he0  = n_hep*(alpha_hep + alpha_d) / (gamma_ehe0);
    n_hepp = n_hep*(gamma_ehep)/alpha_hepp;
    n_e    = n_hp + n_hep + 2*n_hepp;
  }

  // using number densities, calculate cooling rates for
  // various processes (Table 1 from Katz 1996)
  le_h0 = (7.50e-19) * exp(-118348.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_h0;
  le_hep = (5.54e-17) * pow(T,(-0.397)) * exp(-473638.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_hep;
  li_h0 = (1.27e-21) * sqrt(T) * exp(-157809.1/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_h0;
  li_he0 = (9.38e-22) * sqrt(T) * exp(-285335.4/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_he0;
  li_hep = (4.95e-22) * sqrt(T) * exp(-631515.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_hep;
  lr_hp = (8.70e-27) * sqrt(T) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7)))) * n_e * n_hp;
  lr_hep = (1.55e-26) * pow(T,(0.3647)) * n_e * n_hep;
  lr_hepp = (3.48e-26) * sqrt(T) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7)))) * n_e * n_hepp;
  ld_hep = (1.24e-13) * pow(T,(-1.5)) * exp(-470000.0/T) * (1.0 + 0.3*exp(-94000.0/T)) * n_e * n_hep;
  g_ff = 1.1 + 0.34*exp(-(5.5-log(T))*(5.5-log(T))/3.0); // Gaunt factor
  l_ff = (1.42e-27) * g_ff * sqrt(T) * (n_hp + n_hep + 4*n_hepp) * n_e;

  // calculate total cooling rate (erg s^-1 cm^-3)
  cool = le_h0 + le_hep + li_h0 + li_he0 + li_hep + lr_hp + lr_hep + lr_hepp + ld_hep + l_ff;

  // calculate total photoionization heating rate
  H = 0.0;
  if (heat_flag) {
    H = n_h0*e_h0 + n_he0*e_he0 + n_hep*e_hep; 
  }
  
  cool -= H;

  return cool;

}


/* \fn __device__ Real CIE_cool(Real n, Real T)
 * \brief Analytic fit to a solar metallicity CIE cooling curve 
          calculated using Cloudy. */
__device__ Real CIE_cool(Real n, Real T)
{
  Real lambda = 0.0; //cooling rate, erg s^-1 cm^3
  Real cool = 0.0; //cooling per unit volume, erg /s / cm^3
  
  // fit to CIE cooling function 
  if (log10(T) < 4.0) {
    lambda = 0.0;
  }
  else if (log10(T) >= 4.0 && log10(T) < 5.9) {
    lambda = pow(10.0, (-1.3 * (log10(T) - 5.25) * (log10(T) - 5.25) - 21.25));
  }
  else if (log10(T) >= 5.9 && log10(T) < 7.4) {
    lambda = pow(10.0, (0.7 * (log10(T) - 7.1) * (log10(T) - 7.1) - 22.8));
  }
  else {
    lambda = pow(10.0, (0.45*log10(T) - 26.065));
  }

  // cooling rate per unit volume
  cool = n*n*lambda;

  return cool;

}


#ifdef CLOUDY_COOL
/* \fn __device__ Real Cloudy_cool(Real n, Real T)
 * \brief Uses texture mapping to interpolate Cloudy cooling/heating 
          tables at z = 0 with solar metallicity and an HM05 UV background. */
__device__ Real Cloudy_cool(Real n, Real T)
{
  Real lambda = 0.0; //cooling rate, erg s^-1 cm^3
  Real H = 0.0; //heating rate, erg s^-1 cm^3
  Real cool = 0.0; //cooling per unit volume, erg /s / cm^3
  float log_n, log_T;
  log_n = log10(n);
  log_T = log10(T);

  // remap coordinates for texture
  log_T = (log_T - 1.0)/8.1;
  log_n = (log_n + 6.0)/12.1; 
 
  // don't cool below 10 K
  if (log10(T) > 1.0) {
  lambda = tex2D<float>(coolTexObj, log_T, log_n);
  }
  else lambda = 0.0;
  H = tex2D<float>(heatTexObj, log_T, log_n);

  // cooling rate per unit volume
  cool = n*n*(powf(10, lambda) - powf(10, H));

  return cool;
}
#endif


#endif //COOLING_GPU
#endif //CUDA
