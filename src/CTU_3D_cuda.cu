
#include <hip/hip_runtime.h>
/*! \file CTU_3D_cuda.cu
 *  \brief Definitions of the cuda 3D CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"gpu.hpp"
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_3D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_3D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_3D.h"
#include"io.h"


__global__ void Evolve_Interface_States_3D(Real *dev_conserved, Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           Real *dev_Q_Lz, Real *dev_Q_Rz, Real *dev_F_z,
                                           int nx, int ny, int nz, int n_ghost, 
                                           Real dx, Real dy, Real dz, Real dt, int n_fields);


Real CTU_Algorithm_3D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt, int n_fields , Real density_floor, Real U_floor, Real *host_grav_potential, Real max_dti_slow )
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array
  //host_conserved0 contains the values at time n,
  //host_conserved1 contains the values at time n+1
  
  // Initialize dt values 
  Real max_dti = 0;
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  #endif  

  if ( !block_size ) {
    // calculate the dimensions for the subgrid blocks
    sub_dimensions_3D(nx, ny, nz, n_ghost, &nx_s, &ny_s, &nz_s, &block1_tot, &block2_tot, &block3_tot, &remainder1, &remainder2, &remainder3, n_fields);
    //printf("Subgrid dimensions set: %d %d %d %d %d %d %d %d %d\n", nx_s, ny_s, nz_s, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3);
    //fflush(stdout);
    block_tot = block1_tot*block2_tot*block3_tot;
    // number of cells in one subgrid block
    BLOCK_VOL = nx_s*ny_s*nz_s;
    // dimensions for the 1D GPU grid
    ngrid = (BLOCK_VOL + TPB - 1) / TPB;
    #ifndef DYNAMIC_GPU_ALLOC
    block_size = true;
    #endif
  }
  // set values for GPU kernels
  // number of blocks per 1D grid  
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block   
  dim3 dim1dBlock(TPB, 1, 1);

  // Set up pointers for the location to copy from and to
  if (block_tot == 1) {
    tmp1 = host_conserved0;
    tmp2 = host_conserved1;
    //host_grav_potential is NULL if not using GRAVITY
    temp_potential = host_grav_potential;
  }

  if ( !memory_allocated ) {

    // allocate buffer to copy conserved variable blocks to/from
    if (block_tot > 1) {
      if ( cudaSuccess != cudaHostAlloc(&buffer, n_fields*BLOCK_VOL*sizeof(Real), cudaHostAllocDefault) ) {
        printf("Failed to allocate CPU buffer.\n");
      }
      tmp1 = buffer;
      tmp2 = buffer;
      
      #if defined( GRAVITY ) 
      if ( cudaSuccess != cudaHostAlloc(&buffer_potential, BLOCK_VOL*sizeof(Real), cudaHostAllocDefault) ) {
        printf("Failed to allocate CPU Grav_Potential buffer.\n");
      }
      #else
      buffer_potential = NULL;
      #endif
      temp_potential = buffer_potential;
    }

    // allocate an array on the CPU to hold max_dti returned from each thread block
    CudaSafeCall( cudaHostAlloc(&host_dti_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaHostAlloc(&host_dt_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #endif

    // allocate memory on the GPU
    CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif
    
    #if defined( GRAVITY ) 
    CudaSafeCall( cudaMalloc((void**)&dev_grav_potential, BLOCK_VOL*sizeof(Real)) );
    #else
    dev_grav_potential = NULL;
    #endif
        
    #ifndef DYNAMIC_GPU_ALLOC 
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif 
  }  

  // counter for which block we're on
  int block = 0;


  // START LOOP OVER SUBGRID BLOCKS
  while (block < block_tot) {

    // copy the conserved variable block to the buffer
    host_copy_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved0, buffer, n_fields, host_grav_potential, buffer_potential);

    get_offsets_3D(nx_s, ny_s, nz_s, n_ghost, x_off, y_off, z_off, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, &x_off_s, &y_off_s, &z_off_s);

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );
    #if defined( GRAVITY ) 
    CudaSafeCall( cudaMemcpy(dev_grav_potential, temp_potential, BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );
    #endif
   

    // Step 1: Do the reconstruction
    #ifdef PCM
    hipLaunchKernelGGL(PCM_Reconstruction_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    #endif //PCM
    #ifdef PLMP
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PLMP 
    #ifdef PLMC
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PLMC 
    #ifdef PPMP
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMP
    #ifdef PPMC
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMC
    CudaCheckError();
   

    // Step 2: Calculate the fluxes
    #ifdef EXACT
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    #ifdef HLL 
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    CudaCheckError();


    #ifdef CTU
    // Step 3: Evolve the interface states
    hipLaunchKernelGGL(Evolve_Interface_States_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, F_x, Q_Ly, Q_Ry, F_y, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dt, n_fields);
    CudaCheckError();


    // Step 4: Calculate the fluxes again
    #ifdef EXACT
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    #ifdef HLL 
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    CudaCheckError();
    #endif //CTU
    
    #ifdef DE
    // Compute the divergence of Vel before updating the conserved array, this solves sincronization issues when adding this term on Update_Conserved_Variables_3D
    hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0,  dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dt, gama, n_fields );
    CudaCheckError();
    #endif

  
    // Step 5: Update the conserved variable array
    hipLaunchKernelGGL(Update_Conserved_Variables_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, F_x, F_y, F_z, nx_s, ny_s, nz_s, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, dt, gama, n_fields, density_floor, dev_grav_potential );
    CudaCheckError();


    // Synchronize the total and internal energies
    #ifdef DE
    hipLaunchKernelGGL(Select_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields);
    hipLaunchKernelGGL(Sync_Energies_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    CudaCheckError();
    #endif //DE
    
    #ifdef TEMPERATURE_FLOOR
    hipLaunchKernelGGL(Apply_Temperature_Floor, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, U_floor );
    CudaCheckError();
    #endif //TEMPERATURE_FLOOR


    // Apply cooling
    #ifdef COOLING_GPU
    hipLaunchKernelGGL(cooling_kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, dt, gama, dev_dt_array);
    CudaCheckError();
    #endif


    // Step 6: Calculate the next timestep
    hipLaunchKernelGGL(Calc_dt_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dev_dti_array, gama, max_dti_slow );
    CudaCheckError();

  

    // copy the updated conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );
    CudaCheckError();

    // copy the updated conserved variable array from the buffer into the host_conserved array on the CPU
    host_return_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved1, buffer, n_fields);

    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }
    #ifdef COOLING_GPU
    // copy the dt array from cooling onto the CPU
    CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // find maximum inverse timestep from cooling time
    for (int i=0; i<ngrid; i++) {
      min_dt = fmin(min_dt, host_dt_array[i]);
    }  
    if (min_dt < C_cfl/max_dti) {
      max_dti = C_cfl/min_dt;
    }
    #endif

    // add one to the counter
    block++;

  }


  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_CTU_3D();
  #endif


  // return the maximum inverse timestep
  return max_dti;

}


void Free_Memory_CTU_3D() {

  // free CPU memory
  if (block_tot > 1) CudaSafeCall( cudaFreeHost(buffer) );
  CudaSafeCall( cudaFreeHost(host_dti_array) );
  #ifdef COOLING_GPU
  CudaSafeCall( cudaFeeHost(host_dt_array) );  
  #endif

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif
  #if defined( GRAVITY ) 
  cudaFree(dev_grav_potential);
  if (block_tot > 1) CudaSafeCall( cudaFreeHost(buffer_potential) );
  #endif

}


__global__ void Evolve_Interface_States_3D(Real *dev_conserved, Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           Real *dev_Q_Lz, Real *dev_Q_Rz, Real *dev_F_z,
                                           int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, int n_fields)
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  int n_cells = nx*ny*nz;

  // get a thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;
  int id = xid + yid*nx + zid*nx*ny;

  if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid > n_ghost-2 && yid < ny-n_ghost+1 && zid > n_ghost-2 && zid < nz-n_ghost+1)
  {
    // set the new x interface states
    // left
    int ipo = xid+1 + yid*nx + zid*nx*ny;
    int jmo = xid + (yid-1)*nx + zid*nx*ny;
    int kmo = xid + yid*nx + (zid-1)*nx*ny;
    int ipojmo = xid+1 + (yid-1)*nx + zid*nx*ny;
    int ipokmo = xid+1 + yid*nx + (zid-1)*nx*ny;
    dev_Q_Lx[            id] += 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id])
                              + 0.5*dtodz*(dev_F_z[            kmo] - dev_F_z[            id]);
    dev_Q_Lx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                              + 0.5*dtodz*(dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_Q_Lx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_Q_Lx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_Q_Lx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Lx[(5+i)*n_cells + id] += 0.5*dtody*(dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                + 0.5*dtodz*(dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
    }                          
    #endif
    #ifdef DE
    dev_Q_Lx[(n_fields-1)*n_cells + id] += 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id]);
    #endif

    // right
    dev_Q_Rx[            id] += 0.5*dtody*(dev_F_y[            ipojmo] - dev_F_y[            ipo])
                              + 0.5*dtodz*(dev_F_z[            ipokmo] - dev_F_z[            ipo]); 
    dev_Q_Rx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + ipojmo] - dev_F_y[  n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[  n_cells + ipokmo] - dev_F_z[  n_cells + ipo]);
    dev_Q_Rx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + ipojmo] - dev_F_y[2*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[2*n_cells + ipokmo] - dev_F_z[2*n_cells + ipo]);
    dev_Q_Rx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + ipojmo] - dev_F_y[3*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[3*n_cells + ipokmo] - dev_F_z[3*n_cells + ipo]);
    dev_Q_Rx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + ipojmo] - dev_F_y[4*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[4*n_cells + ipokmo] - dev_F_z[4*n_cells + ipo]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Rx[(5+i)*n_cells + id] += 0.5*dtody*(dev_F_y[(5+i)*n_cells + ipojmo] - dev_F_y[(5+i)*n_cells + ipo])
                                + 0.5*dtodz*(dev_F_z[(5+i)*n_cells + ipokmo] - dev_F_z[(5+i)*n_cells + ipo]);
    }                          
    #endif
    #ifdef DE
    dev_Q_Rx[(n_fields-1)*n_cells + id] += 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + ipojmo] - dev_F_y[(n_fields-1)*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + ipokmo] - dev_F_z[(n_fields-1)*n_cells + ipo]);
    #endif
  }
  if (yid > n_ghost-3 && yid < ny-n_ghost+1 && xid > n_ghost-2 && xid < nx-n_ghost+1 && zid > n_ghost-2 && zid < nz-n_ghost+1)
  {
    // set the new y interface states
    // left
    int jpo = xid + (yid+1)*nx + zid*nx*ny;
    int imo = xid-1 + yid*nx + zid*nx*ny;
    int kmo = xid + yid*nx + (zid-1)*nx*ny;
    int jpoimo = xid-1 + (yid+1)*nx + zid*nx*ny;
    int jpokmo = xid + (yid+1)*nx + (zid-1)*nx*ny;
    dev_Q_Ly[            id] += 0.5*dtodz*(dev_F_z[            kmo] - dev_F_z[            id])
                              + 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id]);
    dev_Q_Ly[  n_cells + id] += 0.5*dtodz*(dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id])
                              + 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]);
    dev_Q_Ly[2*n_cells + id] += 0.5*dtodz*(dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]);
    dev_Q_Ly[3*n_cells + id] += 0.5*dtodz*(dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id]);
    dev_Q_Ly[4*n_cells + id] += 0.5*dtodz*(dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Ly[(5+i)*n_cells + id] += 0.5*dtodz*(dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id])
                                + 0.5*dtodx*(dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_Q_Ly[(n_fields-1)*n_cells + id] += 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id]);
    #endif

    // right
    dev_Q_Ry[            id] += 0.5*dtodz*(dev_F_z[            jpokmo] - dev_F_z[            jpo])
                              + 0.5*dtodx*(dev_F_x[            jpoimo] - dev_F_x[            jpo]); 
    dev_Q_Ry[  n_cells + id] += 0.5*dtodz*(dev_F_z[  n_cells + jpokmo] - dev_F_z[  n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[  n_cells + jpoimo] - dev_F_x[  n_cells + jpo]);
    dev_Q_Ry[2*n_cells + id] += 0.5*dtodz*(dev_F_z[2*n_cells + jpokmo] - dev_F_z[2*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[2*n_cells + jpoimo] - dev_F_x[2*n_cells + jpo]);
    dev_Q_Ry[3*n_cells + id] += 0.5*dtodz*(dev_F_z[3*n_cells + jpokmo] - dev_F_z[3*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[3*n_cells + jpoimo] - dev_F_x[3*n_cells + jpo]);
    dev_Q_Ry[4*n_cells + id] += 0.5*dtodz*(dev_F_z[4*n_cells + jpokmo] - dev_F_z[4*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[4*n_cells + jpoimo] - dev_F_x[4*n_cells + jpo]);    
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Ry[(5+i)*n_cells + id] += 0.5*dtodz*(dev_F_z[(5+i)*n_cells + jpokmo] - dev_F_z[(5+i)*n_cells + jpo])
                                + 0.5*dtodx*(dev_F_x[(5+i)*n_cells + jpoimo] - dev_F_x[(5+i)*n_cells + jpo]);    
    }                            
    #endif
    #ifdef DE
    dev_Q_Ry[(n_fields-1)*n_cells + id] += 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + jpokmo] - dev_F_z[(n_fields-1)*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + jpoimo] - dev_F_x[(n_fields-1)*n_cells + jpo]);    
    #endif
  }
  if (zid > n_ghost-3 && zid < nz-n_ghost+1 && xid > n_ghost-2 && xid < nx-n_ghost+1 && yid > n_ghost-2 && yid < ny-n_ghost+1)
  {
    // set the new z interface states
    // left
    int kpo = xid + yid*nx + (zid+1)*nx*ny;
    int imo = xid-1 + yid*nx + zid*nx*ny;
    int jmo = xid + (yid-1)*nx + zid*nx*ny;
    int kpoimo = xid-1 + yid*nx + (zid+1)*nx*ny;
    int kpojmo = xid + (yid-1)*nx + (zid+1)*nx*ny;
    dev_Q_Lz[            id] += 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id])
                              + 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id]);
    dev_Q_Lz[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                              + 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_Q_Lz[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                              + 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]);
    dev_Q_Lz[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                              + 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_Q_Lz[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                              + 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Lz[(5+i)*n_cells + id] += 0.5*dtodx*(dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                + 0.5*dtody*(dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_Q_Lz[(n_fields-1)*n_cells + id] += 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                              + 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id]);
    #endif
    // right
    dev_Q_Rz[            id] += 0.5*dtodx*(dev_F_x[            kpoimo] - dev_F_x[            kpo])
                              + 0.5*dtody*(dev_F_y[            kpojmo] - dev_F_y[            kpo]); 
    dev_Q_Rz[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + kpoimo] - dev_F_x[  n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[  n_cells + kpojmo] - dev_F_y[  n_cells + kpo]);
    dev_Q_Rz[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + kpoimo] - dev_F_x[2*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[2*n_cells + kpojmo] - dev_F_y[2*n_cells + kpo]);
    dev_Q_Rz[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + kpoimo] - dev_F_x[3*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[3*n_cells + kpojmo] - dev_F_y[3*n_cells + kpo]);
    dev_Q_Rz[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + kpoimo] - dev_F_x[4*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[4*n_cells + kpojmo] - dev_F_y[4*n_cells + kpo]);    
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Rz[(5+i)*n_cells + id] += 0.5*dtodx*(dev_F_x[(5+i)*n_cells + kpoimo] - dev_F_x[(5+i)*n_cells + kpo])
                                + 0.5*dtody*(dev_F_y[(5+i)*n_cells + kpojmo] - dev_F_y[(5+i)*n_cells + kpo]);    
    }                            
    #endif
    #ifdef DE
    dev_Q_Rz[(n_fields-1)*n_cells + id] += 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + kpoimo] - dev_F_x[(n_fields-1)*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + kpojmo] - dev_F_y[(n_fields-1)*n_cells + kpo]);    
    #endif
  }

}



#endif //CUDA
