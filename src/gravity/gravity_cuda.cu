
#include <hip/hip_runtime.h>
/*! \file gravity_cuda.cu
 *  \brief Definitions of functions to calculate gravitational
           acceleration in 1, 2, and 3D. Called in Update_Conserved_Variables
           functions in hydro_cuda.cu. */
#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include"gpu.hpp"
#include"global.h"
#include"global_cuda.h"
#include"gravity_cuda.h"

// Work around lack of pow(Real,int) in Hip Clang for Rocm 3.5
static inline __device__ Real pow2(const Real x) { return x*x; }

__device__ void calc_g_1D(int xid, int x_off, int n_ghost, Real dx, Real xbound, Real *gx)
{
  Real x_pos, r_disk, r_halo;
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;

  // for disk components, calculate polar r
  //r_disk = 0.220970869121;
  //r_disk = 6.85009694274;
  r_disk = 13.9211647546;
  //r_disk = 20.9922325665;
  // for halo, calculate spherical r
  r_halo = sqrt(x_pos*x_pos + r_disk*r_disk);

  // set properties of halo and disk (these must match initial conditions)
  Real a_disk_z, a_halo, M_vir, M_d, R_vir, R_d, z_d, R_h, M_h, c_vir, phi_0_h, x;
  M_vir = 1.0e12; // viral mass of MW in M_sun
  M_d = 6.5e10; // mass of disk in M_sun
  M_h = M_vir - M_d; // halo mass in M_sun
  R_vir = 261; // viral radius in kpc
  c_vir = 20.0; // halo concentration
  R_h = R_vir / c_vir; // halo scale length in kpc
  R_d = 3.5; // disk scale length in kpc
  z_d = 3.5/5.0; // disk scale height in kpc
  phi_0_h = GN * M_h / (log(1.0+c_vir) - c_vir / (1.0+c_vir));
  x = r_halo / R_h;
  
  // calculate acceleration due to NFW halo & Miyamoto-Nagai disk
  a_halo = - phi_0_h * (log(1+x) - x/(1+x)) / (r_halo*r_halo);
  a_disk_z = - GN * M_d * x_pos * (R_d + sqrt(x_pos*x_pos + z_d*z_d)) / ( pow(r_disk*r_disk + pow2(R_d + sqrt(x_pos*x_pos + z_d*z_d)), 1.5) * sqrt(x_pos*x_pos + z_d*z_d) );

  // total acceleration is the sum of the halo + disk components
  *gx = (x_pos/r_halo)*a_halo + a_disk_z;

  return;

}


__device__ void calc_g_2D(int xid, int yid, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real *gx, Real *gy)
{
  Real x_pos, y_pos, r, phi;
  // use the subgrid offset and global boundaries to calculate absolute positions on the grid
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;
  y_pos = (y_off + yid - n_ghost + 0.5)*dy + ybound;

  // for Gresho, also need r & phi
  r = sqrt(x_pos*x_pos + y_pos*y_pos);
  phi = atan2(y_pos, x_pos);

/*
  // set acceleration to balance v_phi in Gresho problem
  if (r < 0.2) {
    *gx = -cos(phi)*25.0*r;
    *gy = -sin(phi)*25.0*r;
  }
  else if (r >= 0.2 && r < 0.4) {
    *gx = -cos(phi)*(4.0 - 20.0*r + 25.0*r*r)/r;
    *gy = -sin(phi)*(4.0 - 20.0*r + 25.0*r*r)/r;
  }
  else {
    *gx = 0.0;
    *gy = 0.0;
  }
*/
/*
  // set gravitational acceleration for Keplarian potential
  Real M;
  M = 1*Msun;
  *gx = -cos(phi)*GN*M/(r*r);
  *gy = -sin(phi)*GN*M/(r*r);
*/
  // set gravitational acceleration for Kuzmin disk + NFW halo
  Real a_d, a_h, a, M_vir, M_d, R_vir, R_d, R_s, M_h, c_vir, x;
  M_vir = 1.0e12; // viral mass of MW in M_sun
  M_d = 6.5e10; // mass of disk in M_sun (assume all gas)
  M_h = M_vir - M_d; // halo mass in M_sun
  R_vir = 261; // viral radius in kpc
  c_vir = 20; // halo concentration
  R_s = R_vir / c_vir; // halo scale length in kpc
  R_d = 3.5; // disk scale length in kpc
  
  // calculate acceleration
  x = r / R_s;
  a_d = GN * M_d * r * pow(r*r + R_d*R_d, -1.5);
  a_h = GN * M_h * (log(1+x)- x / (1+x)) / ((log(1+c_vir) - c_vir / (1+c_vir)) * r*r);
  a = a_d + a_h;

  *gx = -cos(phi)*a;
  *gy = -sin(phi)*a;

  return;
}


__device__ void calc_g_3D(int xid, int yid, int zid, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real *gx, Real *gy, Real *gz)
{
  Real x_pos, y_pos, z_pos, r_disk, r_halo;
  // use the subgrid offset and global boundaries to calculate absolute positions on the grid
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;
  y_pos = (y_off + yid - n_ghost + 0.5)*dy + ybound;
  z_pos = (z_off + zid - n_ghost + 0.5)*dz + zbound;

  // for disk components, calculate polar r
  r_disk = sqrt(x_pos*x_pos + y_pos*y_pos);
  // for halo, calculate spherical r
  r_halo = sqrt(x_pos*x_pos + y_pos*y_pos + z_pos*z_pos);

  // set properties of halo and disk (these must match initial conditions)
  Real a_disk_r, a_disk_z, a_halo, a_halo_r, a_halo_z;
  Real M_vir, M_d, R_vir, R_d, z_d, R_h, M_h, c_vir, phi_0_h, x;
  // MW model
  M_vir = 1.0e12; // viral mass of in M_sun
  M_d = 6.5e10; // viral mass of in M_sun
  R_d = 3.5; // disk scale length in kpc
  z_d = 3.5/5.0; // disk scale height in kpc
  R_vir = 261.; // virial radius in kpc
  c_vir = 20.0; // halo concentration
  // M82 model
  //M_vir = 5.0e10; // viral mass of in M_sun
  //M_d = 1.0e10; // mass of disk in M_sun
  //R_d = 0.8; // disk scale length in kpc
  //z_d = 0.15; // disk scale height in kpc
  //R_vir = R_d/0.015; // viral radius in kpc
  //c_vir = 10.0; // halo concentration

  M_h = M_vir - M_d; // halo mass in M_sun
  R_h = R_vir / c_vir; // halo scale length in kpc
  phi_0_h = GN * M_h / (log(1.0+c_vir) - c_vir / (1.0+c_vir));
  x = r_halo / R_h;
  
  // calculate acceleration due to NFW halo & Miyamoto-Nagai disk
  a_halo = - phi_0_h * (log(1+x) - x/(1+x)) / (r_halo*r_halo);
  a_halo_r = a_halo*(r_disk/r_halo);
  a_halo_z = a_halo*(z_pos/r_halo);
  a_disk_r = - GN * M_d * r_disk * pow(r_disk*r_disk+ pow2(R_d + sqrt(z_pos*z_pos + z_d*z_d)), -1.5);
  a_disk_z = - GN * M_d * z_pos * (R_d + sqrt(z_pos*z_pos + z_d*z_d)) / ( pow(r_disk*r_disk + pow2(R_d + sqrt(z_pos*z_pos + z_d*z_d)), 1.5) * sqrt(z_pos*z_pos + z_d*z_d) );

  // total acceleration is the sum of the halo + disk components
  *gx = (x_pos/r_disk)*(a_disk_r+a_halo_r);
  *gy = (y_pos/r_disk)*(a_disk_r+a_halo_r);
  *gz = a_disk_z+a_halo_z;

  return;
}

#endif //CUDA

