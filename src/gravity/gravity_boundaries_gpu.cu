
#include <hip/hip_runtime.h>
#if defined(GRAVITY) && defined(GRAVITY_GPU)

  #include <cmath>

  #include "../gravity/grav3D.h"
  #include "../grid/grid3D.h"
  #include "../io/io.h"

  #if defined(GRAV_ISOLATED_BOUNDARY_X) || defined(GRAV_ISOLATED_BOUNDARY_Y) || defined(GRAV_ISOLATED_BOUNDARY_Z)

void __global__ Set_Potential_Boundaries_Isolated_kernel(int direction, int side, int size_buffer, int n_i, int n_j,
                                                         int nx, int ny, int nz, int n_ghost, Real *potential_d,
                                                         Real *pot_boundary_d)
{
  // get a global thread ID
  int tid, tid_i, tid_j, tid_k, tid_buffer, tid_pot;
  tid   = threadIdx.x + blockIdx.x * blockDim.x;
  tid_k = tid / (n_i * n_j);
  tid_j = (tid - tid_k * n_i * n_j) / n_i;
  tid_i = tid - tid_k * n_i * n_j - tid_j * n_i;

  if (tid_i < 0 || tid_i >= n_i || tid_j < 0 || tid_j >= n_j || tid_k < 0 || tid_k >= n_ghost) {
    return;
  }

  tid_buffer = tid_i + tid_j * n_i + tid_k * n_i * n_j;

  if (direction == 0) {
    if (side == 0) {
      tid_pot = (tid_k) + (tid_i + n_ghost) * nx + (tid_j + n_ghost) * nx * ny;
    }
    if (side == 1) {
      tid_pot = (nx - n_ghost + tid_k) + (tid_i + n_ghost) * nx + (tid_j + n_ghost) * nx * ny;
    }
  }
  if (direction == 1) {
    if (side == 0) {
      tid_pot = (tid_i + n_ghost) + (tid_k)*nx + (tid_j + n_ghost) * nx * ny;
    }
    if (side == 1) {
      tid_pot = (tid_i + n_ghost) + (ny - n_ghost + tid_k) * nx + (tid_j + n_ghost) * nx * ny;
    }
  }
  if (direction == 2) {
    if (side == 0) {
      tid_pot = (tid_i + n_ghost) + (tid_j + n_ghost) * nx + (tid_k)*nx * ny;
    }
    if (side == 1) {
      tid_pot = (tid_i + n_ghost) + (tid_j + n_ghost) * nx + (nz - n_ghost + tid_k) * nx * ny;
    }
  }

  potential_d[tid_pot] = pot_boundary_d[tid_buffer];
}

void Grid3D::Set_Potential_Boundaries_Isolated_GPU(int direction, int side, int *flags)
{
  int n_i, n_j, n_ghost, size_buffer;
  int nx_g, ny_g, nz_g;
  n_ghost = N_GHOST_POTENTIAL;
  nx_g    = Grav.nx_local + 2 * n_ghost;
  ny_g    = Grav.ny_local + 2 * n_ghost;
  nz_g    = Grav.nz_local + 2 * n_ghost;

  Real *pot_boundary_h, *pot_boundary_d;
    #ifdef GRAV_ISOLATED_BOUNDARY_X
  if (direction == 0) {
    n_i = Grav.ny_local;
    n_j = Grav.nz_local;
    if (side == 0) {
      pot_boundary_h = Grav.F.pot_boundary_x0;
    }
    if (side == 1) {
      pot_boundary_h = Grav.F.pot_boundary_x1;
    }
    if (side == 0) {
      pot_boundary_d = Grav.F.pot_boundary_x0_d;
    }
    if (side == 1) {
      pot_boundary_d = Grav.F.pot_boundary_x1_d;
    }
  }
    #endif
    #ifdef GRAV_ISOLATED_BOUNDARY_Y
  if (direction == 1) {
    n_i = Grav.nx_local;
    n_j = Grav.nz_local;
    if (side == 0) {
      pot_boundary_h = Grav.F.pot_boundary_y0;
    }
    if (side == 1) {
      pot_boundary_h = Grav.F.pot_boundary_y1;
    }
    if (side == 0) {
      pot_boundary_d = Grav.F.pot_boundary_y0_d;
    }
    if (side == 1) {
      pot_boundary_d = Grav.F.pot_boundary_y1_d;
    }
  }
    #endif
    #ifdef GRAV_ISOLATED_BOUNDARY_Z
  if (direction == 2) {
    n_i = Grav.nx_local;
    n_j = Grav.ny_local;
    if (side == 0) {
      pot_boundary_h = Grav.F.pot_boundary_z0;
    }
    if (side == 1) {
      pot_boundary_h = Grav.F.pot_boundary_z1;
    }
    if (side == 0) {
      pot_boundary_d = Grav.F.pot_boundary_z0_d;
    }
    if (side == 1) {
      pot_boundary_d = Grav.F.pot_boundary_z1_d;
    }
  }
    #endif

  size_buffer = N_GHOST_POTENTIAL * n_i * n_j;

  // set values for GPU kernels
  int ngrid = (size_buffer - 1) / TPB_GRAV + 1;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB_GRAV, 1, 1);

  // Copy the boundary array from host to device
  cudaMemcpy(pot_boundary_d, pot_boundary_h, size_buffer * sizeof(Real), cudaMemcpyHostToDevice);
  cudaDeviceSynchronize();

  // Copy the potential boundary from buffer to potential array
  hipLaunchKernelGGL(Set_Potential_Boundaries_Isolated_kernel, dim1dGrid, dim1dBlock, 0, 0, direction, side,
                     size_buffer, n_i, n_j, nx_g, ny_g, nz_g, n_ghost, Grav.F.potential_d, pot_boundary_d);
}

  #endif  // GRAV_ISOLATED_BOUNDARY

void __global__ Set_Potential_Boundaries_Periodic_kernel(int direction, int side, int n_i, int n_j, int nx, int ny,
                                                         int nz, int n_ghost, Real *potential_d)
{
  // get a global thread ID
  int tid, tid_i, tid_j, tid_k, tid_src, tid_dst;
  tid   = threadIdx.x + blockIdx.x * blockDim.x;
  tid_k = tid / (n_i * n_j);
  tid_j = (tid - tid_k * n_i * n_j) / n_i;
  tid_i = tid - tid_k * n_i * n_j - tid_j * n_i;

  if (tid_i < 0 || tid_i >= n_i || tid_j < 0 || tid_j >= n_j || tid_k < 0 || tid_k >= n_ghost) {
    return;
  }

  if (direction == 0) {
    if (side == 0) {
      tid_src = (nx - 2 * n_ghost + tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
    if (side == 0) {
      tid_dst = (tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_src = (n_ghost + tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_dst = (nx - n_ghost + tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
  }
  if (direction == 1) {
    if (side == 0) {
      tid_src = (tid_i) + (ny - 2 * n_ghost + tid_k) * nx + (tid_j)*nx * ny;
    }
    if (side == 0) {
      tid_dst = (tid_i) + (tid_k)*nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_src = (tid_i) + (n_ghost + tid_k) * nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_dst = (tid_i) + (ny - n_ghost + tid_k) * nx + (tid_j)*nx * ny;
    }
  }
  if (direction == 2) {
    if (side == 0) {
      tid_src = (tid_i) + (tid_j)*nx + (nz - 2 * n_ghost + tid_k) * nx * ny;
    }
    if (side == 0) {
      tid_dst = (tid_i) + (tid_j)*nx + (tid_k)*nx * ny;
    }
    if (side == 1) {
      tid_src = (tid_i) + (tid_j)*nx + (n_ghost + tid_k) * nx * ny;
    }
    if (side == 1) {
      tid_dst = (tid_i) + (tid_j)*nx + (nz - n_ghost + tid_k) * nx * ny;
    }
  }

  potential_d[tid_dst] = potential_d[tid_src];
}

void Grid3D::Set_Potential_Boundaries_Periodic_GPU(int direction, int side, int *flags)
{
  int n_i, n_j, n_ghost, size;
  int nx_g, ny_g, nz_g;
  n_ghost = N_GHOST_POTENTIAL;
  nx_g    = Grav.nx_local + 2 * n_ghost;
  ny_g    = Grav.ny_local + 2 * n_ghost;
  nz_g    = Grav.nz_local + 2 * n_ghost;

  if (direction == 0) {
    n_i = ny_g;
    n_j = nz_g;
  }
  if (direction == 1) {
    n_i = nx_g;
    n_j = nz_g;
  }
  if (direction == 2) {
    n_i = nx_g;
    n_j = ny_g;
  }

  size = N_GHOST_POTENTIAL * n_i * n_j;

  // set values for GPU kernels
  int ngrid = (size - 1) / TPB_GRAV + 1;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB_GRAV, 1, 1);

  // Copy the potential boundary from buffer to potential array
  hipLaunchKernelGGL(Set_Potential_Boundaries_Periodic_kernel, dim1dGrid, dim1dBlock, 0, 0, direction, side, n_i, n_j,
                     nx_g, ny_g, nz_g, n_ghost, Grav.F.potential_d);
}

__global__ void Load_Transfer_Buffer_GPU_kernel(int direction, int side, int size_buffer, int n_i, int n_j, int nx,
                                                int ny, int nz, int n_ghost_transfer, int n_ghost_potential,
                                                Real *potential_d, Real *transfer_buffer_d)
{
  // get a global thread ID
  int tid, tid_i, tid_j, tid_k, tid_buffer, tid_pot;
  tid   = threadIdx.x + blockIdx.x * blockDim.x;
  tid_k = tid / (n_i * n_j);
  tid_j = (tid - tid_k * n_i * n_j) / n_i;
  tid_i = tid - tid_k * n_i * n_j - tid_j * n_i;

  if (tid_i < 0 || tid_i >= n_i || tid_j < 0 || tid_j >= n_j || tid_k < 0 || tid_k >= n_ghost_transfer) {
    return;
  }

  tid_buffer = tid_i + tid_j * n_i + tid_k * n_i * n_j;

  if (direction == 0) {
    if (side == 0) {
      tid_pot = (n_ghost_potential + tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_pot = (nx - n_ghost_potential - n_ghost_transfer + tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
  }
  if (direction == 1) {
    if (side == 0) {
      tid_pot = (tid_i) + (n_ghost_potential + tid_k) * nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_pot = (tid_i) + (ny - n_ghost_potential - n_ghost_transfer + tid_k) * nx + (tid_j)*nx * ny;
    }
  }
  if (direction == 2) {
    if (side == 0) {
      tid_pot = (tid_i) + (tid_j)*nx + (n_ghost_potential + tid_k) * nx * ny;
    }
    if (side == 1) {
      tid_pot = (tid_i) + (tid_j)*nx + (nz - n_ghost_potential - n_ghost_transfer + tid_k) * nx * ny;
    }
  }
  transfer_buffer_d[tid_buffer] = potential_d[tid_pot];
}

int Grid3D::Load_Gravity_Potential_To_Buffer_GPU(int direction, int side, Real *buffer, int buffer_start)
{
  // printf( "Loading Gravity Buffer: Dir %d  side: %d \n", direction, side );
  int nx_pot, ny_pot, nz_pot, size_buffer, n_ghost_potential, n_ghost_transfer, n_i, n_j, ngrid;
  ;
  n_ghost_potential = N_GHOST_POTENTIAL;
  n_ghost_transfer  = N_GHOST_POTENTIAL;
  nx_pot            = Grav.nx_local + 2 * n_ghost_potential;
  ny_pot            = Grav.ny_local + 2 * n_ghost_potential;
  nz_pot            = Grav.nz_local + 2 * n_ghost_potential;

  if (direction == 0) {
    n_i = ny_pot;
    n_j = nz_pot;
  }
  if (direction == 1) {
    n_i = nx_pot;
    n_j = nz_pot;
  }
  if (direction == 2) {
    n_i = nx_pot;
    n_j = ny_pot;
  }

  size_buffer = n_ghost_transfer * n_i * n_j;

  // set values for GPU kernels
  ngrid = (size_buffer - 1) / TPB_GRAV + 1;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB_GRAV, 1, 1);

  Real *potential_d;
  potential_d = (Real *)Grav.F.potential_d;

  Real *send_buffer_d;
  send_buffer_d = buffer;

  hipLaunchKernelGGL(Load_Transfer_Buffer_GPU_kernel, dim1dGrid, dim1dBlock, 0, 0, direction, side, size_buffer, n_i,
                     n_j, nx_pot, ny_pot, nz_pot, n_ghost_transfer, n_ghost_potential, potential_d, send_buffer_d);
  GPU_Error_Check(cudaDeviceSynchronize());

  return size_buffer;
}

__global__ void Unload_Transfer_Buffer_GPU_kernel(int direction, int side, int size_buffer, int n_i, int n_j, int nx,
                                                  int ny, int nz, int n_ghost_transfer, int n_ghost_potential,
                                                  Real *potential_d, Real *transfer_buffer_d)
{
  // get a global thread ID
  int tid, tid_i, tid_j, tid_k, tid_buffer, tid_pot;
  tid   = threadIdx.x + blockIdx.x * blockDim.x;
  tid_k = tid / (n_i * n_j);
  tid_j = (tid - tid_k * n_i * n_j) / n_i;
  tid_i = tid - tid_k * n_i * n_j - tid_j * n_i;

  if (tid_i < 0 || tid_i >= n_i || tid_j < 0 || tid_j >= n_j || tid_k < 0 || tid_k >= n_ghost_transfer) {
    return;
  }

  tid_buffer = tid_i + tid_j * n_i + tid_k * n_i * n_j;

  if (direction == 0) {
    if (side == 0) {
      tid_pot = (n_ghost_potential - n_ghost_transfer + tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_pot = (nx - n_ghost_potential + tid_k) + (tid_i)*nx + (tid_j)*nx * ny;
    }
  }
  if (direction == 1) {
    if (side == 0) {
      tid_pot = (tid_i) + (n_ghost_potential - n_ghost_transfer + tid_k) * nx + (tid_j)*nx * ny;
    }
    if (side == 1) {
      tid_pot = (tid_i) + (ny - n_ghost_potential + tid_k) * nx + (tid_j)*nx * ny;
    }
  }
  if (direction == 2) {
    if (side == 0) {
      tid_pot = (tid_i) + (tid_j)*nx + (n_ghost_potential - n_ghost_transfer + tid_k) * nx * ny;
    }
    if (side == 1) {
      tid_pot = (tid_i) + (tid_j)*nx + (nz - n_ghost_potential + tid_k) * nx * ny;
    }
  }
  potential_d[tid_pot] = transfer_buffer_d[tid_buffer];
}

void Grid3D::Unload_Gravity_Potential_from_Buffer_GPU(int direction, int side, Real *buffer, int buffer_start)
{
  // printf( "Loading Gravity Buffer: Dir %d  side: %d \n", direction, side );
  int nx_pot, ny_pot, nz_pot, size_buffer, n_ghost_potential, n_ghost_transfer, n_i, n_j, ngrid;
  ;
  n_ghost_potential = N_GHOST_POTENTIAL;
  n_ghost_transfer  = N_GHOST_POTENTIAL;
  nx_pot            = Grav.nx_local + 2 * n_ghost_potential;
  ny_pot            = Grav.ny_local + 2 * n_ghost_potential;
  nz_pot            = Grav.nz_local + 2 * n_ghost_potential;

  if (direction == 0) {
    n_i = ny_pot;
    n_j = nz_pot;
  }
  if (direction == 1) {
    n_i = nx_pot;
    n_j = nz_pot;
  }
  if (direction == 2) {
    n_i = nx_pot;
    n_j = ny_pot;
  }

  size_buffer = n_ghost_transfer * n_i * n_j;

  // set values for GPU kernels
  ngrid = (size_buffer - 1) / TPB_GRAV + 1;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB_GRAV, 1, 1);

  Real *potential_d;
  potential_d = (Real *)Grav.F.potential_d;

  Real *recv_buffer_d;
  recv_buffer_d = buffer;

  hipLaunchKernelGGL(Unload_Transfer_Buffer_GPU_kernel, dim1dGrid, dim1dBlock, 0, 0, direction, side, size_buffer, n_i,
                     n_j, nx_pot, ny_pot, nz_pot, n_ghost_transfer, n_ghost_potential, potential_d, recv_buffer_d);
}

#endif  // GRAVITY
