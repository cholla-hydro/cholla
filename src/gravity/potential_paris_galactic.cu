
#include <hip/hip_runtime.h>
#ifdef PARIS_GALACTIC

  #include <cassert>

  #include "../gravity/potential_paris_galactic.h"
  #include "../io/io.h"
  #include "../utils/gpu.hpp"

PotentialParisGalactic::PotentialParisGalactic()
    : dn_{0, 0, 0},
      dr_{0, 0, 0},
      lo_{0, 0, 0},
      lr_{0, 0, 0},
      myLo_{0, 0, 0},
      pp_(nullptr),
      densityBytes_(0),
      minBytes_(0),
      da_(nullptr),
      db_(nullptr)
  #ifndef GRAVITY_GPU
      ,
      potentialBytes_(0),
      dc_(nullptr)
  #endif
{
}

PotentialParisGalactic::~PotentialParisGalactic() { Reset(); }

void PotentialParisGalactic::Get_Potential(const Real *const density, Real *const potential, const Real g,
                                           const DiskGalaxy &galaxy)
{
  const Real scale = Real(4) * M_PI * g;

  assert(da_);
  Real *const da = da_;
  Real *const db = db_;
  assert(density);

  const int ni = dn_[2];
  const int nj = dn_[1];
  const int nk = dn_[0];

  const int ngi = ni + N_GHOST_POTENTIAL + N_GHOST_POTENTIAL;
  const int ngj = nj + N_GHOST_POTENTIAL + N_GHOST_POTENTIAL;

  #ifdef GRAVITY_GPU
  const Real *const rho = density;
  Real *const phi       = potential;
  #else
  GPU_Error_Check(cudaMemcpyAsync(da, density, densityBytes_, cudaMemcpyHostToDevice, 0));
  GPU_Error_Check(cudaMemcpyAsync(dc_, potential, potentialBytes_, cudaMemcpyHostToDevice, 0));
  const Real *const rho = da;
  Real *const phi       = dc_;
  #endif

  const Real xMin = myLo_[2];
  const Real yMin = myLo_[1];
  const Real zMin = myLo_[0];

  const Real dx = dr_[2];
  const Real dy = dr_[1];
  const Real dz = dr_[0];

  const Real md = SIMULATED_FRACTION * galaxy.getM_d();
  const Real rd = galaxy.getR_d();
  const Real zd = galaxy.getZ_d();

  const Real rho0 = md * zd * zd / (4.0 * M_PI);
  gpuFor(
      nk, nj, ni, GPU_LAMBDA(const int k, const int j, const int i) {
        const int ia = i + ni * (j + nj * k);

        const Real x = xMin + i * dx;
        const Real y = yMin + j * dy;
        const Real z = zMin + k * dz;

        const Real r    = sqrt(x * x + y * y);
        const Real a    = sqrt(z * z + zd * zd);
        const Real b    = rd + a;
        const Real c    = r * r + b * b;
        const Real dRho = rho0 * (rd * c + 3.0 * a * b * b) / (a * a * a * pow(c, 2.5));

        da[ia] = scale * (rho[ia] - dRho);
      });

  pp_->solve(minBytes_, da, db);

  const Real phi0 = -g * md;
  gpuFor(
      nk, nj, ni, GPU_LAMBDA(const int k, const int j, const int i) {
        const int ia = i + ni * (j + nj * k);
        const int ib = i + N_GHOST_POTENTIAL + ngi * (j + N_GHOST_POTENTIAL + ngj * (k + N_GHOST_POTENTIAL));

        const Real x = xMin + i * dx;
        const Real y = yMin + j * dy;
        const Real z = zMin + k * dz;

        const Real r    = sqrt(x * x + y * y);
        const Real a    = sqrt(z * z + zd * zd);
        const Real b    = a + rd;
        const Real c    = sqrt(r * r + b * b);
        const Real dPhi = phi0 / c;

        phi[ib] = db[ia] + dPhi;
      });

  #ifndef GRAVITY_GPU
  GPU_Error_Check(cudaMemcpy(potential, dc_, potentialBytes_, cudaMemcpyDeviceToHost));
  #endif
}

void PotentialParisGalactic::Initialize(const Real lx, const Real ly, const Real lz, const Real xMin, const Real yMin,
                                        const Real zMin, const int nx, const int ny, const int nz, const int nxReal,
                                        const int nyReal, const int nzReal, const Real dx, const Real dy, const Real dz)
{
  const long nl012 = long(nxReal) * long(nyReal) * long(nzReal);
  assert(nl012 <= INT_MAX);

  dn_[0] = nzReal;
  dn_[1] = nyReal;
  dn_[2] = nxReal;

  dr_[0] = dz;
  dr_[1] = dy;
  dr_[2] = dx;

  lr_[0] = lz;
  lr_[1] = ly;
  lr_[2] = lx;

  myLo_[0] = zMin + 0.5 * dr_[0];
  myLo_[1] = yMin + 0.5 * dr_[1];
  myLo_[2] = xMin + 0.5 * dr_[2];
  MPI_Allreduce(myLo_, lo_, 3, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

  const Real hi[3] = {lo_[0] + lr_[0] - dr_[0], lo_[1] + lr_[1] - dr_[1], lo_[2] + lr_[1] - dr_[2]};
  const int n[3]   = {nz, ny, nx};
  const int m[3]   = {n[0] / nzReal, n[1] / nyReal, n[2] / nxReal};
  const int id[3]  = {int(round((myLo_[0] - lo_[0]) / (dn_[0] * dr_[0]))),
                      int(round((myLo_[1] - lo_[1]) / (dn_[1] * dr_[1]))),
                      int(round((myLo_[2] - lo_[2]) / (dn_[2] * dr_[2])))};
  chprintf(
      " Paris Galactic: [ %g %g %g ]-[ %g %g %g ] n_local[ %d %d %d ] tasks[ "
      "%d %d %d ]\n",
      lo_[2], lo_[1], lo_[0], hi[2], hi[1], hi[0], dn_[2], dn_[1], dn_[0], m[2], m[1], m[0]);

  assert(dn_[0] == n[0] / m[0]);
  assert(dn_[1] == n[1] / m[1]);
  assert(dn_[2] == n[2] / m[2]);

  pp_ = new PoissonZero3DBlockedGPU(n, lo_, hi, m, id);
  assert(pp_);
  minBytes_     = pp_->bytes();
  densityBytes_ = long(sizeof(Real)) * dn_[0] * dn_[1] * dn_[2];

  GPU_Error_Check(cudaMalloc(reinterpret_cast<void **>(&da_), std::max(minBytes_, densityBytes_)));
  GPU_Error_Check(cudaMalloc(reinterpret_cast<void **>(&db_), std::max(minBytes_, densityBytes_)));

  #ifndef GRAVITY_GPU
  const long gg   = N_GHOST_POTENTIAL + N_GHOST_POTENTIAL;
  potentialBytes_ = long(sizeof(Real)) * (dn_[0] + gg) * (dn_[1] + gg) * (dn_[2] + gg);
  GPU_Error_Check(cudaMalloc(reinterpret_cast<void **>(&dc_), potentialBytes_));
  #endif
}

void PotentialParisGalactic::Reset()
{
  #ifndef GRAVITY_GPU
  if (dc_) {
    GPU_Error_Check(cudaFree(dc_));
  }
  dc_             = nullptr;
  potentialBytes_ = 0;
  #endif

  if (db_) {
    GPU_Error_Check(cudaFree(db_));
  }
  db_ = nullptr;

  if (da_) {
    GPU_Error_Check(cudaFree(da_));
  }
  da_ = nullptr;

  densityBytes_ = minBytes_ = 0;

  if (pp_) {
    delete pp_;
  }
  pp_ = nullptr;

  myLo_[2] = myLo_[1] = myLo_[0] = 0;
  lr_[2] = lr_[1] = lr_[0] = 0;
  lo_[2] = lo_[1] = lo_[0] = 0;
  dr_[2] = dr_[1] = dr_[0] = 0;
  dn_[2] = dn_[1] = dn_[0] = 0;
}

#endif
