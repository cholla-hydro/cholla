
#include <hip/hip_runtime.h>
/*! \file subgrid_routines_3D.cu
 *  \brief Definitions of the routines for subgrid gpu staging for 3D CTU. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include"global.h"
#include"mpi_routines.h"
#include"subgrid_routines_3D.h"





void sub_dimensions_3D(int nx, int ny, int nz, int n_ghost, int *nx_s, int *ny_s, int *nz_s, int *block1_tot, int *block2_tot, int *block3_tot, int *remainder1, int *remainder2, int *remainder3, int n_fields) {

  int sx = 2;
  int sy = 2;
  int sz = 2;
  size_t free;
  size_t total;
  int cell_mem, max_vol;

  *nx_s = nx;
  *ny_s = ny;
  *nz_s = nz;

  // determine the amount of free memory available on the device
  cudaMemGetInfo(&free, &total);

  // use that to determine the maximum subgrid block volume
  // memory used per cell (arrays allocated on GPU)
  cell_mem = 11*n_fields*sizeof(Real);
  #ifdef H_CORRECTION
  cell_mem += 6*sizeof(Real);
  #endif
  #if defined( GRAVITY ) 
  cell_mem += sizeof(Real);
  #endif
  max_vol = free / cell_mem; 
  // plus a buffer for dti array
  max_vol = max_vol - 400;

  // split if necessary - subgrid block volume cannot exceed MAX_VOL_3D
  // try to keep the ratio of the y & z dimensions close to 1
  // do not let the ratio of the geometric mean of the y&z dimensions to the
  // x dimension exceed 5 - we don't want cubes, but we don't want
  // REALLY NOT CUBES
  while ((*nx_s)*(*ny_s)*(*nz_s) > max_vol) {

    // if the aspect ratio has gotten too large, split in x 
    if ((*nx_s) / sqrt((*ny_s)*(*nz_s)) > 5) {
        *nx_s = ceil(Real (nx-2*n_ghost) / Real (sx)) + 2*n_ghost;
        sx++;
    }
    else {
      if (*ny_s > *nz_s) {
        *ny_s = ceil(Real (ny-2*n_ghost) / Real (sy)) + 2*n_ghost;
        sy++;
      }
      else {
        *nz_s = ceil(Real (nz-2*n_ghost) / Real (sz)) + 2*n_ghost;
        sz++;
      }
    }

  }

  // determine the number of blocks needed
  // not splitting
  if (*nx_s == nx && *ny_s == ny && *nz_s == nz) {
    *block1_tot = 1;
    *block2_tot = 1;
    *block3_tot = 1;
    *remainder1 = 0;
    *remainder2 = 0;
    *remainder3 = 0;
    return;
  }
  // splitting in x
  else if (*nx_s < nx && *ny_s == ny && *nz_s == nz) {
    *block1_tot = ceil(Real (nx-2*n_ghost) / Real (*nx_s-2*n_ghost) );
    *block2_tot = 1;
    *block3_tot = 1;
    // calculate the remainder
    *remainder1 = (nx-2*n_ghost)%(*nx_s-2*n_ghost);
    *remainder2 = 0;
    *remainder3 = 0;
  }  
  // splitting in y
  else if (*nx_s == nx && *ny_s < ny && *nz_s == nz) {
    *block1_tot = 1;
    *block2_tot = ceil(Real (ny-2*n_ghost) / Real (*ny_s-2*n_ghost) );
    *block3_tot = 1;
    // calculate the remainder
    *remainder1 = 0;
    *remainder2 = (ny-2*n_ghost)%(*ny_s-2*n_ghost);
    *remainder3 = 0;
  }  
  // splitting in z
  else if (*nx_s == nx && *ny_s == ny && *nz_s < nz) {
    *block1_tot = 1;
    *block2_tot = 1;
    *block3_tot = ceil(Real (nz-2*n_ghost) / Real (*nz_s-2*n_ghost) );
    // calculate the remainder
    *remainder1 = 0;
    *remainder2 = 0;
    *remainder3 = (nz-2*n_ghost)%(*nz_s-2*n_ghost);
  }
  // splitting in x & y
  else if (*nx_s < nx && *ny_s < ny && *nz_s == nz) {
    *block1_tot = ceil(Real (nx-2*n_ghost) / Real (*nx_s-2*n_ghost) );
    *block2_tot = ceil(Real (ny-2*n_ghost) / Real (*ny_s-2*n_ghost) );
    *block3_tot = 1;
    // calculate the remainder
    *remainder1 = (nx-2*n_ghost)%(*nx_s-2*n_ghost);
    *remainder2 = (ny-2*n_ghost)%(*ny_s-2*n_ghost);
    *remainder3 = 0;
  }  
  // splitting in y & z
  else if (*nx_s == nx && *ny_s < ny && *nz_s < nz) {
    *block1_tot = 1;
    *block2_tot = ceil(Real (ny-2*n_ghost) / Real (*ny_s-2*n_ghost) );
    *block3_tot = ceil(Real (nz-2*n_ghost) / Real (*nz_s-2*n_ghost) );
    // calculate the remainders
    *remainder1 = 0;
    *remainder2 = (ny-2*n_ghost)%(*ny_s-2*n_ghost);
    *remainder3 = (nz-2*n_ghost)%(*nz_s-2*n_ghost);
  }
  // splitting in x & z
  else if (*nx_s < nx && *ny_s == ny && *nz_s < nz) {
    *block1_tot = ceil(Real (nx-2*n_ghost) / Real (*nx_s-2*n_ghost) );
    *block2_tot = 1;
    *block3_tot = ceil(Real (nz-2*n_ghost) / Real (*nz_s-2*n_ghost) );
    // calculate the remainder
    *remainder1 = (nx-2*n_ghost)%(*nx_s-2*n_ghost);
    *remainder2 = 0;
    *remainder3 = (nz-2*n_ghost)%(*nz_s-2*n_ghost);
  }  
  // splitting in x, y & z
  else if (*nx_s < nx && *ny_s < ny && *nz_s < nz) {
    *block1_tot = ceil(Real (nx-2*n_ghost) / Real (*nx_s-2*n_ghost) );
    *block2_tot = ceil(Real (ny-2*n_ghost) / Real (*ny_s-2*n_ghost) );
    *block3_tot = ceil(Real (nz-2*n_ghost) / Real (*nz_s-2*n_ghost) );
    // calculate the remainders
    *remainder1 = (nx-2*n_ghost)%(*nx_s-2*n_ghost);
    *remainder2 = (ny-2*n_ghost)%(*ny_s-2*n_ghost);
    *remainder3 = (nz-2*n_ghost)%(*nz_s-2*n_ghost);
  }  
  else {
    printf("Error determining number and size of subgrid blocks.\n");
    exit(0);
  }


}


void get_offsets_3D(int nx_s, int ny_s, int nz_s, int n_ghost, int x_off, int y_off, int z_off, int block, int block1_tot, int block2_tot, int block3_tot, int remainder1, int remainder2, int remainder3, int *x_off_s, int *y_off_s, int *z_off_s) {

  int block1;
  int block2;
  int block3;

  // determine which row of subgrid blocks we're on for each dimension
  block3 = block / (block2_tot*block1_tot); // zid of current block
  block2 = (block - block3*block2_tot*block1_tot) / block1_tot; // yid of current block
  block1 = block - block3*block2_tot*block1_tot - block2*block1_tot; // xid of current block
  // calculate global offsets
  *x_off_s = x_off + (nx_s-2*n_ghost)*block1;
  *y_off_s = y_off + (ny_s-2*n_ghost)*block2;
  *z_off_s = z_off + (nz_s-2*n_ghost)*block3;
  // need to be careful on the last block due to remainder offsets
  if (remainder1 != 0 && block1 == block1_tot-1) *x_off_s = x_off + (nx_s-2*n_ghost)*(block1-1) + remainder1;
  if (remainder2 != 0 && block2 == block2_tot-1) *y_off_s = y_off + (ny_s-2*n_ghost)*(block2-1) + remainder2;
  if (remainder3 != 0 && block3 == block3_tot-1) *z_off_s = z_off + (nz_s-2*n_ghost)*(block3-1) + remainder3;
  
}





// copy the conserved variable block into the buffer
void host_copy_block_3D(int nx, int ny, int nz, int nx_s, int ny_s, int nz_s, int n_ghost, int block, int block1_tot, int block2_tot, int block3_tot, int remainder1, int remainder2, int remainder3, int BLOCK_VOL, Real *host_conserved, Real *buffer, int n_fields, Real *host_grav_potential, Real *buffer_potential ) {
  
  int n_cells = nx*ny*nz;
  int block1, block2, block3;
  int x_offset, y_offset, z_offset;
  int x_host, y_host, z_host;

  // if no subgrid blocks, do nothing
  if (nx_s == nx && ny_s == ny && nz_s == nz) return;


  // splitting only in x
  else if (nx_s < nx && ny_s == ny && nz_s == nz) {

    block1 = block; // xid of block
  
    // if we are on the last block, make sure it doesn't go past 
    // the bounds of the host array
    x_offset = 0;
    if (block1 == block1_tot-1 && remainder1 != 0) {
      x_offset = nx_s - 2*n_ghost - remainder1;
    }
    // calculate the x location in the host array to copy from
    x_host = block1*(nx_s-2*n_ghost) - x_offset;

    // copy data from host conserved array into buffer
    for (int k=0; k<nz_s; k++) {
      for (int j=0; j<ny_s; j++) {
        for (int ii=0; ii<n_fields; ii++) {
          memcpy(&buffer[ii*BLOCK_VOL + j*nx_s + k*nx_s*ny_s], &host_conserved[x_host + ii*n_cells + j*nx + k*nx*ny], nx_s*sizeof(Real)); 
        }
        #if defined( GRAVITY ) 
        memcpy(&buffer_potential[j*nx_s + k*nx_s*ny_s], &host_grav_potential[x_host + j*nx + k*nx*ny], nx_s*sizeof(Real));
        #endif
      }
    }

    return;

  }

  // splitting only in y
  else if (nx_s == nx && ny_s < ny && nz_s == nz) {

    block2 = block; // yid of block
  
    // if we are on the last block, make sure it doesn't go past 
    // the bounds of the host array
    y_offset = 0;
    if (block2 == block2_tot-1 && remainder2 != 0) {
      y_offset = ny_s - 2*n_ghost - remainder2;
    }
    // calculate the y location in the host array to copy from
    y_host = block2*nx*(ny_s-2*n_ghost) - nx*y_offset;

    // copy data from host conserved array into buffer
    for (int k=0; k<nz_s; k++) {
      for (int ii=0; ii<n_fields; ii++) {
        memcpy(&buffer[ii*BLOCK_VOL + k*nx_s*ny_s], &host_conserved[y_host + ii*n_cells + k*nx*ny], nx_s*ny_s*sizeof(Real)); 
      }
      #if defined( GRAVITY )
      memcpy(&buffer_potential[k*nx_s*ny_s], &host_grav_potential[y_host + k*nx*ny], nx_s*ny_s*sizeof(Real));
      #endif
    }

    return;

  }


  // splitting only in z
  else if (nx_s == nx && ny_s == ny && nz_s < nz) {

    block3 = block; // zid of block
  
    // if we are on the last block, make sure it doesn't go past 
    // the bounds of the host array
    z_offset = 0;
    if (block3 == block3_tot-1 && remainder3 != 0) {
      z_offset = nz_s - 2*n_ghost - remainder3;
    }
    // calculate the z location in the host array to copy from
    z_host = block3*nx*ny*(nz_s-2*n_ghost) - nx*ny*(z_offset);

    // copy data from host conserved array into buffer
    for (int ii=0; ii<n_fields; ii++) {
      memcpy(&buffer[ii*BLOCK_VOL], &host_conserved[z_host + ii*n_cells], BLOCK_VOL*sizeof(Real));
    }
    #if defined( GRAVITY ) 
    memcpy(&buffer_potential[0], &host_grav_potential[z_host], BLOCK_VOL*sizeof(Real));
    #endif

    return;
  }


  // splitting in y and z
  else if (nx_s == nx && ny_s < ny && nz_s < nz) {

    block3 = block / block2_tot; // zid of current block
    block2 = block - block3*block2_tot; // yid of current block

    // if we are on the last y block, make sure it doesn't go past 
    // the bounds of the host array
    y_offset = 0;
    if (block2 == block2_tot-1 && remainder2 != 0) {
      y_offset = ny_s - 2*n_ghost - remainder2;
    }
    // calculate the y location in the host array to copy from
    y_host = block2*nx*(ny_s-2*n_ghost) - nx*y_offset;


    // if we are on the last z block, make sure it doesn't go past 
    // the bounds of the host array
    z_offset = 0;
    if (block3 == block3_tot-1 && remainder3 != 0) {
      z_offset = nz_s - 2*n_ghost - remainder3;
    }
    // calculate the z location in the host array to copy from
    z_host = block3*nx*ny*(nz_s-2*n_ghost) - nx*ny*(z_offset);

    // copy data from host conserved array into buffer
    for (int k=0; k<nz_s; k++) {
      for (int ii=0; ii<n_fields; ii++) {
        memcpy(&buffer[ii*BLOCK_VOL + k*nx_s*ny_s], &host_conserved[z_host + y_host + ii*n_cells + k*nx*ny], nx_s*ny_s*sizeof(Real)); 
      }
      #if defined( GRAVITY )
      memcpy(&buffer_potential[k*nx_s*ny_s], &host_grav_potential[z_host + y_host + k*nx*ny], nx_s*ny_s*sizeof(Real)); 
      #endif
    }

    return;
  }

  // splitting in x, y, and z
  else if (nx_s < nx && ny_s < ny && nz_s < nz) {

    block3 = block / (block2_tot*block1_tot); // zid of current block
    block2 = (block - block3*block2_tot*block1_tot) / block1_tot; // yid of current block
    block1 = block - block3*block2_tot*block1_tot - block2*block1_tot; // xid of current block

    // if we are on the last x block, make sure it doesn't go past 
    // the bounds of the host array
    x_offset = 0;
    if (block1 == block1_tot-1 && remainder1 != 0) {
      x_offset = nx_s - 2*n_ghost - remainder1;
    }
    // calculate the x location in the host array to copy from
    x_host = block1*(nx_s-2*n_ghost) - x_offset;

    // if we are on the last y block, make sure it doesn't go past 
    // the bounds of the host array
    y_offset = 0;
    if (block2 == block2_tot-1 && remainder2 != 0) {
      y_offset = ny_s - 2*n_ghost - remainder2;
    }
    // calculate the y location in the host array to copy from
    y_host = block2*nx*(ny_s-2*n_ghost) - nx*y_offset;

    // if we are on the last z block, make sure it doesn't go past 
    // the bounds of the host array
    z_offset = 0;
    if (block3 == block3_tot-1 && remainder3 != 0) {
      z_offset = nz_s - 2*n_ghost - remainder3;
    }
    // calculate the z location in the host array to copy from
    z_host = block3*nx*ny*(nz_s-2*n_ghost) - nx*ny*(z_offset);

    // copy data from host conserved array into buffer
    for (int k=0; k<nz_s; k++) {
      for (int j=0; j<ny_s; j++) {
        for (int ii=0; ii<n_fields; ii++) {
          memcpy(&buffer[ii*BLOCK_VOL + j*nx_s + k*nx_s*ny_s], &host_conserved[x_host + y_host + z_host + ii*n_cells + j*nx + k*nx*ny], nx_s*sizeof(Real)); 
        }
        #if defined( GRAVITY ) 
        memcpy(&buffer_potential[j*nx_s + k*nx_s*ny_s], &host_grav_potential[x_host + y_host + z_host + j*nx + k*nx*ny], nx_s*sizeof(Real)); 
        #endif
      }
    }

    return;
  }

  else {
    printf("Error copying into buffer. Unsupported grid dimensions.\n");
    printf("nx: %d  nx_s: %d  ny: %d  ny_s: %d  nz: %d  nz_s: %d.\n", nx, nx_s, ny, ny_s, nz, nz_s);
    exit(0);
  }


}




// return the values from buffer to the host_conserved array
void host_return_block_3D(int nx, int ny, int nz, int nx_s, int ny_s, int nz_s, int n_ghost, int block, int block1_tot, int block2_tot, int block3_tot, int remainder1, int remainder2, int remainder3, int BLOCK_VOL, Real *host_conserved, Real *buffer, int n_fields) {

  int n_cells = nx*ny*nz;
  int block1, block2, block3;
  int x_offset, y_offset, z_offset;
  int x_host, y_host, z_host, x_gpu, y_gpu, z_gpu, host_loc, gpu_loc;
  int length, hid, gid;

  // if no subgrid blocks, do nothing
  if (nx_s == nx && ny_s == ny && nz_s == nz) return;

  // splitting only in x
  else if (nx_s < nx && ny_s == ny && nz_s == nz) {

    // return values based on current block id
    block1 = block;

    // if we just did the last x block, make sure to copy the cells to the right place
    x_offset = 0;
    if (block1 == block1_tot-1 && remainder1 != 0) {
      x_offset = nx_s - 2*n_ghost - remainder1;
    }

    x_host = block1*(nx_s-2*n_ghost) + (n_ghost-x_offset);
    y_host = n_ghost*nx;
    z_host = n_ghost*nx*ny;
    host_loc = x_host + y_host + z_host;
    x_gpu = n_ghost;
    y_gpu = n_ghost*nx_s;
    z_gpu = n_ghost*nx_s*ny_s;
    gpu_loc = x_gpu + y_gpu + z_gpu;
    length = (nx_s-2*n_ghost); // number of cells to copy back

    for (int k=0; k<nz_s-2*n_ghost; k++) {
      for (int j=0; j<ny_s-2*n_ghost; j++) {
        hid = j*nx + k*nx*ny;
        gid = j*nx_s + k*nx_s*ny_s;
        for (int ii=0; ii<n_fields; ii++) {
          memcpy(&host_conserved[host_loc + hid + ii*n_cells], &buffer[gpu_loc + gid + ii*BLOCK_VOL], length*sizeof(Real));
        }
      }
    }

    return;
  }

  // splitting only in y
  else if (nx_s == nx && ny_s < ny && nz_s == nz) {

    // return values based on current block id
    block2 = block;

    // if we just did the last slice, make sure to copy the cells to the right place
    y_offset = 0;
    if (block2 == block2_tot-1 && remainder2 != 0) {
      y_offset = ny_s - 2*n_ghost - remainder2;
    }

    y_host = block2*nx*(ny_s-2*n_ghost) + nx*(n_ghost-y_offset);
    z_host = n_ghost*nx*ny;
    host_loc = y_host + z_host;
    y_gpu = n_ghost*nx_s;
    z_gpu = n_ghost*nx_s*ny_s;
    gpu_loc = y_gpu + z_gpu;
    length = nx_s*(ny_s-2*n_ghost); // number of cells to copy back

    for (int k=0; k<nz_s-2*n_ghost; k++) {
      hid = k*nx*ny;
      gid = k*nx_s*ny_s;
      for (int ii=0; ii<n_fields; ii++) {
        memcpy(&host_conserved[host_loc + hid + ii*n_cells], &buffer[gpu_loc + gid + ii*BLOCK_VOL], length*sizeof(Real));
      }
    }

    return;
  }

  // splitting only in z
  else if (nx_s == nx && ny_s == ny && nz_s < nz) {

    // return values based on current block id
    block3 = block;

    z_offset = 0;
    // if we just did the last slice, make sure to copy the cells to the right place
    if (block3 == block3_tot-1 && remainder3 != 0) {
      z_offset = nz_s - 2*n_ghost - remainder3;
    }

    z_host = block3*nx*ny*(nz_s-2*n_ghost) + nx*ny*(n_ghost-z_offset);
    z_gpu = n_ghost*nx_s*ny_s;
    length = nx_s*ny_s*(nz_s-2*n_ghost); // number of cells to copy back

    for (int ii=0; ii<n_fields; ii++) {
      memcpy(&host_conserved[z_host + ii*n_cells], &buffer[z_gpu + ii*BLOCK_VOL], length*sizeof(Real));
    }

    return;
  }


  // splitting in y and z
  else if (nx_s == nx && ny_s < ny && nz_s < nz) {

    // return values based on current block id
    block3 = block / block2_tot; // zid of current block
    block2 = block - block3*block2_tot; // yid of current block

    z_offset = 0;
    // if we just did the z last slice, make sure to copy the cells to the right place
    if (block3 == block3_tot-1 && remainder3 != 0) {
      z_offset = nz_s - 2*n_ghost - remainder3;
    }

    y_offset = 0;
    // if we just did the y last slice, make sure to copy the cells to the right place
    if (block2 == block2_tot-1 && remainder2 != 0) {
      y_offset = ny_s - 2*n_ghost - remainder2;
    }

    z_host = block3*nx*ny*(nz_s-2*n_ghost) + nx*ny*(n_ghost-z_offset);
    y_host = block2*nx*(ny_s-2*n_ghost) + nx*(n_ghost-y_offset);
    host_loc = y_host + z_host;
    z_gpu  = n_ghost*nx_s*ny_s;
    y_gpu  = n_ghost*nx_s;
    gpu_loc = y_gpu + z_gpu;
    length = nx_s*(ny_s-2*n_ghost); // number of cells to copy back
    
    for (int k=0; k<nz_s-2*n_ghost; k++) {
      hid = k*nx*ny;
      gid = k*nx_s*ny_s;
      for (int ii=0; ii<n_fields; ii++) {
        memcpy(&host_conserved[host_loc + hid + ii*n_cells], &buffer[gpu_loc + gid + ii*BLOCK_VOL], length*sizeof(Real));
      }
    }

    return;
  }

  // splitting in x, y, and z
  else if (nx_s < nx && ny_s < ny && nz_s < nz) {

    // return values based on current block id
    block3 = block / (block2_tot*block1_tot); // zid of current block
    block2 = (block - block3*block2_tot*block1_tot) / block1_tot; // yid of current block
    block1 = block - block3*block2_tot*block1_tot - block2*block1_tot; // xid of current block

    z_offset = 0;
    // if we just did the z last slice, make sure to copy the cells to the right place
    if (block3 == block3_tot-1 && remainder3 != 0) {
      z_offset = nz_s - 2*n_ghost - remainder3;
    }

    y_offset = 0;
    // if we just did the y last slice, make sure to copy the cells to the right place
    if (block2 == block2_tot-1 && remainder2 != 0) {
      y_offset = ny_s - 2*n_ghost - remainder2;
    }

    x_offset = 0;
    // if we just did the x last slice, make sure to copy the cells to the right place
    if (block1 == block1_tot-1 && remainder1 != 0) {
      x_offset = nx_s - 2*n_ghost - remainder1;
    }

    z_host = block3*nx*ny*(nz_s-2*n_ghost) + nx*ny*(n_ghost-z_offset);
    y_host = block2*nx*(ny_s-2*n_ghost) + nx*(n_ghost-y_offset);
    x_host = block1*(nx_s-2*n_ghost) + (n_ghost-x_offset);
    host_loc = x_host + y_host + z_host;
    z_gpu = n_ghost*nx_s*ny_s;
    y_gpu = n_ghost*nx_s;
    x_gpu = n_ghost;
    gpu_loc = x_gpu + y_gpu + z_gpu;
    length = (nx_s-2*n_ghost); // number of cells to copy back
    
    for (int k=0; k<nz_s-2*n_ghost; k++) {
      for (int j=0; j<ny_s-2*n_ghost; j++) {
        hid = j*nx + k*nx*ny;
        gid = j*nx_s + k*nx_s*ny_s;
        for (int ii=0; ii<n_fields; ii++) {
          memcpy(&host_conserved[host_loc + hid + ii*n_cells], &buffer[gpu_loc + gid + ii*BLOCK_VOL], length*sizeof(Real));
        }
      }
    }

    return;
  }


  else {
    printf("Error returning values to host. Unsupported grid dimensions.\n");
    printf("nx: %d  nx_s: %d  ny: %d  ny_s: %d  nz: %d  nz_s: %d.\n", nx, nx_s, ny, ny_s, nz, nz_s);
    exit(0);
  }


}




#endif //CUDA
