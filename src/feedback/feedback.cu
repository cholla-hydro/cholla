#include "hip/hip_runtime.h"

  #include <math.h>
  #include <stdio.h>
  #include <stdlib.h>
  #include <unistd.h>

  #include <cstring>
  #include <fstream>
  #include <sstream>
  #include <vector>

  #include "../global/global.h"
  #include "../global/global_cuda.h"
  #include "../grid/grid3D.h"
  #include "../io/io.h"
  #include "../utils/DeviceVector.h"
  #include "../utils/error_handling.h"
  #include "../utils/reduction_utilities.h"
  #include "../feedback/ratecalc.h"
  #include "../feedback/feedback_model.h"
  #include "feedback.h"

  #define TPB_FEEDBACK 128



/** This function used for debugging potential race conditions.  Feedback from neighboring
    particles could simultaneously alter one hydro cell's conserved quantities.
 */
inline __device__ bool Particle_Is_Alone(Real* pos_x_dev, Real* pos_y_dev, Real* pos_z_dev, part_int_t n_local,
                                         int gtid, Real dx)
{
  Real x0 = pos_x_dev[gtid];
  Real y0 = pos_y_dev[gtid];
  Real z0 = pos_z_dev[gtid];
  // Brute force loop to see if particle is alone
  for (int i = 0; i < n_local; i++) {
    if (i == gtid) continue;
    if (abs(x0 - pos_x_dev[i]) > dx) continue;
    if (abs(y0 - pos_y_dev[i]) > dx) continue;
    if (abs(z0 - pos_z_dev[i]) > dx) continue;
    // If we made it here, something is too close.
    return false;
  }
  return true;
}


template<typename FeedbackModel>
__global__ void Cluster_Feedback_Kernel(part_int_t n_local, part_int_t* id_dev, Real* pos_x_dev, Real* pos_y_dev,
                                        Real* pos_z_dev, Real* mass_dev, Real* age_dev, Real xMin, Real yMin, Real zMin,
                                        Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx_g, int ny_g,
                                        int nz_g, int n_ghost, Real t, Real dt, Real* info, Real* conserved_dev,
                                        Real gamma, int* num_SN_dev, int n_step, FeedbackModel feedback_model)
{
  const int tid = threadIdx.x;
  const int gtid = blockIdx.x * blockDim.x + tid;

  // prologoue: setup buffer for collecting SN feedback information
  __shared__ Real s_info[feedinfoLUT::LEN * TPB_FEEDBACK];
  for (unsigned int cur_ind = 0; cur_ind < feedinfoLUT::LEN; cur_ind++) {
    s_info[feedinfoLUT::LEN * tid + cur_ind] = 0;
  }

  // do the main work:
  {
    // reduce branching
    part_int_t tmp_gtid_ = min(n_local - 1, part_int_t(gtid));

    Real pos_x    = pos_x_dev[tmp_gtid_];
    Real pos_y    = pos_y_dev[tmp_gtid_];
    Real pos_z    = pos_z_dev[tmp_gtid_];

    // compute the position in index-units (appropriate for a field with a ghost-zone)
    // - an integer value corresponds to the left edge of a cell
    const Real pos_x_indU = (pos_x - xMin) / dx + n_ghost;
    const Real pos_y_indU = (pos_y - yMin) / dy + n_ghost;
    const Real pos_z_indU = (pos_z - zMin) / dz + n_ghost;

    bool ignore = (((pos_x_indU < n_ghost) or (pos_x_indU >= (nx_g - n_ghost))) or
                   ((pos_y_indU < n_ghost) or (pos_y_indU >= (ny_g - n_ghost))) or
                   ((pos_z_indU < n_ghost) or (pos_z_indU >= (ny_g - n_ghost))));

    if ((not ignore) and (n_local > gtid)) {
      // note age_dev is actually the time of birth
      Real age = t - age_dev[gtid];

      feedback_model.apply_feedback(pos_x_indU, pos_y_indU, pos_z_indU, age, mass_dev, id_dev, dx, dy, dz,
                                    nx_g, ny_g, nz_g, n_ghost, num_SN_dev[gtid], s_info, conserved_dev);
    }
  }


  // epilogue: sum the info from all threads (in all blocks) and add it into info
  __syncthreads();
  reduction_utilities::blockAccumulateIntoNReals<feedinfoLUT::LEN,TPB_FEEDBACK>(info, s_info);
}

/* determine the number of supernovae during the current step */
__global__ void Get_SN_Count_Kernel(part_int_t n_local, part_int_t* id_dev, Real* mass_dev,
                                    Real* age_dev, Real t, Real dt,
                                    const feedback::SNRateCalc snr_calc, int n_step, int* num_SN_dev)
{
  int tid = threadIdx.x;

  int gtid = blockIdx.x * blockDim.x + tid;
  // Bounds check on particle arrays
  if (gtid >= n_local) return;

  // note age_dev is actually the time of birth
  Real age = t - age_dev[gtid];

  Real average_num_sn = snr_calc.Get_SN_Rate(age) * mass_dev[gtid] * dt;
  num_SN_dev[gtid]    = snr_calc.Get_Number_Of_SNe_In_Cluster(average_num_sn, n_step, id_dev[gtid]);
}

namespace { // anonymous namespace

/* This functor is the callback used in the main part of cholla
 */
template<typename FeedbackModel>
struct ClusterFeedbackMethod {

  ClusterFeedbackMethod(FeedbackAnalysis& analysis, bool use_snr_calc, feedback::SNRateCalc snr_calc)
    : analysis(analysis), use_snr_calc_(use_snr_calc), snr_calc_(snr_calc)
{ }

  /* Actually apply the stellar feedback (SNe and stellar winds) */
  void operator() (Grid3D& G);

private: // attributes

  FeedbackAnalysis& analysis;
  /* When false, ignore the snr_calc_ attribute. Instead, assume all clusters undergo a single
   * supernova during the very first cycle and then never have a supernova again. */
  const bool use_snr_calc_;
  feedback::SNRateCalc snr_calc_;
};

} // close anonymous namespace

/**
 * @brief Stellar feedback function (SNe and stellar winds)
 *
 * @param G
 */
template<typename FeedbackModel>
void ClusterFeedbackMethod<FeedbackModel>::operator()(Grid3D& G)
{
#if !(defined(PARTICLES_GPU) && defined(PARTICLE_AGE) && defined(PARTICLE_IDS))
  CHOLLA_ERROR("This function can't be called with the current compiler flags");
#else
  #ifdef CPU_TIME
  G.Timer.Feedback.Start();
  #endif

  if (max(fabs(G.H.dy - G.H.dx), fabs(G.H.dz - G.H.dx))  > fabs(1e-15 * G.H.dx)) {
    CHOLLA_ERROR("dx, dy, dz must all approximately be the same with the current feedback prescriptions");
  }

  if (G.H.dt == 0) return;

  // h_info is used to store feedback summary info on the host. The following
  // syntax sets all entries to 0 -- important if a process has no particles
  // (this is valid C++ syntax, but historically wasn't valid C syntax)
  Real h_info[feedinfoLUT::LEN] = {};

  // only apply feedback if we have clusters
  if (G.Particles.n_local > 0) {
    // compute the grid-size or the number of thread-blocks per grid. The number of threads in a block is
    // given by TPB_FEEDBACK
    int ngrid = (G.Particles.n_local - 1) / TPB_FEEDBACK + 1;

    // Declare/allocate device buffer for holding the number of supernovae per particle in the current cycle
    // (The following behavior can be accomplished without any memory allocations if we employ templates)
    cuda_utilities::DeviceVector<int> d_num_SN(G.Particles.n_local, true);  // initialized to 0

    if (use_snr_calc_) {
      hipLaunchKernelGGL(Get_SN_Count_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local,
                         G.Particles.partIDs_dev, G.Particles.mass_dev, G.Particles.age_dev, G.H.t, G.H.dt,
                         snr_calc_, G.H.n_step, d_num_SN.data());
      CHECK(hipDeviceSynchronize());
    } else {
      // in this branch, ``this->use_snr_calc_ == false``. This means that we assume all particles undergo
      // a supernova during the very first cycle. Then there is never another supernova
      if (G.H.n_step == 0) {
        std::vector<int> tmp(G.Particles.n_local, 1);
        CHECK(hipMemcpy(d_num_SN.data(), tmp.data(), sizeof(int)*G.Particles.n_local, hipMemcpyHostToDevice));
      } else {
        // do nothing - the number of supernovae is already zero
      }
    }

    // Declare/allocate device buffer for accumulating summary information about feedback
    cuda_utilities::DeviceVector<Real> d_info(feedinfoLUT::LEN, true);  // initialized to 0

    // initialize feedback_model
    FeedbackModel feedback_model{};

    hipLaunchKernelGGL(Cluster_Feedback_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local,
                       G.Particles.partIDs_dev, G.Particles.pos_x_dev, G.Particles.pos_y_dev, G.Particles.pos_z_dev,
                       G.Particles.mass_dev, G.Particles.age_dev, G.H.xblocal, G.H.yblocal, G.H.zblocal,
                       G.H.xblocal_max, G.H.yblocal_max, G.H.zblocal_max, G.H.dx, G.H.dy, G.H.dz, G.H.nx, G.H.ny,
                       G.H.nz, G.H.n_ghost, G.H.t, G.H.dt, d_info.data(), G.C.d_density, gama, 
                       d_num_SN.data(), G.H.n_step, feedback_model);

    // copy summary data back to the host
    CHECK(hipMemcpy(&h_info, d_info.data(), feedinfoLUT::LEN * sizeof(Real), hipMemcpyDeviceToHost));
  }

  // now gather the feedback summary info into an array called info.
  #ifdef MPI_CHOLLA
  Real info[feedinfoLUT::LEN];
  MPI_Reduce(&h_info, &info, feedinfoLUT::LEN, MPI_CHREAL, MPI_SUM, root, world);
  #else
  Real* info = h_info;
  #endif

  #ifdef MPI_CHOLLA  // only do stats gathering on root rank
  if (procID == 0) {
  #endif

    analysis.countSN += (long)info[feedinfoLUT::countSN];
    analysis.countResolved += (long)info[feedinfoLUT::countResolved];
    analysis.countUnresolved += (long)info[feedinfoLUT::countUnresolved];
    analysis.totalEnergy += info[feedinfoLUT::totalEnergy];
    analysis.totalMomentum += info[feedinfoLUT::totalMomentum];
    analysis.totalUnresEnergy += info[feedinfoLUT::totalUnresEnergy];
    analysis.totalWindMomentum += info[feedinfoLUT::totalWindMomentum];
    analysis.totalWindEnergy += info[feedinfoLUT::totalWindEnergy];

    chprintf("iteration %d, t %.4e, dt %.4e", G.H.n_step, G.H.t, G.H.dt);

  #ifndef NO_SN_FEEDBACK
    Real global_resolved_ratio = 0.0;
    if (analysis.countResolved > 0 || analysis.countUnresolved > 0) {
      global_resolved_ratio = analysis.countResolved / (analysis.countResolved + analysis.countUnresolved);
    }
    chprintf(": number of SN: %d,(R: %d, UR: %d)\n", (int)info[feedinfoLUT::countSN], (long)info[feedinfoLUT::countResolved],
             (long)info[feedinfoLUT::countUnresolved]);
    chprintf("    cummulative: #SN: %d, ratio of resolved (R: %d, UR: %d) = %.3e\n", (long)analysis.countSN,
             (long)analysis.countResolved, (long)analysis.countUnresolved, global_resolved_ratio);
    chprintf("    sn  r energy  : %.5e erg, cumulative: %.5e erg\n", info[feedinfoLUT::totalEnergy] * FORCE_UNIT * LENGTH_UNIT,
             analysis.totalEnergy * FORCE_UNIT * LENGTH_UNIT);
    chprintf("    sn ur energy  : %.5e erg, cumulative: %.5e erg\n",
             info[feedinfoLUT::totalUnresEnergy] * FORCE_UNIT * LENGTH_UNIT,
             analysis.totalUnresEnergy * FORCE_UNIT * LENGTH_UNIT);
    chprintf("    sn momentum  : %.5e SM km/s, cumulative: %.5e SM km/s\n",
             info[feedinfoLUT::totalMomentum] * VELOCITY_UNIT / 1e5, analysis.totalMomentum * VELOCITY_UNIT / 1e5);
  #endif  // NO_SN_FEEDBACK

  #ifndef NO_WIND_FEEDBACK
    chprintf("    wind momentum: %.5e S.M. km/s,  cumulative: %.5e S.M. km/s\n",
             info[feedinfoLUT::totalWindMomentum] * VELOCITY_UNIT / 1e5, analysis.totalWindMomentum * VELOCITY_UNIT / 1e5);
    chprintf("    wind energy  : %.5e erg,  cumulative: %.5e erg\n", info[feedinfoLUT::totalWindEnergy] * FORCE_UNIT * LENGTH_UNIT,
             analysis.totalWindEnergy * FORCE_UNIT * LENGTH_UNIT);
  #endif  // NO_WIND_FEEDBACK

  #ifdef MPI_CHOLLA
  }  //   end if procID == 0
  #endif

  #ifdef CPU_TIME
  G.Timer.Feedback.End();
  #endif
#endif // the ifdef statement for Particle-stuff
}

std::function<void(Grid3D&)> feedback::configure_feedback_callback(struct parameters& P,
                                                                   FeedbackAnalysis& analysis)
{
#if !(defined(FEEDBACK) && defined(PARTICLES_GPU) && defined(PARTICLE_AGE) && defined(PARTICLE_IDS))
  const bool supports_feedback = false;
#else
  const bool supports_feedback = true;
#endif

  // retrieve the supernova-feedback model name
  std::string sn_model = P.feedback_sn_model;
  if (sn_model.empty() and (not supports_feedback)) {
    sn_model = "none";
  } else if (sn_model.empty()) {
#ifdef ONLY_RESOLVED
    sn_model = "resolvedCiC";
#else
    sn_model = "legacy";
#endif
    chprintf("the feedback_sn_model was not supplied. Right now, we are defaulting to \"%s\" (based "
             "on compiler flags) - in the future we will abort with an error instead",
             sn_model.c_str());
  }


  // handle the case when there is no feedback (or if the code can't support feedback)
  if (sn_model == "none") {  // return an empty objec
    return {};
  } else if (not supports_feedback) {
    CHOLLA_ERROR("The way that cholla was compiled does not currently support feedback");
  }


  // parse the supernova-rate-model to initialize some values
  SNRateCalc snr_calc{};
  bool use_snr_calc;

  const std::string sn_rate_model = P.feedback_sn_rate;
  if (sn_rate_model.empty() or (sn_rate_model == "table")) {
    use_snr_calc = true;
    snr_calc = feedback::SNRateCalc(P);
  } else if (sn_rate_model == "immediate_sn") {
    use_snr_calc = false;
  } else {
    CHOLLA_ERROR("Unrecognized option passed to sn_rate_model: %s", sn_rate_model.c_str());
  }

  // now lets initialize ClusterFeedbackMethod<> and return
  std::function<void(Grid3D&)> out;
  if (sn_model == "legacy") {
    out = ClusterFeedbackMethod<feedback_model::LegacySNe<feedback_model::CiCResolvedSNPrescription>>(analysis, use_snr_calc, snr_calc);
  } else if (sn_model == "resolvedCiC") {
    out = ClusterFeedbackMethod<feedback_model::CiCResolvedSNPrescription>(analysis, use_snr_calc, snr_calc);
  } else if (sn_model == "resolved27cell") {
    out = ClusterFeedbackMethod<feedback_model::Sphere27ResolvedSNPrescription>(analysis, use_snr_calc, snr_calc);
  } else if (sn_model == "resolvedExperimentalBinarySphere"){
    out = ClusterFeedbackMethod<feedback_model::SphereBinaryResolvedSNPrescription>(analysis, use_snr_calc, snr_calc);
  } else {
    CHOLLA_ERROR("Unrecognized sn_model: %s", sn_model.c_str());
  }
  return out;
}


