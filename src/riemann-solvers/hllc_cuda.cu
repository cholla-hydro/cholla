
#include <hip/hip_runtime.h>
/*! \file hllc_cuda.cu
 *  \brief Function definitions for the cuda HLLC Riemann solver.*/

#ifdef CUDA

#include"gpu.hpp"
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"hllc_cuda.h"

#ifdef DE //PRESSURE_DE
#include"hydro_cuda.h"
#endif


/*! \fn Calculate_HLLC_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir, int n_fields)
 *  \brief HLLC Riemann solver based on the version described in Toro (2006), Sec. 10.4. */
__global__ void Calculate_HLLC_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir, int n_fields)
{
  // get a thread index
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;

  int n_cells = nx*ny*nz;

  Real dl, vxl, mxl, vyl, myl, vzl, mzl, pl, El;
  Real dr, vxr, mxr, vyr, myr, vzr, mzr, pr, Er;

  Real g1 = gamma - 1.0; 
  Real Hl, Hr;
  Real sqrtdl, sqrtdr, vx, vy, vz, H;
  Real vsq, asq, a;
  Real lambda_m, lambda_p;
  Real f_d_l, f_mx_l, f_my_l, f_mz_l, f_E_l;
  Real f_d_r, f_mx_r, f_my_r, f_mz_r, f_E_r;
  Real dls, drs, mxls, mxrs, myls, myrs, mzls, mzrs, Els, Ers;
  Real f_d, f_mx, f_my, f_mz, f_E;
  Real Sl, Sr, Sm, cfl, cfr, ps;
  #ifdef DE
  Real dgel, dger, gel, ger, gels, gers, f_ge_l, f_ge_r, f_ge, E_kin;
  #endif
  #ifdef SCALAR
  Real dscl[NSCALARS], dscr[NSCALARS], scl[NSCALARS], scr[NSCALARS], scls[NSCALARS], scrs[NSCALARS], f_sc_l[NSCALARS], f_sc_r[NSCALARS], f_sc[NSCALARS];
  #endif
  
  Real etah = 0;

  int o1, o2, o3;
  if (dir==0) {
    o1 = 1; o2 = 2; o3 = 3;
  }
  if (dir==1) {
    o1 = 2; o2 = 3; o3 = 1;
  }
  if (dir==2) {
    o1 = 3; o2 = 1; o3 = 2;
  }

  // Each thread executes the solver independently
  //if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid < ny && zid < nz) 
  if (xid < nx && yid < ny && zid < nz) 
  {
    // retrieve conserved variables
    dl  = dev_bounds_L[             tid];
    mxl = dev_bounds_L[o1*n_cells + tid];
    myl = dev_bounds_L[o2*n_cells + tid];
    mzl = dev_bounds_L[o3*n_cells + tid];
    El  = dev_bounds_L[4*n_cells + tid];
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dscl[i] = dev_bounds_L[(5+i)*n_cells + tid];
    }
    #endif
    #ifdef DE
    dgel = dev_bounds_L[(n_fields-1)*n_cells + tid];
    #endif

    dr  = dev_bounds_R[            tid];
    mxr = dev_bounds_R[o1*n_cells + tid];
    myr = dev_bounds_R[o2*n_cells + tid];
    mzr = dev_bounds_R[o3*n_cells + tid];
    Er  = dev_bounds_R[4*n_cells + tid]; 
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dscr[i] = dev_bounds_R[(5+i)*n_cells + tid];
    }
    #endif
    #ifdef DE
    dger = dev_bounds_R[(n_fields-1)*n_cells + tid];
    #endif

    // calculate primative variables
    vxl = mxl / dl;
    vyl = myl / dl;
    vzl = mzl / dl;
    #ifdef DE //PRESSURE_DE
    E_kin = 0.5 * dl * ( vxl*vxl + vyl*vyl + vzl*vzl );
    pl = Get_Pressure_From_DE( El, El - E_kin, dgel, gamma ); 
    #else
    pl  = (El - 0.5*dl*(vxl*vxl + vyl*vyl + vzl*vzl)) * (gamma - 1.0);
    #endif //PRESSURE_DE
    pl  = fmax(pl, (Real) TINY_NUMBER);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scl[i] = dscl[i] / dl;
    }
    #endif
    #ifdef DE
    gel = dgel / dl;
    #endif
    vxr = mxr / dr;
    vyr = myr / dr;
    vzr = mzr / dr;
    #ifdef DE //PRESSURE_DE
    E_kin = 0.5 * dr * ( vxr*vxr + vyr*vyr + vzr*vzr );
    pr = Get_Pressure_From_DE( Er, Er - E_kin, dger, gamma );
    #else
    pr  = (Er - 0.5*dr*(vxr*vxr + vyr*vyr + vzr*vzr)) * (gamma - 1.0);
    #endif //PRESSURE_DE
    pr  = fmax(pr, (Real) TINY_NUMBER);    
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scr[i] = dscr[i] / dr;
    }
    #endif
    #ifdef DE
    ger = dger / dr;
    #endif

    // calculate the enthalpy in each cell
    Hl = (El + pl) / dl;
    Hr = (Er + pr) / dr;

    // calculate averages of the variables needed for the Roe Jacobian 
    // (see Stone et al., 2008, Eqn 65, or Toro 2009, 11.118)
    sqrtdl = sqrt(dl);
    sqrtdr = sqrt(dr);
    vx = (sqrtdl*vxl + sqrtdr*vxr) / (sqrtdl + sqrtdr);
    vy = (sqrtdl*vyl + sqrtdr*vyr) / (sqrtdl + sqrtdr);
    vz = (sqrtdl*vzl + sqrtdr*vzr) / (sqrtdl + sqrtdr);
    H  = (sqrtdl*Hl  + sqrtdr*Hr)  / (sqrtdl + sqrtdr); 

    // calculate the sound speed squared (Stone B2)
    vsq = (vx*vx + vy*vy + vz*vz);
    asq = g1*(H - 0.5*vsq);
    a = sqrt(asq);

    // calculate the averaged eigenvectors of the Roe matrix (Stone Eqn B2, Toro 11.107)
    lambda_m = vx - a; 
    lambda_p = vx + a;


    // compute max and min wave speeds
    cfl = sqrt(gamma*pl/dl);  // sound speed in left state
    cfr = sqrt(gamma*pr/dr);  // sound speed in right state

    // for signal speeds, take max/min of Roe eigenvalues and left and right sound speeds
    // Batten eqn. 48
    Sl = fmin(lambda_m, vxl - cfl);
    Sr = fmax(lambda_p, vxr + cfr);

    // if the H-correction is turned on, add cross-flux dissipation
    Sl = sgn_CUDA(Sl)*fmax(fabs(Sl), etah);
    Sr = sgn_CUDA(Sr)*fmax(fabs(Sr), etah);

 
    // left and right fluxes 
    f_d_l  = mxl;
    f_mx_l = mxl*vxl + pl;
    f_my_l = myl*vxl;
    f_mz_l = mzl*vxl;
    f_E_l  = (El + pl)*vxl;
    #ifdef DE
    f_ge_l = dgel*vxl;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      f_sc_l[i] = dscl[i]*vxl;
    }
    #endif

    f_d_r  = mxr;
    f_mx_r = mxr*vxr + pr;
    f_my_r = myr*vxr;
    f_mz_r = mzr*vxr;
    f_E_r  = (Er + pr)*vxr;
    #ifdef DE
    f_ge_r = dger*vxr;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      f_sc_r[i] = dscr[i]*vxr;
    }
    #endif

    // return upwind flux if flow is supersonic 
    if (Sl > 0.0) {
      dev_flux[           tid] = f_d_l;
      dev_flux[o1*n_cells+tid] = f_mx_l;
      dev_flux[o2*n_cells+tid] = f_my_l;
      dev_flux[o3*n_cells+tid] = f_mz_l;
      dev_flux[4*n_cells+tid]  = f_E_l;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells+tid]  = f_sc_l[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells+tid]  = f_ge_l;
      #endif
      return;
    }
    else if (Sr < 0.0) {
      dev_flux[           tid] = f_d_r;
      dev_flux[o1*n_cells+tid] = f_mx_r;
      dev_flux[o2*n_cells+tid] = f_my_r;
      dev_flux[o3*n_cells+tid] = f_mz_r;
      dev_flux[4*n_cells+tid]  = f_E_r;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells+tid]  = f_sc_r[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells+tid]  = f_ge_r;
      #endif
      return;
    }
    // otherwise compute subsonic flux
    else { 

      // compute contact wave speed and pressure in star region (Batten eqns 34 & 36)
      Sm = (dr*vxr*(Sr - vxr) - dl*vxl*(Sl - vxl) + pl - pr) / (dr*(Sr - vxr) - dl*(Sl - vxl));
      ps = dl*(vxl - Sl)*(vxl - Sm) + pl;

      // conserved variables in the left star state (Batten eqns 35 - 40)
      dls = dl * (Sl - vxl) / (Sl - Sm);
      mxls = (mxl*(Sl - vxl) + ps - pl) / (Sl - Sm);
      myls = dls*vyl;
      mzls = dls*vzl;
      Els = (El*(Sl - vxl) - pl*vxl + ps*Sm) / (Sl - Sm);
      #ifdef DE
      gels = dls*gel;
      #endif
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        scls[i] = dls*scl[i];
      }
      #endif

      // conserved variables in the right star state
      drs = dr * (Sr - vxr) / (Sr - Sm);
      mxrs = (mxr*(Sr - vxr) + ps - pr) / (Sr - Sm);
      myrs = drs*vyr;
      mzrs = drs*vzr;
      Ers = (Er*(Sr - vxr) - pr*vxr + ps*Sm) / (Sr - Sm);
      #ifdef DE
      gers = drs*ger;
      #endif
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        scrs[i] = drs*scr[i];
      }
      #endif


      // compute the hllc flux (Batten eqn 27)
      f_d  = 0.5*(f_d_l  + f_d_r  + (Sr - fabs(Sm))*drs  + (Sl + fabs(Sm))*dls  - Sl*dl  - Sr*dr);
      f_mx = 0.5*(f_mx_l + f_mx_r + (Sr - fabs(Sm))*mxrs + (Sl + fabs(Sm))*mxls - Sl*mxl - Sr*mxr);
      f_my = 0.5*(f_my_l + f_my_r + (Sr - fabs(Sm))*myrs + (Sl + fabs(Sm))*myls - Sl*myl - Sr*myr);
      f_mz = 0.5*(f_mz_l + f_mz_r + (Sr - fabs(Sm))*mzrs + (Sl + fabs(Sm))*mzls - Sl*mzl - Sr*mzr);
      f_E  = 0.5*(f_E_l  + f_E_r  + (Sr - fabs(Sm))*Ers  + (Sl + fabs(Sm))*Els  - Sl*El  - Sr*Er);
      #ifdef DE
      f_ge = 0.5*(f_ge_l + f_ge_r + (Sr - fabs(Sm))*gers + (Sl + fabs(Sm))*gels - Sl*dgel - Sr*dger);
      #endif
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        f_sc[i] = 0.5*(f_sc_l[i] + f_sc_r[i] + (Sr - fabs(Sm))*scrs[i] + (Sl + fabs(Sm))*scls[i] - Sl*dscl[i] - Sr*dscr[i]);
      }
      #endif


      // return the hllc fluxes
      dev_flux[           tid] = f_d;
      dev_flux[o1*n_cells+tid] = f_mx;
      dev_flux[o2*n_cells+tid] = f_my;
      dev_flux[o3*n_cells+tid] = f_mz;
      dev_flux[4*n_cells+tid]  = f_E;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells+tid]  = f_sc[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells+tid]  = f_ge;
      #endif

    }
  }

}


#endif //CUDA
