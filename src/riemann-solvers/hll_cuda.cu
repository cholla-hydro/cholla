
#include <hip/hip_runtime.h>
/*! \file hllc_cuda.cu
 *  \brief Function definitions for the cuda HLLC Riemann solver.*/

#ifdef CUDA

#include"gpu.hpp"
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"hll_cuda.h"

#ifdef DE //PRESSURE_DE
#include"hydro_cuda.h"
#endif



/*! \fn Calculate_HLLC_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir, int n_fields)
 *  \brief HLLC Riemann solver based on the version described in Toro (2006), Sec. 10.4. */
__global__ void Calculate_HLL_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir, int n_fields)
{
  // get a thread index
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;

  int n_cells = nx*ny*nz;

  Real dl, vxl, mxl, vyl, myl, vzl, mzl, pl, El;
  Real dr, vxr, mxr, vyr, myr, vzr, mzr, pr, Er;

  // Real g1 = gamma - 1.0; 
  // Real Hl, Hr;
  // Real sqrtdl, sqrtdr, vx, vy, vz, H;
  // Real vsq, asq, a;
  // Real lambda_m, lambda_p;
  Real f_d_l, f_mx_l, f_my_l, f_mz_l, f_E_l;
  Real f_d_r, f_mx_r, f_my_r, f_mz_r, f_E_r;
  // Real dls, drs, mxls, mxrs, myls, myrs, mzls, mzrs, Els, Ers;
  Real f_d, f_mx, f_my, f_mz, f_E;
  Real Sl, Sr, cfl, cfr;
  #ifdef DE
  Real dgel, dger, f_ge_l, f_ge_r, f_ge, E_kin;
  #endif
  #ifdef SCALAR
  Real dscl[NSCALARS], dscr[NSCALARS], f_sc_l[NSCALARS], f_sc_r[NSCALARS], f_sc[NSCALARS];
  #endif
  
  // Real etah = 0;

  int o1, o2, o3;
  if (dir==0) {
    o1 = 1; o2 = 2; o3 = 3;
  }
  if (dir==1) {
    o1 = 2; o2 = 3; o3 = 1;
  }
  if (dir==2) {
    o1 = 3; o2 = 1; o3 = 2;
  }

  // Each thread executes the solver independently
  //if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid < ny && zid < nz) 
  if (xid < nx && yid < ny && zid < nz) 
  {
    // retrieve conserved variables
    dl  = dev_bounds_L[             tid];
    mxl = dev_bounds_L[o1*n_cells + tid];
    myl = dev_bounds_L[o2*n_cells + tid];
    mzl = dev_bounds_L[o3*n_cells + tid];
    El  = dev_bounds_L[4*n_cells + tid];
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dscl[i] = dev_bounds_L[(5+i)*n_cells + tid];
    }
    #endif
    #ifdef DE
    dgel = dev_bounds_L[(n_fields-1)*n_cells + tid];
    #endif

    dr  = dev_bounds_R[            tid];
    mxr = dev_bounds_R[o1*n_cells + tid];
    myr = dev_bounds_R[o2*n_cells + tid];
    mzr = dev_bounds_R[o3*n_cells + tid];
    Er  = dev_bounds_R[4*n_cells + tid]; 
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dscr[i] = dev_bounds_R[(5+i)*n_cells + tid];
    }
    #endif
    #ifdef DE
    dger = dev_bounds_R[(n_fields-1)*n_cells + tid];
    #endif

    // calculate primative variables
    vxl = mxl / dl;
    vyl = myl / dl;
    vzl = mzl / dl;
    #ifdef DE //PRESSURE_DE
    E_kin = 0.5 * dl * ( vxl*vxl + vyl*vyl + vzl*vzl );
    pl = Get_Pressure_From_DE( El, El - E_kin, dgel, gamma ); 
    #else
    pl  = (El - 0.5*dl*(vxl*vxl + vyl*vyl + vzl*vzl)) * (gamma - 1.0);
    #endif//DE
    pl  = fmax(pl, (Real) TINY_NUMBER);
    // #ifdef SCALAR
    // for (int i=0; i<NSCALARS; i++) {
    //   scl[i] = dscl[i] / dl;
    // }
    // #endif
    // #ifdef DE
    // gel = dgel / dl;
    // #endif
    vxr = mxr / dr;
    vyr = myr / dr;
    vzr = mzr / dr;
    #ifdef DE //PRESSURE_DE
    E_kin = 0.5 * dr * ( vxr*vxr + vyr*vyr + vzr*vzr );
    pr = Get_Pressure_From_DE( Er, Er - E_kin, dger, gamma ); 
    #else
    pr  = (Er - 0.5*dr*(vxr*vxr + vyr*vyr + vzr*vzr)) * (gamma - 1.0);
    #endif//DE
    pr  = fmax(pr, (Real) TINY_NUMBER);    
    // #ifdef SCALAR
    // for (int i=0; i<NSCALARS; i++) {
    //   scr[i] = dscr[i] / dr;
    // }
    // #endif
    // #ifdef DE
    // ger = dger / dr;
    // #endif

    // calculate the enthalpy in each cell
    // Hl = (El + pl) / dl;
    // Hr = (Er + pr) / dr;

    // calculate averages of the variables needed for the Roe Jacobian 
    // (see Stone et al., 2008, Eqn 65, or Toro 2009, 11.118)
    // sqrtdl = sqrt(dl);
    // sqrtdr = sqrt(dr);
    // vx = (sqrtdl*vxl + sqrtdr*vxr) / (sqrtdl + sqrtdr);
    // vy = (sqrtdl*vyl + sqrtdr*vyr) / (sqrtdl + sqrtdr);
    // vz = (sqrtdl*vzl + sqrtdr*vzr) / (sqrtdl + sqrtdr);
    // H  = (sqrtdl*Hl  + sqrtdr*Hr)  / (sqrtdl + sqrtdr); 

    // calculate the sound speed squared (Stone B2)
    // vsq = (vx*vx + vy*vy + vz*vz);
    // asq = g1*(H - 0.5*vsq);
    // a = sqrt(asq);

    // calculate the averaged eigenvectors of the Roe matrix (Stone Eqn B2, Toro 11.107)
    // lambda_m = vx - a; 
    // lambda_p = vx + a;


    // compute max and min wave speeds
    cfl = sqrt(gamma*pl/dl);  // sound speed in left state
    cfr = sqrt(gamma*pr/dr);  // sound speed in right state

    // for signal speeds, take max/min of Roe eigenvalues and left and right sound speeds
    // Batten eqn. 48
    // Sl = fmin(lambda_m, vxl - cfl);
    // Sr = fmax(lambda_p, vxr + cfr);

    // if the H-correction is turned on, add cross-flux dissipation
    // Sl = sgn_CUDA(Sl)*fmax(fabs(Sl), etah);
    // Sr = sgn_CUDA(Sr)*fmax(fabs(Sr), etah);
    Sl = fmin(vxr - cfr, vxl - cfl);
    Sr = fmax(vxl + cfl, vxr + cfr);

 
    // left and right fluxes 
    f_d_l  = mxl;
    f_mx_l = mxl*vxl + pl;
    f_my_l = myl*vxl;
    f_mz_l = mzl*vxl;
    f_E_l  = (El + pl)*vxl;
    #ifdef DE
    f_ge_l = dgel*vxl;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      f_sc_l[i] = dscl[i]*vxl;
    }
    #endif

    f_d_r  = mxr;
    f_mx_r = mxr*vxr + pr;
    f_my_r = myr*vxr;
    f_mz_r = mzr*vxr;
    f_E_r  = (Er + pr)*vxr;
    #ifdef DE
    f_ge_r = dger*vxr;
    #endif
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      f_sc_r[i] = dscr[i]*vxr;
    }
    #endif

    // return upwind flux if flow is supersonic 
    if (Sl > 0.0) {
      dev_flux[           tid] = f_d_l;
      dev_flux[o1*n_cells+tid] = f_mx_l;
      dev_flux[o2*n_cells+tid] = f_my_l;
      dev_flux[o3*n_cells+tid] = f_mz_l;
      dev_flux[4*n_cells+tid]  = f_E_l;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells+tid]  = f_sc_l[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells+tid]  = f_ge_l;
      #endif
      return;
    }
    else if (Sr < 0.0) {
      dev_flux[           tid] = f_d_r;
      dev_flux[o1*n_cells+tid] = f_mx_r;
      dev_flux[o2*n_cells+tid] = f_my_r;
      dev_flux[o3*n_cells+tid] = f_mz_r;
      dev_flux[4*n_cells+tid]  = f_E_r;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells+tid]  = f_sc_r[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells+tid]  = f_ge_r;
      #endif
      return;
    }
    // otherwise compute subsonic flux
    else { 
      
      f_d = ( ( Sr * f_d_l ) - ( Sl * f_d_r ) + Sl*Sr*( dr - dl ) ) / ( Sr - Sl );  
      f_mx = ( ( Sr * f_mx_l ) - ( Sl * f_mx_r ) + Sl*Sr*( mxr - mxl ) ) / ( Sr - Sl );  
      f_my = ( ( Sr * f_my_l ) - ( Sl * f_my_r ) + Sl*Sr*( myr - myl ) ) / ( Sr - Sl );  
      f_mz = ( ( Sr * f_mz_l ) - ( Sl * f_mz_r ) + Sl*Sr*( mzr - mzl ) ) / ( Sr - Sl );
      f_E = ( ( Sr * f_E_l ) - ( Sl * f_E_r ) + Sl*Sr*( Er - El ) ) / ( Sr - Sl );    
      #ifdef DE
      f_ge = ( ( Sr * f_ge_l ) - ( Sl * f_ge_r ) + Sl*Sr*( dger - dgel ) ) / ( Sr - Sl );  
      #endif
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
      f_sc[i] = ( ( Sr * f_sc_l[i] ) - ( Sl * f_sc_r[i] ) + Sl*Sr*( dscr[i] - dscl[i] ) ) / ( Sr - Sl );
      }
      #endif



      // return the hllc fluxes
      dev_flux[           tid] = f_d;
      dev_flux[o1*n_cells+tid] = f_mx;
      dev_flux[o2*n_cells+tid] = f_my;
      dev_flux[o3*n_cells+tid] = f_mz;
      dev_flux[4*n_cells+tid]  = f_E;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells+tid]  = f_sc[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells+tid]  = f_ge;
      #endif

    }
  }

}


#endif //CUDA
