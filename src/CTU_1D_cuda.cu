
#include <hip/hip_runtime.h>
/*! \file CTU_1D_cuda.cu
 *  \brief Definitions of the cuda CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"gpu.hpp"
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_1D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"cooling_cuda.h"
#include"error_handling.h"
#include"io.h"



Real CTU_Algorithm_1D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int x_off, int n_ghost, Real dx, Real xbound, Real dt, int n_fields)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //host_conserved0 contains the values at time n
  //host_conserved1 will contain the values at time n+1

  // Initialize dt values
  Real max_dti = 0;
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  #endif  

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  // set the dimensions of the cuda grid
  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  if ( !memory_allocated ) {

    // allocate an array on the CPU to hold max_dti returned from each thread block
    CudaSafeCall( cudaHostAlloc(&host_dti_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaHostAlloc(&host_dt_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #endif

    // allocate memory on the GPU
    CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   (n_fields)*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #if defined COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif  

    #ifndef DYNAMIC_GPU_ALLOC 
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif 
  }

  // copy the conserved variable array onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, host_conserved0, n_fields*n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
  CudaCheckError();


  // Step 1: Do the reconstruction
  #ifdef PCM
  hipLaunchKernelGGL(PCM_Reconstruction_1D, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMP
  hipLaunchKernelGGL(PLMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMC
  hipLaunchKernelGGL(PLMC_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMP
  hipLaunchKernelGGL(PPMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMC
  hipLaunchKernelGGL(PPMC_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif

  
  // Step 2: Calculate the fluxes
  #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef HLLC 
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();

  #ifdef DE
  // Compute the divergence of Vel before updating the conserved array, this solves syncronization issues when adding this term on Update_Conserved_Variables
  hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_1D, dimGrid, dimBlock, 0, 0,  dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, dx, dt, gama, n_fields );
  #endif


  // Step 3: Update the conserved variable array
  hipLaunchKernelGGL(Update_Conserved_Variables_1D, dimGrid, dimBlock, 0, 0, dev_conserved, F_x, n_cells, x_off, n_ghost, dx, xbound, dt, gama, n_fields);
  CudaCheckError();
   

  // Sychronize the total and internal energy, if using dual-energy formalism
  #ifdef DE
  hipLaunchKernelGGL(Select_Internal_Energy_1D, dimGrid, dimBlock, 0, 0, dev_conserved, nx, n_ghost, n_fields);  
  hipLaunchKernelGGL(Sync_Energies_1D, dimGrid, dimBlock, 0, 0, dev_conserved, n_cells, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif


  // Apply cooling
  #ifdef COOLING_GPU
  hipLaunchKernelGGL(cooling_kernel, dimGrid, dimBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields, dt, gama, dev_dti_array);
  CudaCheckError();
  #endif

  // Calculate the next timestep
  hipLaunchKernelGGL(Calc_dt_1D, dimGrid, dimBlock, 0, 0, dev_conserved, n_cells, n_ghost, dx, dev_dti_array, gama);
  CudaCheckError();


  // copy the conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(host_conserved1, dev_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );

  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }
  #if defined COOLING_GPU
  // copy the dt array from cooling onto the CPU
  CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // find maximum inverse timestep from cooling time
  for (int i=0; i<ngrid; i++) {
    min_dt = fmin(min_dt, host_dt_array[i]);
  }  
  if (min_dt < C_cfl/max_dti) {
    max_dti = C_cfl/min_dt;
  }
  #endif

  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_CTU_1D();
  #endif
  

  // return the maximum inverse timestep
  return max_dti;


}

void Free_Memory_CTU_1D() {

  // free the CPU memory
  CudaSafeCall( cudaFreeHost(host_dti_array) );
  #if defined COOLING_GPU
  CudaSafeCall( cudaFreeHost(host_dt_array) );  
  #endif

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(F_x);
  cudaFree(dev_dti_array);
  #if defined COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

}


#endif //CUDA
