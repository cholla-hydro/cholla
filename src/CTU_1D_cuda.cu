
#include <hip/hip_runtime.h>
/*! \file CTU_1D_cuda.cu
 *  \brief Definitions of the cuda CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"CTU_1D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_ctu_cuda.h"
#include"plmc_ctu_cuda.h"
#include"ppmp_ctu_cuda.h"
#include"ppmc_ctu_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"cooling_cuda.h"
#include"error_handling.h"
#include"io.h"


//#define TEST


__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int n_ghost, 
                                              Real dx, Real dt, Real gamma);

__global__ void Calc_dt_1D(Real *dev_conserved, int n_cells, int n_ghost, Real dx, Real *dti_array, Real gamma);

__global__ void Sync_Energies_1D(Real *dev_conserved, int n_cells, int n_ghost, Real gamma);



Real CTU_Algorithm_1D_CUDA(Real *host_conserved, int nx, int n_ghost, Real dx, Real dt)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid

  // capture the start time
  #ifdef TIME
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  float elapsedTime;
  #endif

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  int n_fields = 5;
  #ifdef DE
  n_fields = 7;
  #endif

  // set the dimensions of the cuda grid
  int  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));

  #ifdef TEST
  Real *test1, *test2;
  test1 = (Real *) malloc(5*n_cells*sizeof(Real));
  test2 = (Real *) malloc(5*n_cells*sizeof(Real));
  #endif

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved;
  // initial input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_L, *Q_R, *F;
  // array to hold zero values for H correction (necessary to pass to Roe solver)
  Real *etah;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;


  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_L, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_R, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F,   (n_fields)*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah, n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );

  // zero all the GPU arrays
  cudaMemset(dev_conserved, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(Q_L, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(Q_R, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(F, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(etah, 0, n_cells*sizeof(Real));
  cudaMemset(dev_dti_array, 0, ngrid*sizeof(Real));
  CudaCheckError();


  // copy the conserved variable array onto the GPU
  #ifdef TIME
  cudaEventRecord(start, 0);
  #endif
  CudaSafeCall( cudaMemcpy(dev_conserved, host_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
  #ifdef TIME
  // get stop time and display the timing results
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU copy: %5.3f ms\n", elapsedTime);
  #endif
  CudaCheckError();


  // Step 1: Do the reconstruction
  #ifdef TIME
  cudaEventRecord(start, 0);
  #endif
  #ifdef PCM
  PCM_Reconstruction_1D<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, n_ghost, gama);
  CudaCheckError();
  #endif
  #ifdef PLMP
  PLMP_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif
  #ifdef PLMC
  PLMC_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif
  #ifdef PPMP
  PPMP_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif
  #ifdef PPMC
  PPMC_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif
  #ifdef TEST
  CudaSafeCall( cudaMemcpy(test1, Q_L, 5*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );  
  CudaSafeCall( cudaMemcpy(test2, Q_R, 5*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );  
  for (int i=0; i<nx; i++) {
    printf("%d %f %f\n", i, test1[i], test2[i]);
  }
  #endif

  #ifdef TIME
  // get stop time and display the timing results
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsedTime, start, stop);
  printf("Time to do reconstruction: %5.3f ms\n", elapsedTime);
  #endif

  
  // Step 2: Calculate the fluxes
  #ifdef TIME
  cudaEventRecord(start, 0);
  #endif
  #ifdef EXACT
  Calculate_Exact_Fluxes<<<dimGrid,dimBlock>>>(Q_L, Q_R, F, nx, ny, nz, n_ghost, gama, 0);
  #endif
  #ifdef ROE
  Calculate_Roe_Fluxes<<<dimGrid,dimBlock>>>(Q_L, Q_R, F, nx, ny, nz, n_ghost, gama, etah, 0);
  #endif
  CudaCheckError();
  #ifdef TIME
  // get stop time, and display the timing results
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsedTime, start, stop);
  printf("Time to do riemann problem:  %5.3f ms\n", elapsedTime);
  #endif


  // Step 3: Update the conserved variable array
  #ifdef TIME
  cudaEventRecord(start, 0);
  #endif      
  Update_Conserved_Variables_1D<<<dimGrid,dimBlock>>>(dev_conserved, F, n_cells, n_ghost, dx, dt, gama);
  CudaCheckError();
  #ifdef TIME
  // get stop time and display the timing results
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsedTime, start, stop);
  printf("conserved variable update: %5.3f ms\n", elapsedTime);
  #endif    

  #ifdef DE
  Sync_Energies_1D<<<dimGrid,dimBlock>>>(dev_conserved, n_cells, n_ghost, gama);
  #endif


  #ifdef COOLING_GPU
  cooling_kernel<<<dimGrid,dimBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dt, gama);
  #endif

  // Calculate the next timestep
  Calc_dt_1D<<<dimGrid,dimBlock>>>(dev_conserved, n_cells, n_ghost, dx, dev_dti_array, gama);
  CudaCheckError();


  // copy the conserved variable array back to the CPU
  #ifdef TIME
  cudaEventRecord(start, 0);
  #endif
  CudaSafeCall( cudaMemcpy(host_conserved, dev_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );
  #ifdef TIME
  // get stop time and display the timing results
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU return: %5.3f ms\n", elapsedTime);
  #endif


  #ifdef TIME
  cudaEventRecord(start, 0);
  #endif      
  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }
  #ifdef TIME
  // get stop time and display the timing results
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsedTime, start, stop);
  printf("dti copying & calc: %5.3f ms\n", elapsedTime);
  #endif     


  // free the CPU memory
  free(host_dti_array);

  #ifdef TEST
  free(test1);
  free(test2);
  #endif

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_L);
  cudaFree(Q_R);
  cudaFree(F);
  cudaFree(etah);

  #ifdef TIME
  cudaEventDestroy(start);
  cudaEventDestroy(stop);
  #endif


  // return the maximum inverse timestep
  return max_dti;


}


__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int n_ghost, Real dx, Real dt, Real gamma)
{
  int id;
  Real d, d_inv, vx, vy, vz, P;  
  //#ifdef DE
  Real vx_imo, vx_ipo, e_old;
  //#endif

  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //#ifdef DE
    vx_imo = dev_conserved[1*n_cells + id-1]/dev_conserved[id-1];
    vx_ipo = dev_conserved[1*n_cells + id+1]/dev_conserved[id+1];
    //e_old = dev_conserved[5*n_cells + id];
    //#endif
  
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F[            id-1] - dev_F[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F[  n_cells + id-1] - dev_F[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F[2*n_cells + id-1] - dev_F[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F[3*n_cells + id-1] - dev_F[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F[4*n_cells + id-1] - dev_F[4*n_cells + id]);
    #ifdef DE
    dev_conserved[5*n_cells + id] += dtodx * (dev_F[5*n_cells + id-1] - dev_F[5*n_cells + id])
    //                              +  dtodx * P * (dev_F[6*n_cells + id-1] - dev_F[6*n_cells + id]);
                                  +  dtodx * P * 0.5 * (vx_imo - vx_ipo);
    #endif
    //if (dev_F[id-1] != dev_F[id]) printf("%3d density flux: %f %f\n", id, dev_F[id-1], dev_F[id]);
    //if (dev_F[4*n_cells + id-1] != dev_F[4*n_cells + id]) printf("%3d Energy flux: %f %f\n", id, dev_F[4*n_cells+id-1], dev_F[4*n_cells+id]);
    //if (dev_F[5*n_cells + id-1] != dev_F[5*n_cells + id]) printf("%3d energy flux: %f %f\n", id, dev_F[5*n_cells+id-1], dev_F[5*n_cells+id]);
    //if (dev_F[6*n_cells + id-1] != dev_F[6*n_cells + id]) printf("%3d velocities: %f %f\n", id, dev_F[6*n_cells+id-1], dev_F[6*n_cells+id]);
    //if (vx_imo != vx_ipo) printf("%3d vx_imo %f vx_ipo %f\n", id, vx_imo, vx_ipo);
    //if (e_old != dev_conserved[5*n_cells + id]) printf("%3d New e is different\n", id);
    if (dev_conserved[id] != dev_conserved[id]) printf("%3d Thread crashed in final update.\n", id);
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%d Negative pressure after final update.\n", id);
    //printf("%3d %f %f %f\n", id, P/(gamma-1.0), dev_conserved[4*n_cells+id],  P/(gamma-1.0)/dev_conserved[4*n_cells+id]);
  }


}



__global__ void Sync_Energies_1D(Real *dev_conserved, int n_cells, int n_ghost, Real gamma)
{
  int id;
  Real d, d_inv, vx, vy, vz, P, E;
  Real ge1, ge2, Emax;
  int im1, ip1;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  
  im1 = max(id-1, n_ghost);
  ip1 = min(id+1, n_cells-n_ghost-1);

  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    // separately tracked internal energy 
    ge1 = dev_conserved[5*n_cells + id];
    // internal energy calculated from total energy
    ge2 = dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz);
    // if the ratio of conservatively calculated internal energy to total energy
    // is greater than 1/1000, use the conservatively calculated internal energy
    // to do the internal energy update
    if (ge2/E > 0.001) {
      dev_conserved[5*n_cells + id] = ge2;
      ge1 = ge2;
    }     
    // find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + im1], E);
    Emax = fmax(dev_conserved[4*n_cells + ip1], Emax);
    // if the ratio of conservatively calculated internal energy to max nearby total energy
    // is greater than 1/10, continue to use the conservatively calculated internal energy 
    if (ge2/Emax > 0.1) {
      dev_conserved[5*n_cells + id] = ge2;
    }
    // sync the total energy with the internal energy 
    else {
      dev_conserved[4*n_cells + id] += ge1 - ge2;
    }
    /*
    // if the conservatively calculated internal energy is greater than the estimate of the truncation error,
    // use the internal energy computed from the total energy to do the update
    //find the max nearby velocity difference (estimate of truncation error) 
    vmax = fmax(fabs(vx-dev_conserved[1*n_cells + im1]/dev_conserved[im1]), fabs(dev_conserved[1*n_cells + ip1]/dev_conserved[ip1]-vx));
    //printf("%3d %f %f %f %f\n", id, ge1, ge2, vmax, 0.25*d*vmax*vmax);
    if (ge2 > 0.25*d*vmax*vmax) {
      dev_conserved[5*n_cells + id] = ge2;
      ge1 = ge2;
    }
    //else printf("%d Using ge1 %f %f %f %f\n", id, ge1, ge2, vmax, 0.25*d*vmax*vmax);
    */
    // update the total energy
     
    // recalculate the pressure 
    P = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);    
    if (P < 0.0) printf("%d Negative pressure after final update. %f %f \n", id, ge1, ge2);    
  }

}


__global__ void Calc_dt_1D(Real *dev_conserved, int n_cells, int n_ghost, Real dx, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // start timestep calculation here
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) TINY_NUMBER);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = (fabs(vx)+cs)/dx;
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];


}





#endif //CUDA
