#include "hip/hip_runtime.h"
/*!
 * \file pcm_cuda_tests.cu
 * \brief Contains the tests for the code in pcm_cuda.h and pcm_cuda.cu
 */

// STL Includes
#include <random>

// External Includes
#include <gtest/gtest.h>  // Include GoogleTest and related libraries/headers

// Local Includes
#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../io/io.h"
#include "../reconstruction/reconstruction.h"
#include "../utils/DeviceVector.h"
#include "../utils/basic_structs.h"
#include "../utils/testing_utilities.h"

template <int reconstruction, uint direction>
__global__ void Reconstruction_Runner(Real const *dev_grid, size_t const xid, size_t const yid, size_t const zid,
                                      size_t const nx, size_t const ny, size_t const n_cells, Real const gamma,
                                      reconstruction::InterfaceState *left_interface,
                                      reconstruction::InterfaceState *right_interface)
{
  reconstruction::Reconstruct_Interface_States<reconstruction, direction>(
      dev_grid, xid, yid, zid, nx, ny, n_cells, gamma, 0.0, 0.0, *left_interface, *right_interface);
}

TEST(tAllReconstructInterfaceStates, PcmCorrectInputExpectCorrectOutput)
{
  // Set up PRNG to use
  std::mt19937_64 prng(42);
  std::uniform_real_distribution<double> doubleRand(0.1, 5);

  // Mock up needed information
  size_t const nx      = 7;
  size_t const ny      = 7;
  size_t const nz      = 7;
  size_t const xid     = 3;
  size_t const yid     = 3;
  size_t const zid     = 3;
  size_t const n_cells = nx * ny * nz;
  double const dx      = doubleRand(prng);
  double const dt      = doubleRand(prng);
  double const gamma   = 5.0 / 3.0;

  // Setup host grid. Fill host grid with random values and randomly assign values
  std::vector<double> host_grid(n_cells * grid_enum::num_fields);
  for (Real &val : host_grid) {
    val = doubleRand(prng);
  }

  // Copy data to GPU
  cuda_utilities::DeviceVector<double> dev_grid(host_grid.size());
  dev_grid.cpyHostToDevice(host_grid);

  // Test each direction
  cuda_utilities::DeviceVector<reconstruction::InterfaceState> test_interface_pcm_l_0{1}, test_interface_pcm_r_0{1},
      test_interface_pcm_l_1{1}, test_interface_pcm_r_1{1}, test_interface_pcm_l_2{1}, test_interface_pcm_r_2{1};

  hipLaunchKernelGGL(HIP_KERNEL_NAME(Reconstruction_Runner<reconstruction::Kind::pcm, 0>), 1, 1, 0, 0, dev_grid.data(),
                     xid, yid, zid, nx, ny, n_cells, gamma, test_interface_pcm_l_0.data(),
                     test_interface_pcm_r_0.data());
  hipLaunchKernelGGL(HIP_KERNEL_NAME(Reconstruction_Runner<reconstruction::Kind::pcm, 1>), 1, 1, 0, 0, dev_grid.data(),
                     xid, yid, zid, nx, ny, n_cells, gamma, test_interface_pcm_l_1.data(),
                     test_interface_pcm_r_1.data());
  hipLaunchKernelGGL(HIP_KERNEL_NAME(Reconstruction_Runner<reconstruction::Kind::pcm, 2>), 1, 1, 0, 0, dev_grid.data(),
                     xid, yid, zid, nx, ny, n_cells, gamma, test_interface_pcm_l_2.data(),
                     test_interface_pcm_r_2.data());

  // Fiducial values
  reconstruction::InterfaceState fiducial_interface_pcm_l_0{
      1.6206985712721595,     {1.9275471960012214, 2.0380692774425846, 1.9771827902007457}, 4.5791453055608384,
      9.9999999999999995e-21, {4.1622274705137627, 2.1906071705977261, 3.1997462690190144}, 16.180636739137334},
      fiducial_interface_pcm_r_0{
          1.5162490166443841,     {0.74079082506491523, 1.4295471037207337, 0.49525487240256766}, 1.6382470722683291,
          9.9999999999999995e-21, {2.6539699941465473, 2.6775840565878508, 2.4794718891665037},   10.180396979545293},
      fiducial_interface_pcm_l_1{
          1.6206985712721595,     {2.0380692774425846, 1.9771827902007457, 1.9275471960012214}, 4.5791453055608384,
          9.9999999999999995e-21, {2.1906071705977261, 3.1997462690190144, 4.1622274705137627}, 16.180636739137338},
      fiducial_interface_pcm_r_1{
          3.8412847012400144,     {1.1260155024295584, 0.37985902941387084, 0.31356489904284668}, 4.1037970369599064,
          9.9999999999999995e-21, {2.7361340285756826, 4.5077114382460621, 3.2694920805403553},   19.247735148770136},
      fiducial_interface_pcm_l_2{
          1.6206985712721595,     {1.9771827902007457, 1.9275471960012214, 2.0380692774425846}, 4.5791453055608384,
          9.9999999999999995e-21, {3.1997462690190144, 4.1622274705137627, 2.1906071705977261}, 16.180636739137334},
      fiducial_interface_pcm_r_2{
          0.75619040256911529,    {4.3870709307030475, 0.53201818469160067, 3.0376042247856248}, 4.181424078824616,
          9.9999999999999995e-21, {3.0291890161755175, 3.4009589976457022, 2.1042141607876181},  12.585112716922399};

  // Check correctness
  testing_utilities::Check_Interface(test_interface_pcm_l_0.at(0), fiducial_interface_pcm_l_0, 0);
  testing_utilities::Check_Interface(test_interface_pcm_r_0.at(0), fiducial_interface_pcm_r_0, 0);
  testing_utilities::Check_Interface(test_interface_pcm_l_1.at(0), fiducial_interface_pcm_l_1, 0);
  testing_utilities::Check_Interface(test_interface_pcm_r_1.at(0), fiducial_interface_pcm_r_1, 0);
  testing_utilities::Check_Interface(test_interface_pcm_l_2.at(0), fiducial_interface_pcm_l_2, 0);
  testing_utilities::Check_Interface(test_interface_pcm_r_2.at(0), fiducial_interface_pcm_r_2, 0);
}

TEST(tAllReconstructInterfaceStates, PlmcCorrectInputExpectCorrectOutput)
{
  // Set up PRNG to use
  std::mt19937_64 prng(42);
  std::uniform_real_distribution<double> doubleRand(0.1, 5);

  // Mock up needed information
  size_t const nx      = 7;
  size_t const ny      = 7;
  size_t const nz      = 7;
  size_t const xid     = 3;
  size_t const yid     = 3;
  size_t const zid     = 3;
  size_t const n_cells = nx * ny * nz;
  double const dx      = doubleRand(prng);
  double const dt      = doubleRand(prng);
  double const gamma   = 5.0 / 3.0;

  // Setup host grid. Fill host grid with random values and randomly assign values
  std::vector<double> host_grid(n_cells * grid_enum::num_fields);
  for (Real &val : host_grid) {
    val = doubleRand(prng);
  }

  // Copy data to GPU
  cuda_utilities::DeviceVector<double> dev_grid(host_grid.size());
  dev_grid.cpyHostToDevice(host_grid);

  // Test each direction
  cuda_utilities::DeviceVector<reconstruction::InterfaceState> test_interface_pcm_l_0{1}, test_interface_pcm_r_0{1},
      test_interface_pcm_l_1{1}, test_interface_pcm_r_1{1}, test_interface_pcm_l_2{1}, test_interface_pcm_r_2{1};

  hipLaunchKernelGGL(HIP_KERNEL_NAME(Reconstruction_Runner<reconstruction::Kind::plmc, 0>), 1, 1, 0, 0, dev_grid.data(),
                     xid, yid, zid, nx, ny, n_cells, gamma, test_interface_pcm_l_0.data(),
                     test_interface_pcm_r_0.data());
  hipLaunchKernelGGL(HIP_KERNEL_NAME(Reconstruction_Runner<reconstruction::Kind::plmc, 1>), 1, 1, 0, 0, dev_grid.data(),
                     xid, yid, zid, nx, ny, n_cells, gamma, test_interface_pcm_l_1.data(),
                     test_interface_pcm_r_1.data());
  hipLaunchKernelGGL(HIP_KERNEL_NAME(Reconstruction_Runner<reconstruction::Kind::plmc, 2>), 1, 1, 0, 0, dev_grid.data(),
                     xid, yid, zid, nx, ny, n_cells, gamma, test_interface_pcm_l_2.data(),
                     test_interface_pcm_r_2.data());

  // Fiducial values
  reconstruction::InterfaceState

      fiducial_interface_pcm_l_0{1.5298689023089278,
                                 {1.9275471960012214, 2.0380692774425846, 1.9771827902007457},
                                 16.528284433518245,
                                 9.9999999999999995e-21,
                                 {0, 2.1906071705977261, 3.1997462690190144},
                                 7.5185679809876387},
      fiducial_interface_pcm_r_0{1.5162490166443841,
                                 {0.76563553460272882, 1.4295471037207337, 0.64944643555588033},
                                 8.9024945527419099,
                                 9.9999999999999995e-21,
                                 {0, 2.4592281743421247, 2.6702482307282183},
                                 6.5890144135926398},
      fiducial_interface_pcm_l_1{1.64438973366853,
                                 {1.9727493343641314, 1.9771827902007457, 1.9108297140717538},
                                 23.703295056609775,
                                 9.9999999999999995e-21,
                                 {0, 3.5385062482728076, 4.0066921167531317},
                                 14.287304093758696},
      fiducial_interface_pcm_r_1{3.5062733063981497,
                                 {1.2716820435726639, 0.37985902941387084, 0.31356489904284668},
                                 19.327079853814666,
                                 9.9999999999999995e-21,
                                 {0, 4.5077114382460621, 3.43711530062046},
                                 16.066612000126828},
      fiducial_interface_pcm_l_2{1.6206985712721595,
                                 {1.9771827902007457, 1.2463253265840999, 2.2261940114076708},
                                 19.318566303601784,
                                 9.9999999999999995e-21,
                                 {0, 4.1622274705137627, 2.1042141607876181},
                                 10.875927375379268},
      fiducial_interface_pcm_r_2{1.0495762526649557,
                                 {2.7223874400155843, 0.80996750866511347, 2.5883656102600812},
                                 18.185136420763868,
                                 9.9999999999999995e-21,
                                 {0, 4.0550440172866837, 2.1042141607876181},
                                 10.435549608295833};

  // Check correctness
  testing_utilities::Check_Interface(test_interface_pcm_l_0.at(0), fiducial_interface_pcm_l_0, 0);
  testing_utilities::Check_Interface(test_interface_pcm_r_0.at(0), fiducial_interface_pcm_r_0, 0);
  testing_utilities::Check_Interface(test_interface_pcm_l_1.at(0), fiducial_interface_pcm_l_1, 0);
  testing_utilities::Check_Interface(test_interface_pcm_r_1.at(0), fiducial_interface_pcm_r_1, 0);
  testing_utilities::Check_Interface(test_interface_pcm_l_2.at(0), fiducial_interface_pcm_l_2, 0);
  testing_utilities::Check_Interface(test_interface_pcm_r_2.at(0), fiducial_interface_pcm_r_2, 0);
}