#include "hip/hip_runtime.h"
/*! \file plmc_cuda.cu
 *  \brief Definitions of the piecewise linear reconstruction functions with
           limiting applied in the characteristic variables, as described
           in Stone et al., 2008. */

#include <math.h>

#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../reconstruction/plmc_cuda.h"
#include "../reconstruction/reconstruction.h"
#include "../utils/cuda_utilities.h"
#include "../utils/gpu.hpp"

#ifdef DE  // PRESSURE_DE
  #include "../utils/hydro_utilities.h"
#endif  // DE

/*! \fn __global__ void PLMC_cuda(Real *dev_conserved, Real *dev_bounds_L, Real
 *dev_bounds_R, int nx, int ny, int nz, Real dx, Real dt, Real
 gamma, int dir)
 *  \brief When passed a stencil of conserved variables, returns the left and
 right boundary values for the interface calculated using plm. */
__global__ __launch_bounds__(TPB) void PLMC_cuda(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx,
                                                 int ny, int nz, Real dx, Real dt, Real gamma, int dir, int n_fields)
{
  // get a thread ID
  int const thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(thread_id, nx, ny, xid, yid, zid);

  // Ensure that we are only operating on cells that will be used
  if (reconstruction::Thread_Guard<2>(nx, ny, nz, xid, yid, zid)) {
    return;
  }

  // Compute the total number of cells
  int const n_cells = nx * ny * nz;

  // Set the field indices for the various directions
  int o1, o2, o3;
  switch (dir) {
    case 0:
      o1 = grid_enum::momentum_x;
      o2 = grid_enum::momentum_y;
      o3 = grid_enum::momentum_z;
      break;
    case 1:
      o1 = grid_enum::momentum_y;
      o2 = grid_enum::momentum_z;
      o3 = grid_enum::momentum_x;
      break;
    case 2:
      o1 = grid_enum::momentum_z;
      o2 = grid_enum::momentum_x;
      o3 = grid_enum::momentum_y;
      break;
  }

  // load the 3-cell stencil into registers
  // cell i
  reconstruction::Primitive const cell_i =
      reconstruction::Load_Data(dev_conserved, xid, yid, zid, nx, ny, n_cells, o1, o2, o3, gamma);

  // cell i-1. The equality checks the direction and will subtract one from the correct direction
  reconstruction::Primitive const cell_imo = reconstruction::Load_Data(
      dev_conserved, xid - int(dir == 0), yid - int(dir == 1), zid - int(dir == 2), nx, ny, n_cells, o1, o2, o3, gamma);

  // cell i+1. The equality checks the direction and add one to the correct direction
  reconstruction::Primitive const cell_ipo = reconstruction::Load_Data(
      dev_conserved, xid + int(dir == 0), yid + int(dir == 1), zid + int(dir == 2), nx, ny, n_cells, o1, o2, o3, gamma);

  // calculate the adiabatic sound speed in cell i
  Real const sound_speed         = hydro_utilities::Calc_Sound_Speed(cell_i.pressure, cell_i.density, gamma);
  Real const sound_speed_squared = sound_speed * sound_speed;

// Compute the eigenvectors
#ifdef MHD
  reconstruction::EigenVecs const eigenvectors =
      reconstruction::Compute_Eigenvectors(cell_i, sound_speed, sound_speed_squared, gamma);
#else
  reconstruction::EigenVecs eigenvectors;
#endif  // MHD

  // Compute the left, right, centered, and van Leer differences of the
  // primitive variables Note that here L and R refer to locations relative to
  // the cell center

  // left
  reconstruction::Primitive const del_L = reconstruction::Compute_Slope(cell_imo, cell_i);

  // right
  reconstruction::Primitive const del_R = reconstruction::Compute_Slope(cell_i, cell_ipo);

  // centered
  reconstruction::Primitive const del_C = reconstruction::Compute_Slope(cell_imo, cell_ipo, 0.5);

  // Van Leer
  reconstruction::Primitive const del_G = reconstruction::Van_Leer_Slope(del_L, del_R);

  // Project the left, right, centered and van Leer differences onto the
  // characteristic variables Stone Eqn 37 (del_a are differences in
  // characteristic variables, see Stone for notation) Use the eigenvectors
  // given in Stone 2008, Appendix A
  reconstruction::Characteristic const del_a_L =
      reconstruction::Primitive_To_Characteristic(cell_i, del_L, eigenvectors, sound_speed, sound_speed_squared, gamma);

  reconstruction::Characteristic const del_a_R =
      reconstruction::Primitive_To_Characteristic(cell_i, del_R, eigenvectors, sound_speed, sound_speed_squared, gamma);

  reconstruction::Characteristic const del_a_C =
      reconstruction::Primitive_To_Characteristic(cell_i, del_C, eigenvectors, sound_speed, sound_speed_squared, gamma);

  reconstruction::Characteristic const del_a_G =
      reconstruction::Primitive_To_Characteristic(cell_i, del_G, eigenvectors, sound_speed, sound_speed_squared, gamma);

  // Apply monotonicity constraints to the differences in the characteristic variables and project the monotonized
  // difference in the characteristic variables back onto the primitive variables Stone Eqn 39
  reconstruction::Primitive del_m_i = reconstruction::Monotonize_Characteristic_Return_Primitive(
      cell_i, del_L, del_R, del_C, del_G, del_a_L, del_a_R, del_a_C, del_a_G, eigenvectors, sound_speed,
      sound_speed_squared, gamma);

  // Compute the left and right interface values using the monotonized difference in the primitive variables
  reconstruction::Primitive interface_L_iph = reconstruction::Calc_Interface_Linear(cell_i, del_m_i, 1.0);
  reconstruction::Primitive interface_R_imh = reconstruction::Calc_Interface_Linear(cell_i, del_m_i, -1.0);

  // Limit the interfaces
  reconstruction::Plm_Limit_Interfaces(interface_L_iph, interface_R_imh, cell_imo, cell_i, cell_ipo);

#ifndef VL

  Real const dtodx = dt / dx;

  // Compute the eigenvalues of the linearized equations in the
  // primitive variables using the cell-centered primitive variables
  Real const lambda_m = cell_i.velocity_x - sound_speed;
  Real const lambda_0 = cell_i.velocity_x;
  Real const lambda_p = cell_i.velocity_x + sound_speed;

  // Integrate linear interpolation function over domain of dependence
  // defined by max(min) eigenvalue
  Real qx                    = -0.5 * fmin(lambda_m, 0.0) * dtodx;
  interface_R_imh.density    = interface_R_imh.density + qx * del_m_i.density;
  interface_R_imh.velocity_x = interface_R_imh.velocity_x + qx * del_m_i.velocity_x;
  interface_R_imh.velocity_y = interface_R_imh.velocity_y + qx * del_m_i.velocity_y;
  interface_R_imh.velocity_z = interface_R_imh.velocity_z + qx * del_m_i.velocity_z;
  interface_R_imh.pressure   = interface_R_imh.pressure + qx * del_m_i.pressure;

  qx                         = 0.5 * fmax(lambda_p, 0.0) * dtodx;
  interface_L_iph.density    = interface_L_iph.density - qx * del_m_i.density;
  interface_L_iph.velocity_x = interface_L_iph.velocity_x - qx * del_m_i.velocity_x;
  interface_L_iph.velocity_y = interface_L_iph.velocity_y - qx * del_m_i.velocity_y;
  interface_L_iph.velocity_z = interface_L_iph.velocity_z - qx * del_m_i.velocity_z;
  interface_L_iph.pressure   = interface_L_iph.pressure - qx * del_m_i.pressure;

  #ifdef DE
  interface_R_imh.gas_energy = interface_R_imh.gas_energy + qx * del_m_i.gas_energy;
  interface_L_iph.gas_energy = interface_L_iph.gas_energy - qx * del_m_i.gas_energy;
  #endif  // DE

  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_R_imh.scalar[i] = interface_R_imh.scalar[i] + qx * del_m_i.scalar[i];
    interface_L_iph.scalar[i] = interface_L_iph.scalar[i] - qx * del_m_i.scalar[i];
  }
  #endif  // SCALAR

  // Perform the characteristic tracing
  // Stone Eqns 42 & 43

  // left-hand interface value, i+1/2
  Real sum_0 = 0.0, sum_1 = 0.0, sum_2 = 0.0, sum_3 = 0.0, sum_4 = 0.0;
  #ifdef DE
  Real sum_ge = 0;
  #endif  // DE
  #ifdef SCALAR
  Real sum_scalar[NSCALARS];
  for (int i = 0; i < NSCALARS; i++) {
    sum_scalar[i] = 0.0;
  }
  #endif  // SCALAR
  if (lambda_m >= 0) {
    Real lamdiff = lambda_p - lambda_m;

    sum_0 += lamdiff *
             (-cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 - del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (-cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }
  if (lambda_0 >= 0) {
    Real lamdiff = lambda_p - lambda_0;

    sum_0 += lamdiff * (del_m_i.density - del_m_i.pressure / (sound_speed_squared));
    sum_2 += lamdiff * del_m_i.velocity_y;
    sum_3 += lamdiff * del_m_i.velocity_z;
  #ifdef DE
    sum_ge += lamdiff * del_m_i.gas_energy;
  #endif  // DE
  #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      sum_scalar[i] += lamdiff * del_m_i.scalar[i];
    }
  #endif  // SCALAR
  }
  if (lambda_p >= 0) {
    Real lamdiff = lambda_p - lambda_p;

    sum_0 += lamdiff *
             (cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 + del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }

  // add the corrections to the initial guesses for the interface values
  interface_L_iph.density += 0.5 * dtodx * sum_0;
  interface_L_iph.velocity_x += 0.5 * dtodx * sum_1;
  interface_L_iph.velocity_y += 0.5 * dtodx * sum_2;
  interface_L_iph.velocity_z += 0.5 * dtodx * sum_3;
  interface_L_iph.pressure += 0.5 * dtodx * sum_4;
  #ifdef DE
  interface_L_iph.gas_energy += 0.5 * dtodx * sum_ge;
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_L_iph.scalar[i] += 0.5 * dtodx * sum_scalar[i];
  }
  #endif  // SCALAR

  // right-hand interface value, i-1/2
  sum_0 = sum_1 = sum_2 = sum_3 = sum_4 = 0;
  #ifdef DE
  sum_ge = 0;
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    sum_scalar[i] = 0;
  }
  #endif  // SCALAR
  if (lambda_m <= 0) {
    Real lamdiff = lambda_m - lambda_m;

    sum_0 += lamdiff *
             (-cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 - del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (-cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }
  if (lambda_0 <= 0) {
    Real lamdiff = lambda_m - lambda_0;

    sum_0 += lamdiff * (del_m_i.density - del_m_i.pressure / (sound_speed_squared));
    sum_2 += lamdiff * del_m_i.velocity_y;
    sum_3 += lamdiff * del_m_i.velocity_z;
  #ifdef DE
    sum_ge += lamdiff * del_m_i.gas_energy;
  #endif  // DE
  #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      sum_scalar[i] += lamdiff * del_m_i.scalar[i];
    }
  #endif  // SCALAR
  }
  if (lambda_p <= 0) {
    Real lamdiff = lambda_m - lambda_p;

    sum_0 += lamdiff *
             (cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 + del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }

  // add the corrections
  interface_R_imh.density += 0.5 * dtodx * sum_0;
  interface_R_imh.velocity_x += 0.5 * dtodx * sum_1;
  interface_R_imh.velocity_y += 0.5 * dtodx * sum_2;
  interface_R_imh.velocity_z += 0.5 * dtodx * sum_3;
  interface_R_imh.pressure += 0.5 * dtodx * sum_4;
  #ifdef DE
  interface_R_imh.gas_energy += 0.5 * dtodx * sum_ge;
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_R_imh.scalar[i] += 0.5 * dtodx * sum_scalar[i];
  }
  #endif  // SCALAR
#endif    // CTU

  // apply minimum constraints
  interface_R_imh.density  = fmax(interface_R_imh.density, (Real)TINY_NUMBER);
  interface_L_iph.density  = fmax(interface_L_iph.density, (Real)TINY_NUMBER);
  interface_R_imh.pressure = fmax(interface_R_imh.pressure, (Real)TINY_NUMBER);
  interface_L_iph.pressure = fmax(interface_L_iph.pressure, (Real)TINY_NUMBER);

  // Convert the left and right states in the primitive to the conserved variables send final values back from kernel
  // bounds_R refers to the right side of the i-1/2 interface
  size_t id = cuda_utilities::compute1DIndex(xid, yid, zid, nx, ny);
  reconstruction::Write_Data(interface_L_iph, dev_bounds_L, dev_conserved, id, n_cells, o1, o2, o3, gamma);

  id = cuda_utilities::compute1DIndex(xid - int(dir == 0), yid - int(dir == 1), zid - int(dir == 2), nx, ny);
  reconstruction::Write_Data(interface_R_imh, dev_bounds_R, dev_conserved, id, n_cells, o1, o2, o3, gamma);
}
