/*!
 * \file plmc_cuda_tests.cu
 * \brief Tests for the contents of plmc_cuda.h and plmc_cuda.cu
 *
 */

// STL Includes
#include <random>
#include <string>
#include <unordered_map>
#include <vector>

// External Includes
#include <gtest/gtest.h>  // Include GoogleTest and related libraries/headers

// Local Includes
#include <algorithm>

#include "../global/global.h"
#include "../io/io.h"
#include "../reconstruction/plmc_cuda.h"
#include "../utils/DeviceVector.h"
#include "../utils/hydro_utilities.h"
#include "../utils/testing_utilities.h"

TEST(tHYDROPlmcReconstructor, CorrectInputExpectCorrectOutput)
{
#ifndef VL
  std::cerr << "Warning: The tHYDROPlmcReconstructor.CorrectInputExpectCorrectOutput only supports the Van Leer (VL) "
               "integrator"
            << std::endl;
  return;
#endif  // VL
  // Set up PRNG to use
  std::mt19937_64 prng(42);
  std::uniform_real_distribution<double> doubleRand(0.1, 5);

  // Mock up needed information
  size_t const nx       = 5;
  size_t const ny       = 4;
  size_t const nz       = 4;
  size_t const n_fields = 5;
  double const dx       = doubleRand(prng);
  double const dt       = doubleRand(prng);
  double const gamma    = 5.0 / 3.0;

  // Setup host grid. Fill host grid with random values and randomly assign maximum value
  std::vector<double> host_grid(nx * ny * nz * n_fields);
  for (Real &val : host_grid) {
    val = doubleRand(prng);
  }

  // Allocating and copying to device
  cuda_utilities::DeviceVector<double> dev_grid(host_grid.size());
  dev_grid.cpyHostToDevice(host_grid);

  // Fiducial Data
  std::vector<std::unordered_map<int, double>> fiducial_interface_left = {{{26, 3.8877922383184833},
                                                                           {27, 0.70033864721549188},
                                                                           {106, 5.6625525038177784},
                                                                           {107, 3.0633780053857027},
                                                                           {186, 4.0069556576401011},
                                                                           {187, 2.1015872413794123},
                                                                           {266, 5.1729859852329314},
                                                                           {267, 3.9675148506537838},
                                                                           {346, 9.6301414677176531},
                                                                           {347, 21.091316282933843}},
                                                                          {{21, 0.74780807318015607},
                                                                           {37, 0.19457128219588618},
                                                                           {101, 5.6515522777659895},
                                                                           {117, 4.4286255636679313},
                                                                           {181, 0.13115998072061905},
                                                                           {197, 2.2851440769830953},
                                                                           {261, 1.5834637771067519},
                                                                           {277, 2.697375839048191},
                                                                           {341, 23.043749364531674},
                                                                           {357, 82.515887983144168}},
                                                                          {{25, 2.2863650183226212},
                                                                           {29, 1.686415421301841},
                                                                           {105, 0.72340346106443465},
                                                                           {109, 5.9563546443402542},
                                                                           {185, 3.6128571662018358},
                                                                           {189, 5.3735653401079038},
                                                                           {265, 0.95177493689267167},
                                                                           {269, 0.46056494878491938},
                                                                           {345, 3.1670194578067843},
                                                                           {349, 19.142817472509272}}};

  std::vector<std::unordered_map<int, double>> fiducial_interface_right =

      {{{25, 3.8877922383184833},
        {26, 0.70033864721549188},
        {105, 1.594778794367564},
        {106, 3.0633780053857027},
        {185, 4.0069556576401011},
        {186, 2.1015872413794123},
        {265, 1.7883678016935782},
        {266, 3.9675148506537838},
        {345, 2.8032969746372531},
        {346, 21.091316282933843}},
       {{17, 0.43265217076853835},
        {33, 0.19457128219588618},
        {97, 3.2697645945288754},
        {113, 4.4286255636679313},
        {177, 0.07588397666718491},
        {193, 2.2851440769830953},
        {257, 0.91612950577699748},
        {273, 2.697375839048191},
        {337, 13.332201861384396},
        {353, 82.515887983144168}},
       {{5, 2.2863650183226212},
        {9, 1.686415421301841},
        {85, 0.72340346106443465},
        {89, 1.77925054463361},
        {165, 5.3997753452111859},
        {169, 1.4379190463124141},
        {245, 0.95177493689267167},
        {249, 0.46056494878491938},
        {325, 6.6889498465051398},
        {329, 1.6145084086614285}}}

  ;

  // Loop over different directions
  for (size_t direction = 0; direction < 3; direction++) {
    // Assign the shape
    size_t nx_rot, ny_rot, nz_rot;
    switch (direction) {
      case 0:
        nx_rot = nx;
        ny_rot = ny;
        nz_rot = nz;
        break;
      case 1:
        nx_rot = ny;
        ny_rot = nz;
        nz_rot = nx;
        break;
      case 2:
        nx_rot = nz;
        ny_rot = nx;
        nz_rot = ny;
        break;
    }

    // Allocate device buffers
    cuda_utilities::DeviceVector<double> dev_interface_left(host_grid.size(), true);
    cuda_utilities::DeviceVector<double> dev_interface_right(host_grid.size(), true);

    // Launch kernel
    switch (direction) {
      case 0:
        hipLaunchKernelGGL(PLMC_cuda<0>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx_rot, ny_rot, nz_rot, dx, dt, gamma, n_fields);
        break;
      case 1:
        hipLaunchKernelGGL(PLMC_cuda<1>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx_rot, ny_rot, nz_rot, dx, dt, gamma, n_fields);
        break;
      case 2:
        hipLaunchKernelGGL(PLMC_cuda<2>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx_rot, ny_rot, nz_rot, dx, dt, gamma, n_fields);
        break;
    }
    GPU_Error_Check();
    GPU_Error_Check(hipDeviceSynchronize());

    // Perform Comparison
    for (size_t i = 0; i < host_grid.size(); i++) {
      // Check the left interface
      double test_val = dev_interface_left.at(i);
      double fiducial_val =
          (fiducial_interface_left.at(direction).find(i) == fiducial_interface_left.at(direction).end())
              ? 0.0
              : fiducial_interface_left.at(direction)[i];

      testing_utilities::Check_Results(
          fiducial_val, test_val,
          "left interface at i=" + std::to_string(i) + ", in direction " + std::to_string(direction));

      // Check the right interface
      test_val     = dev_interface_right.at(i);
      fiducial_val = (fiducial_interface_right.at(direction).find(i) == fiducial_interface_right.at(direction).end())
                         ? 0.0
                         : fiducial_interface_right.at(direction)[i];

      // if (test_val != 0.0) std::cout << "{" << i << ", " << to_string_exact(test_val) << "}," << std::endl;

      testing_utilities::Check_Results(
          fiducial_val, test_val,
          "right interface at i=" + std::to_string(i) + ", in direction " + std::to_string(direction));
    }
  }
}

TEST(tMHDPlmcReconstructor, CorrectInputExpectCorrectOutput)
{
  // Set up PRNG to use
  std::mt19937_64 prng(42);
  std::uniform_real_distribution<double> doubleRand(0.1, 5);

  // Mock up needed information
  size_t const nx = 4, ny = nx, nz = nx;
  size_t const n_fields          = 8;
  size_t const n_cells_grid      = nx * ny * nz * n_fields;
  size_t const n_cells_interface = nx * ny * nz * (n_fields - 1);
  double const dx                = doubleRand(prng);
  double const dt                = doubleRand(prng);
  double const gamma             = 5.0 / 3.0;

  // Setup host grid. Fill host grid with random values and randomly assign maximum value
  std::vector<double> host_grid(n_cells_grid);
  for (Real &val : host_grid) {
    val = doubleRand(prng);
  }

  // Allocating and copying to device
  cuda_utilities::DeviceVector<double> dev_grid(host_grid.size());
  dev_grid.cpyHostToDevice(host_grid);

  // Fiducial Data
  std::vector<std::unordered_map<int, double>> fiducial_interface_left  = {{{21, 0.59023012197434721},
                                                                            {85, 3.0043379408547275},
                                                                            {149, 2.6320759184913625},
                                                                            {213, 0.9487867623146744},
                                                                            {277, 18.551193003661723},
                                                                            {341, 1.8587936590169301},
                                                                            {405, 2.1583975283044725}},
                                                                           {{21, 0.73640639402573249},
                                                                            {85, 3.3462413154443715},
                                                                            {149, 2.1945584994458125},
                                                                            {213, 0.67418839414138987},
                                                                            {277, 16.909618487528142},
                                                                            {341, 2.1533768050263267},
                                                                            {405, 1.6994195863331925}},
                                                                           {{21, 0.25340904981266843},
                                                                            {85, 2.0441984720128734},
                                                                            {149, 1.9959059157695584},
                                                                            {213, 0.45377591914009824},
                                                                            {277, 23.677832869261188},
                                                                            {341, 1.5437923271692418},
                                                                            {405, 1.8141353672443383}}};
  std::vector<std::unordered_map<int, double>> fiducial_interface_right = {{{20, 0.59023012197434721},
                                                                            {84, 3.0043379408547275},
                                                                            {148, 2.6320759184913625},
                                                                            {212, 0.9487867623146744},
                                                                            {276, 22.111134849009044},
                                                                            {340, 1.8587936590169301},
                                                                            {404, 2.1583975283044725}},
                                                                           {
                                                                               {17, 0.44405384992296193},
                                                                               {81, 2.5027813113931279},
                                                                               {145, 2.6371119205792346},
                                                                               {209, 1.0210845222961809},
                                                                               {273, 21.360010722689488},
                                                                               {337, 2.1634182515826184},
                                                                               {401, 1.7073441775673177},
                                                                           },
                                                                           {
                                                                               {5, 0.92705119413602599},
                                                                               {69, 1.9592598982258778},
                                                                               {133, 0.96653490574340428},
                                                                               {197, 1.3203867992383289},
                                                                               {261, 8.0057564947791793},
                                                                               {325, 1.8629714367312684},
                                                                               {389, 1.9034519507895218},
                                                                           }};

  // Loop over different directions
  for (size_t direction = 0; direction < 3; direction++) {
    // Allocate device buffers
    cuda_utilities::DeviceVector<double> dev_interface_left(n_cells_interface, true);
    cuda_utilities::DeviceVector<double> dev_interface_right(n_cells_interface, true);

    // Launch kernel
    switch (direction) {
      case 0:
        hipLaunchKernelGGL(PLMC_cuda<0>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx, ny, nz, dx, dt, gamma, n_fields);
        break;
      case 1:
        hipLaunchKernelGGL(PLMC_cuda<1>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx, ny, nz, dx, dt, gamma, n_fields);
        break;
      case 2:
        hipLaunchKernelGGL(PLMC_cuda<2>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx, ny, nz, dx, dt, gamma, n_fields);
        break;
    }
    GPU_Error_Check();
    GPU_Error_Check(hipDeviceSynchronize());

    // Perform Comparison
    for (size_t i = 0; i < dev_interface_right.size(); i++) {
      // Check the left interface
      double test_val = dev_interface_left.at(i);
      double fiducial_val =
          (fiducial_interface_left.at(direction).find(i) == fiducial_interface_left.at(direction).end())
              ? 0.0
              : fiducial_interface_left.at(direction)[i];

      testing_utilities::Check_Results(
          fiducial_val, test_val,
          "left interface at i=" + std::to_string(i) + ", in direction " + std::to_string(direction));

      // Check the right interface
      test_val     = dev_interface_right.at(i);
      fiducial_val = (fiducial_interface_right.at(direction).find(i) == fiducial_interface_right.at(direction).end())
                         ? 0.0
                         : fiducial_interface_right.at(direction)[i];

      testing_utilities::Check_Results(
          fiducial_val, test_val,
          "right interface at i=" + std::to_string(i) + ", in direction " + std::to_string(direction));
    }
  }
}
