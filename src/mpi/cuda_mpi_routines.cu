#include<stdio.h>
#include"gpu.hpp"
#include"io.h"
#include"cuda_mpi_routines.h"

// #define PRINT_DEVICE_IDS

/*! \fn int initialize_cuda_mpi(int myid, int nprocs);
 *  \brief CUDA initialization within MPI. */
int initialize_cuda_mpi(int myid, int nprocs)
{
  int i_device = 0;   //GPU device for this process
  int n_device;   //number of GPU devices available

  hipError_t flag_error;

  //get the number of cuda devices    
  flag_error = hipGetDeviceCount(&n_device);

  //check for errors
  if(flag_error!=hipSuccess)
  {
    if(flag_error==hipErrorNoDevice)
      fprintf(stderr,"hipGetDeviceCount: Error! for myid = %d and n_device = %d; hipErrorNoDevice\n",myid,n_device);
    if(flag_error==hipErrorInsufficientDriver)
      fprintf(stderr,"hipGetDeviceCount: Error! for myid = %d and n_device = %d; hipErrorInsufficientDriver\n",myid,n_device);
    fflush(stderr);
    return 1;
  }
  
  //get host name
  char pname[MPI_MAX_PROCESSOR_NAME];     //node hostname
  int  pname_length;          //length of node hostname
  MPI_Get_processor_name(pname, &pname_length);

  //set a cuda device for each process
  hipSetDevice(myid%n_device);

  //double check
  hipGetDevice(&i_device);

  #ifdef PRINT_DEVICE_IDS
  printf("In initialize_cuda_mpi: name:%s myid = %d, i_device = %d, n_device = %d\n",pname, myid,i_device,n_device);
  fflush(stdout);
  MPI_Barrier(world);
  #endif
  
  return 0;
    
}
