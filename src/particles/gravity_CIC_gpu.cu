
#include <hip/hip_runtime.h>
#ifdef PARTICLES

  #include <math.h>
  #include <stdio.h>
  #include <stdlib.h>
  #include <unistd.h>

  #include "../global/global.h"
  #include "../global/global_cuda.h"
  #include "../utils/gpu.hpp"
  #include "particles_3D.h"

  #ifdef GRAVITY_GPU
    #include "../grid/grid3D.h"
  #endif

  #ifdef PARTICLES_GPU

// Copy the potential from host to device
void Particles3D::Copy_Potential_To_GPU(Real *potential_host, Real *potential_dev, int n_cells_potential)
{
  GPU_Error_Check(cudaMemcpy(potential_dev, potential_host, n_cells_potential * sizeof(Real), cudaMemcpyHostToDevice));
}

// Kernel to compute the gradient of the potential
__global__ void Get_Gravity_Field_Particles_Kernel(Real *potential_dev, Real *gravity_x_dev, Real *gravity_y_dev,
                                                   Real *gravity_z_dev, int nx, int ny, int nz,
                                                   int n_ghost_particles_grid, int n_ghost_potential, Real dx, Real dy,
                                                   Real dz)
{
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
  int tid_z = blockIdx.z * blockDim.z + threadIdx.z;

  int nx_grav, ny_grav, nz_grav;
  nx_grav = nx + 2 * n_ghost_particles_grid;
  ny_grav = ny + 2 * n_ghost_particles_grid;
  nz_grav = nz + 2 * n_ghost_particles_grid;

  if (tid_x >= nx_grav || tid_y >= ny_grav || tid_z >= nz_grav) {
    return;
  }
  int tid = tid_x + tid_y * nx_grav + tid_z * nx_grav * ny_grav;

  int nx_pot, ny_pot;
  nx_pot = nx + 2 * n_ghost_potential;
  ny_pot = ny + 2 * n_ghost_potential;

  int nGHST = n_ghost_potential - n_ghost_particles_grid;

  Real phi_l, phi_r;
  int id_l, id_r;
    #ifdef GRAVITY_5_POINTS_GRADIENT
  Real phi_ll, phi_rr;
  int id_ll, id_rr;
    #endif

  // Get Potential Gradient X
  id_l  = (tid_x - 1 + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  id_r  = (tid_x + 1 + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  phi_l = potential_dev[id_l];
  phi_r = potential_dev[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
  id_ll              = (tid_x - 2 + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  id_rr              = (tid_x + 2 + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  phi_ll             = potential_dev[id_ll];
  phi_rr             = potential_dev[id_rr];
  gravity_x_dev[tid] = -1 * (-phi_rr + 8 * phi_r - 8 * phi_l + phi_ll) / (12 * dx);
    #else
  gravity_x_dev[tid] = -0.5 * (phi_r - phi_l) / dx;
    #endif

  // Get Potential Gradient Y
  id_l  = (tid_x + nGHST) + (tid_y - 1 + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  id_r  = (tid_x + nGHST) + (tid_y + 1 + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  phi_l = potential_dev[id_l];
  phi_r = potential_dev[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
  id_ll              = (tid_x + nGHST) + (tid_y - 2 + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  id_rr              = (tid_x + nGHST) + (tid_y + 2 + nGHST) * nx_pot + (tid_z + nGHST) * ny_pot * nx_pot;
  phi_ll             = potential_dev[id_ll];
  phi_rr             = potential_dev[id_rr];
  gravity_y_dev[tid] = -1 * (-phi_rr + 8 * phi_r - 8 * phi_l + phi_ll) / (12 * dy);
    #else
  gravity_y_dev[tid] = -0.5 * (phi_r - phi_l) / dy;
    #endif

  // Get Potential Gradient Z
  id_l  = (tid_x + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z - 1 + nGHST) * ny_pot * nx_pot;
  id_r  = (tid_x + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z + 1 + nGHST) * ny_pot * nx_pot;
  phi_l = potential_dev[id_l];
  phi_r = potential_dev[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
  id_ll              = (tid_x + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z - 2 + nGHST) * ny_pot * nx_pot;
  id_rr              = (tid_x + nGHST) + (tid_y + nGHST) * nx_pot + (tid_z + 2 + nGHST) * ny_pot * nx_pot;
  phi_ll             = potential_dev[id_ll];
  phi_rr             = potential_dev[id_rr];
  gravity_z_dev[tid] = -1 * (-phi_rr + 8 * phi_r - 8 * phi_l + phi_ll) / (12 * dz);
    #else
  gravity_z_dev[tid] = -0.5 * (phi_r - phi_l) / dz;
    #endif
}

// Call the kernel to compute the gradient of the potential
void Particles3D::Get_Gravity_Field_Particles_GPU_function(int nx_local, int ny_local, int nz_local,
                                                           int n_ghost_particles_grid, int n_cells_potential, Real dx,
                                                           Real dy, Real dz, Real *potential_host, Real *potential_dev,
                                                           Real *gravity_x_dev, Real *gravity_y_dev,
                                                           Real *gravity_z_dev)
{
    #ifndef GRAVITY_GPU
  Copy_Potential_To_GPU(potential_host, potential_dev, n_cells_potential);
    #endif

  int nx_g, ny_g, nz_g;
  nx_g = nx_local + 2 * N_GHOST_POTENTIAL;
  ny_g = ny_local + 2 * N_GHOST_POTENTIAL;
  nz_g = nz_local + 2 * N_GHOST_POTENTIAL;

  // set values for GPU kernels
  int tpb_x   = 8;
  int tpb_y   = 8;
  int tpb_z   = 8;
  int ngrid_x = (nx_g + tpb_x - 1) / tpb_x;
  int ngrid_y = (ny_g + tpb_y - 1) / tpb_y;
  int ngrid_z = (nz_g + tpb_z - 1) / tpb_z;
  // number of blocks per 1D grid
  dim3 dim3dGrid(ngrid_x, ngrid_y, ngrid_z);
  //  number of threads per 1D block
  dim3 dim3dBlock(tpb_x, tpb_y, tpb_z);

  hipLaunchKernelGGL(Get_Gravity_Field_Particles_Kernel, dim3dGrid, dim3dBlock, 0, 0, potential_dev, gravity_x_dev,
                     gravity_y_dev, gravity_z_dev, nx_local, ny_local, nz_local, n_ghost_particles_grid,
                     N_GHOST_POTENTIAL, dx, dy, dz);
  GPU_Error_Check();
}

// Get CIC indexes from the particles positions
__device__ void Get_Indexes_CIC_Gravity(Real xMin, Real yMin, Real zMin, Real dx, Real dy, Real dz, Real pos_x,
                                        Real pos_y, Real pos_z, int &indx_x, int &indx_y, int &indx_z)
{
  indx_x = (int)floor((pos_x - xMin - 0.5 * dx) / dx);
  indx_y = (int)floor((pos_y - yMin - 0.5 * dy) / dy);
  indx_z = (int)floor((pos_z - zMin - 0.5 * dz) / dz);
}

// Kernel to compute the gravitational field at the particles positions via
// Cloud-In-Cell
__global__ void Get_Gravity_CIC_Kernel(part_int_t n_local, Real *gravity_x_dev, Real *gravity_y_dev,
                                       Real *gravity_z_dev, Real *pos_x_dev, Real *pos_y_dev, Real *pos_z_dev,
                                       Real *grav_x_dev, Real *grav_y_dev, Real *grav_z_dev, Real xMin, Real yMin,
                                       Real zMin, Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx,
                                       int ny, int nz, int n_ghost)
{
  part_int_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= n_local) {
    return;
  }

  int nx_g, ny_g;
  nx_g = nx + 2 * n_ghost;
  ny_g = ny + 2 * n_ghost;

  Real pos_x, pos_y, pos_z;
  Real cell_center_x, cell_center_y, cell_center_z;
  Real delta_x, delta_y, delta_z;
  Real g_x_bl, g_x_br, g_x_bu, g_x_bru, g_x_tl, g_x_tr, g_x_tu, g_x_tru;
  Real g_y_bl, g_y_br, g_y_bu, g_y_bru, g_y_tl, g_y_tr, g_y_tu, g_y_tru;
  Real g_z_bl, g_z_br, g_z_bu, g_z_bru, g_z_tl, g_z_tr, g_z_tu, g_z_tru;
  Real g_x, g_y, g_z;

  pos_x = pos_x_dev[tid];
  pos_y = pos_y_dev[tid];
  pos_z = pos_z_dev[tid];

  int indx_x, indx_y, indx_z, indx;
  Get_Indexes_CIC_Gravity(xMin, yMin, zMin, dx, dy, dz, pos_x, pos_y, pos_z, indx_x, indx_y, indx_z);

  bool in_local = true;

  if (pos_x < xMin || pos_x >= xMax) {
    in_local = false;
  }
  if (pos_y < yMin || pos_y >= yMax) {
    in_local = false;
  }
  if (pos_z < zMin || pos_z >= zMax) {
    in_local = false;
  }
  if (!in_local) {
    printf(" Gravity CIC Error: Particle outside local domain");
    return;
  }

  cell_center_x = xMin + indx_x * dx + 0.5 * dx;
  cell_center_y = yMin + indx_y * dy + 0.5 * dy;
  cell_center_z = zMin + indx_z * dz + 0.5 * dz;
  delta_x       = 1 - (pos_x - cell_center_x) / dx;
  delta_y       = 1 - (pos_y - cell_center_y) / dy;
  delta_z       = 1 - (pos_z - cell_center_z) / dz;
  indx_x += n_ghost;
  indx_y += n_ghost;
  indx_z += n_ghost;

  indx   = indx_x + indx_y * nx_g + indx_z * nx_g * ny_g;
  g_x_bl = gravity_x_dev[indx];
  g_y_bl = gravity_y_dev[indx];
  g_z_bl = gravity_z_dev[indx];

  indx   = (indx_x + 1) + (indx_y)*nx_g + (indx_z)*nx_g * ny_g;
  g_x_br = gravity_x_dev[indx];
  g_y_br = gravity_y_dev[indx];
  g_z_br = gravity_z_dev[indx];

  indx   = (indx_x) + (indx_y + 1) * nx_g + (indx_z)*nx_g * ny_g;
  g_x_bu = gravity_x_dev[indx];
  g_y_bu = gravity_y_dev[indx];
  g_z_bu = gravity_z_dev[indx];

  indx    = (indx_x + 1) + (indx_y + 1) * nx_g + (indx_z)*nx_g * ny_g;
  g_x_bru = gravity_x_dev[indx];
  g_y_bru = gravity_y_dev[indx];
  g_z_bru = gravity_z_dev[indx];

  indx   = (indx_x) + (indx_y)*nx_g + (indx_z + 1) * nx_g * ny_g;
  g_x_tl = gravity_x_dev[indx];
  g_y_tl = gravity_y_dev[indx];
  g_z_tl = gravity_z_dev[indx];

  indx   = (indx_x + 1) + (indx_y)*nx_g + (indx_z + 1) * nx_g * ny_g;
  g_x_tr = gravity_x_dev[indx];
  g_y_tr = gravity_y_dev[indx];
  g_z_tr = gravity_z_dev[indx];

  indx   = (indx_x) + (indx_y + 1) * nx_g + (indx_z + 1) * nx_g * ny_g;
  g_x_tu = gravity_x_dev[indx];
  g_y_tu = gravity_y_dev[indx];
  g_z_tu = gravity_z_dev[indx];

  indx    = (indx_x + 1) + (indx_y + 1) * nx_g + (indx_z + 1) * nx_g * ny_g;
  g_x_tru = gravity_x_dev[indx];
  g_y_tru = gravity_y_dev[indx];
  g_z_tru = gravity_z_dev[indx];

  g_x = g_x_bl * (delta_x) * (delta_y) * (delta_z) + g_x_br * (1 - delta_x) * (delta_y) * (delta_z) +
        g_x_bu * (delta_x) * (1 - delta_y) * (delta_z) + g_x_bru * (1 - delta_x) * (1 - delta_y) * (delta_z) +
        g_x_tl * (delta_x) * (delta_y) * (1 - delta_z) + g_x_tr * (1 - delta_x) * (delta_y) * (1 - delta_z) +
        g_x_tu * (delta_x) * (1 - delta_y) * (1 - delta_z) + g_x_tru * (1 - delta_x) * (1 - delta_y) * (1 - delta_z);

  g_y = g_y_bl * (delta_x) * (delta_y) * (delta_z) + g_y_br * (1 - delta_x) * (delta_y) * (delta_z) +
        g_y_bu * (delta_x) * (1 - delta_y) * (delta_z) + g_y_bru * (1 - delta_x) * (1 - delta_y) * (delta_z) +
        g_y_tl * (delta_x) * (delta_y) * (1 - delta_z) + g_y_tr * (1 - delta_x) * (delta_y) * (1 - delta_z) +
        g_y_tu * (delta_x) * (1 - delta_y) * (1 - delta_z) + g_y_tru * (1 - delta_x) * (1 - delta_y) * (1 - delta_z);

  g_z = g_z_bl * (delta_x) * (delta_y) * (delta_z) + g_z_br * (1 - delta_x) * (delta_y) * (delta_z) +
        g_z_bu * (delta_x) * (1 - delta_y) * (delta_z) + g_z_bru * (1 - delta_x) * (1 - delta_y) * (delta_z) +
        g_z_tl * (delta_x) * (delta_y) * (1 - delta_z) + g_z_tr * (1 - delta_x) * (delta_y) * (1 - delta_z) +
        g_z_tu * (delta_x) * (1 - delta_y) * (1 - delta_z) + g_z_tru * (1 - delta_x) * (1 - delta_y) * (1 - delta_z);

  grav_x_dev[tid] = g_x;
  grav_y_dev[tid] = g_y;
  grav_z_dev[tid] = g_z;
}

// Call the kernel to compote the gravitational field at the particles positions
// ( CIC )
void Particles3D::Get_Gravity_CIC_GPU_function(part_int_t n_local, int nx_local, int ny_local, int nz_local,
                                               int n_ghost_particles_grid, Real xMin, Real xMax, Real yMin, Real yMax,
                                               Real zMin, Real zMax, Real dx, Real dy, Real dz, Real *pos_x_dev,
                                               Real *pos_y_dev, Real *pos_z_dev, Real *grav_x_dev, Real *grav_y_dev,
                                               Real *grav_z_dev, Real *gravity_x_dev, Real *gravity_y_dev,
                                               Real *gravity_z_dev)
{
  // set values for GPU kernels
  int ngrid = (n_local - 1) / TPB_PARTICLES + 1;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB_PARTICLES, 1, 1);

  // Only runs if there are local particles
  if (n_local > 0) {
    hipLaunchKernelGGL(Get_Gravity_CIC_Kernel, dim1dGrid, dim1dBlock, 0, 0, n_local, gravity_x_dev, gravity_y_dev,
                       gravity_z_dev, pos_x_dev, pos_y_dev, pos_z_dev, grav_x_dev, grav_y_dev, grav_z_dev, xMin, yMin,
                       zMin, xMax, yMax, zMax, dx, dy, dz, nx_local, ny_local, nz_local, n_ghost_particles_grid);
    GPU_Error_Check();
  }
}

  #endif  // PARTICLES_GPU

  #ifdef GRAVITY_GPU

void __global__ Copy_Particles_Density_Kernel(Real *dst_density, Real *src_density, int nx_local, int ny_local,
                                              int nz_local, int n_ghost)
{
  int tid_x, tid_y, tid_z, tid_CIC, tid_dens;
  tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  tid_y = blockIdx.y * blockDim.y + threadIdx.y;
  tid_z = blockIdx.z * blockDim.z + threadIdx.z;

  if (tid_x >= nx_local || tid_y >= ny_local || tid_z >= nz_local) {
    return;
  }

  tid_dens = tid_x + tid_y * nx_local + tid_z * nx_local * ny_local;

  tid_x += n_ghost;
  tid_y += n_ghost;
  tid_z += n_ghost;

  int nx_CIC, ny_CIC;
  nx_CIC  = nx_local + 2 * n_ghost;
  ny_CIC  = ny_local + 2 * n_ghost;
  tid_CIC = tid_x + tid_y * nx_CIC + tid_z * nx_CIC * ny_CIC;

  dst_density[tid_dens] = src_density[tid_CIC];
}

// Copy the particles density to the density array in Grav to compute the
// potential
void Grid3D::Copy_Particles_Density_GPU()
{
  int nx_local, ny_local, nz_local, n_ghost;
  n_ghost  = Particles.G.n_ghost_particles_grid;
  nx_local = Grav.nx_local;
  ny_local = Grav.ny_local;
  nz_local = Grav.nz_local;

  // set values for GPU kernels
  int tpb_x   = 16;
  int tpb_y   = 8;
  int tpb_z   = 8;
  int ngrid_x = (nx_local - 1) / tpb_x + 1;
  int ngrid_y = (ny_local - 1) / tpb_y + 1;
  int ngrid_z = (nz_local - 1) / tpb_z + 1;
  // number of blocks per 1D grid
  dim3 dim3dGrid(ngrid_x, ngrid_y, ngrid_z);
  //  number of threads per 1D block
  dim3 dim3dBlock(tpb_x, tpb_y, tpb_z);

  hipLaunchKernelGGL(Copy_Particles_Density_Kernel, dim3dGrid, dim3dBlock, 0, 0, Grav.F.density_d,
                     Particles.G.density_dev, nx_local, ny_local, nz_local, n_ghost);
}

  #endif  // GRAVITY_GPU

#endif  // PARTICLES
