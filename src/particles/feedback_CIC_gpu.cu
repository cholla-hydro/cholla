
#include <hip/hip_runtime.h>
#if defined(SUPERNOVA) && defined(PARTICLES_GPU) && defined(PARTICLE_AGE) && defined(PARTICLE_IDS)

  #include <math.h>
  #include <stdio.h>
  #include <stdlib.h>
  #include <unistd.h>

  #include <cstring>
  #include <fstream>
  #include <sstream>
  #include <vector>

  #include "../global/global.h"
  #include "../global/global_cuda.h"
  #include "../grid/grid3D.h"
  #include "../io/io.h"
  #include "supernova.h"

  #define TPB_FEEDBACK   128
  #define FEED_INFO_N    6
  #define I_RES          1  // unused
  #define I_UNRES        2  // unused
  #define I_ENERGY       3  // unused
  #define I_MOMENTUM     4  // unused
  #define I_UNRES_ENERGY 5  // used

typedef curandStateMRG32k3a_t FeedbackPrng;
// typedef curandStatePhilox4_32_10_t FeedbackPrng;

namespace supernova
{
FeedbackPrng* randStates;
part_int_t n_states;
Real *dev_snr, snr_dt, time_sn_start, time_sn_end;
int snr_n;
}  // namespace supernova

  #ifndef O_HIP
// NOLINTNEXTLINE(readability-identifier-naming)
__device__ double atomicMax(double* address, double val)
{
  auto* address_as_ull       = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}
  #endif  // O_HIP

__global__ void Init_State_Kernel(unsigned int seed, FeedbackPrng* states)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  curand_init(seed, id, 0, &states[id]);
}

/**
 * @brief Does 2 things:
 * -# Read in SN rate data from Starburst 99. If no file exists, assume a
 * constant rate.
 * -# Initialize the cuRAND state, which is analogous to the concept of
 * generators in CPU code. The state object maintains configuration and status
 * the cuRAND context for each thread on the GPU. Initialize more than the
 * number of local particles since the latter will change through MPI transfers.
 *
 * @param P pointer to parameters struct. Passes in starburst 99 filename and
 * random number gen seed.
 * @param n_local  number of local particles on the GPU
 * @param allocation_factor
 */
void supernova::initState(struct Parameters* P, part_int_t n_local, Real allocation_factor)
{
  chprintf("supernova::initState start\n");
  std::string snr_filename(P->snr_filename);
  if (not snr_filename.empty()) {
    chprintf("Specified a SNR filename %s.\n", snr_filename.data());

    // read in array of supernova rate values.
    std::ifstream snr_in(snr_filename);
    if (!snr_in.is_open()) {
      chprintf("ERROR: but couldn't read SNR file.\n");
      exit(-1);
    }

    std::vector<Real> snr_time;
    std::vector<Real> snr;

    const int N_HEADER    = 7;    // S'99 has 7 rows of header information
    const char* s99_delim = " ";  // S'99 data separator
    std::string line;
    int line_counter = 0;

    while (snr_in.good()) {
      std::getline(snr_in, line);
      if (line_counter++ < N_HEADER) {
        continue;
      }  // skip header processing

      int i      = 0;
      char* data = strtok(line.data(), s99_delim);
      while (data != nullptr) {
        if (i == 0) {
          // in the following divide by # years per kyr (1000)
          snr_time.push_back(std::stof(std::string(data)) / 1000);
        } else if (i == 1) {
          snr.push_back(pow(10, std::stof(std::string(data))) / 1000);
        }
        if (i > 0) {
          break;  // only care about the first 2 items.  Once i = 1 can break
        }         // here.

        data = strtok(nullptr, s99_delim);
        i++;
      }
    }

    time_sn_end   = snr_time[snr_time.size() - 1];
    time_sn_start = snr_time[0];
    // the following is the time interval between data points
    // (i.e. assumes regular temporal spacing)
    snr_dt = (time_sn_end - time_sn_start) / (snr.size() - 1);

    GPU_Error_Check(cudaMalloc((void**)&dev_snr, snr.size() * sizeof(Real)));
    GPU_Error_Check(cudaMemcpy(dev_snr, snr.data(), snr.size() * sizeof(Real), cudaMemcpyHostToDevice));

  } else {
    chprintf("No SN rate file specified.  Using constant rate\n");
    time_sn_start = DEFAULT_SN_START;
    time_sn_end   = DEFAULT_SN_END;
  }

  // Now initialize the poisson random number generator state.
  n_states = n_local * allocation_factor;
  GPU_Error_Check(cudaMalloc((void**)&randStates, n_states * sizeof(FeedbackPrng)));

  int ngrid = (n_states - 1) / TPB_FEEDBACK + 1;
  dim3 grid(ngrid);
  dim3 block(TPB_FEEDBACK);

  hipLaunchKernelGGL(Init_State_Kernel, grid, block, 0, 0, P->prng_seed, randStates);
  GPU_Error_Check(cudaDeviceSynchronize());
  chprintf("supernova::initState end: n_states=%ld, ngrid=%d, threads=%d\n", n_states, ngrid, TPB_FEEDBACK);
}

__device__ Real GetSNRate(Real t, Real* dev_snr, Real snr_dt, Real t_start, Real t_end)
{
  if (t < t_start || t >= t_end) {
    return 0;
  }
  if (dev_snr == nullptr) {
    return supernova::DEFAULT_SNR;
  }

  int index = (int)((t - t_start) / snr_dt);
  return dev_snr[index] + (t - index * snr_dt) * (dev_snr[index + 1] - dev_snr[index]) / snr_dt;
}

__device__ Real Calc_Timestep(Real gamma, Real* density, Real* momentum_x, Real* momentum_y, Real* momentum_z,
                              Real* energy, int index, Real dx, Real dy, Real dz, Real density_floor)
{
  Real dens  = fmax(density[index], density_floor);
  Real d_inv = 1.0 / dens;
  Real vx    = momentum_x[index] * d_inv;
  Real vy    = momentum_y[index] * d_inv;
  Real vz    = momentum_z[index] * d_inv;
  Real P     = fmax((energy[index] - 0.5 * dens * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0), TINY_NUMBER);
  Real cs    = sqrt(gamma * P * d_inv);
  return fmax(fmax((fabs(vx) + cs) / dx, (fabs(vy) + cs) / dy), (fabs(vz) + cs) / dz);
}

/** the prescription for dividing a scalar quantity between 3x3x3 cells is done
   by imagining a 2x2x2 cell volume around the SN.  These fractions, then,
   represent the linear extent of this volume into the cell in question. For i=0
   this should be 1*1/2. For i=-1 this should be (1-dx)*1/2. For i=+1 this
   should be dx*1/2. In the above the 1/2 factor is normalize over 2
   cells/direction.
  */
__device__ Real Frac(int i, Real dx) { return (-0.5 * i * i - 0.5 * i + 1 + i * dx) * 0.5; }

__device__ Real D_Fr(int i, Real dx)
{
  return (dx > 0.5) * i * (1 - 2 * dx) + ((i + 1) * dx + 0.5 * (i - 1)) - 3 * (i - 1) * (i + 1) * (0.5 - dx);
}

__device__ Real GetAverageDensity(Real* density, int xi, int yi, int zi, int nx_grid, int ny_grid, int n_ghost)
{
  Real d_average = 0.0;
  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      for (int k = -1; k < 2; k++) {
        d_average +=
            density[(xi + n_ghost + i) + (yi + n_ghost + j) * nx_grid + (zi + n_ghost + k) * nx_grid * ny_grid];
      }
    }
  }
  return d_average / 27;
}

__device__ Real GetAverageNumberDensity_CGS(Real* density, int xi, int yi, int zi, int nx_grid, int ny_grid,
                                            int n_ghost)
{
  return GetAverageDensity(density, xi, yi, zi, nx_grid, ny_grid, n_ghost) * DENSITY_UNIT / (supernova::MU * MP);
}

__device__ bool Particle_Is_Alone(Real* pos_x_dev, Real* pos_y_dev, Real* pos_z_dev, part_int_t n_local, int gtid,
                                  Real dx)
{
  Real x0 = pos_x_dev[gtid];
  Real y0 = pos_y_dev[gtid];
  Real z0 = pos_z_dev[gtid];
  // Brute force loop to see if particle is alone
  for (int i = 0; i < n_local; i++) {
    if (i == gtid) {
      continue;
    }
    if (abs(x0 - pos_x_dev[i]) > dx) {
      continue;
    }
    if (abs(y0 - pos_y_dev[i]) > dx) {
      continue;
    }
    if (abs(z0 - pos_z_dev[i]) > dx) {
      continue;
    }
    // If we made it here, something is too close.
    return false;
  }
  return true;
}

__global__ void Cluster_Feedback_Kernel(part_int_t n_local, part_int_t* id, Real* pos_x_dev, Real* pos_y_dev,
                                        Real* pos_z_dev, Real* mass_dev, Real* age_dev, Real xMin, Real yMin, Real zMin,
                                        Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx_g, int ny_g,
                                        int nz_g, int n_ghost, Real t, Real dt, Real* dti, Real* info, Real* density,
                                        Real* gasEnergy, Real* energy, Real* momentum_x, Real* momentum_y,
                                        Real* momentum_z, Real gamma, FeedbackPrng* states, Real* prev_dens,
                                        int* prev_N, short direction, Real* dev_snr, Real snr_dt, Real time_sn_start,
                                        Real time_sn_end, int n_step, Real density_floor)
{
  __shared__ Real s_info[FEED_INFO_N * TPB_FEEDBACK];  // for collecting SN feedback information, like #
                                                       // of SNe or # resolved.
  int tid  = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + tid;

  s_info[FEED_INFO_N * tid]     = 0;  // number of supernovae
  s_info[FEED_INFO_N * tid + 1] = 0;  // number of resolved events
  s_info[FEED_INFO_N * tid + 2] = 0;  // number of unresolved events
  s_info[FEED_INFO_N * tid + 3] = 0;  // resolved energy
  s_info[FEED_INFO_N * tid + 4] = 0;  // unresolved momentum
  s_info[FEED_INFO_N * tid + 5] = 0;  // unresolved KE added via momentum injection

  if (gtid < n_local) {
    Real pos_x, pos_y, pos_z;
    Real cell_center_x, cell_center_y, cell_center_z;
    Real delta_x, delta_y, delta_z;
    Real x_frac, y_frac, z_frac;
    Real px, py, pz, d;
    // Real t_b, t_a, v_1, v_2, d_b, d_a, p_b, p_a, e;
    Real feedback_energy = 0, feedback_density = 0, feedback_momentum = 0, n_0, shell_radius;
    bool is_resolved = false;
    Real dV          = dx * dy * dz;
    Real local_dti   = 0.0;

    pos_x = pos_x_dev[gtid];
    pos_y = pos_y_dev[gtid];
    pos_z = pos_z_dev[gtid];
    // kernel_printf("(%d): pos:(%.4e, %.4e, %.4e)\n", gtid, pos_x, pos_y,
    // pos_z); kernel_printf("(%d): MIN:(%.4e, %.4e, %.4e)\n", gtid, xMin, yMin,
    // xMin);

    bool in_local =
        (pos_x >= xMin && pos_x < xMax) && (pos_y >= yMin && pos_y < yMax) && (pos_z >= zMin && pos_z < zMax);
    if (!in_local) {
      kernel_printf(
          " Feedback GPU: Particle outside local domain [%f  %f  %f]  [%f %f] "
          "[%f %f] [%f %f]\n ",
          pos_x, pos_y, pos_z, xMin, xMax, yMin, yMax, zMin, zMax);
    }

    int indx_x = (int)floor((pos_x - xMin) / dx);
    int indx_y = (int)floor((pos_y - yMin) / dy);
    int indx_z = (int)floor((pos_z - zMin) / dz);
    // kernel_printf("(%d): indx:(%d, %d, %d)\n", gtid, indx_x, indx_y, indx_z);

    bool ignore = indx_x < 0 || indx_y < 0 || indx_z < 0 || indx_x >= nx_g - 2 * n_ghost ||
                  indx_y >= ny_g - 2 * n_ghost || indx_z >= nz_g - 2 * n_ghost;
    if (ignore) {
      kernel_printf(
          " Feedback GPU: Particle CIC index err [%f  %f  %f]  [%d %d %d] [%d "
          "%d %d] \n ",
          pos_x, pos_y, pos_z, indx_x, indx_y, indx_z, nx_g, ny_g, nz_g);
    }

    // Avoid overlap issues for now
    // bool is_alone = Particle_Is_Alone(pos_x_dev, pos_y_dev, pos_z_dev,
    // n_local,
    //                                  gtid, 6 * dx);

    if (!ignore && in_local) {
      int N = 0;
      // only calculate this if there will be SN feedback
      if ((t - age_dev[gtid]) <= time_sn_end) {
        if (direction == -1) {
          N = -prev_N[gtid];
        } else {
          Real average_num_sn =
              GetSNRate(t - age_dev[gtid], dev_snr, snr_dt, time_sn_start, time_sn_end) * mass_dev[gtid] * dt;

          // N = (int) (average_num_sn + 0.5);

          FeedbackPrng state;  // = states[0]; // load initial state

          curand_init(42, 0, 0, &state);
          unsigned long long skip = n_step * 10000 + id[gtid];
          skipahead(skip, &state);  // provided by curand
          // unsigned int debug_state = curand(&state);

          N = (int)curand_poisson(&state, average_num_sn);

          // states[gtid] = state; // don't write back to state, keep it
          // pristine
          prev_N[gtid] = N;
        }
        if (N != 0) {
          mass_dev[gtid] -= N * supernova::MASS_PER_SN;
          feedback_energy  = N * supernova::ENERGY_PER_SN / dV;
          feedback_density = N * supernova::MASS_PER_SN / dV;
          if (direction == -1) {
            n_0 = prev_dens[gtid];
          } else {
            n_0             = GetAverageNumberDensity_CGS(density, indx_x, indx_y, indx_z, nx_g, ny_g, n_ghost);
            prev_dens[gtid] = n_0;
          }
          // int devcount;
          // cudaGetDeviceCount(&devcount);
          // int devId;
          // cudaGetDevice(&devId);
          // kernel_printf("[%d: %d] N: %d, time: %.4e, dt: %.4e, e: %.4e, n_0:
          // %.4e\n", devId, gtid, N, t, dt, feedback_energy, n_0);

          feedback_momentum = direction * supernova::FINAL_MOMENTUM * pow(n_0, -0.17) * pow(fabsf(N), 0.93) / dV;
          shell_radius      = supernova::R_SH * pow(n_0, -0.46) * pow(fabsf(N), 0.29);
          is_resolved       = 3 * max(dx, max(dy, dz)) <= shell_radius;
          if (!is_resolved) {
            kernel_printf(
                "UR[%f] at (%d, %d, %d)  id=%d, N=%d, shell_rad=%0.4e, "
                "n_0=%0.4e\n",
                t, indx_x + n_ghost, indx_y + n_ghost, indx_z + n_ghost, (int)id[gtid], N, shell_radius, n_0);
          }

          s_info[FEED_INFO_N * tid] = 1. * N;
          if (is_resolved) {
            s_info[FEED_INFO_N * tid + 1] = direction * 1.0;
          } else {
            s_info[FEED_INFO_N * tid + 2] = direction * 1.0;
          }

          int indx;

          if (is_resolved) {  // if resolved inject energy and density
            s_info[FEED_INFO_N * tid + 3] = feedback_energy * dV;

            indx_x = (int)floor((pos_x - xMin - 0.5 * dx) / dx);
            indx_y = (int)floor((pos_y - yMin - 0.5 * dy) / dy);
            indx_z = (int)floor((pos_z - zMin - 0.5 * dz) / dz);

            cell_center_x = xMin + indx_x * dx + 0.5 * dx;
            cell_center_y = yMin + indx_y * dy + 0.5 * dy;
            cell_center_z = zMin + indx_z * dz + 0.5 * dz;

            delta_x = 1 - (pos_x - cell_center_x) / dx;
            delta_y = 1 - (pos_y - cell_center_y) / dy;
            delta_z = 1 - (pos_z - cell_center_z) / dz;
            indx_x += n_ghost;
            indx_y += n_ghost;
            indx_z += n_ghost;

            for (int i = 0; i < 2; i++) {
              for (int j = 0; j < 2; j++) {
                for (int k = 0; k < 2; k++) {
                  indx = (indx_x + i) + (indx_y + j) * nx_g + (indx_z + k) * nx_g * ny_g;

                  if (abs(momentum_x[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Rb: (%d, %d, %d) vx = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_x[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_y[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Rb: (%d, %d, %d) vy = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_y[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_z[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Rb: (%d, %d, %d) vz = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_z[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }

                  // i_frac are the fractions of energy/density to be allocated
                  // to each of the 8 cells.
                  x_frac = i * (1 - delta_x) + (1 - i) * delta_x;
                  y_frac = j * (1 - delta_y) + (1 - j) * delta_y;
                  z_frac = k * (1 - delta_z) + (1 - k) * delta_z;

                  atomicAdd(&density[indx], x_frac * y_frac * z_frac * feedback_density);
                  atomicAdd(&gasEnergy[indx], x_frac * y_frac * z_frac * feedback_energy);
                  atomicAdd(&energy[indx], x_frac * y_frac * z_frac * feedback_energy);

                  if (abs(momentum_x[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ra: (%d, %d, %d) vx = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_x[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_y[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ra: (%d, %d, %d) vy = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_y[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_z[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ra: (%d, %d, %d) vz = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_z[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }

                  if (direction > 0) {
                    local_dti = fmax(local_dti, Calc_Timestep(gamma, density, momentum_x, momentum_y, momentum_z,
                                                              energy, indx, dx, dy, dz, density_floor));
                  }
                }
              }
            }
          } else {  // if not resolved, inject momentum and density
            s_info[FEED_INFO_N * tid + 4] = feedback_momentum * dV;

            delta_x = (pos_x - xMin - indx_x * dx) / dx;
            delta_y = (pos_y - yMin - indx_y * dy) / dy;
            delta_z = (pos_z - zMin - indx_z * dz) / dz;
            // kernel_printf("(%d):indx:(%d, %d, %d)\n", gtid, indx_x, indx_y,
            // indx_z); kernel_printf("(%d): pos:(%.4e, %.4e, %.4e), delta_x
            // (%.2e, %.2e,
            // %.2e)\n", gtid, pos_x, pos_y, pos_z, delta_x, delta_y, delta_z);

            indx_x += n_ghost;
            indx_y += n_ghost;
            indx_z += n_ghost;

            if (abs(feedback_momentum / feedback_density * VELOCITY_UNIT * 1e-5) >
                40000) {  // injected speeds are greater than 4e4 km/s
              kernel_printf("**** (%d, %d, %d) injected speeds are %.3e km/s\n", indx_x, indx_y, indx_z,
                            feedback_momentum / feedback_density * VELOCITY_UNIT * 1e-5);
            }
            feedback_momentum /= sqrt(3.0);

            for (int i = -1; i < 2; i++) {
              for (int j = -1; j < 2; j++) {
                for (int k = -1; k < 2; k++) {
                  // index in array of conserved quantities
                  indx = (indx_x + i) + (indx_y + j) * nx_g + (indx_z + k) * nx_g * ny_g;

                  x_frac = D_Fr(i, delta_x) * Frac(j, delta_y) * Frac(k, delta_z);
                  y_frac = Frac(i, delta_x) * D_Fr(j, delta_y) * Frac(k, delta_z);
                  z_frac = Frac(i, delta_x) * Frac(j, delta_y) * D_Fr(k, delta_z);

                  px = x_frac * feedback_momentum;
                  py = y_frac * feedback_momentum;
                  pz = z_frac * feedback_momentum;
                  d  = (abs(x_frac) + abs(y_frac) + abs(z_frac)) / 6 * feedback_density +
                      n_0 * supernova::MU * MP / DENSITY_UNIT;

                  // d  = frac(i, delta_x) * frac(j, delta_y) * frac(k, delta_z)
                  // * feedback_density; e  = frac(i, delta_x) * frac(j,
                  // delta_y) * frac(k, delta_z) * feedback_energy;
                  // kernel_printf("(%d, %d, %d): delta:(%.4e, %.4e, %.4e),
                  // frac: %.4e\n", indx_x, indx_y, indx_z, delta_x, delta_y,
                  // delta_z, frac(i, delta_x)*frac(j, delta_y)*frac(k,
                  // delta_z)); kernel_printf("(%d, %d, %d):(%d SN) (i:%d, j:%d,
                  // k:%d) before: %.4e\n", indx_x, indx_y, indx_z, N, i, j, k,
                  // density[indx]*DENSITY_UNIT/0.6/MP);

                  // v_1 = sqrt((momentum_x[indx]*momentum_x[indx] +
                  // momentum_y[indx]*momentum_y[indx] +
                  // momentum_z[indx]*momentum_z[indx])/density[indx]/density[indx])*VELOCITY_UNIT/1e5;
                  // t_b = gasEnergy[indx]*ENERGY_UNIT*(gamma -
                  // 1)/(density[indx]*DENSITY_UNIT/0.6/MP*KB); p_b =
                  // sqrt(momentum_x[indx]*momentum_x[indx] +
                  // momentum_y[indx]*momentum_y[indx] +
                  // momentum_z[indx]*momentum_z[indx])*VELOCITY_UNIT/1e5; d_b =
                  // density[indx]*DENSITY_UNIT/0.6/MP;

                  if (abs(momentum_x[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ub: (%d, %d, %d) vx = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_x[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_y[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ub: (%d, %d, %d) vy = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_y[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_z[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ub: (%d, %d, %d) vz = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_z[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }

                  atomicAdd(&momentum_x[indx], px);
                  atomicAdd(&momentum_y[indx], py);
                  atomicAdd(&momentum_z[indx], pz);

                  density[indx] = d;
                  energy[indx]  = (momentum_x[indx] * momentum_x[indx] + momentum_y[indx] * momentum_y[indx] +
                                  momentum_z[indx] * momentum_z[indx]) /
                                     2 / density[indx] +
                                 gasEnergy[indx];

                  // atomicAdd(    &energy[indx], e );
                  // atomicAdd(   &density[indx], d );

                  s_info[FEED_INFO_N * tid + I_UNRES_ENERGY] +=
                      direction * (px * px + py * py + pz * pz) / 2 / density[indx] * dV;

                  if (abs(momentum_x[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ua: (%d, %d, %d) vx = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_x[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_y[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ua: (%d, %d, %d) vy = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_y[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  if (abs(momentum_z[indx] / density[indx]) >= C_L) {
                    kernel_printf(
                        "%d, Ua: (%d, %d, %d) vz = %.3e, d = %.3e, n_0 = "
                        "%.3e\n",
                        direction, indx_x + i, indx_y + j, indx_z + k,
                        momentum_z[indx] / density[indx] * VELOCITY_UNIT * 1e-5,
                        density[indx] * DENSITY_UNIT / 0.6 / MP, n_0);
                  }
                  // gasEnergy[indx] = energy[indx] -
                  // (momentum_x[indx]*momentum_x[indx] +
                  // momentum_y[indx]*momentum_y[indx] +
                  // momentum_z[indx]*momentum_z[indx])/2/density[indx]; v_2 =
                  // sqrt((momentum_x[indx]*momentum_x[indx] +
                  // momentum_y[indx]*momentum_y[indx] +
                  // momentum_z[indx]*momentum_z[indx])/density[indx]/density[indx])
                  // * VELOCITY_UNIT/1e5; t_a =
                  // gasEnergy[indx]*ENERGY_UNIT*(gamma -
                  // 1)/(density[indx]*DENSITY_UNIT/0.6/MP*KB); d_a =
                  // density[indx]*DENSITY_UNIT/0.6/MP; p_a =
                  // sqrt(momentum_x[indx]*momentum_x[indx] +
                  // momentum_y[indx]*momentum_y[indx] +
                  // momentum_z[indx]*momentum_z[indx])*VELOCITY_UNIT/1e5;

                  // kernel_printf("(%d, %d, %d):(CM: %.2e, SN: %d) (i:%d, j:%d,
                  // k:%d) v_1: %.5e v_2: %.5e   V_DIFF-> %.4f %%\n", indx_x,
                  // indx_y, indx_z, mass_dev[gtid], N, i, j, k, v_1, v_2,
                  // (v_2-v_1)/v_1*100); kernel_printf("   (%d, %d, %d):(%d SN)
                  // (i:%d, j:%d, k:%d) T_b: %.5e T_a: %.5e   T_DIFF-> %.4f
                  // %%\n", indx_x, indx_y, indx_z, N, i, j, k, t_b, t_a,
                  // (t_a-t_b)/t_b*100); kernel_printf("      (%d, %d, %d):(%d
                  // SN) (i:%d, j:%d, k:%d) d_b: %.5e d_a: %.5e   D_DIFF-> %.1f
                  // %%\n", indx_x, indx_y, indx_z, N, i, j, k, d_b, d_a,
                  // (d_a-d_b)/d_b*100); kernel_printf("         (%d, %d,
                  // %d):(%d SN) (i:%d, j:%d, k:%d) p_b: %.5e p_a: %.5e P_DIFF->
                  // %.4f
                  // %%\n", indx_x, indx_y, indx_z, N, i, j, k, p_b, p_a,
                  // (p_a-p_b)/p_b*100);

                  if (direction > 0) {
                    // kernel_printf("urs time:%.3e id:%d N:%d d:%.5e\n", t,
                    // id[gtid], N, n_0);
                    local_dti = fmax(local_dti, Calc_Timestep(gamma, density, momentum_x, momentum_y, momentum_z,
                                                              energy, indx, dx, dy, dz, density_floor));
                  }
                }
              }
            }
          }
          if (direction > 0) {
            atomicMax(dti, local_dti);
          }
        }
      }
    }
  }

  __syncthreads();

  // reduce the info from all the threads in the block
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      s_info[FEED_INFO_N * tid] += s_info[FEED_INFO_N * (tid + s)];
      s_info[FEED_INFO_N * tid + 1] += s_info[FEED_INFO_N * (tid + s) + 1];
      s_info[FEED_INFO_N * tid + 2] += s_info[FEED_INFO_N * (tid + s) + 2];
      s_info[FEED_INFO_N * tid + 3] += s_info[FEED_INFO_N * (tid + s) + 3];
      s_info[FEED_INFO_N * tid + 4] += s_info[FEED_INFO_N * (tid + s) + 4];
      s_info[FEED_INFO_N * tid + 5] += s_info[FEED_INFO_N * (tid + s) + 5];
    }
    __syncthreads();
  }

  if (tid == 0) {
    info[FEED_INFO_N * blockIdx.x]     = s_info[0];
    info[FEED_INFO_N * blockIdx.x + 1] = s_info[1];
    info[FEED_INFO_N * blockIdx.x + 2] = s_info[2];
    info[FEED_INFO_N * blockIdx.x + 3] = s_info[3];
    info[FEED_INFO_N * blockIdx.x + 4] = s_info[4];
    info[FEED_INFO_N * blockIdx.x + 5] = s_info[5];
  }
}

Real supernova::Cluster_Feedback(Grid3D& G, FeedbackAnalysis& analysis)
{
  #ifdef CPU_TIME
  G.Timer.Feedback.Start();
  #endif

  if (G.H.dt == 0) {
    return 0.0;
  }

  /*
  if (G.Particles.n_local > supernova::n_states) {
    printf("ERROR: not enough cuRAND states (%ld) for %ld local particles\n",
           supernova::n_states, G.Particles.n_local);
    exit(-1);
  }
  */

  Real h_dti = 0.0;
  int direction, ngrid;
  Real h_info[6] = {0, 0, 0, 0, 0, 0};
  Real info[6];
  Real *d_dti, *d_info;
  // require d_prev_dens & d_prev_N in case we have to undo feedback if the time
  // step is too large.
  Real* d_prev_dens;
  int* d_prev_N;

  if (G.Particles.n_local > 0) {
    GPU_Error_Check(cudaMalloc(&d_dti, sizeof(Real)));
    GPU_Error_Check(cudaMemcpy(d_dti, &h_dti, sizeof(Real), cudaMemcpyHostToDevice));
    GPU_Error_Check(cudaMalloc(&d_prev_dens, G.Particles.n_local * sizeof(Real)));
    GPU_Error_Check(cudaMalloc(&d_prev_N, G.Particles.n_local * sizeof(int)));
    GPU_Error_Check(cudaMemset(d_prev_dens, 0, G.Particles.n_local * sizeof(Real)));
    GPU_Error_Check(cudaMemset(d_prev_N, 0, G.Particles.n_local * sizeof(int)));

    ngrid = std::ceil((1. * G.Particles.n_local) / TPB_FEEDBACK);
    GPU_Error_Check(cudaMalloc((void**)&d_info, FEED_INFO_N * ngrid * sizeof(Real)));
  }
  // TODO: info collection and max dti calculation
  // assumes ngrid is 1.  The reason being that reduction of
  // d_info is currently done on each block.  Only the first block reduction
  // is used

  do {
    direction = 1;
    if (G.Particles.n_local > 0) {
      hipLaunchKernelGGL(Cluster_Feedback_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local,
                         G.Particles.partIDs_dev, G.Particles.pos_x_dev, G.Particles.pos_y_dev, G.Particles.pos_z_dev,
                         G.Particles.mass_dev, G.Particles.age_dev, G.H.xblocal, G.H.yblocal, G.H.zblocal,
                         G.H.xblocal_max, G.H.yblocal_max, G.H.zblocal_max, G.H.dx, G.H.dy, G.H.dz, G.H.nx, G.H.ny,
                         G.H.nz, G.H.n_ghost, G.H.t, G.H.dt, d_dti, d_info, G.C.d_density, G.C.d_GasEnergy,
                         G.C.d_Energy, G.C.d_momentum_x, G.C.d_momentum_y, G.C.d_momentum_z, gama,
                         supernova::randStates, d_prev_dens, d_prev_N, direction, dev_snr, snr_dt, time_sn_start,
                         time_sn_end, G.H.n_step, G.H.density_floor);

      GPU_Error_Check(cudaMemcpy(&h_dti, d_dti, sizeof(Real), cudaMemcpyDeviceToHost));
    }

  #ifdef MPI_CHOLLA
    h_dti = ReduceRealMax(h_dti);
    MPI_Barrier(world);
  #endif  // MPI_CHOLLA

    if (h_dti != 0 && (C_cfl / h_dti < G.H.dt)) {
      // timestep too big: need to undo the last operation
      direction = -1;
      if (G.Particles.n_local > 0) {
        hipLaunchKernelGGL(Cluster_Feedback_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local,
                           G.Particles.partIDs_dev, G.Particles.pos_x_dev, G.Particles.pos_y_dev, G.Particles.pos_z_dev,
                           G.Particles.mass_dev, G.Particles.age_dev, G.H.xblocal, G.H.yblocal, G.H.zblocal,
                           G.H.xblocal_max, G.H.yblocal_max, G.H.zblocal_max, G.H.dx, G.H.dy, G.H.dz, G.H.nx, G.H.ny,
                           G.H.nz, G.H.n_ghost, G.H.t, G.H.dt, d_dti, d_info, G.C.d_density, G.C.d_GasEnergy,
                           G.C.d_Energy, G.C.d_momentum_x, G.C.d_momentum_y, G.C.d_momentum_z, gama,
                           supernova::randStates, d_prev_dens, d_prev_N, direction, dev_snr, snr_dt, time_sn_start,
                           time_sn_end, G.H.n_step, G.H.density_floor);

        GPU_Error_Check(cudaDeviceSynchronize());
      }
      G.H.dt = C_cfl / h_dti;
    }

  } while (direction == -1);

  if (G.Particles.n_local > 0) {
    GPU_Error_Check(cudaMemcpy(&h_info, d_info, FEED_INFO_N * sizeof(Real), cudaMemcpyDeviceToHost));
    GPU_Error_Check(cudaFree(d_dti));
    GPU_Error_Check(cudaFree(d_info));
    GPU_Error_Check(cudaFree(d_prev_dens));
    GPU_Error_Check(cudaFree(d_prev_N));
  }

  #ifdef MPI_CHOLLA
  MPI_Reduce(&h_info, &info, FEED_INFO_N, MPI_CHREAL, MPI_SUM, root, world);
  #else
  info = h_info;
  #endif

  analysis.countSN += (int)info[supernova::SN];
  analysis.countResolved += (int)info[supernova::RESOLVED];
  analysis.countUnresolved += (int)info[supernova::NOT_RESOLVED];
  analysis.totalEnergy += info[supernova::ENERGY];
  analysis.totalMomentum += info[supernova::MOMENTUM];
  analysis.totalUnresEnergy += info[supernova::UNRES_ENERGY];

  Real resolved_ratio = 0.0;
  if (info[supernova::RESOLVED] > 0 || info[supernova::NOT_RESOLVED] > 0) {
    resolved_ratio = info[supernova::RESOLVED] / (info[supernova::RESOLVED] + info[supernova::NOT_RESOLVED]);
  }
  Real global_resolved_ratio = 0.0;
  if (analysis.countResolved > 0 || analysis.countUnresolved > 0) {
    global_resolved_ratio = (Real)(analysis.countResolved) / (Real)(analysis.countResolved + analysis.countUnresolved);
  }

  chprintf("iteration %d: number of SN: %d, ratio of resolved %.3e\n", G.H.n_step, (long)info[supernova::SN],
           resolved_ratio);
  chprintf(
      "    this iteration: energy: %.5e erg.  momentum: %.5e S.M. km/s  "
      "unres_energy: %.5e erg\n",
      info[supernova::ENERGY] * MASS_UNIT * LENGTH_UNIT * LENGTH_UNIT / TIME_UNIT / TIME_UNIT,
      info[supernova::MOMENTUM] * VELOCITY_UNIT / 1e5,
      info[supernova::UNRES_ENERGY] * MASS_UNIT * LENGTH_UNIT * LENGTH_UNIT / TIME_UNIT / TIME_UNIT);
  chprintf("    cummulative: #SN: %d, ratio of resolved (R: %d, UR: %d) = %.3e\n", (long)analysis.countSN,
           (long)analysis.countResolved, (long)analysis.countUnresolved, global_resolved_ratio);
  chprintf(
      "    energy: %.5e erg.  Total momentum: %.5e S.M. km/s, Total unres "
      "energy: %.5e\n",
      analysis.totalEnergy * MASS_UNIT * LENGTH_UNIT * LENGTH_UNIT / TIME_UNIT / TIME_UNIT,
      analysis.totalMomentum * VELOCITY_UNIT / 1e5,
      analysis.totalUnresEnergy * MASS_UNIT * LENGTH_UNIT * LENGTH_UNIT / TIME_UNIT / TIME_UNIT);

  #ifdef CPU_TIME
  G.Timer.Feedback.End();
  #endif

  return h_dti;
}

#endif  // SUPERNOVA & PARTICLES_GPU & PARTICLE_IDS & PARTICLE_AGE
