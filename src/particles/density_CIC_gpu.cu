
#include <hip/hip_runtime.h>
#ifdef PARTICLES

  #include <math.h>
  #include <stdio.h>
  #include <stdlib.h>
  #include <unistd.h>

  #include "../global/global.h"
  #include "../global/global_cuda.h"
  #include "../grid/grid3D.h"
  #include "../particles/particles_3D.h"
  #include "../utils/gpu.hpp"

  #ifdef GRAVITY_GPU
void Grid3D::Copy_Particles_Density_to_GPU()
{
  GPU_Error_Check(cudaMemcpy(Particles.G.density_dev, Particles.G.density, Particles.G.n_cells * sizeof(Real),
                             cudaMemcpyHostToDevice));
}

  #endif

  #ifdef PARTICLES_GPU

    // Define atomic_add if it's not supported
    #if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
    #else
__device__ double atomicAdd(double *address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old             = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
    #endif

// Get the CIC index from the particle position ( device function )
__device__ void Get_Indexes_CIC(Real xMin, Real yMin, Real zMin, Real dx, Real dy, Real dz, Real pos_x, Real pos_y,
                                Real pos_z, int &indx_x, int &indx_y, int &indx_z)
{
  indx_x = (int)floor((pos_x - xMin - 0.5 * dx) / dx);
  indx_y = (int)floor((pos_y - yMin - 0.5 * dy) / dy);
  indx_z = (int)floor((pos_z - zMin - 0.5 * dz) / dz);
}

// CUDA Kernel to compute the CIC density from the particles positions
__global__ void Get_Density_CIC_Kernel(part_int_t n_local, Real particle_mass, Real *density_dev, Real *pos_x_dev,
                                       Real *pos_y_dev, Real *pos_z_dev, Real *mass_dev, Real xMin, Real yMin,
                                       Real zMin, Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx,
                                       int ny, int nz, int n_ghost)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n_local) {
    return;
  }

  int nx_g, ny_g;
  nx_g = nx + 2 * n_ghost;
  ny_g = ny + 2 * n_ghost;

  Real pos_x, pos_y, pos_z, pMass;
  Real cell_center_x, cell_center_y, cell_center_z;
  Real delta_x, delta_y, delta_z;
  Real dV_inv = 1. / (dx * dy * dz);

  pos_x = pos_x_dev[tid];
  pos_y = pos_y_dev[tid];
  pos_z = pos_z_dev[tid];

    #ifdef SINGLE_PARTICLE_MASS
  pMass = particle_mass * dV_inv;
    #else
  pMass = mass_dev[tid] * dV_inv;
    #endif

  int indx_x, indx_y, indx_z, indx;
  Get_Indexes_CIC(xMin, yMin, zMin, dx, dy, dz, pos_x, pos_y, pos_z, indx_x, indx_y, indx_z);

  bool in_local = true;

  if (pos_x < xMin || pos_x >= xMax) {
    in_local = false;
  }
  if (pos_y < yMin || pos_y >= yMax) {
    in_local = false;
  }
  if (pos_z < zMin || pos_z >= zMax) {
    in_local = false;
  }
  if (!in_local) {
    printf(
        " Density CIC Error: Particle outside local domain [%f  %f  %f]  [%f "
        "%f] [%f %f] [%f %f]\n ",
        pos_x, pos_y, pos_z, xMin, xMax, yMin, yMax, zMin, zMax);
    return;
  }

  cell_center_x = xMin + indx_x * dx + 0.5 * dx;
  cell_center_y = yMin + indx_y * dy + 0.5 * dy;
  cell_center_z = zMin + indx_z * dz + 0.5 * dz;
  delta_x       = 1 - (pos_x - cell_center_x) / dx;
  delta_y       = 1 - (pos_y - cell_center_y) / dy;
  delta_z       = 1 - (pos_z - cell_center_z) / dz;
  indx_x += n_ghost;
  indx_y += n_ghost;
  indx_z += n_ghost;

  indx = indx_x + indx_y * nx_g + indx_z * nx_g * ny_g;
  // density_dev[indx] += pMass  * delta_x * delta_y * delta_z;
  atomicAdd(&density_dev[indx], pMass * delta_x * delta_y * delta_z);

  indx = (indx_x + 1) + indx_y * nx_g + indx_z * nx_g * ny_g;
  // density_dev[indx] += pMass  * (1-delta_x) * delta_y * delta_z;
  atomicAdd(&density_dev[indx], pMass * (1 - delta_x) * delta_y * delta_z);

  indx = indx_x + (indx_y + 1) * nx_g + indx_z * nx_g * ny_g;
  // density_dev[indx] += pMass  * delta_x * (1-delta_y) * delta_z;
  atomicAdd(&density_dev[indx], pMass * delta_x * (1 - delta_y) * delta_z);
  //
  indx = indx_x + indx_y * nx_g + (indx_z + 1) * nx_g * ny_g;
  // density_dev[indx] += pMass  * delta_x * delta_y * (1-delta_z);
  atomicAdd(&density_dev[indx], pMass * delta_x * delta_y * (1 - delta_z));

  indx = (indx_x + 1) + (indx_y + 1) * nx_g + indx_z * nx_g * ny_g;
  // density_dev[indx] += pMass  * (1-delta_x) * (1-delta_y) * delta_z;
  atomicAdd(&density_dev[indx], pMass * (1 - delta_x) * (1 - delta_y) * delta_z);

  indx = (indx_x + 1) + indx_y * nx_g + (indx_z + 1) * nx_g * ny_g;
  // density_dev[indx] += pMass  * (1-delta_x) * delta_y * (1-delta_z);
  atomicAdd(&density_dev[indx], pMass * (1 - delta_x) * delta_y * (1 - delta_z));

  indx = indx_x + (indx_y + 1) * nx_g + (indx_z + 1) * nx_g * ny_g;
  // density_dev[indx] += pMass  * delta_x * (1-delta_y) * (1-delta_z);
  atomicAdd(&density_dev[indx], pMass * delta_x * (1 - delta_y) * (1 - delta_z));

  indx = (indx_x + 1) + (indx_y + 1) * nx_g + (indx_z + 1) * nx_g * ny_g;
  // density_dev[indx] += pMass * (1-delta_x) * (1-delta_y) * (1-delta_z);
  atomicAdd(&density_dev[indx], pMass * (1 - delta_x) * (1 - delta_y) * (1 - delta_z));
}

// Clear the density array: density=0
void Particles3D::Clear_Density_GPU_function(Real *density_dev, int n_cells)
{
  Set_Particles_Array_Real(0.0, density_dev, n_cells);
}

// Call the CIC density kernel to get the particles density
void Particles3D::Get_Density_CIC_GPU_function(part_int_t n_local, Real particle_mass, Real xMin, Real xMax, Real yMin,
                                               Real yMax, Real zMin, Real zMax, Real dx, Real dy, Real dz, int nx_local,
                                               int ny_local, int nz_local, int n_ghost_particles_grid, int n_cells,
                                               Real *density_h, Real *density_dev, Real *pos_x_dev, Real *pos_y_dev,
                                               Real *pos_z_dev, Real *mass_dev)
{
  // set values for GPU kernels
  int ngrid = (n_local - 1) / TPB_PARTICLES + 1;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB_PARTICLES, 1, 1);

  // Only runs if there are local particles
  if (n_local > 0) {
    hipLaunchKernelGGL(Get_Density_CIC_Kernel, dim1dGrid, dim1dBlock, 0, 0, n_local, particle_mass, density_dev,
                       pos_x_dev, pos_y_dev, pos_z_dev, mass_dev, xMin, yMin, zMin, xMax, yMax, zMax, dx, dy, dz,
                       nx_local, ny_local, nz_local, n_ghost_particles_grid);
    GPU_Error_Check();
    cudaDeviceSynchronize();
  }

    #if !defined(GRAVITY_GPU)
  // Copy the density from device to host
  GPU_Error_Check(cudaMemcpy(density_h, density_dev, n_cells * sizeof(Real), cudaMemcpyDeviceToHost));
    #endif
}

  #endif  // PARTICLES_GPU
#endif    // PARTICLES
