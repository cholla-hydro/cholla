
#include <hip/hip_runtime.h>
// Require HDF5
#ifdef HDF5

  #include <hdf5.h>

  #include "../grid/grid3D.h"
  #include "../io/io.h"
  #include "../utils/cuda_utilities.h"

// Note that the HDF5 file and buffer will have size nx_real * ny_real * nz_real
// whereas the conserved variables have size nx,ny,nz.

// Note that magnetic fields
// add +1 to nx_real ny_real nz_real since an extra face needs to be output, but
// also has the same size nx ny nz.

// For the magnetic field case, a different
// nx_real+1 ny_real+1 nz_real+1 n_ghost-1 are provided as inputs.

// 2D version of CopyReal3D_GPU_Kernel. Note that magnetic fields and float32 output are not enabled in 2-D so this is a
// simpler kernel
__global__ void CopyReal2D_GPU_Kernel(int nx, int ny, int nx_real, int ny_real, int nz_real, int n_ghost,
                                      Real* destination, Real* source)
{
  int const id = threadIdx.x + blockIdx.x * blockDim.x;

  int i, j, k;
  cuda_utilities::compute3DIndices(id, nx_real, ny_real, i, j, k);
  // i goes up to nx_real
  // j goes up to ny_real
  // for 2D, k should be 0
  if (k >= 1) {
    return;
  }

  // This converts into HDF5 indexing that plays well with Python
  int const dest_id   = j + i * ny_real;
  int const source_id = (i + n_ghost) + (j + n_ghost) * nx;

  destination[dest_id] = source[source_id];
}

// Copy Real (non-ghost) cells from source to a double destination (for writing
// HDF5 in double precision)
__global__ void CopyReal3D_GPU_Kernel(int nx, int ny, int nx_real, int ny_real, int nz_real, int n_ghost,
                                      double* destination, Real* source, int mhd_direction)
{
  int const id = threadIdx.x + blockIdx.x * blockDim.x;

  int i, j, k;
  cuda_utilities::compute3DIndices(id, nx_real, ny_real, i, j, k);

  if (k >= nz_real) {
    return;
  }

  // This converts into HDF5 indexing that plays well with Python
  int const dest_id   = k + j * nz_real + i * ny_real * nz_real;
  int const source_id = (i + n_ghost - int(mhd_direction == 0)) + (j + n_ghost - int(mhd_direction == 1)) * nx +
                        (k + n_ghost - int(mhd_direction == 2)) * nx * ny;

  destination[dest_id] = (double)source[source_id];
}

// Copy Real (non-ghost) cells from source to a float destination (for writing
// HDF5 in float precision)
__global__ void CopyReal3D_GPU_Kernel(int nx, int ny, int nx_real, int ny_real, int nz_real, int n_ghost,
                                      float* destination, Real* source, int mhd_direction)
{
  int const id = threadIdx.x + blockIdx.x * blockDim.x;

  int i, j, k;
  cuda_utilities::compute3DIndices(id, nx_real, ny_real, i, j, k);

  if (k >= nz_real) {
    return;
  }

  // This converts into HDF5 indexing that plays well with Python.
  // The `int(mhd_direction == NUM)` sections provide appropriate shifts for writing out the magnetic fields since they
  // need an extra cell in the same direction as the field
  int const dest_id   = k + j * nz_real + i * ny_real * nz_real;
  int const source_id = (i + n_ghost - int(mhd_direction == 0)) + (j + n_ghost - int(mhd_direction == 1)) * nx +
                        (k + n_ghost - int(mhd_direction == 2)) * nx * ny;

  destination[dest_id] = (float)source[source_id];
}

// When buffer is double, automatically use the double version of everything
// using function overloading
void Write_HDF5_Field_3D(int nx, int ny, int nx_real, int ny_real, int nz_real, int n_ghost, hid_t file_id,
                         double* buffer, double* device_buffer, Real* device_source, const char* name,
                         int mhd_direction)
{
  herr_t status;
  hsize_t dims[3];
  dims[0]            = nx_real;
  dims[1]            = ny_real;
  dims[2]            = nz_real;
  hid_t dataspace_id = H5Screate_simple(3, dims, NULL);

  // Copy non-ghost parts of source to buffer
  dim3 dim1dGrid((nx_real * ny_real * nz_real + TPB - 1) / TPB, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);
  hipLaunchKernelGGL(CopyReal3D_GPU_Kernel, dim1dGrid, dim1dBlock, 0, 0, nx, ny, nx_real, ny_real, nz_real, n_ghost,
                     device_buffer, device_source, mhd_direction);
  GPU_Error_Check(
      cudaMemcpy(buffer, device_buffer, nx_real * ny_real * nz_real * sizeof(double), cudaMemcpyDeviceToHost));

  // Write Buffer to HDF5
  status = Write_HDF5_Dataset(file_id, dataspace_id, buffer, name);

  status = H5Sclose(dataspace_id);
  if (status < 0) {
    printf("File write failed.\n");
  }
}

// When buffer is float, automatically use the float version of everything using
// function overloading
void Write_HDF5_Field_3D(int nx, int ny, int nx_real, int ny_real, int nz_real, int n_ghost, hid_t file_id,
                         float* buffer, float* device_buffer, Real* device_source, const char* name, int mhd_direction)
{
  herr_t status;
  hsize_t dims[3];
  dims[0]            = nx_real;
  dims[1]            = ny_real;
  dims[2]            = nz_real;
  hid_t dataspace_id = H5Screate_simple(3, dims, NULL);

  // Copy non-ghost parts of source to buffer
  dim3 dim1dGrid((nx_real * ny_real * nz_real + TPB - 1) / TPB, 1, 1);
  dim3 dim1dBlock(TPB, 1, 1);
  hipLaunchKernelGGL(CopyReal3D_GPU_Kernel, dim1dGrid, dim1dBlock, 0, 0, nx, ny, nx_real, ny_real, nz_real, n_ghost,
                     device_buffer, device_source, mhd_direction);
  GPU_Error_Check(
      cudaMemcpy(buffer, device_buffer, nx_real * ny_real * nz_real * sizeof(float), cudaMemcpyDeviceToHost));

  // Write Buffer to HDF5
  status = Write_HDF5_Dataset(file_id, dataspace_id, buffer, name);

  status = H5Sclose(dataspace_id);
  if (status < 0) {
    printf("File write failed.\n");
  }
}
void Fill_HDF5_Buffer_From_Grid_GPU(int nx, int ny, int nz, int nx_real, int ny_real, int nz_real, int n_ghost,
                                    Real* hdf5_buffer, Real* device_hdf5_buffer, Real* device_grid_buffer)
{
  int mhd_direction = -1;

  // 3D case
  if (nx > 1 && ny > 1 && nz > 1) {
    dim3 dim1dGrid((nx_real * ny_real * nz_real + TPB - 1) / TPB, 1, 1);
    dim3 dim1dBlock(TPB, 1, 1);
    hipLaunchKernelGGL(CopyReal3D_GPU_Kernel, dim1dGrid, dim1dBlock, 0, 0, nx, ny, nx_real, ny_real, nz_real, n_ghost,
                       device_hdf5_buffer, device_grid_buffer, mhd_direction);
    GPU_Error_Check(cudaMemcpy(hdf5_buffer, device_hdf5_buffer, nx_real * ny_real * nz_real * sizeof(Real),
                               cudaMemcpyDeviceToHost));
    return;
  }

  // 2D case
  if (nx > 1 && ny > 1 && nz == 1) {
    dim3 dim1dGrid((nx_real * ny_real + TPB - 1) / TPB, 1, 1);
    dim3 dim1dBlock(TPB, 1, 1);
    hipLaunchKernelGGL(CopyReal2D_GPU_Kernel, dim1dGrid, dim1dBlock, 0, 0, nx, ny, nx_real, ny_real, nz_real, n_ghost,
                       device_hdf5_buffer, device_grid_buffer);
    GPU_Error_Check(
        cudaMemcpy(hdf5_buffer, device_hdf5_buffer, nx_real * ny_real * sizeof(Real), cudaMemcpyDeviceToHost));
    return;
  }

  // 1D case
  if (nx > 1 && ny == 1 && nz == 1) {
    GPU_Error_Check(
        cudaMemcpy(hdf5_buffer, device_grid_buffer + n_ghost, nx_real * sizeof(Real), cudaMemcpyDeviceToHost));
    return;
  }
}

#endif  // HDF5
